#include "hip/hip_runtime.h"
//                             -*- Mode: C++ -*-
// Filename         : libdscuda.cu
// Description      : DS-CUDA client node library for Remote Procedure Call.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-09-15 11:45:48
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include <unistd.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/ioctl.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <net/if.h>
#include <ctype.h>
#include <pwd.h>
#include <netdb.h>
#include <pthread.h>
#include "dscuda.h"
#include "dscudautil.h"
#include "libdscuda.h"

static void  extractENV(bool &bool_var, const char *envname);
static void  extractENV(int  &int_var,  const char *envname, int undef=0);
static void  extractENV(char *str_var,  const char *envname, int len);
static void  getenvDSCUDA_WARNLEVEL(void);
static void  updateSpareServerList(void);
static char* readServerConf(char *fname);

static int   VdevidIndexMax = 0; //# of pthreads which utilize virtual devices.
const  char *DEFAULT_SVRIP = "localhost";

static pthread_mutex_t VdevidMutex = PTHREAD_MUTEX_INITIALIZER;
static pthread_t       VdevidIndex2ptid[RC_NPTHREADMAX]; // convert an Vdevid index into pthread id.
// CheckPointing mutual exclusion
int cp_thread_exit=0;
       pthread_mutex_t cudaMemcpyD2H_mutex = PTHREAD_MUTEX_INITIALIZER;
       pthread_mutex_t cudaMemcpyH2D_mutex = PTHREAD_MUTEX_INITIALIZER;
       pthread_mutex_t cudaKernelRun_mutex = PTHREAD_MUTEX_INITIALIZER;
       pthread_mutex_t cudaElse_mutex      = PTHREAD_MUTEX_INITIALIZER;

       pthread_mutex_t Tc_reset_mutex      = PTHREAD_MUTEX_INITIALIZER;
       int             Tc_reset_req;

       RCmappedMem    *RCmappedMemListTop     = NULL;
       RCmappedMem    *RCmappedMemListTail    = NULL;

//#if RC_SUPPORT_STREAM
static RCstreamArray  *RCstreamArrayListTop   = NULL;
static RCstreamArray  *RCstreamArrayListTail  = NULL;
//#endif

static RCeventArray   *RCeventArrayListTop    = NULL;
static RCeventArray   *RCeventArrayListTail   = NULL;

static RCcuarrayArray *RCcuarrayArrayListTop  = NULL;
static RCcuarrayArray *RCcuarrayArrayListTail = NULL;

int    Vdevid[RC_NPTHREADMAX] = {0};   // the virtual device currently in use.

/*
 * Physical GPU device server
 */
ServerArray SvrCand;
ServerArray SvrSpare;   // Alternative GPU Device Servers.
ServerArray SvrIgnore;  // Forbidden GPU Device Servers.

void (*errorHandler)(void *arg) = NULL;
void *errorHandlerArg = NULL;

//struct rdma_cm_id *Cmid[RC_NVDEVMAX][RC_NREDUNDANCYMAX];

ClientState   St;
PtxStore      Ptx;
/*
 * Client initializer.
 * This function may be executed in parallel threads, so need mutex lock.
 */
ClientState::ClientState(void) {
    //<-- Open dscuda output file.
    char curr_time[80];
    dscuda::sprintfDate( curr_time );
    sprintf( this->dslog_filename, "c%s.dslog", curr_time );
    sprintf( this->dserr_filename, "c%s.dserr", curr_time );
    sprintf( this->dschp_filename, "c%s.dschp", curr_time );
    
    dscuda_stdout = fopen( dslog_filename, "w" );
    if (dscuda_stdout == NULL) {
	fprintf(stderr, "dscuda: failed to open file %s.", dslog_filename);
	exit(EXIT_FAILURE);
    } else {
	fprintf(stderr, "dscuda: log file ==> %s\n", dslog_filename);
    }
    //
    dscuda_stderr = fopen( dserr_filename, "w" );
    if (dscuda_stderr == NULL) {
	fprintf(stderr, "dscuda: failed to open file %s.", dserr_filename);
	exit(EXIT_FAILURE);
    } else {
	fprintf(stderr, "dscuda: err file ==> %s\n", dserr_filename);
    }
    //
    dscuda_chkpnt = fopen( dschp_filename, "w" );
    if (dscuda_chkpnt == NULL) {
	fprintf(stderr, "dscuda: failed to open file %s.", dschp_filename);
	exit(EXIT_FAILURE);
    } else {
	fprintf(stderr, "dscuda: err file ==> %s\n", dschp_filename);
    }
    //--> Open dscuda output file.
	
    INFO0("\
###******************************************************************************\n\
###***                                                                          *\n\
###***   Start process of DS-CUDA client library.                               *\n\
###***                                                                          *\n\
###******************************************************************************\n");
    INFO0("[ DS-CUDA Version      ] %s\n", RC_DSCUDA_VER);
    {
	/* Capture start time and print to logfile. */
	char s_time[80];
	struct tm *timebuf;
	start_time = time(NULL);
	timebuf = localtime(&start_time);
	strftime(s_time, 80, "%T (%F)", timebuf);
	INFO0("[ Start time           ] %s\n", s_time);
    }
    {
	/* Print IP address of DS-CUDA client host. */
	sockaddr_in addrin;
	get_myaddress(&addrin);
	setMyIPAddr(addrin.sin_addr.s_addr);
	INFO0("[ IP address of client ] %s\n",dscudaGetIpaddrString(St.getIpAddress()));
    }
    {
	char path[1024];
	getcwd(path, 1024);
	INFO0("[ Working Directory    ] %s\n", path);
	INFO0("[ Process ID (PID)     ] %d\n", getpid());
    }


    use_ibv     = 0;
    autoverb    = 0;
    daemon      = 0;
    this->unsetRollbackCalling();

    extractENV( dscuda_path, "DSCUDA_PATH", 512 );
    INFO0("[ Environment varialbe ] DSCUDA_PATH      = %s\n", dscuda_path);

    getenvDSCUDA_WARNLEVEL();       /* set from DSCUDA_WARNLEVEL */
    INFO0("[ Environment variable ] DSCUDA_WARNLEVEL = %d\n", dscuda::getWarnLevel());

    this->configFT();
    INFO0("[ Environment variable ] DSCUDA_USEDAEMON = %d\n", daemon);
    INFO0("[ Environment variable ] DSCUDA_AUTOVERB  = %d\n", autoverb);
    INFO0("[ Environment variable ] DSCUDA_CP_PERIOD = %d\n", cp_period);
    INFO0("[ Fault Tolerant Mode  ] ");
    switch (ft_mode) {
    case FT_NONE:
	INFO0("\"FT_NONE\"\n");
	break;
    case FT_ERRSTAT:
	INFO0("\"FT_ERRSTAT\"\n");
	break;
    case FT_BYCPY:
	INFO0("\"FT_BYCPY\"\n");
	break;
    case FT_BYTIMER:
	INFO0("\"FT_BYTIMER\"\n");
	break;
    case FT_OPTION:
	INFO0("\"FT_OPTION\"\n");
	break;
    default:
	WARN0(0, "(UNKNOWN).\n");
	exit(EXIT_FAILURE);
    }
    INFO0("[Environment var] DSCUDA_FT0  = %d (d2h_simple)\n",   ft.d2h_simple   );
    INFO0("[Environment var] DSCUDA_FT1  = %d (d2h_reduncpy)\n", ft.d2h_reduncpy );
    INFO0("[Environment var] DSCUDA_FT2  = %d (d2h_compare)\n",  ft.d2h_compare  );
    INFO0("[Environment var] DSCUDA_FT3  = %d (d2h_statics)\n",  ft.d2h_statics  );
    INFO0("[Environment var] DSCUDA_FT4  = %d (d2h_rollback)\n", ft.d2h_rollback );
    
    INFO0("[Environment var] DSCUDA_FT8  = %d (cp_periodic)\n",  ft.cp_periodic );
    INFO0("[Environment var] DSCUDA_FT9  = %d (cp_reduncpy)\n",  ft.cp_reduncpy );
    INFO0("[Environment var] DSCUDA_FT10 = %d (cp_compare)\n",   ft.cp_compare  );
    INFO0("[Environment var] DSCUDA_FT11 = %d (cp_statics)\n",   ft.cp_statics  );
    INFO0("[Environment var] DSCUDA_FT12 = %d (cp_rollback)\n",  ft.cp_rollback );
    
    INFO0("[Environment var] DSCUDA_FT16 = %d (rec_en)\n", ft.rec_en );
    INFO0("[Environment var] DSCUDA_FT24 = %d (migrate)\n", ft.gpu_migrate );

    this->initVirtualDevice();  /* Update the list of virtual devices */
    WARN0(0, "\n");

    // Search around cluster.
    WARN(2, "   <---Start Searching DS-CUDA daemon program                            *\n");
    dscuda::searchDaemon();
    WARN(2, "   --->Stop  Searching DS-CUDA daemon program                            *\n\n");

    ServerArray svr_array;
    svr_array.captureEnv("DSCUDA_SERVER_IGNORE", hl_BAD);
    svr_array.print();

    updateSpareServerList();
    svr_array.captureEnv("DSCUDA_SERVER_SPARE", hl_GOOD);
    svr_array.print();
    for (int i=0; i<svr_array.num; i++) {
	SvrSpare.append( &svr_array.svr[i] );
    }
    
    printVirtualDeviceList(); /* Print result to terminal. */

    WARN(2, "method of remote procedure call: ");
    switch ( dscudaRemoteCallType() ) {
    case RC_REMOTECALL_TYPE_RPC:
	WARN0(2, "RPC\n");
	break;
    case RC_REMOTECALL_TYPE_IBV:
	WARN0(2, "InfiniBand Verbs\n");
	break;
    default:
	WARN0(0, "(Unkown)\n"); exit(1);
    }

    /*
     * Establish connections of all physical devices.
     */
    for (int i=0; i<Nvdev; i++) {
	for (int j=0; j<Vdev[i].nredundancy; j++) {
	    Vdev[i].server[j].setupConnection();
	    Vdev[i].server[j].setID( j );
	    WARN(1, "setupConn. Vdev[%d].server[%d].Clnt=%p\n",
		 i, j, Vdev[i].server[j].Clnt);
        }
    }
    
    if (ft.d2h_statics) {
	if (ft.cp_statics) { WARN(1, "[ERRORSTATICS] count both @D2H and @CP.\n" ); }
	else               { WARN(1, "[ERRORSTATICS] count @D2H but @CP.\n" );      }
    } else {
	if (ft.cp_statics) { WARN(1, "[ERRORSTATICS] count @CP  but @D2H.\n" );     }
	else               { WARN(1, "[ERRORSTATICS] not counted.\n" );             }
    }

    if (ft.cp_periodic) {
	WARN(1, "Starts Automatic CheckPointing Threads.\n" );	
	pthread_create(&tid, NULL, periodicCheckpoint, (void *)&cp_period);
    }
    
    INFO0("\
###******************************************************************************\n\
###***   Start user application process.                                        *\n\
###******************************************************************************\n");
} //--> ClientState::ClientState(void)
//--
//--
//--
ClientState::~ClientState(void) {
    INFO0("\
###******************************************************************************\n\
###***   Completed user application process.                                    *\n\
###******************************************************************************\n");

    PhyDev  *svr;
    time_t     exe_time;
    char       my_tfmt[64];	      
    struct tm *my_local;

    //--- Terminate the checkpointing thread.
    cp_thread_exit = 1;
    if (ft.cp_periodic) {
	WARN(1, "Stops Automatic CheckPointing Threads.\n" );
	//pthread_cancel(tid);
	pthread_join(tid, NULL);
    }
	
    stop_time = time( NULL);
    exe_time = stop_time - start_time;

    //--- Report start time.
    my_local = localtime( &start_time);
    strftime( my_tfmt, 64, "%c", my_local);
    WARN0(1, "    Start_time: %s\n", my_tfmt);
    //--- Report stop time.
    my_local = localtime( &stop_time);
    strftime( my_tfmt, 64, "%c", my_local);
    WARN0(1, "    Stop_time:  %s\n", my_tfmt);
    //--- Report run time.
    my_local = localtime( &exe_time);
    strftime( my_tfmt, 64, "%s", my_local);
    WARN0(1, "    Run_time:   %s (sec)\n", my_tfmt);

    if (ft.d2h_statics) {
	if (ft.cp_statics) {
	    WARN_CP(1, "[ERRORSTATICS] count both @D2H and @CP.\n" );
	    for (int i=0; i<Nvdev; i++) {
		WARN_CP(1, "    Virtual[%d]'s total error = %d [times]\n", i, Vdev[i].ft_unmatch_total);
	    }
	}
	else {
	    WARN0(1, "[ERRORSTATICS] count @D2H not @CP.\n" );
	    for (int i=0; i<Nvdev; i++) {
		WARN0(1, "    [ERRORSTAT]  Virtual[%2d]\n", i);
		for (int j=0; j<Vdev[i].nredundancy; j++) {
		    svr = &Vdev[i].server[j];
		    WARN0(1, "    [ERRORSTAT]  + Physical[%2d]:%s:%s: ErrorCount= %d , MatchCount= %d\n",
			  j, svr->ip, svr->hostname, svr->stat_error, svr->stat_correct);
		}
	    }
	}
    }
    else {
	if (ft.cp_statics) { WARN(1, "[ERRORSTATICS] TODO.\n" );        }
	else               { WARN(1, "[ERRORSTATICS] not counted.\n" ); }
    }

    INFO0("\
###******************************************************************************\n\
###***                                                                          *\n\
###***   Completed process of DS-CUDA client library.                           *\n\
###***                                                                          *\n\
###******************************************************************************\n");
} //--> ClientState::~ClientState(void)

void
ClientState::configFT(void) {
    extractENV( this->daemon,    "DSCUDA_USEDAEMON",  0 );
    extractENV( this->cp_period, "DSCUDA_CP_PERIOD", 60 );
    extractENV( this->autoverb,  "DSCUDA_AUTOVERB",   0 );
    //<--- Define Fault Tolerant behavior from env.var.
    switch (autoverb) {
    case -1:
	ft_mode = FT_OPTION;
	break;
    case 0:
	ft_mode = FT_NONE;
	break;
    case 1:
	ft_mode = FT_ERRSTAT;
	break;
    case 2:
	ft_mode = FT_BYCPY;
	break;
    case 3:
	ft_mode = FT_BYTIMER;
	break;
    default:
	WARN(0, "Found invalid setting of DSCUDA_AUTOVERB=%d\n", autoverb);
	exit(EXIT_FAILURE);
    }
    //---> Define Fault Tolerant behavior from env.var.
    extractENV( ft.d2h_simple,    "DSCUDA_FT0" );
    extractENV( ft.d2h_reduncpy,  "DSCUDA_FT1" );
    extractENV( ft.d2h_compare,   "DSCUDA_FT2" );
    extractENV( ft.d2h_statics,   "DSCUDA_FT3" );
    extractENV( ft.d2h_rollback,  "DSCUDA_FT4" );
    //
    extractENV( ft.cp_periodic,   "DSCUDA_FT8" ); // 2nd: checkpointing
    extractENV( ft.cp_reduncpy,   "DSCUDA_FT9" );
    extractENV( ft.cp_compare,    "DSCUDA_FT10" );
    extractENV( ft.cp_statics,    "DSCUDA_FT11" );
    extractENV( ft.cp_rollback,   "DSCUDA_FT12" );
    //
    extractENV( ft.rec_en,        "DSCUDA_FT16" ); // 3rd: CUDA API recording
    //
    extractENV( ft.gpu_migrate,   "DSCUDA_FT24" ); // 4th: GPU Migration

    //<--- copy same value to virtual and physical device.
    for (int i=0; i<RC_NVDEVMAX; i++) {
	this->Vdev[i].ft_mode = this->ft_mode;
	this->Vdev[i].ft      = this->ft;
	for (int k=0; k<Vdev[i].nredundancy; k++) {
	    Vdev[i].server[k].ft_mode = this->ft_mode;
	    Vdev[i].server[k].ft      = this->ft;
	}
    }
    //---> copy same value to virtual and physical device.

    if (ft.rec_en) {
	for (int i=0; i<RC_NVDEVMAX; i++) {
	    Vdev[i].recordON();
	}
    }
}//--> void ClientState::configsFT(void)
/*
 *
 */
void
ClientState::initVirtualDevice(void) {
    char   *ip;
    char    ips[RC_NVDEVMAX][256];
    char    buf[1024*RC_NVDEVMAX];
    PhyDev *sp;
    char   *vdev_token;
    char   *pdev_token;
    {// DSCUDA_SERVER
	char *sconfname;
	char *env;    
	if (sconfname = getenv("DSCUDA_SERVER_CONF")) {
	    env = readServerConf(sconfname);
	    INFO0("[ Environment variable ] DSCUDA_SERVER_CONF = %s\n", env);
	} else {
	    env = getenv("DSCUDA_SERVER");
	    INFO0("[ Environment variable ] DSCUDA_SERVER    = %s\n", env);
	}
	// check DSCUDA_SERVER (1)
	if (env == NULL) {
	    Nvdev = 1;
	    Vdev[0].nredundancy = 1;
	    sp = Vdev[0].server;
	    sp->id = 0;
	    strncpy(sp->ip, DEFAULT_SVRIP, sizeof(sp->ip));
	    return;
	}
	
	// check DSCUDA_SERVER (2)
	if (sizeof(buf) < strlen(env)) {
	    WARN(0, "Too long length of DSCUDA_SERVER.\n");
	    exit(EXIT_FAILURE);
	}
	strncpy( buf, env, sizeof(buf) );
    }
    //<-- set "Nvdev", # of virtual device count.
    Nvdev = 0;
    vdev_token = strtok(buf, DELIM_VDEV); // a list of IPs which consist a single vdev.
    while (vdev_token != NULL) {
	strcpy(ips[Nvdev], vdev_token);
	Nvdev++;
	if (RC_NVDEVMAX < Nvdev) {
	    WARN(0, "number of devices exceeds the limit, RC_NVDEVMAX (=%d).\n",
		 RC_NVDEVMAX);
	    exit(EXIT_FAILURE);
	}
	vdev_token = strtok(NULL, DELIM_VDEV);
    }
    //--> set "Nvdev", # of virtual device count.
    
    for (int i=0; i<Nvdev; i++) {
	int nred=0;
	int uniq=0; // begin with 0.
	pdev_token = strtok(ips[i], DELIM_REDUN); // an IP (optionally with devid preceded by a comma) of
	// a single element of the vdev.
	while (pdev_token != NULL) {
	    strcpy(Vdev[i].server[nred].ip, pdev_token);
	    pdev_token = strtok(NULL, DELIM_REDUN);
	    nred++;
	}
	/*
	 * update Vdev.info.
	 */
	Vdev[i].setConfInfo(nred);
	
	for (int j=0; j<nred; j++) {
	    sp = &Vdev[i].server[j];
	    strncpy(buf, sp->ip, sizeof(buf));
	    ip = strtok(buf, ":");
	    sp->setIP(ip);
	    ip = strtok(NULL, ":");
	    sp->setCID(ip);
	    sp->setUNIQ(uniq);
	    uniq++;
	}
    } //for ( int i=0; ...
    /* convert hostname to ip address. */
    char *hostname;
    int  det_abc;
    char letter;
    char *ip_ref;
    struct hostent *hostent0;
    for (int i=0; i<Nvdev; i++) {
	Vdev[i].id = i;
	for (int j=0; j < Vdev[i].nredundancy; j++) {
	    ip = Vdev[i].server[j].ip;
	    hostname = Vdev[i].server[j].hostname;
	    det_abc=1;
	    for (int k=0; k < strlen(ip); k++) {
		letter = ip[k];
		if (isdigit((unsigned char)letter || letter=='.')) {
		    det_abc = 0;
		    printf("%c", letter);
		} else {
		    det_abc = 1;
		    break;
		}
		printf("\n");
	    }
	    if (det_abc == 1) {
		strcpy( hostname, ip );
		hostent0 = gethostbyname( hostname );
		if ( hostent0 == NULL ) {
		    WARN( 0, "May be set invalid hostname \"%s\" to DSCUDA_SERVER or something.\n", hostname );
		    WARN( 0, "Program terminated.\n\n\n\n" );
		    exit(EXIT_FAILURE);
		} else {
		    ip_ref = inet_ntoa( *(in_addr*)hostent0->h_addr_list[0] );
		    strcpy( ip, ip_ref );
		}
	    }
	}
    } // for (int i=0; ...
} //---> void ClientState::initVirtualDevice(void)
unsigned
ClientState::getIpAddress(void) {
    return this->ip_addr;
}
void
ClientState::useIbv(void) {
    this->use_ibv = true;
}
void
ClientState::useRpc(void) {
    this->use_ibv = false;
}
bool
ClientState::isIbv(void) {
    return this->use_ibv;     
}
bool
ClientState::isRpc(void) {
    return !this->use_ibv;     
}
void
ClientState::setRollbackCalling(void) {
    this->rollback_calling = true;
}
void
ClientState::unsetRollbackCalling(void) {
    this->rollback_calling = false;
}
bool
ClientState::isRollbackCalling(void) {
    return this->rollback_calling;
}
void
ClientState::setMyIPAddr(unsigned val) {
    this->ip_addr = val;
}

ServerArray::ServerArray(void) {
    num = 0;
}
//*********************************************************
//*** CLASS: PtxRecord
//*********************************************************
PtxRecord::PtxRecord(void) {
    if (RC_KMODULENAMELEN < 16) {
	WARN(0, "%s():RC_KMODULENAMELEN is too small.\n", __func__);
	exit(1);
    }
    if (RC_KMODULEIMAGELEN < 16) {
	WARN(0, "%s():RC_KMODULEIMAGELEN is too small.\n", __func__);
	exit(1);
    }
    strcpy(name, "unknown");
    strcpy(ptx_image, "empty");
    valid = 0;
}
void
PtxRecord::invalidate(void) {
    strncpy(name, "unknown", RC_KMODULENAMELEN);
    strncpy(ptx_image, "empty", RC_KMODULEIMAGELEN);
    valid = 0;
}

void PtxRecord::set(char *name0, char *ptx_image0) {
    strncpy(name,      name0,      RC_KMODULENAMELEN);
    strncpy(ptx_image, ptx_image0, RC_KMODULEIMAGELEN);
    valid = 1;
    
    if (name[RC_KMODULENAMELEN-1] != '\0') {
	WARN(0, "%s():RC_KMODULENAMELEN is too small.\n");
	exit(1);
    }
    if (ptx_image[RC_KMODULEIMAGELEN-1] != '\0') {
	WARN(0, "%s():RC_KMODULEIMAGELEN is too small.\n");
	exit(1);
    }
}
//*********************************************************
//*** CLASS: PtxStore
//*********************************************************
PtxStore::PtxStore(void) {
    used_count = 0;
}
PtxRecord*
PtxStore::add(char *name0, char *ptx_image0) {
    PtxRecord *ptx_ptr = &ptx_record[used_count];
    if (used_count > RC_NKMODULEMAX) {
	WARN(0, "PtxStore::%s(): PtxStore array FULL!\n");
	exit(1);
    }
    ptx_ptr->set(name0, ptx_image0);
    used_count++;
    return ptx_ptr;
}

PtxRecord*
PtxStore::query(char *name0) {
    PtxRecord *ptx_ptr;
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	ptx_ptr = &ptx_record[i];
	if ( strcmp(name0, ptx_ptr->name)==0 ) {/*found*/
	    WARN(9, "      +PtxStore::query(): Found ptx.\n")
	    return ptx_ptr;
	}
    }
    WARN(5, "      + PtxStore::query(): Not found ptx.\n");
    return NULL;
}

void
PtxStore::print(int n) {
    for (int i=0; i<n; i++) {
	WARN(1, "ptx_record[%d]: valid=%d, name=%s.\n",
	     i, ptx_record[i].valid, ptx_record[i].name); 
    }
}
ClientModule::ClientModule(void) {
    valid    = -1;
    id       = -1;
    ptx_data = NULL;
}
int
ClientModule::isValid(void) {
    if (valid<-1 || valid>1) {
	fprintf(stderr, "Unexpected error. %s:%d\n", __FILE__, __LINE__);
	exit(1);
    } else if (valid==1) {
	return 1;
    } else {
	return 0;
    }
}

int
ClientModule::isInvalid(void) {
    if (valid<-1 || valid>1) {
	fprintf(stderr, "Unexpected error. %s:%d\n", __FILE__, __LINE__);
	exit(1);
    } else if (valid==1) {
	return 0;
    } else {
	return 1;
    }
}

int
ServerArray::append(const char *ip, int ndev, const char *hname) {
    if ( num >= (RC_NVDEVMAX - 1) ) {
	WARN(0, "(+_+) Too many DS-CUDA daemons, exceeds RC_NVDEVMAX(=%d)\n",
	     RC_NVDEVMAX);
	exit(EXIT_FAILURE);
    }
    svr[num].setIP( ip );
    strcpy(svr[num].hostname, hname);
    svr[num].setID( ndev );
    svr[num].setCID( ndev );
    svr[num].uniq = RC_UNIQ_CANDBASE + num;
    num++;
    return 0;
}
int
ServerArray::append(PhyDev *svrptr) {
    if ( num >= (RC_NVDEVMAX - 1) ) {
	WARN(0, "(+_+) Too many DS-CUDA daemons, exceeds RC_NVDEVMAX(=%d)\n",
	     RC_NVDEVMAX);
	exit(EXIT_FAILURE);
    }
    svr[num].setIP( svrptr->ip );
    strcpy(svr[num].hostname, svrptr->hostname);
    svr[num].setID( svrptr->id );
    svr[num].setCID( svrptr->cid );
    svr[num].setUNIQ( RC_UNIQ_CANDBASE + num );
    svr[num].setFTMODE( svrptr->ft_mode );
    num++;
    return 0;
}
PhyDev*
ServerArray::findSpareOne(void) {
    PhyDev *sp = NULL;
    for (int i=0; i<num; i++) {
	if (svr[i].ft_health==hl_GOOD || svr[i].ft_health==hl_RECYCLED) {
	    sp = &svr[i];
	}
    }
    return sp;
}
PhyDev*
ServerArray::findBrokenOne(void) {
    PhyDev *sp = NULL;
    for (int i=0; i<num; i++) {
	if (svr[i].ft_health==hl_BAD) {
	    sp = &svr[i];
	}
    }
    return sp;
}
void
ServerArray::captureEnv(char *env_str, FThealth cond) {
    char buf[1024*RC_NVDEVMAX];
    char *svr_token;
    char svr_token_ar[RC_NVDEVMAX][256];

    {/* update buf[] */
	char *env = getenv(env_str);
	if (env == NULL) {
	    INFO0("[ Environment variable ] %s = (Not found)\n", env_str);
	    return;
	}
	if (sizeof(buf) < strlen(env)) {
	    WARN(0, "Too long length of DSCUDA_SERVER.\n");
	    exit(EXIT_FAILURE);
	}
	strncpy(buf, env, sizeof(buf));
    }

    //<--- svr_token_sr[x]="hostname:n"
    int  svr_count = 0;    
    svr_token = strtok(buf, " ");
    while (svr_token != NULL) {
	strcpy(svr_token_ar[svr_count], svr_token);
	svr_count++;
	if (svr_count > RC_NVDEVMAX) {
	    WARN(0, "number of devices exceeds the limit, RC_NVDEVMAX (=%d).\n",
		 RC_NVDEVMAX);
	    exit(EXIT_FAILURE);
	}
	svr_token = strtok(NULL, " ");
    }

    for (int i=0; i<svr_count; i++) {
	svr_token = strtok( svr_token_ar[i], ":" );
	this->svr[i].setIP( svr_token );
	svr_token = strtok( NULL, ":" );
	svr[i].setCID( svr_token );
	svr[i].setHealth(cond);
    }
    this->num = svr_count;
}//-->captureEnv()
void
ServerArray::print(void) {
    WARN(5, "ServerArray.num = %d\n", num);
    for (int i=0; i<num; i++) {
	WARN(1, "      + svrarr[%d].id= %d\n", i, svr[i].id);
	WARN(1, "      + svrarr[%d].cid= %d\n", i, svr[i].cid);
	WARN(1, "      + svrarr[%d].ip= %s\n", i, svr[i].ip);
	WARN(1, "      + svrarr[%d].hostname= %s\n", i, svr[i].hostname);
    }
}

void
FToption::infoD2H(void) {
    WARN(1, "d2h_simple  =%d\n", (d2h_simple)?   1:0 );
    WARN(1, "d2h_reduncpy=%d\n", (d2h_reduncpy)? 1:0 );
    WARN(1, "d2h_compare =%d\n", (d2h_compare)?  1:0 );
    WARN(1, "d2h_statics =%d\n", (d2h_statics)?  1:0 );
    WARN(1, "d2h_rollback=%d\n", (d2h_rollback)? 1:0 );
}

int
requestDaemonForDevice(char *ip, int devid, bool useibv) {
    int dsock; // socket for side-band communication with the daemon & server.
    int sport; // port number of the server. given by the daemon.
    char msg[256];
    sockaddr_in sockaddr;

    sockaddr = setupSockaddr( ip, RC_DAEMON_IP_PORT );
    dsock = socket(AF_INET, SOCK_STREAM, 0);
    if (dsock < 0) {
        perror("socket");
        exit(1);
    }
    
    if ( connect(dsock, (struct sockaddr *)&sockaddr, sizeof(sockaddr)) == -1 ) {
        perror("(;_;) connect(...)");
	WARN(0, "+--- Program terminated at %s:L%d\n", __FILE__, __LINE__ );
	WARN(0, "+--- Maybe DS-CUDA daemon is not running...\n" );
        //exit(1);
	return -1;
    }
    sprintf(msg, "deviceid:%d", devid);
    sendMsgBySocket(dsock, msg);
    WARN(1, "<--- Send message: \"%s\".\n", msg);

    memset(msg, 0, strlen(msg));
    recvMsgBySocket(dsock, msg, sizeof(msg));
    WARN(1, "---> Recv message: \"%s\".\n", msg);    
    sscanf(msg, "sport:%d", &sport);

    if (sport < 0) {
        WARN(0, "max possible ports on %s already in use.\n", ip);
        exit(1);
    }

    WARN(3, "server port: %d  daemon socket: %d\n", sport, dsock);

    if (useibv) {
        sprintf(msg, "remotecall:ibv");
    } else {
        sprintf(msg, "remotecall:rpc");
    }
    WARN(3, "send \"%s\" to the server.\n", msg);
    sendMsgBySocket(dsock, msg);

    WARN(2, "waiting for the server to be set up...\n");
    memset(msg, 0, strlen(msg));
    recvMsgBySocket(dsock, msg, sizeof(msg)); // wait for "ready" from the server.
    if (strncmp("ready", msg, strlen("ready"))) {
        WARN(0, "unexpected message (\"%s\") from the server. abort.\n", msg);
        exit(1);
    }
    return sport;
}

/*
 * Obtain a small integer unique for each thread.
 * The integer is used as an index to 'Vdevid[]'.
 */
int
vdevidIndex(void) {
    int i;
    pthread_t ptid = pthread_self();

    for (i=0; i<VdevidIndexMax; i++) {
        if (VdevidIndex2ptid[i] == ptid) {
            return i;
        }
    }

    pthread_mutex_lock(&VdevidMutex);
    i = VdevidIndexMax;
    VdevidIndex2ptid[i] = ptid;
    VdevidIndexMax++;
    pthread_mutex_unlock(&VdevidMutex);

    if (RC_NPTHREADMAX <= VdevidIndexMax) {
        fprintf(stderr, "vdevidIndex():device requests from too many (more than %d) pthreads.\n", RC_NPTHREADMAX);
        exit(1);
    }

    return i;
}

void
RCmappedMemRegister(void *pHost, void* pDevice, size_t size) {
    RCmappedMem *mem = (RCmappedMem *)malloc(sizeof(RCmappedMem));
    if (!mem) {
        perror("RCmappedMemRegister");
    }
    mem->pHost = pHost;
    mem->pDevice = pDevice;
    mem->size = size;
    mem->prev = RCmappedMemListTail;
    mem->next = NULL;
    if ( !RCmappedMemListTop ) { // mem will be the 1st entry.
        RCmappedMemListTop = mem;
    } else {
        RCmappedMemListTail->next = mem;
    }
    RCmappedMemListTail = mem;
}

RCmappedMem*
RCmappedMemQuery(void *pHost) {
    RCmappedMem *mem = RCmappedMemListTop;
    while (mem) {
        if (mem->pHost == pHost) {
            return mem;
        }
        mem = mem->next;
    }
    return NULL; // pHost not found in the list.
}

void
RCmappedMemUnregister(void *pHost) {
    RCmappedMem *mem = RCmappedMemQuery(pHost);
    if (!mem) return;

    if (mem->prev) { // reconnect the linked list.
        mem->prev->next = mem->next;
    } else { // mem was the 1st entry.
        RCmappedMemListTop = mem->next;
        if (mem->next) {
            mem->next->prev = NULL;
        }
    }
    if ( !mem->next ) { // mem was the last entry.
        RCmappedMemListTail = mem->prev;
    }
    free( mem );
}

//#if RC_SUPPORT_STREAM
/*
 * Register a stream array. each component is associated to a stream
 * on each Server[]. User see only the 1st element, streams[0].
 * Others, i.e., streams[1..Nredunddancy-1], are used by this library
 * to handle redundant calculation mechanism.
 */
static void
RCstreamArrayRegister(hipStream_t *streams) {
    RCstreamArray *st = (RCstreamArray *)malloc(sizeof(RCstreamArray));
    if (!st) {
        perror("RCstreamArrayRegister");
    }
    for (int i=0; i<RC_NREDUNDANCYMAX; i++) {
        st->s[i] = streams[i];
    }
    st->prev = RCstreamArrayListTail;
    st->next = NULL;
    if (!RCstreamArrayListTop) { // st will be the 1st entry.
        RCstreamArrayListTop = st;
    } else {
        RCstreamArrayListTail->next = st;
    }
    RCstreamArrayListTail = st;
}

#if 0
static void
showsta(void) {
    RCstreamArray *st = RCstreamArrayListTop;
    while (st) {
        fprintf(stderr, ">>> 0x%08llx    prev:%p  next:%p\n", st, st->prev, st->next);
        st = st->next;
    }
}
#endif

RCstreamArray*
RCstreamArrayQuery(hipStream_t stream0) {
    static RCstreamArray default_stream = { 0,};

    if (stream0 == 0) {
        return &default_stream;
    }

    RCstreamArray *st = RCstreamArrayListTop;
    while (st) {
        if (st->s[0] == stream0) {
            return st;
        }
        st = st->next;
    }
    return NULL;
}

static void
RCstreamArrayUnregister(hipStream_t stream0) {
    RCstreamArray *st = RCstreamArrayQuery(stream0);
    if (!st) return;

    if (st->prev) { // reconnect the linked list.
        st->prev->next = st->next;
    } else { // st was the 1st entry.
        RCstreamArrayListTop = st->next;
        if (st->next) {
            st->next->prev = NULL;
        }
    }
    if (!st->next) { // st was the last entry.
        RCstreamArrayListTail = st->prev;
    }
    free(st);
    //    showsta();
}
//#endif


/*
 * Register a hipArray array. each component is associated to a hipArray
 * on each Server[]. User see only the 1st element, cuarrays[0].
 * Others, i.e., cuarrays[1..Nredunddancy-1], are used by this library
 * to handle redundant calculation mechanism.
 */
void
RCcuarrayArrayRegister(hipArray **cuarrays) {
    RCcuarrayArray *ca = (RCcuarrayArray *)malloc(sizeof(RCcuarrayArray));
    if (!ca) {
        perror("RCcuarrayArrayRegister");
    }
    for (int i = 0; i < RC_NREDUNDANCYMAX; i++) {
        ca->ap[i] = cuarrays[i];
    }
    ca->prev = RCcuarrayArrayListTail;
    ca->next = NULL;
    if (!RCcuarrayArrayListTop) { // ca will be the 1st entry.
        RCcuarrayArrayListTop = ca;
    } else {
        RCcuarrayArrayListTail->next = ca;
    }
    RCcuarrayArrayListTail = ca;
}

RCcuarrayArray*
RCcuarrayArrayQuery(hipArray *cuarray0) {
    RCcuarrayArray *ca = RCcuarrayArrayListTop;
    while (ca) {
        if (ca->ap[0] == cuarray0) {
            return ca;
        }
        ca = ca->next;
    }
    return NULL;
}

void
RCcuarrayArrayUnregister(hipArray *cuarray0) {
    RCcuarrayArray *ca = RCcuarrayArrayQuery(cuarray0);
    if (!ca) return;

    if (ca->prev) { // reconnect the linked list.
        ca->prev->next = ca->next;
    } else { // ca was the 1st entry.
        RCcuarrayArrayListTop = ca->next;
        if (ca->next) {
            ca->next->prev = NULL;
        }
    }
    if (!ca->next) { // ca was the last entry.
        RCcuarrayArrayListTail = ca->prev;
    }
    free(ca);
}


/*
 * Register an event array. each component is associated to an event
 * on each Server[]. User see only the 1st element, events[0].
 * Others, i.e., events[1..Nredunddancy-1], are used by this library
 * to handle redundant calculation mechanism.
 */
void
RCeventArrayRegister(hipEvent_t *events) {
    RCeventArray *ev = (RCeventArray *)malloc(sizeof(RCeventArray));
    if (!ev) {
        perror("RCeventArrayRegister");
    }
    for (int i = 0; i < RC_NREDUNDANCYMAX; i++) {
        ev->e[i] = events[i];
    }
    ev->prev = RCeventArrayListTail;
    ev->next = NULL;
    if (!RCeventArrayListTop) { // ev will be the 1st entry.
        RCeventArrayListTop = ev;
    } else {
        RCeventArrayListTail->next = ev;
    }
    RCeventArrayListTail = ev;
}

RCeventArray*
RCeventArrayQuery(hipEvent_t event0) {
    RCeventArray *ev = RCeventArrayListTop;
    while (ev) {
        if (ev->e[0] == event0) {
            return ev;
        }
        ev = ev->next;
    }
    return NULL;
}

void
RCeventArrayUnregister(hipEvent_t event0) {
    RCeventArray *ev = RCeventArrayQuery(event0);
    if (!ev) return;

    if (ev->prev) { // reconnect the linked list.
        ev->prev->next = ev->next;
    } else { // ev was the 1st entry.
        RCeventArrayListTop = ev->next;
        if (ev->next) {
            ev->next->prev = NULL;
        }
    }
    if (!ev->next) { // ev was the last entry.
        RCeventArrayListTail = ev->prev;
    }
    free(ev);
}
/*
 * Compose UVA from GPU local address and its deviceID.
 */
void*
dscudaUvaOfAdr( void *adr, int devid ) {
    DscudaUva_t adri = (DscudaUva_t)adr;
#if __LP64__
    adri |= ((DscudaUva_t)devid << 48);
#endif
    return (void *)adri;
}
/*====================================================================
 * Get GPU deviceID from UVA.
 */
int
dscudaDevidOfUva( void *adr ) {
#if __LP64__
    DscudaUva_t adri = (DscudaUva_t)adr;
    int devid = adri >> 48;
    return devid;
#else
    return 0;
#endif
}
/*
 * Get GPU local address from UVA.
 */
void*
dscudaAdrOfUva( void *adr ) {
    DscudaUva_t adri = (DscudaUva_t)adr;
#if __LP64__
    adri &= 0x0000ffffffffffffLL;
#endif
    return (void *)adri;
}

static char*
readServerConf(char *fname) {
    FILE *fp = fopen(fname, "r");
    char linebuf[1024];
    int len;
    static char buf[1024*RC_NVDEVMAX];

    buf[0] = 0;
    if (!fp) {
        WARN(0, "cannot open file '%s'\n", fname);
        exit(1);
    }

    while (!feof(fp)) {
        char *s = fgets(linebuf, sizeof(linebuf), fp);
        if (!s) break;
        len = strlen(linebuf);
        if (linebuf[len-1] == '\n') {
            linebuf[len-1] = 0;
        }
        if (sizeof(buf) < strlen(buf) + len) {
            WARN(0, "readServerConf:file %s too long.\n", fname);
            exit(1);
        }
        strncat(buf, linebuf, sizeof(linebuf));
        strcat(buf, " ");
    }
    fclose(fp);
    return buf;
}

/*
 *
 */
void
printVirtualDeviceList( void ) {
    VirDev     *pVdev;
    PhyDev   *pSvr;
    int         i,j;
    
    INFO0("###***   <---Start Virtual Device Construction. (Total %d device%s)\n", St.Nvdev, (St.Nvdev>1)? "s":"" );
    for (i=0, pVdev=St.Vdev; i<St.Nvdev; i++, pVdev++) {
	if (i >= RC_NVDEVMAX) {
	    INFO0("(;_;) Too many virtual devices. %s().\nexit.", __func__);
	    exit(EXIT_FAILURE);
	}
	if (pVdev->nredundancy == 1) {
	    INFO0("    Virt[%d] (MONO)\n", i );
	} else if ( pVdev->nredundancy > 1 ) {
	    INFO0("    Virt[%d] (POLY:%d)\n", i, pVdev->nredundancy );
	} else {
	    INFO0("    Virt[%d] (????:%d)\n", i, pVdev->nredundancy );
	}
	
	for (j=0, pSvr=pVdev->server; j<pVdev->nredundancy; j++, pSvr++) {
	    if (j >= RC_NREDUNDANCYMAX) {
		WARN(0, "(;_;) Too many redundant devices %d. %s().\nexit.\n", __func__);
		exit( EXIT_FAILURE );
	    }
	    INFO0("    +  Phy[%d]: id=%d, cid=%d, IP=%s(%s), uniq=%d.\n", j,
		   pSvr->id, pSvr->cid, pSvr->ip, pSvr->hostname, pSvr->uniq);
	}
    }
    INFO0("###***   --->Stop Virtual Device Construction.\n\n");

    if (St.ft_mode==FT_BYCPY || St.ft_mode==FT_BYTIMER) {
	/*
	 * Device Candidates
	 */
	INFO0("###*** Physical Device Candidates. (Ncand=%d)\n", SvrCand.num );
	for( i=0, pSvr=SvrCand.svr; i < SvrCand.num; i++, pSvr++ ){
	    if (i >= RC_NVDEVMAX) {
		WARN(0, "(;_;) Too many candidate devices. %s().\nexit.", __func__);
		exit( EXIT_FAILURE );
	    }
	    INFO0("###***    - Cand[%2d]: id=%d, cid=%d, IP=%s, uniq=%d.\n", i,
		  pSvr->id, pSvr->cid, pSvr->ip, pSvr->uniq);
	}
	/*
	 * Alternate Devices
	 */
	INFO0("###*** Spare Server Info.(Nspare=%d)\n", SvrSpare.num);
	for( i=0, pSvr=SvrSpare.svr; i < SvrSpare.num; i++, pSvr++ ){
	    if (i >= RC_NVDEVMAX) {
		WARN(0, "(;_;) Too many spare devices. %s().\nexit.", __func__);
		exit( EXIT_FAILURE );
	    }
	    INFO0("###***    - Spare[%d]: id=%d, cid=%d, IP=%s, uniq=%d.\n", i,
		  pSvr->id, pSvr->cid, pSvr->ip, pSvr->uniq);
	}
    }
}

void
VirDev::setConfInfo(int redun) {
    nredundancy = redun; //Update Vdev.nredundancy.
    if (redun == 1) {
	conf = VDEV_MONO;
	sprintf(info, "MONO");
    } else if (redun > 1) {
	conf = VDEV_POLY;
	sprintf(info, "POLY%d", redun);
    } else {
	WARN(0, "Detect invalid nredundancy = %d.\n", redun);
	exit(EXIT_FAILURE);
    }
}

void
VirDev::printModuleList(void) {
    const int len = 256;
    char printbuf[len];
    int valid_cnt = 0;
    
    WARN(5, "====================================================\n");
    WARN(5, "===  VirDev::%s(void)\n", __func__ );
    WARN(5, "====================================================\n");
    WARN(5, "RC_NKMODULEMAX= %d\n", RC_NKMODULEMAX);
    
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	if( modulelist[i].valid==1 || modulelist[i].valid==0 ) {
	    WARN( 5, "Virtual[%d]:modulelist[%d]:\n", id, i);
	    WARN( 5, "    + name= %s\n", modulelist[i].ptx_data->name);
	    //printf("    + send_time= \n", sent_time., sent_time.);
	    //strncpy(printbuf, modulelist[i].ptx_data->ptx_image, len - 1 );
	    //printbuf[255]='\0';
	    //printf("# %s():    + ptx_image=\n%s\n", __func__, printbuf);
	    valid_cnt++;
	}
    }
    WARN(5, "%d valid modules registered.\n",  valid_cnt);
    WARN(5, "====================================================\n");
    
}

void
printModuleList(void) {
    for (int i=0; i<St.Nvdev; i++) {
	St.Vdev[i].printModuleList();
    }
}
uint32_t
dscuda::calcChecksum(void *sta, size_t size_byte) {
    uint32_t *p = (uint32_t *)sta;
    uint32_t  s           = 0;
    uint32_t  s_remain    = 0; // zero padding, ignore sign bit.
    size_t    sum_count   = size_byte / sizeof(uint32_t);
    size_t    size_remain = size_byte - (sum_count * sizeof(uint32_t));
    for (int i=0; i<sum_count; i++) {
	s += *p;
	p++;
    }
    if (size_remain >= sizeof(s_remain)) {
	fprintf(stderr, "Unexpected ERROR: %s()\n", __func__);
	exit(1);
    }
    memcpy( &s_remain, p, size_remain );
    s += s_remain;
    return s;
}
int
dscuda::searchDaemon(void) {
    int sendsock;
    int recvsock;

    char sendbuf[SEARCH_BUFLEN_TX];
    char recvbuf[SEARCH_BUFLEN_RX];
    
    int recvlen;
    int num_daemon = 0;
    int num_device = 0;
    int num_ignore = 0;

    unsigned int adr, mask;
    socklen_t    sin_size;
    int          setsockopt_ret;

    sockaddr_in addr, svr;
    struct ifreq ifr[2];
    struct ifconf ifc;
    struct passwd *pwd;

    INFO0("[ Constant             ] RC_DAEMON_IP_PORT = %d\n", RC_DAEMON_IP_PORT);
    sendsock = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
    recvsock = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
    if ( sendsock == -1 || recvsock == -1 ) {
	perror("searchDaemon: socket()");
	exit(1);
    }
    {
	int val = 1;
	setsockopt_ret = setsockopt(sendsock, SOL_SOCKET, SO_BROADCAST, &val, sizeof(val));
	if (setsockopt_ret != 0) {
	    perror("searchDaemon: setsockopt()");
	    exit(1);
	}
    }
    ifc.ifc_len = sizeof(ifr) * 2;
    ifc.ifc_ifcu.ifcu_buf = (char *)ifr;
    ioctl(sendsock, SIOCGIFCONF, &ifc);

    ifr[1].ifr_addr.sa_family = AF_INET;
    ioctl(sendsock, SIOCGIFADDR, &ifr[1]);
    adr = ((sockaddr_in *)(&ifr[1].ifr_addr))->sin_addr.s_addr;
    ioctl(sendsock, SIOCGIFNETMASK, &ifr[1]);
    mask = ((sockaddr_in *)(&ifr[1].ifr_netmask))->sin_addr.s_addr;

    addr.sin_family      = AF_INET;
    addr.sin_port        = htons(RC_DAEMON_IP_PORT - 1);
    addr.sin_addr.s_addr = adr | ~mask;

    strncpy( sendbuf, SEARCH_PING, SEARCH_BUFLEN_TX - 1 );
    sendto( sendsock, sendbuf, SEARCH_BUFLEN_TX, 0, (sockaddr *)&addr, sizeof(addr));
    INFO0("Broadcast \"%s\" message\n", SEARCH_PING);
    sin_size = sizeof(sockaddr_in);

    svr.sin_family      = AF_INET;
    svr.sin_port        = htons(RC_DAEMON_IP_PORT - 2);
    svr.sin_addr.s_addr = htonl(INADDR_ANY);
    
    // Set timeout for recvsock.
    {
	timeval tout;
	tout.tv_sec  = RC_SEARCH_DAEMON_TIMEOUT ;
	tout.tv_usec = 0;
	setsockopt_ret = setsockopt(recvsock, SOL_SOCKET, SO_RCVTIMEO, (char *)&tout, sizeof(tout));
	if (setsockopt_ret != 0) {
	    perror("searchDaemon: setsockopt(recvsock)");
	    exit(1);
	}
    }

    {
	int bind_ret = bind( recvsock, (struct sockaddr *)&svr, sizeof(svr) );
	if( bind_ret != 0 ) {
	    fprintf(stderr, "Error: bind() returned %d. recvsock=%d, port=%d\n",
		    bind_ret, recvsock, svr.sin_port); //port:38655
	    perror("searchDaemon: bind()");
	    return -1;
	}
    }
    
    pwd = getpwuid( getuid() );

    /* Recieve ack message from dscudad running at other host. */
    char *magic_word;
    char *user_name;
    char *host_name;
    char *dev_count;
    char  ipaddr[32];
    int   num_eachdev;

    SvrCand.num = 0;

    memset( recvbuf, 0, SEARCH_BUFLEN_RX );
    while(( recvlen = recvfrom( recvsock, recvbuf, SEARCH_BUFLEN_RX - 1, 0, (struct sockaddr *)&svr, &sin_size)) > 0) {
	INFO0(" + Detect ACK \"%s\" ", recvbuf);
	/*
	 * Analyze message.
	 */
	magic_word = strtok( recvbuf, SEARCH_DELIM );
	user_name  = strtok( NULL,    SEARCH_DELIM );
	host_name  = strtok( NULL,    SEARCH_DELIM );
	dev_count  = strtok( NULL,    SEARCH_DELIM ); // Ndev=4
	sscanf( dev_count, "Ndev=%d", &num_eachdev );
	sprintf( ipaddr, "%s", inet_ntoa( svr.sin_addr )); //192.168.1.1
	if ( magic_word == NULL ) {
	    WARN(0, "\n\n###(ERROR) Unexpected token in %s().\n\n", __func__);
	    exit(1);
	} else {
	    INFO0("from \"%s\" ", ipaddr );
	    if ( strcmp( magic_word, SEARCH_ACK   )==0 &&
		 strcmp( user_name,  pwd->pw_name )==0 ) { /* Found */
		INFO0("valid.\n");
		/*
		 * Updata SvrCand;
		 */
		for (int d=0; d<num_eachdev; d++) {
		    SvrCand.append(ipaddr, d, host_name);
		}
		num_daemon += 1;
		num_device += num_eachdev;
	    } else {
		INFO0("ignored.\n");
		num_ignore++;
	    }
	}
	memset( recvbuf, 0, SEARCH_BUFLEN_RX );
    }

    {
	int close_ret = close( sendsock );
	if ( close_ret != 0 ) {
	    WARN(0, "close(sendsock) failed.\n");
	    exit(EXIT_FAILURE);
	}
	
	close_ret = close( recvsock );
	if ( close_ret != 0 ) {
	    WARN(0, "close(recvsock) failed.\n");
	    exit(EXIT_FAILURE);
	}
    }

    if (num_daemon > 0) {
	INFO0("Found %d valid DSCUDA daemon%s. (%d ignored).\n",
	      num_daemon, (num_daemon>1)? "s":"", num_ignore );
    }
    else if (num_daemon==0) {
	//
	// Even if no daemons found, the servers defined in DSCUDA_SERVER are available.
	//
	WARN( 0, "%s(): Not found DS-CUDA daemon in this network.\n", __func__ );
	WARN( 0, "%s(): And Trying to continue execution.\n", __func__ );
    }
    else { 	/* Terminate program and exit. */
	WARN( 0, "%s(): Detected unexpected trouble; num_daemon=%d?\n", __func__, num_daemon );
	exit(EXIT_FAILURE);
    }
    return num_daemon;
} //---> int dscuda::searchDaemon(void)
//
//
//
#if 0
void
ServerArray::removeArray(ServerArray *sub) {

    for (int i=0; i<num; i++) {
	for (int k=0; k<sub->num; k++) {
	}
    }
}
#endif
static void
updateSpareServerList(void) {
    int         spare_count = 0;;
    VirDev     *pVdev;
    PhyDev *pSvr;

    for (int i=0; i<SvrCand.num; i++) {    // Sweep all Vdev.server[] and compare.
	int found = 0;
	pVdev = St.Vdev;
	for (int j=0; j<St.Nvdev; j++) {
	    pSvr = pVdev->server;
	    for (int k=0; k < pVdev->nredundancy; k++) {
		if ( strcmp( SvrCand.svr[i].ip,  pSvr->ip  )==0 &&
		     SvrCand.svr[i].cid==pSvr->cid ) { /* check same IP */
		    found=1;
		}
		pSvr++;
	    }
	    pVdev++;
	}
	if (found==0) { /* not found */
	    SvrSpare.svr[spare_count].id   = SvrCand.svr[i].id;
	    SvrSpare.svr[spare_count].cid  = SvrCand.svr[i].cid;
	    SvrSpare.svr[spare_count].uniq = SvrCand.svr[i].uniq;
	    SvrSpare.svr[spare_count].ft_health = hl_GOOD;
	    strcpy(SvrSpare.svr[spare_count].ip, SvrCand.svr[i].ip);
	    spare_count++;
	}
    }
    SvrSpare.num = spare_count;
}

static void
getenvDSCUDA_WARNLEVEL(void) {
    char *env = getenv("DSCUDA_WARNLEVEL");
    int val;
    if ( env ) {
        val = atoi(strtok(env, " "));
        if ( val >= 0 ) {
	    dscuda::setWarnLevel( val );
	} else {
	    WARN(0, "(;_;) Invalid DSCUDA_WARNLEVEL(%d), set 0 or positive integer.\n", val);
	    exit(EXIT_FAILURE);
	}
    } else {
	dscuda::setWarnLevel(RC_WARNLEVEL_DEFAULT);
    }

}
static void
extractENV(bool &bool_var, const char *envname) {
    char *env = getenv(envname);
    if (env==NULL) {
	bool_var = false;
    } else {
	int val = atoi(env);
	if (val==0) {
	    bool_var = false;
	} else {
	    bool_var = true;
	}
    }
}
static void
extractENV(int &int_var, const char *envname, int undef) {
    char *env = getenv(envname);
    if (env==NULL) {
	int_var = undef;
    } else {
	int_var = atoi(env);
    }
}
static void
extractENV(char *str_var, const char *envname, int len) {
    char *env = getenv(envname);
    if (env==NULL) {
	strncpy(str_var, "(undef)", len);  
    } else {
	strncpy(str_var, env, len);  
    }
}
/****** CHECK-POINTING THREAD ****************************************
 * Take the data backups of each virtualized GPU to client's host
 * memory after verifying between redundant physical GPUs every
 * specified wall clock
 * time period. The period is defined in second.
 */

void*
periodicCheckpoint(void *arg) {
    int cp_period = *(int *)arg;
    int cp_trim = 3;
    int trim_grid_usec = 100000; //100msec
    int correct_count = 0;
    int faulted_count = 0;
    int cp_count = 1;
    int Tc_reset_req0 = 0;
    //<-- timer
    double Tc_exp,Tc_exp_l=(double)cp_period*0.7, Tc_exp_h=(double)cp_period*1.3;
    int    Tc_exp_sec;
    double Tc_exp_usec;
    double Ta, Ta_sum=0.0, Ta_sta, Ta_avr=-1.0, Ta_min=1.0e6, Ta_max=0.0; //all
    double Tm, Tm_sum=0.0, Tm_sta, Tm_avr=0.0,  Tm_min=1.0e6, Tm_max=0.0; //mutexlock
    double Ts, Ts_sum=0.0, Ts_sta, Ts_avr=-1.0, Ts_min=1.0e6, Ts_max=0.0; //store
    double Tc, Tc_sum=0.0, Tc_sta, Tc_avr=-1.0, Tc_min=1.0e6, Tc_max=0.0; //cp sleep
    double Tr, Tr_sum=0.0, Tr_sta, Tr_avr=-1.0, Tr_min=1.0e6, Tr_max=0.0; //restre mem
    double Tx, Tx_sum=0.0, Tx_sta, Tx_avr=-1.0, Tx_min=1.0e6, Tx_max=0.0; //redo api
    //
    double Td2h, Td2h_partial, Td2h_sum=0.0, Td2h_sta, Td2h_avr=-1.0, Td2h_min=1.0e6,
	Td2h_max=0.0; //Tr+Tx in D2h
    //--> timer 
    /*
      Memo: "Tc" is defined by "int cp_period" above.
            "Ts" is not able to defined.
      |<----  Ta  ----->| 
      |<--  Tc  -->| Ts |
      +------------+====+
      |         |Tm|
     */
    while (cp_thread_exit==0) {
	dscuda::stopwatch(&Ta_sta);
	dscuda::stopwatch(&Tc_sta);
	Td2h = 0.0;
	//<-- Wait for specified period (sec) passed.
	Tc_exp = ((double)cp_period * (double)cp_count) - Tc_sum - Tm_avr; //in sec
	if      (Tc_exp < Tc_exp_l) Tc_exp = Tc_exp_l; // saturate to lower bound time.
	else if (Tc_exp > Tc_exp_h) Tc_exp = Tc_exp_h; //             upper bound time.
	Tc_exp_sec  = (int)floor(Tc_exp);
	Tc_exp_usec = (Tc_exp - Tc_exp_sec)*1e6;
	for (int i=0; i<Tc_exp_sec; i++) {
	    for (int j=0; j<10; j++) { // 1.000s = 100ms * 10;
		//<-- mutex lock
		dscuda::stopwatch(&Td2h_sta);
		do {
		    pthread_mutex_lock( &Tc_reset_mutex );
		    Tc_reset_req0 = Tc_reset_req;
		    pthread_mutex_unlock( &Tc_reset_mutex );
		} while (Tc_reset_req0 == 1);
		Td2h_partial = dscuda::stopwatch(&Td2h_sta);
		Td2h += Td2h_partial;
		//--> mutex lock
		usleep( 100000 ); // =100ms
	    }
	}
	//<-- mutex lock
	dscuda::stopwatch(&Td2h_sta);
	do {
	    pthread_mutex_lock( &Tc_reset_mutex );
	    Tc_reset_req0 = Tc_reset_req;
	    pthread_mutex_unlock( &Tc_reset_mutex );
	} while (Tc_reset_req0 == 1);
	Td2h_partial = dscuda::stopwatch(&Td2h_sta);
	Td2h += Td2h_partial; // Fix the value of Td2h.
	//--> mutex lock
	usleep( (int)Tc_exp_usec );
	
	//--> Wait for specified period (sec) passed.

	dscuda::stopwatch(&Tm_sta);
	//<-- mutex locks for avoiding R/W collisions
	pthread_mutex_lock( &cudaMemcpyD2H_mutex );
	pthread_mutex_lock( &cudaMemcpyH2D_mutex );
	pthread_mutex_lock( &cudaKernelRun_mutex );
	pthread_mutex_lock( &cudaElse_mutex );
	//--> mutex locks for avoiding R/W collisions

	//<-- flush all cuda stream
	for (int i=0; i<St.Nvdev; i++) {
	    St.Vdev[i].hipDeviceSynchronize();
	}
	//--> flush all cuda stream
		    
	//*****
	//***** <-- "Ts" starts here.
	//*****
	Tm = dscuda::stopwatch(&Tm_sta, &Tm_min, &Tm_max);
	Tm_sum += Tm;
	Tm_avr =  Tm_sum / (double)cp_count;

	//Tc = dscuda::stopwatch(&Tc_sta, &Tc_min, &Tc_max);
	Tc = dscuda::stopwatch(&Tc_sta) - Td2h;
	if (Tc < Tc_min) Tc_min = Tc;
	if (Tc > Tc_max) Tc_max = Tc;
	Tc_sum += Tc;
	Tc_avr =  Tc_sum / (double)cp_count;

	//Td2h
	if (Td2h < Td2h_min) Td2h_min = Td2h;
	if (Td2h > Td2h_max) Td2h_max = Td2h;
	Td2h_sum += Td2h;
	Td2h_avr = Td2h_sum / (double)cp_count;
	
	dscuda::stopwatch(&Ts_sta);	
	//<-- Output beginning message.
	WARN_CP(0,"==================================================== #%d begin\n", cp_count);
	WARN_CP(0,"periodicCheckpoint( period = %d sec )\n", cp_period );
	//--> Output beginning message.

	//<-- copy from all hipMalloc() regions of all devices.
	St.collectEntireRegions();
	//--> copy from all hipMalloc() regions of all devices.

	bool correct = St.verifyEntireRegions();
#if 0 // force pseudo error
	if (correct_count % 5 == 4) {
	    correct = false;
	}
#endif
	if (correct) {
	    correct_count++;
	    //***
	    //*** All memory regions on all virtual devices are correct.
	    //*** Then, collect clean device memory regions to host memory.
	    //*** and clear CUDA API called history.
	    //***
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].updateMemlist();
	    }
	    WARN_CP(0, "(^_^)Update clean backup region, age=%d\n",
		    St.Vdev[0].memlist.getAge());
	    
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].clearReclist();
	    }
	}
	//*****
	//***** --> "Ts" completes here.
	//*****
	Ts = dscuda::stopwatch(&Ts_sta, &Ts_min, &Ts_max);
	Ts_sum += Ts;
	Ts_avr =  Ts_sum / (double)cp_count;
	
	if (!correct) {
	    faulted_count++;
	    //***
	    //*** Some memory regions on any virtual devices are currupted.
	    //*** Then, restore clean memory regions to all devices, and
	    //*** redo the historical cuda API calls.
	    //***
	    dscuda::stopwatch(&Tr_sta);
	    WARN_CP(0,"(+_+) Detect corrupted region.\n");
	    WARN_CP(0,"%8.3f sec from start. nth=%d\n",
		    Tr_sta - (double)St.start_time, faulted_count);
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].restoreMemlist();
	    }
	    Tr = dscuda::stopwatch(&Tr_sta, &Tr_min, &Tr_max);
	    Tr_sum += Tr;
	    Tr_avr =  Tr_sum / (double)faulted_count;

	    WARN_CP(0, "(._.)Completed restoring the device memory previous backup ");
	    WARN_CP0(0, "age=%d\n", St.Vdev[0].memlist.getAge());
		    
	    WARN_CP(0, "(+_+)Rollback the CUDA APIs by CP.\n");
	    dscuda::stopwatch(&Tx_sta);
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].reclist.print();
		St.Vdev[i].recordOFF();
		WARN_CP(1, "        VirDev[%d]\n", St.Vdev[i].id);
		St.Vdev[i].reclist.recall();
		St.Vdev[i].recordON();
	    }
	    //<-- flush all cuda stream
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].hipDeviceSynchronize();
	    }
	    WARN_CP(0, "Synchronize() Rollbacked CUDA APIs.\n");
	    //--> flush all cuda stream
	    Tx = dscuda::stopwatch(&Tx_sta, &Tx_min, &Tx_max);
	    Tx_sum += Tx;
	    Tx_avr =  Tx_sum / (double)faulted_count;
	}


	Ta = dscuda::stopwatch(&Ta_sta, &Ta_min, &Ta_max);
	Ta_sum  += Ta;
	Ta_avr  =  Ta_sum / (double)cp_count;
	
	//<-- Output ending message.	
	WARN_CP(0,"} elapsed time report #%d (sec)\n", cp_count);
	WARN_CP(0," 'Name' = 'now' { 'min' , 'avr' , 'max' } 'sum'\n");
	WARN_CP(0," Tm = %8.3f { %8.3f , %8.3f , %8.3f } %8.3f\n",
		Tm, Tm_min, Tm_avr, Tm_max, Tm_sum);
	WARN_CP(0," Tc = %8.3f { %8.3f , %8.3f , %8.3f } %8.3f\n",
		Tc, Tc_min, Tc_avr, Tc_max, Tc_sum);
	WARN_CP(0," Ts = %8.3f { %8.3f , %8.3f , %8.3f } %8.3f\n",
		Ts, Ts_min, Ts_avr, Ts_max, Ts_sum);
	WARN_CP(0," Ta = %8.3f { %8.3f , %8.3f , %8.3f } %8.3f\n",
		Ta, Ta_min, Ta_avr, Ta_max, Ta_sum);
	WARN_CP(0," Td2h = %8.3f { %8.3f , %8.3f , %8.3f } %8.3f\n",
		Td2h, Td2h_min, Td2h_avr, Td2h_max, Td2h_sum);
		
	if (faulted_count>0) {
	    WARN_CP(0," *Tr= %8.3f { %8.3f , %8.3f , %8.3f } %8.3f (%d)\n",
		    Tr, Tr_min, Tr_avr, Tr_max, Tr_sum, faulted_count);
	    WARN_CP(0," *Tx= %8.3f { %8.3f , %8.3f , %8.3f } %8.3f (%d)\n",
		    Tx, Tx_min, Tx_avr, Tx_max, Tx_sum, faulted_count);
	}
	else {
	    WARN_CP(0," *Tr= - { - , - , - } %8.3f (%d)\n", Tr_sum, faulted_count);
	    WARN_CP(0," *Tx= - { - , - , - } %8.3f (%d)\n", Tx_sum, faulted_count);
	}
	//<-- flush all cuda stream
	for (int i=0; i<St.Nvdev; i++) {
	    WARN_CP(0," Vdev[%d].ft_unmatch_count= %d\n", i, St.Vdev[i].ft_unmatch_total);
	}
	//--> flush all cuda stream
	pthread_testcancel();/* thread cancelation point */
	WARN_CP(0,"==================================================== #%d end\n", cp_count);
	//--> Output ending message.
	cp_count++;
	//<-- mutex unlocks for following R/W.
	pthread_mutex_unlock( &cudaMemcpyD2H_mutex );
	pthread_mutex_unlock( &cudaMemcpyH2D_mutex );
	pthread_mutex_unlock( &cudaKernelRun_mutex );
	pthread_mutex_unlock( &cudaElse_mutex );
	//--> mutex unlocks for following R/W.
    }//while(true)
    
    WARN_CP0(0,"periodicCheckpoint() thread completed.\n");
    WARN_CP0(0,"====================================================\n");
    WARN_CP0(0,"= Summary\n");
    WARN_CP0(0,"= Total Checkpointed count = %d times.\n", cp_count-1);
    WARN_CP0(0,"=       Correct      count = %d times.\n", correct_count);
    WARN_CP0(0,"=       Fault        count = %d times.\n", faulted_count);
    WARN_CP0(0,"=    : { 'min' , 'avr' , 'max' } 'sum' [sec]\n");
    WARN_CP0(0,"=  Tm: { %8.3f , %8.3f , %8.3f } %8.3f\n", Tm_min, Tm_avr, Tm_max, Tm_sum);
    WARN_CP0(0,"=  Tc: { %8.3f , %8.3f , %8.3f } %8.3f\n", Tc_min, Tc_avr, Tc_max, Tc_sum);
    WARN_CP0(0,"=  Ts: { %8.3f , %8.3f , %8.3f } %8.3f\n", Ts_min, Ts_avr, Ts_max, Ts_sum);
    WARN_CP0(0,"=  Ta: { %8.3f , %8.3f , %8.3f } %8.3f\n", Ta_min, Ta_avr, Ta_max, Ta_sum);
    if (faulted_count > 0) {
	WARN_CP0(0,"=  *Tr:{ %8.3f , %8.3f , %8.3f } %8.3f\n", Tr_min, Tr_avr, Tr_max, Tr_sum);
	WARN_CP0(0,"=  *Tx:{ %8.3f , %8.3f , %8.3f } %8.3f\n", Tx_min, Tx_avr, Tx_max, Tx_sum);
    }
    else {
	WARN_CP0(0,"=  *Tr= - { - , - , - } %8.3f (%d)\n", Tr_sum, faulted_count);
	WARN_CP0(0,"=  *Tx= - { - , - , - } %8.3f (%d)\n", Tx_sum, faulted_count);
    }
    WARN_CP0(0, "====================================================\n");
    sleep(1);
    return NULL;
} // periodicCheckpoint()

void
VirDev::invalidateAllModuleCache(void) {
    for (int i=0; i<RC_NKMODULEMAX; i++) {
        if( modulelist[i].isValid() ){
	    modulelist[i].invalidate();
	} else { 
	    continue;
	}
    }
}

/*
 * public functions
 */
int
dscudaNredundancy(void) {
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    return vdev->nredundancy;
}

void
dscudaSetErrorHandler(void (*handler)(void *), void *handler_arg) {
    errorHandler = handler;
    errorHandlerArg = handler_arg;
}

/*
 * Obtain a mangled symbol name of a function, whose
 * interface is given by 'funcif' and is defined somewhere in 'ptxdata'.
 * The obtained symbol name is returned to 'name'.
 *
 * eg) funcif  : void dscudavecAdd(dim3, dim3, size_t, ihipStream_t*, float*, float*, float*)
 *     ptxdata : .version 1.4
 *               .target sm_10, map_f64_to_f32
 *               ...
 *               .entry _Z6vecAddPfS_S_ (
 *               ...
 *               } // _Z6vecMulPfS_fS_iPi
 */
void
dscudaGetMangledFunctionName(char *name, const char *funcif, const char *ptxdata)
{
    WARN(10, "<---Entering %s()\n", __func__);
    static char mangler[256] = {0, };
    char cmd[4096];
    FILE *outpipe;
    FILE *tmpfp;
    char ptxfile[1024];

    WARN(4, "getMangledFunctionName(%p, %p, %p)  funcif:\"%s\"\n",
         name, funcif, ptxdata, funcif);

    // create a tmporary file that contains 'ptxdata'.
    system("/bin/mkdir /tmp/dscuda 1> /dev/null  2> /dev/null");
    // do not use >& since /bin/sh on some distro does not recognize it.

    sprintf(ptxfile, "/tmp/dscuda/mgl%d", getpid());
    tmpfp = fopen(ptxfile, "w");
    fprintf(tmpfp, "%s", ptxdata);
    fclose(tmpfp);

    // exec 'ptx2symbol' to obtain the mangled name.
    // command output is stored to name.
    if (!mangler[0]) {
        sprintf(mangler, "%s/bin/ptx2symbol", St.dscuda_path);
    }
    sprintf(cmd, "%s %s << EOF\n%s\nEOF", mangler, ptxfile, funcif);
    outpipe = popen(cmd, "r");
    if (!outpipe) {
        perror("getMangledFunctionName()");
        exit(1);
    }
    fgets(name, 256, outpipe);
    pclose(outpipe);
    if (!strlen(name)) {
        WARN(0, "getMangledFunctionName() : %s returned an error. "
             "it could not found any entry, or found multiple candidates. "
             "set DSCUDA_WARNLEVEL 4 or higher and try again to see "
             "error messages from %s.\n", mangler, mangler);
        exit(1);
    }
    WARN(10, "--->Exiting %s()\n", __func__);
}


static pthread_mutex_t LoadModuleMutex = PTHREAD_MUTEX_INITIALIZER;
/*
 * Load a cuda module from a .ptx file, and then, send it to the server.
 * returns id for the module.
 * the module is cached and sent only once for a certain period.
 */

int
dscudaLoadModule(char *name, char *strdata) { // 'strdata' must be NULL terminated.
    WARN(5, "dscudaLoadModule( name=%p(%s), strdata=%p ) {\n", name, name, strdata);
    int idx = vdevidIndex();
    VirDev *vdev = St.Vdev + Vdevid[idx];
    int module_index;

    module_index = vdev->loadModule(name, strdata);
    
    //printModuleList();
    WARN(5, "} //dscudaLoadModule() returned %d.\n", module_index);
    WARN(5, "\n");
    return module_index;
}

#if 0 // backup
int*
dscudaLoadModule(char *name, char *strdata) {// 'strdata' must be NULL terminated.
    int i, j, mid;
    ClientModule *mp;
    int idx;

    if (name != NULL) {
	WARN(5, "dscudaLoadModule(%p) modulename:%s  ...\n", name, name);
#if RC_CACHE_MODULE
	// look for modulename in the module list.
	for (i=0, mp=CltModulelist; i < RC_NKMODULEMAX; i++, mp++) {
	    if ( mp->isInvalid() ) {
		continue;
	    }
	    
	    idx = vdevidIndex();
	    if (mp->vdevid != Vdevid[idx]) {
		continue;
	    }
	    if ( !strcmp(name, mp->name) ) {
		if ( mp->isAlive() ) {
		    WARN(5, "done. found a cached one. id:%d  age:%d  name:%s\n",
			 mp->id[i], time(NULL) - mp->sent_time, mp->name);
		    return mp->id; // module found. i.e, it's already loaded.
		} else {
		    WARN(5, "found a cached one with id:%d, but it is too old (age:%d). resend it.\n",
			 mp->id[i], time(NULL) - mp->sent_time);
		    mp->invalidate(); // invalidate the cache.
		}
	    }
	} //for
#endif // RC_CACHE_MODULE
    } else {
	WARN(5, "dscudaLoadModule(%p) modulename:-\n", name);
    }

    //<---
    char *strdata_found = NULL;
    char *name_found=NULL;
    if (name==NULL && strdata==NULL) {
        for (i=0, mp=CltModulelist; i<RC_NKMODULEMAX; i++, mp++) {
	    WARN(10, "i=%d\n", i);
	    if (mp->isInvalid()) continue;
	    idx = vdevidIndex();
	    if (mp->vdevid != Vdevid[idx]) continue;
	    if (!strcmp(name, mp->name)) {     /* matched */
		strdata_found = mp->ptx_image;
		name_found = mp->name;
		break;
	    }
	}
    } else {
	strdata_found = strdata;
	name_found = name;
    }
    //--->

    // module not found in the module list.
    // really need to send it to the server.
    idx = vdevidIndex();
    VirDev *vdev = St.Vdev + Vdevid[idx];
    PhyDev *sp = vdev->server;

    for (i=0; i < vdev->nredundancy; i++) {
	// mid = dscudaLoadModuleLocal(St.getIpAddress(), getpid(), name, strdata, Vdevid[vi], i);
	// mid = dscudaLoadModuleLocal(St.getIpAddress(), getpid(), name_found, strdata_found, Vdevid[idx], i);
	mid = sp[i].loadModule(St.getIpAddress(), getpid(), name_found, strdata_found);
        WARN(3, "(info) dscudaLoadModuleLocal() returns mid=%d as Vdevid[%d], Redun[%d].\n", mid, idx, i);

        // register a new module into the list,
        // and then, return a module id assigned by the server.
        if (i==0) {
            for (j=0; j<RC_NKMODULEMAX; j++) { /* Search vacant sheet. */
                if( CltModulelist[j].isInvalid() ) break;
            }
	    if( j >= RC_NKMODULEMAX ) {
		WARN(0, "\n\n### (+_+) ERROR in DS-CUDA!\n");
		WARN(0,     "### (+_+) module send buffer is full. and exit.\n");
		WARN(0,     "### (+_+) Check if the array length of CltModulelist[%d].\n\n\n", j);
		exit(1);
	    }
            CltModulelist[j].validate();
            CltModulelist[j].sent_time = time(NULL);
//            CltModulelist[j].setPtxPath(name_found);
//	    CltModulelist[j].setPtxImage(strdata_found);
	    CltModulelist[j].linkPtxData(name_found, strdata_found, &PtxStore);
	    
            WARN(5, "New client module item was registered. id:%d\n", mid);
        }
        CltModulelist[j].id[i] = mid;
    }
    CltModulelist[j].vdevid = Vdevid[idx];
    printModuleList();

    return CltModulelist[j].id; //mp->id;
}
#endif

hipError_t
dscudaFuncGetAttributesWrapper(int *moduleid, struct hipFuncAttributes *attr, const char *func)
{
    hipError_t err = hipSuccess;
    dscudaFuncGetAttributesResult *rp;

    WARN(3, "dscudaFuncGetAttributesWrapper(%d, 0x%08llx, %s)...",
         moduleid, (unsigned long long)attr, func);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {
#warning fill this part in dscudaFuncGetAttributesWrapper().
        } else {
            rp = dscudafuncgetattributesid_1(moduleid[i], (char*)func, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            if (i == 0) {
                attr->binaryVersion      = rp->attr.binaryVersion;
                attr->constSizeBytes     = rp->attr.constSizeBytes;
                attr->localSizeBytes     = rp->attr.localSizeBytes;
                attr->maxThreadsPerBlock = rp->attr.maxThreadsPerBlock;
                attr->numRegs            = rp->attr.numRegs;
                attr->ptxVersion         = rp->attr.ptxVersion;
                attr->sharedSizeBytes    = rp->attr.sharedSizeBytes;
            }
            xdr_free((xdrproc_t)xdr_dscudaFuncGetAttributesResult, (char *)rp);
        }
    }

    WARN(3, "done.\n");
    WARN(3, "  attr->binaryVersion: %d\n", attr->binaryVersion);
    WARN(3, "  attr->constSizeBytes: %zu\n", attr->constSizeBytes);
    WARN(3, "  attr->localSizeBytes: %zu\n", attr->localSizeBytes);
    WARN(3, "  attr->maxThreadsPerBlock: %d\n", attr->maxThreadsPerBlock);
    WARN(3, "  attr->numRegs: %d\n", attr->numRegs);
    WARN(3, "  attr->ptxVersion: %d\n", attr->ptxVersion);
    WARN(3, "  attr->sharedSizeBytes: %zu\n", attr->sharedSizeBytes);

    return err;
}

hipError_t
dscudaMemcpyToSymbolWrapper(int *moduleid, const char *symbol, const void *src,
                           size_t count, size_t offset, enum hipMemcpyKind kind) {
    hipError_t err = hipSuccess;
    int nredundancy;

    WARN(3, "dscudaMemcpyToSymbolWrapper(%p, 0x%08lx, 0x%08lx, %zu, %zu, %s)"
         "symbol:%s  ...",
         moduleid, (unsigned long)symbol, (unsigned long)src,
         count, offset, dscudaMemcpyKindName(kind), symbol);

    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyHostToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolH2D(moduleid[i], (char *)symbol, src, count, offset, Vdevid[vdevidIndex()], i);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolD2D(moduleid[i], (char *)symbol, src, count, offset, Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    if ((St.Vdev + Vdevid[vdevidIndex()])->isRecording() &&
	(kind==hipMemcpyHostToDevice || kind==hipMemcpyDeviceToDevice)) {
        CudaMemcpyToSymbolArgs args;
        args.moduleid = moduleid;
        args.symbol = (char *)symbol;
        args.src = (void *)src;
        args.count = count;
        args.offset = offset;
        args.kind = kind;
        //HISTREC.add(dscudaMemcpyToSymbolH2DId, (void *)&args);
    }

    return err;
}//dscudaMemcpyToSymbolWrapper(int *moduleid, const char *symbol, const void *src,

hipError_t
dscudaMemcpyFromSymbolWrapper(int *moduleid, void *dst, const char *symbol,
                             size_t count, size_t offset,
                             enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    int nredundancy;
    void *dstbuf;

    WARN(3, "dscudaMemcpyFromSymbolWrapper(%p, %p, %p, %zu, %zu, %s)"
         "symbol:%s  ...",
         moduleid, dst, symbol, count, offset, dscudaMemcpyKindName(kind), symbol);

    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyDeviceToHost:
        if (St.isIbv()) {
            dstbuf = calloc(1, count);
            if (!dstbuf) {
                WARN(0, "dscudaMemcpyFromSymbolWrapper:calloc() failed.\n");
                exit(1);
            }
        }

        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolD2H(moduleid[i], &dstbuf, (char *)symbol, count, offset, Vdevid[vdevidIndex()], i);
            if (i == 0) {
                memcpy(dst, dstbuf, count);
            } else if (bcmp(dst, dstbuf, count) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpyFromSymbol() data copied from device%d matched with that from device0.\n", i);
            }
        }
        if (St.isIbv()) {
            free(dstbuf);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolD2D(moduleid[i], dst, (char *)symbol, count, offset, Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
dscudaMemcpyToSymbolAsyncWrapper(int *moduleid, const char *symbol, const void *src,
                                 size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream)
{
    hipError_t err = hipSuccess;
    RCstreamArray *st;
    int nredundancy;

    WARN(3, "sym:%s\n", symbol);
    WARN(3, "dscudaMemcpyToSymbolAsyncWrapper(%p, 0x%08lx, 0x%08lx, %zu, %zu, %s, 0x%08lx) "
         "symbol:%s  ...",
         moduleid, (unsigned long)symbol, (unsigned long)src,
         count, offset, dscudaMemcpyKindName(kind), (unsigned long)stream, symbol);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyHostToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolAsyncH2D(moduleid[i], (char *)symbol, src, count, offset,
                                               (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolAsyncD2D(moduleid[i], (char *)symbol, src, count, offset,
                                               (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}


hipError_t
dscudaMemcpyFromSymbolAsyncWrapper(int *moduleid, void *dst, const char *symbol,
                                   size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream)
{
    hipError_t err = hipSuccess;
    RCstreamArray *st;
    int nredundancy;
    void *dstbuf;

    WARN(3, "dscudaMemcpyFromSymbolAsyncWrapper(%d, 0x%08lx, 0x%08lx, %zu, %zu, %s, 0x%08lx)"
         " symbol:%s  ...",
         moduleid, (unsigned long)dst, (unsigned long)symbol,
         count, offset, dscudaMemcpyKindName(kind), (unsigned long)stream, symbol);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyDeviceToHost:
        if (St.isIbv()) {
            dstbuf = calloc(1, count);
            if (!dstbuf) {
                WARN(0, "dscudaMemcpyFromSymbolAsyncWrapper:calloc() failed.\n");
                exit(1);
            }
        }
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolAsyncD2H(moduleid[i], &dstbuf, (char *)symbol, count, offset,
                                                 (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
            if (i == 0) {
                memcpy(dst, dstbuf, count);
            } else if (bcmp(dst, dstbuf, count) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpyFromSymbol() data copied from device%d matched with that from device0.\n", i);
            }
        }
        if (St.isIbv()) {
            free(dstbuf);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolAsyncD2D(moduleid[i], dst, (char *)symbol, count, offset,
                                                 (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

static void
setTextureParams(RCtexture *texbufp, const struct textureReference *tex, const struct hipChannelFormatDesc *desc)
{
    texbufp->normalized = tex->normalized;
    texbufp->filterMode = tex->filterMode;
    texbufp->addressMode[0] = tex->addressMode[0];
    texbufp->addressMode[1] = tex->addressMode[1];
    texbufp->addressMode[2] = tex->addressMode[2];
    if (desc) {
        texbufp->x = desc->x;
        texbufp->y = desc->y;
        texbufp->z = desc->z;
        texbufp->w = desc->w;
        texbufp->f = desc->f;
    } else {
        texbufp->x = tex->channelDesc.x;
        texbufp->y = tex->channelDesc.y;
        texbufp->z = tex->channelDesc.z;
        texbufp->w = tex->channelDesc.w;
        texbufp->f = tex->channelDesc.f;
    }
}

hipError_t
dscudaBindTextureWrapper(int *moduleid, char *texname,
                        size_t *offset,
                        const struct textureReference *tex,
                        const void *devPtr,
                        const struct hipChannelFormatDesc *desc,
                        size_t size)
{
    hipError_t err = hipSuccess;
    dscudaBindTextureResult *rp;
    RCtexture texbuf;

    WARN(3, "dscudaBindTextureWrapper(%p, %s, %p, %p, %p, %p, %zu)...",
         moduleid, texname,
         offset, tex, devPtr, desc, size);

    setTextureParams(&texbuf, tex, desc);

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {

#warning fill this part in dscudaBindTextureWrapper().
        }
        else {
            rp = dscudabindtextureid_1(moduleid[i], texname,
                                       (RCadr)devPtr, size, (RCtexture)texbuf, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            if (i == 0) {
                if (offset) {
                    *offset = rp->offset;
                }
            }
            xdr_free((xdrproc_t)xdr_dscudaBindTextureResult, (char *)rp);
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTexture2DWrapper(int *moduleid, char *texname,
                          size_t *offset,
                          const struct textureReference *tex,
                          const void *devPtr,
                          const struct hipChannelFormatDesc *desc,
                          size_t width, size_t height, size_t pitch)
{
    hipError_t err = hipSuccess;
    dscudaBindTexture2DResult *rp;
    RCtexture texbuf;

    WARN(3, "dscudaBindTexture2DWrapper(%p, %s, %p, %p, %p, %p, %zu, %zu, %zu)...",
         moduleid, texname,
         offset, tex, devPtr, desc, width, height, pitch);

    setTextureParams(&texbuf, tex, desc);

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {

#warning fill this part in dscudaBindTexture2DWrapper().
        } else {

            rp = dscudabindtexture2did_1(moduleid[i], texname,
                                         (RCadr)devPtr, width, height, pitch, (RCtexture)texbuf, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            if (i == 0) {
                if (offset) {
                    *offset = rp->offset;
                }
            }
            xdr_free((xdrproc_t)xdr_dscudaBindTexture2DResult, (char *)rp);
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTextureToArrayWrapper(int *moduleid, char *texname,
                               const struct textureReference *tex,
                               const struct hipArray *array,
                               const struct hipChannelFormatDesc *desc) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCtexture texbuf;
    RCcuarrayArray *ca;

    WARN(3, "dscudaBindTextureToArrayWrapper(%p, %s, %p, %p)...", moduleid, texname, array, desc);

    setTextureParams(&texbuf, tex, desc);

    ca = RCcuarrayArrayQuery((hipArray *)array);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", array);
        exit(1);
    }

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {

#warning fill this part in dscudaBindTextureToArrayWrapper().
        } else {

            rp = dscudabindtexturetoarrayid_1(moduleid[i], texname, (RCadr)ca->ap[i], (RCtexture)texbuf, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
        }
    }
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipGetDevice(int *device) {
    hipError_t err = hipSuccess;

    WARN(3, "hipGetDevice(%p)...", device);
    *device = Vdevid[vdevidIndex()];
    WARN(3, "done.\n");

    return err;
}
hipError_t
cudaSetDevice_clnt(int device, int errcheck) {
    hipError_t cuerr = hipSuccess;
    int         vi    = vdevidIndex();
    
    if (0 <= device && device < St.Nvdev ) {
        Vdevid[vi] = device;
    }
    else {
        cuerr = hipErrorInvalidDevice;
	if (errcheck != 0) {
	    fprintf( stderr, "%s(): failed.\n", __func__);
	    exit(1);
	}
    }
    return cuerr;
}

hipError_t
hipSetDevice(int device) {
    hipError_t cuerr    = hipSuccess;
    int         errcheck = 0; 
    WARN(3, "%s(%d) {\n", __func__, device);
    
#if 0
    // hipSetDevice() is not needed on CUDA called record.
    // active target device is recorded on each cuda*() funcitons.
    if (HISTREC.rec_en > 0) {
        CudaSetDeviceArgs args;
        args.device = device;
        //HISTREC.add(dscudaSetDeviceId, (void *)&args);
    }
#endif
    
    cuerr = cudaSetDevice_clnt( device, errcheck );
    WARN(3, "}\n");
    return cuerr;
}

hipError_t
hipChooseDevice(int *device, const struct hipDeviceProp_t *prop) {
    hipError_t err = hipSuccess;

    WARN(3, "hipChooseDevice(%p, %p)...", device, prop);
    *device = 0;
    WARN(3, "done.\n");
    WARN(3, "Note : The current implementation always returns device 0.\n");

    return err;
}

hipError_t
hipGetDeviceCount(int *count) {
    hipError_t err = hipSuccess;

    *count = St.Nvdev;
    WARN(3, "hipGetDeviceCount(%p)  count:%d ...", count, *count);
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceCanAccessPeer(int *canAccessPeer, int device, int peerDevice) {
    hipError_t err = hipSuccess;

    WARN(3, "hipDeviceCanAccessPeer(%p, %d, %d)...", canAccessPeer, device, peerDevice);
    if (device < 0 || St.Nvdev <= device) {
        err = hipErrorInvalidDevice;
    }
    if (peerDevice < 0 || St.Nvdev <= peerDevice) {
        err = hipErrorInvalidDevice;
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceEnablePeerAccess(int peerDevice, unsigned int flags) {
    hipError_t err = hipSuccess;

    WARN(3, "cudaDeviceEnablePeer(%d, %d)...", peerDevice, flags);
    if (peerDevice < 0 || St.Nvdev <= peerDevice) {
        err = hipErrorInvalidDevice;
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceDisablePeerAccess(int peerDevice) {
    hipError_t err = hipSuccess;

    WARN(3, "cudaDeviceDisablePeer(%d)...", peerDevice);
    if (peerDevice < 0 || St.Nvdev <= peerDevice) {
        err = hipErrorInvalidDevice;
    }
    WARN(3, "done.\n");

    return err;
}

/*
 * MEMO: BkupMemList_t::reallocDeviceRegion(PhyDev *svr)
 */
void
VirDev::remallocRegionsGPU(int num_svr) {
    BkupMem *mem = memlist.headPtr();
    //int     verb = St.isAutoVerb();
    int     copy_count = 0;
    int     i = 0;
    
    WARN(1, "%s(PhyDev *sp).\n", __func__);
    //WARN(1, "Num. of realloc region = %d\n", BKUPMEM.length );
    //St.unsetAutoVerb();
    while ( mem != NULL ) {
	/* TODO: select migrateded virtual device, not all region. */
	WARN(5, "mem[%d]->dst = %p, size= %d\n", i, mem->d_region, mem->size);
	//dscudaVerbMalloc(&mem->d_region, mem->size, svr);
	mem = mem->next;
	i++;
    }
    //St.setAutoVerb(verb);
    WARN(1, "+--- Done.\n");
}
