#include "hip/hip_runtime.h"
//                             -*- Mode: C++ -*-
// Filename         : libdscuda.cu
// Description      : DS-CUDA client node library for Remote Procedure Call.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-09-15 11:45:48
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include <unistd.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/ioctl.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <net/if.h>
#include <ctype.h>
#include <pwd.h>
#include <netdb.h>
#include <pthread.h>
#include "dscuda.h"
#include "dscudautil.h"
#include "libdscuda.h"

static void  extractENV(bool &bool_var, const char *envname);
static void  extractENV(int  &int_var,  const char *envname, int undef=0);
static void  extractENV(char *str_var,  const char *envname, int len);
static void  getenvDSCUDA_WARNLEVEL(void);
static void  updateSpareServerList(void);
static char* readServerConf(char *fname);

static int   VdevidIndexMax = 0; //# of pthreads which utilize virtual devices.
const  char *DEFAULT_SVRIP = "localhost";

static pthread_mutex_t VdevidMutex = PTHREAD_MUTEX_INITIALIZER;
static pthread_t       VdevidIndex2ptid[RC_NPTHREADMAX]; // convert an Vdevid index into pthread id.
// CheckPointing mutual exclusion
       pthread_mutex_t cudaMemcpyD2H_mutex = PTHREAD_MUTEX_INITIALIZER;
       pthread_mutex_t cudaMemcpyH2D_mutex = PTHREAD_MUTEX_INITIALIZER;
       pthread_mutex_t cudaKernelRun_mutex = PTHREAD_MUTEX_INITIALIZER;

       RCmappedMem    *RCmappedMemListTop     = NULL;
       RCmappedMem    *RCmappedMemListTail    = NULL;

//#if RC_SUPPORT_STREAM
static RCstreamArray  *RCstreamArrayListTop   = NULL;
static RCstreamArray  *RCstreamArrayListTail  = NULL;
//#endif

static RCeventArray   *RCeventArrayListTop    = NULL;
static RCeventArray   *RCeventArrayListTail   = NULL;

static RCcuarrayArray *RCcuarrayArrayListTop  = NULL;
static RCcuarrayArray *RCcuarrayArrayListTail = NULL;

int    Vdevid[RC_NPTHREADMAX] = {0};   // the virtual device currently in use.

/*
 * Physical GPU device server
 */
ServerArray SvrCand;
ServerArray SvrSpare;   // Alternative GPU Device Servers.
ServerArray SvrIgnore;  // Forbidden GPU Device Servers.

void (*errorHandler)(void *arg) = NULL;
void *errorHandlerArg = NULL;

//struct rdma_cm_id *Cmid[RC_NVDEVMAX][RC_NREDUNDANCYMAX];

ClientState   St;
PtxStore      Ptx;
/*
 * Client initializer.
 * This function may be executed in parallel threads, so need mutex lock.
 */
ClientState::ClientState(void) {
    //<-- Open dscuda output file.
    char curr_time[80];
    dscuda::sprintfDate( curr_time );
    sprintf( this->dslog_filename, "c%s.dslog", curr_time );
    sprintf( this->dserr_filename, "c%s.dserr", curr_time );
    sprintf( this->dschp_filename, "c%s.dschp", curr_time );
    
    dscuda_stdout = fopen( dslog_filename, "w" );
    if (dscuda_stdout == NULL) {
	fprintf(stderr, "dscuda: failed to open file %s.", dslog_filename);
	exit(EXIT_FAILURE);
    } else {
	fprintf(stderr, "dscuda: log file ==> %s\n", dslog_filename);
    }
    //
    dscuda_stderr = fopen( dserr_filename, "w" );
    if (dscuda_stderr == NULL) {
	fprintf(stderr, "dscuda: failed to open file %s.", dserr_filename);
	exit(EXIT_FAILURE);
    } else {
	fprintf(stderr, "dscuda: err file ==> %s\n", dserr_filename);
    }
    //
    dscuda_chkpnt = fopen( dschp_filename, "w" );
    if (dscuda_chkpnt == NULL) {
	fprintf(stderr, "dscuda: failed to open file %s.", dschp_filename);
	exit(EXIT_FAILURE);
    } else {
	fprintf(stderr, "dscuda: err file ==> %s\n", dschp_filename);
    }
    //--> Open dscuda output file.
	
    INFO0("\
###******************************************************************************\n\
###***                                                                          *\n\
###***   Start process of DS-CUDA client library.                               *\n\
###***                                                                          *\n\
###******************************************************************************\n");
    INFO0("[ DS-CUDA Version      ] %s\n", RC_DSCUDA_VER);
    {
	/* Capture start time and print to logfile. */
	char s_time[80];
	struct tm *timebuf;
	start_time = time(NULL);
	timebuf = localtime(&start_time);
	strftime(s_time, 80, "%T (%F)", timebuf);
	INFO0("[ Start time           ] %s\n", s_time);
    }
    {
	/* Print IP address of DS-CUDA client host. */
	struct sockaddr_in addrin;
	get_myaddress(&addrin);
	setMyIPAddr(addrin.sin_addr.s_addr);
	INFO0("[ IP address of client ] %s\n",dscudaGetIpaddrString(St.getIpAddress()));
    }
    {
	char path[1024];
	getcwd(path, 1024);
	INFO0("[ Working Directory    ] %s\n", path);
	INFO0("[ Process ID (PID)     ] %d\n", getpid());
    }


    use_ibv     = 0;
    autoverb    = 0;
    daemon      = 0;
    this->unsetRollbackCalling();

    extractENV( dscuda_path, "DSCUDA_PATH", 512 );
    INFO0("[ Environment varialbe ] DSCUDA_PATH      = %s\n", dscuda_path);

    getenvDSCUDA_WARNLEVEL();       /* set from DSCUDA_WARNLEVEL */
    INFO0("[ Environment variable ] DSCUDA_WARNLEVEL = %d\n", dscuda::getWarnLevel());

    this->configFT();
    INFO0("[ Environment variable ] DSCUDA_USEDAEMON = %d\n", daemon);
    INFO0("[ Environment variable ] DSCUDA_AUTOVERB  = %d\n", autoverb);
    INFO0("[ Environment variable ] DSCUDA_CP_PERIOD = %d\n", cp_period);
    INFO0("[ Fault Tolerant Mode  ] ");
    switch (ft_mode) {
    case FT_NONE:
	INFO0("\"FT_NONE\"\n");
	break;
    case FT_ERRSTAT:
	INFO0("\"FT_ERRSTAT\"\n");
	break;
    case FT_BYCPY:
	INFO0("\"FT_BYCPY\"\n");
	break;
    case FT_BYTIMER:
	INFO0("\"FT_BYTIMER\"\n");
	break;
    case FT_OPTION:
	INFO0("\"FT_OPTION\"\n");
	break;
    default:
	WARN0(0, "(UNKNOWN).\n");
	exit(EXIT_FAILURE);
    }
    INFO0("[Environment var] DSCUDA_FT0  = %d (d2h_simple)\n",   ft.d2h_simple   );
    INFO0("[Environment var] DSCUDA_FT1  = %d (d2h_reduncpy)\n", ft.d2h_reduncpy );
    INFO0("[Environment var] DSCUDA_FT2  = %d (d2h_compare)\n",  ft.d2h_compare  );
    INFO0("[Environment var] DSCUDA_FT3  = %d (d2h_statics)\n",  ft.d2h_statics  );
    INFO0("[Environment var] DSCUDA_FT4  = %d (d2h_rollback)\n", ft.d2h_rollback );
    
    INFO0("[Environment var] DSCUDA_FT8  = %d (cp_periodic)\n",  ft.cp_periodic );
    INFO0("[Environment var] DSCUDA_FT9  = %d (cp_reduncpy)\n",  ft.cp_reduncpy );
    INFO0("[Environment var] DSCUDA_FT10 = %d (cp_compare)\n",   ft.cp_compare  );
    INFO0("[Environment var] DSCUDA_FT11 = %d (cp_statics)\n",   ft.cp_statics  );
    INFO0("[Environment var] DSCUDA_FT12 = %d (cp_rollback)\n",  ft.cp_rollback );
    
    INFO0("[Environment var] DSCUDA_FT16 = %d (rec_en)\n", ft.rec_en );
    INFO0("[Environment var] DSCUDA_FT24 = %d (migrate)\n", ft.gpu_migrate );

    initVirtualDevice();  /* Update the list of virtual devices */
    WARN0(0, "\n");

    // Search around cluster.
    WARN(2, "   <---Start Searching DS-CUDA daemon program                            *\n");
    dscuda::searchDaemon();
    WARN(2, "   --->Stop  Searching DS-CUDA daemon program                            *\n\n");

    ServerArray svr_array;
    svr_array.captureEnv("DSCUDA_SERVER_IGNORE", hl_BAD);
    svr_array.print();

    updateSpareServerList();
    svr_array.captureEnv("DSCUDA_SERVER_SPARE", hl_GOOD);
    svr_array.print();
    for (int i=0; i<svr_array.num; i++) {
	SvrSpare.append( &svr_array.svr[i] );
    }
    
    printVirtualDeviceList(); /* Print result to terminal. */

    WARN(2, "method of remote procedure call: ");
    switch ( dscudaRemoteCallType() ) {
    case RC_REMOTECALL_TYPE_RPC:
	WARN0(2, "RPC\n");
	break;
    case RC_REMOTECALL_TYPE_IBV:
	WARN0(2, "InfiniBand Verbs\n");
	break;
    default:
	WARN0(0, "(Unkown)\n"); exit(1);
    }

    /*
     * Establish connections of all physical devices.
     */
    for (int i=0; i<Nvdev; i++) {
	for (int j=0; j<Vdev[i].nredundancy; j++) {
	    Vdev[i].server[j].setupConnection();
	    WARN(1, "setupConn. Vdev[%d].server[%d].Clnt=%p\n",
		 i, j, Vdev[i].server[j].Clnt);
        }
    }
    
    if (ft.d2h_statics) {
	if (ft.cp_statics) { WARN(1, "[ERRORSTATICS] count both @D2H and @CP.\n" ); }
	else               { WARN(1, "[ERRORSTATICS] count @D2H but @CP.\n" );      }
    } else {
	if (ft.cp_statics) { WARN(1, "[ERRORSTATICS] count @CP  but @D2H.\n" );     }
	else               { WARN(1, "[ERRORSTATICS] not counted.\n" );             }
    }

    if (ft.cp_periodic) {
	WARN(1, "Starts Automatic CheckPointing Threads.\n" );	
	pthread_create(&tid, NULL, periodicCheckpoint, (void *)&cp_period);
    }
    
    INFO0("\
###******************************************************************************\n\
###***   Start user application process.                                        *\n\
###******************************************************************************\n");
} //--> ClientState::ClientState(void)
//--
//--
//--
ClientState::~ClientState(void) {
    INFO0("\
###******************************************************************************\n\
###***   Completed user application process.                                    *\n\
###******************************************************************************\n");

    PhyDev  *svr;
    time_t     exe_time;
    char       my_tfmt[64];	      
    struct tm *my_local;

    //--- Terminate the checkpointing thread.
    if (ft.cp_periodic) {
	WARN(1, "Stops Automatic CheckPointing Threads.\n" );
	pthread_cancel(tid);
    }
	
    stop_time = time( NULL);
    exe_time = stop_time - start_time;

    //--- Report start time.
    my_local = localtime( &start_time);
    strftime( my_tfmt, 64, "%c", my_local);
    WARN0(1, "    Start_time: %s\n", my_tfmt);
    //--- Report stop time.
    my_local = localtime( &stop_time);
    strftime( my_tfmt, 64, "%c", my_local);
    WARN0(1, "    Stop_time:  %s\n", my_tfmt);
    //--- Report run time.
    my_local = localtime( &exe_time);
    strftime( my_tfmt, 64, "%s", my_local);
    WARN0(1, "    Run_time:   %s (sec)\n", my_tfmt);

    if (ft.d2h_statics) {
	if (ft.cp_statics) {
	    WARN(1, "[ERRORSTATICS] count both @D2H and @CP.\n" );
	} else {
	    WARN(1, "[ERRORSTATICS] count @D2H but @CP.\n" );
	    for (int i=0; i<Nvdev; i++) {
		WARN0(1, "    [ERRORSTAT]  Virtual[%2d]\n", i);
		for (int j=0; j<Vdev[i].nredundancy; j++) {
		    svr = &Vdev[i].server[j];
		    WARN0(1, "    [ERRORSTAT]  + Physical[%2d]:%s:%s: ErrorCount= %d , MatchCount= %d\n",
			  j, svr->ip, svr->hostname, svr->stat_error, svr->stat_correct);
		}
	    }
	}
    }
    else {
	if (ft.cp_statics) { WARN(1, "[ERRORSTATICS] TODO.\n" );        }
	else               { WARN(1, "[ERRORSTATICS] not counted.\n" ); }
    }

    INFO0("\
###******************************************************************************\n\
###***                                                                          *\n\
###***   Completed process of DS-CUDA client library.                           *\n\
###***                                                                          *\n\
###******************************************************************************\n");
} //--> ClientState::~ClientState(void)
void
ClientState::configFT(void) {
    extractENV( this->daemon,    "DSCUDA_USEDAEMON",  0 );
    extractENV( this->cp_period, "DSCUDA_CP_PERIOD", 60 );
    extractENV( this->autoverb,  "DSCUDA_AUTOVERB",   0 );
    //<--- Define Fault Tolerant behavior from env.var.
    switch (autoverb) {
    case -1:
	ft_mode = FT_OPTION;
	break;
    case 0:
	ft_mode = FT_NONE;
	break;
    case 1:
	ft_mode = FT_ERRSTAT;
	break;
    case 2:
	ft_mode = FT_BYCPY;
	break;
    case 3:
	ft_mode = FT_BYTIMER;
	break;
    default:
	WARN(0, "Found invalid setting of DSCUDA_AUTOVERB=%d\n", autoverb);
	exit(EXIT_FAILURE);
    }
    //---> Define Fault Tolerant behavior from env.var.
    extractENV( ft.d2h_simple,    "DSCUDA_FT0" );
    extractENV( ft.d2h_reduncpy,  "DSCUDA_FT1" );
    extractENV( ft.d2h_compare,   "DSCUDA_FT2" );
    extractENV( ft.d2h_statics,   "DSCUDA_FT3" );
    extractENV( ft.d2h_rollback,  "DSCUDA_FT4" );
    //
    extractENV( ft.cp_periodic,   "DSCUDA_FT8" ); // 2nd: checkpointing
    extractENV( ft.cp_reduncpy,   "DSCUDA_FT9" );
    extractENV( ft.cp_compare,    "DSCUDA_FT10" );
    extractENV( ft.cp_statics,    "DSCUDA_FT11" );
    extractENV( ft.cp_rollback,   "DSCUDA_FT12" );
    //
    extractENV( ft.rec_en,        "DSCUDA_FT16" ); // 3rd: CUDA API recording
    //
    extractENV( ft.gpu_migrate,   "DSCUDA_FT24" ); // 4th: GPU Migration

    //<--- copy same value to virtual and physical device.
    for (int i=0; i<RC_NVDEVMAX; i++) {
	Vdev[i].ft_mode = this->ft_mode;
	for (int k=0; k<Vdev[i].nredundancy; k++) {
	    Vdev[i].server[k].ft_mode = this->ft_mode;
	}
    }
    //---> copy same value to virtual and physical device.

    if (ft_mode==FT_BYCPY || ft_mode==FT_BYTIMER) {
	for (int i=0; i<RC_NVDEVMAX; i++) {
	    Vdev[i].recordON();
	}
    }
}//--> void ClientState::configsFT(void)
/*
 *
 */
void
ClientState::initVirtualDevice(void) {
    char   *ip;
    char    ips[RC_NVDEVMAX][256];
    char    buf[1024*RC_NVDEVMAX];
    PhyDev *sp;
    char   *vdev_token;
    char   *pdev_token;
    {// DSCUDA_SERVER
	char *sconfname;
	char *env;    
	if (sconfname = getenv("DSCUDA_SERVER_CONF")) {
	    env = readServerConf(sconfname);
	    INFO0("[ Environment variable ] DSCUDA_SERVER_CONF = %s\n", env);
	} else {
	    env = getenv("DSCUDA_SERVER");
	    INFO0("[ Environment variable ] DSCUDA_SERVER    = %s\n", env);
	}
	// check DSCUDA_SERVER (1)
	if (env == NULL) {
	    Nvdev = 1;
	    Vdev[0].nredundancy = 1;
	    sp = Vdev[0].server;
	    sp->id = 0;
	    strncpy(sp->ip, DEFAULT_SVRIP, sizeof(sp->ip));
	    return;
	}
	
	// check DSCUDA_SERVER (2)
	if (sizeof(buf) < strlen(env)) {
	    WARN(0, "Too long length of DSCUDA_SERVER.\n");
	    exit(EXIT_FAILURE);
	}
	strncpy( buf, env, sizeof(buf) );
    }
    //<-- set "Nvdev", # of virtual device count.
    Nvdev = 0;
    vdev_token = strtok(buf, DELIM_VDEV); // a list of IPs which consist a single vdev.
    while (vdev_token != NULL) {
	strcpy(ips[Nvdev], vdev_token);
	Nvdev++;
	if (RC_NVDEVMAX < Nvdev) {
	    WARN(0, "number of devices exceeds the limit, RC_NVDEVMAX (=%d).\n",
		 RC_NVDEVMAX);
	    exit(EXIT_FAILURE);
	}
	vdev_token = strtok(NULL, DELIM_VDEV);
    }
    //--> set "Nvdev", # of virtual device count.
    
    for (int i=0; i<Nvdev; i++) {
	int nred=0;
	int uniq=0; // begin with 0.
	pdev_token = strtok(ips[i], DELIM_REDUN); // an IP (optionally with devid preceded by a comma) of
	// a single element of the vdev.
	while (pdev_token != NULL) {
	    strcpy(Vdev[i].server[nred].ip, pdev_token);
	    pdev_token = strtok(NULL, DELIM_REDUN);
	    nred++;
	}
	/*
	 * update Vdev.info.
	 */
	Vdev[i].setConfInfo(nred);
	
	for (int j=0; j<nred; j++) {
	    sp = &Vdev[i].server[j];
	    strncpy(buf, sp->ip, sizeof(buf));
	    ip = strtok(buf, ":");
	    sp->setIP(ip);
	    ip = strtok(NULL, ":");
	    sp->setCID(ip);
	    sp->setUNIQ(uniq);
	    uniq++;
	}
    } // for ( int i=0; ...
    /* convert hostname to ip address. */
    char *hostname;
    int  det_abc;
    char letter;
    char *ip_ref;
    struct hostent *hostent0;
    for (int i=0; i<Nvdev; i++) {
	Vdev[i].id = i;
	for (int j=0; j < Vdev[i].nredundancy; j++) {
	    ip = Vdev[i].server[j].ip;
	    hostname = Vdev[i].server[j].hostname;
	    det_abc=1;
	    for (int k=0; k < strlen(ip); k++) {
		letter = ip[k];
		if (isdigit((unsigned char)letter || letter=='.')) {
		    det_abc = 0;
		    printf("%c", letter);
		} else {
		    det_abc = 1;
		    break;
		}
		printf("\n");
	    }
	    if (det_abc == 1) {
		strcpy( hostname, ip );
		hostent0 = gethostbyname( hostname );
		if ( hostent0 == NULL ) {
		    WARN( 0, "May be set invalid hostname \"%s\" to DSCUDA_SERVER or something.\n", hostname );
		    WARN( 0, "Program terminated.\n\n\n\n" );
		    exit(EXIT_FAILURE);
		} else {
		    ip_ref = inet_ntoa( *(struct in_addr*)hostent0->h_addr_list[0] );
		    strcpy( ip, ip_ref );
		}
	    }
	}
    } // for (int i=0; ...
} //---> void ClientState::initVirtualDevice(void)
unsigned
ClientState::getIpAddress(void) {
    return this->ip_addr;
}
void
ClientState::useIbv(void) {
    this->use_ibv = true;
}
void
ClientState::useRpc(void) {
    this->use_ibv = false;
}
bool
ClientState::isIbv(void) {
    return this->use_ibv;     
}
bool
ClientState::isRpc(void) {
    return !this->use_ibv;     
}
void
ClientState::setRollbackCalling(void) {
    this->rollback_calling = true;
}
void
ClientState::unsetRollbackCalling(void) {
    this->rollback_calling = false;
}
bool
ClientState::isRollbackCalling(void) {
    return this->rollback_calling;
}
void
ClientState::setMyIPAddr(unsigned val) {
    this->ip_addr = val;
}

ServerArray::ServerArray(void) {
    num = 0;
}
//*********************************************************
//*** CLASS: PtxRecord
//*********************************************************
PtxRecord::PtxRecord(void) {
    if (RC_KMODULENAMELEN < 16) {
	WARN(0, "%s():RC_KMODULENAMELEN is too small.\n", __func__);
	exit(1);
    }
    if (RC_KMODULEIMAGELEN < 16) {
	WARN(0, "%s():RC_KMODULEIMAGELEN is too small.\n", __func__);
	exit(1);
    }
    strcpy(name, "unknown");
    strcpy(ptx_image, "empty");
    valid = 0;
}
void
PtxRecord::invalidate(void) {
    strncpy(name, "unknown", RC_KMODULENAMELEN);
    strncpy(ptx_image, "empty", RC_KMODULEIMAGELEN);
    valid = 0;
}

void PtxRecord::set(char *name0, char *ptx_image0) {
    strncpy(name,      name0,      RC_KMODULENAMELEN);
    strncpy(ptx_image, ptx_image0, RC_KMODULEIMAGELEN);
    valid = 1;
    
    if (name[RC_KMODULENAMELEN-1] != '\0') {
	WARN(0, "%s():RC_KMODULENAMELEN is too small.\n");
	exit(1);
    }
    if (ptx_image[RC_KMODULEIMAGELEN-1] != '\0') {
	WARN(0, "%s():RC_KMODULEIMAGELEN is too small.\n");
	exit(1);
    }
}
//*********************************************************
//*** CLASS: PtxStore
//*********************************************************
PtxStore::PtxStore(void) {
    used_count = 0;
}
PtxRecord*
PtxStore::add(char *name0, char *ptx_image0) {
    PtxRecord *ptx_ptr = &ptx_record[used_count];
    if (used_count > RC_NKMODULEMAX) {
	WARN(0, "PtxStore::%s(): PtxStore array FULL!\n");
	exit(1);
    }
    ptx_ptr->set(name0, ptx_image0);
    used_count++;
    return ptx_ptr;
}

PtxRecord*
PtxStore::query(char *name0) {
    PtxRecord *ptx_ptr;
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	ptx_ptr = &ptx_record[i];
	if ( strcmp(name0, ptx_ptr->name)==0 ) {/*found*/
	    WARN(9, "      +PtxStore::query(): Found ptx.\n")
	    return ptx_ptr;
	}
    }
    WARN(5, "      + PtxStore::query(): Not found ptx.\n");
    return NULL;
}

void
PtxStore::print(int n) {
    for (int i=0; i<n; i++) {
	WARN(1, "ptx_record[%d]: valid=%d, name=%s.\n",
	     i, ptx_record[i].valid, ptx_record[i].name); 
    }
}
ClientModule::ClientModule(void) {
    valid    = -1;
    id       = -1;
    ptx_data = NULL;
}
int
ClientModule::isValid(void) {
    if (valid<-1 || valid>1) {
	fprintf(stderr, "Unexpected error. %s:%d\n", __FILE__, __LINE__);
	exit(1);
    } else if (valid==1) {
	return 1;
    } else {
	return 0;
    }
}

int
ClientModule::isInvalid(void) {
    if (valid<-1 || valid>1) {
	fprintf(stderr, "Unexpected error. %s:%d\n", __FILE__, __LINE__);
	exit(1);
    } else if (valid==1) {
	return 0;
    } else {
	return 1;
    }
}

int
ServerArray::append(const char *ip, int ndev, const char *hname) {
    if ( num >= (RC_NVDEVMAX - 1) ) {
	WARN(0, "(+_+) Too many DS-CUDA daemons, exceeds RC_NVDEVMAX(=%d)\n",
	     RC_NVDEVMAX);
	exit(EXIT_FAILURE);
    }
    svr[num].setIP( ip );
    strcpy(svr[num].hostname, hname);
    svr[num].setID( ndev );
    svr[num].setCID( ndev );
    svr[num].uniq = RC_UNIQ_CANDBASE + num;
    num++;
    return 0;
}
int
ServerArray::append(PhyDev *svrptr) {
    if ( num >= (RC_NVDEVMAX - 1) ) {
	WARN(0, "(+_+) Too many DS-CUDA daemons, exceeds RC_NVDEVMAX(=%d)\n",
	     RC_NVDEVMAX);
	exit(EXIT_FAILURE);
    }
    svr[num].setIP( svrptr->ip );
    strcpy(svr[num].hostname, svrptr->hostname);
    svr[num].setID( svrptr->id );
    svr[num].setCID( svrptr->cid );
    svr[num].setUNIQ( RC_UNIQ_CANDBASE + num );
    svr[num].setFTMODE( svrptr->ft_mode );
    num++;
    return 0;
}
PhyDev*
ServerArray::findSpareOne(void) {
    PhyDev *sp = NULL;
    for (int i=0; i<num; i++) {
	if (svr[i].ft_health==hl_GOOD || svr[i].ft_health==hl_RECYCLED) {
	    sp = &svr[i];
	}
    }
    return sp;
}
PhyDev*
ServerArray::findBrokenOne(void) {
    PhyDev *sp = NULL;
    for (int i=0; i<num; i++) {
	if (svr[i].ft_health==hl_BAD) {
	    sp = &svr[i];
	}
    }
    return sp;
}
void
ServerArray::captureEnv(char *env_str, FThealth cond) {
    char buf[1024*RC_NVDEVMAX];
    char *svr_token;
    char svr_token_ar[RC_NVDEVMAX][256];

    {/* update buf[] */
	char *env = getenv(env_str);
	if (env == NULL) {
	    INFO0("[ Environment variable ] %s = (Not found)\n", env_str);
	    return;
	}
	if (sizeof(buf) < strlen(env)) {
	    WARN(0, "Too long length of DSCUDA_SERVER.\n");
	    exit(EXIT_FAILURE);
	}
	strncpy(buf, env, sizeof(buf));
    }

    //<--- svr_token_sr[x]="hostname:n"
    int  svr_count = 0;    
    svr_token = strtok(buf, " ");
    while (svr_token != NULL) {
	strcpy(svr_token_ar[svr_count], svr_token);
	svr_count++;
	if (svr_count > RC_NVDEVMAX) {
	    WARN(0, "number of devices exceeds the limit, RC_NVDEVMAX (=%d).\n",
		 RC_NVDEVMAX);
	    exit(EXIT_FAILURE);
	}
	svr_token = strtok(NULL, " ");
    }

    for (int i=0; i<svr_count; i++) {
	svr_token = strtok( svr_token_ar[i], ":" );
	this->svr[i].setIP( svr_token );
	svr_token = strtok( NULL, ":" );
	svr[i].setCID( svr_token );
	svr[i].setHealth(cond);
    }
    this->num = svr_count;
}//-->captureEnv()
void
ServerArray::print(void) {
    WARN(5, "ServerArray.num = %d\n", num);
    for (int i=0; i<num; i++) {
	WARN(1, "      + svrarr[%d].id= %d\n", i, svr[i].id);
	WARN(1, "      + svrarr[%d].cid= %d\n", i, svr[i].cid);
	WARN(1, "      + svrarr[%d].ip= %s\n", i, svr[i].ip);
	WARN(1, "      + svrarr[%d].hostname= %s\n", i, svr[i].hostname);
    }
}

int
requestDaemonForDevice(char *ip, int devid, bool useibv) {
    int dsock; // socket for side-band communication with the daemon & server.
    int sport; // port number of the server. given by the daemon.
    char msg[256];
    struct sockaddr_in sockaddr;

    sockaddr = setupSockaddr( ip, RC_DAEMON_IP_PORT );
    dsock = socket(AF_INET, SOCK_STREAM, 0);
    if (dsock < 0) {
        perror("socket");
        exit(1);
    }
    
    if ( connect(dsock, (struct sockaddr *)&sockaddr, sizeof(sockaddr)) == -1 ) {
        perror("(;_;) connect(...)");
	WARN(0, "+--- Program terminated at %s:L%d\n", __FILE__, __LINE__ );
	WARN(0, "+--- Maybe DS-CUDA daemon is not running...\n" );
        //exit(1);
	return -1;
    }
    sprintf(msg, "deviceid:%d", devid);
    sendMsgBySocket(dsock, msg);
    WARN(1, "<--- Send message: \"%s\".\n", msg);

    memset(msg, 0, strlen(msg));
    recvMsgBySocket(dsock, msg, sizeof(msg));
    WARN(1, "---> Recv message: \"%s\".\n", msg);    
    sscanf(msg, "sport:%d", &sport);

    if (sport < 0) {
        WARN(0, "max possible ports on %s already in use.\n", ip);
        exit(1);
    }

    WARN(3, "server port: %d  daemon socket: %d\n", sport, dsock);

    if (useibv) {
        sprintf(msg, "remotecall:ibv");
    } else {
        sprintf(msg, "remotecall:rpc");
    }
    WARN(3, "send \"%s\" to the server.\n", msg);
    sendMsgBySocket(dsock, msg);

    WARN(2, "waiting for the server to be set up...\n");
    memset(msg, 0, strlen(msg));
    recvMsgBySocket(dsock, msg, sizeof(msg)); // wait for "ready" from the server.
    if (strncmp("ready", msg, strlen("ready"))) {
        WARN(0, "unexpected message (\"%s\") from the server. abort.\n", msg);
        exit(1);
    }
    return sport;
}

/*
 * Obtain a small integer unique for each thread.
 * The integer is used as an index to 'Vdevid[]'.
 */
int
vdevidIndex(void) {
    int i;
    pthread_t ptid = pthread_self();

    for (i=0; i<VdevidIndexMax; i++) {
        if (VdevidIndex2ptid[i] == ptid) {
            return i;
        }
    }

    pthread_mutex_lock(&VdevidMutex);
    i = VdevidIndexMax;
    VdevidIndex2ptid[i] = ptid;
    VdevidIndexMax++;
    pthread_mutex_unlock(&VdevidMutex);

    if (RC_NPTHREADMAX <= VdevidIndexMax) {
        fprintf(stderr, "vdevidIndex():device requests from too many (more than %d) pthreads.\n", RC_NPTHREADMAX);
        exit(1);
    }

    return i;
}

void
RCmappedMemRegister(void *pHost, void* pDevice, size_t size) {
    RCmappedMem *mem = (RCmappedMem *)malloc(sizeof(RCmappedMem));
    if (!mem) {
        perror("RCmappedMemRegister");
    }
    mem->pHost = pHost;
    mem->pDevice = pDevice;
    mem->size = size;
    mem->prev = RCmappedMemListTail;
    mem->next = NULL;
    if ( !RCmappedMemListTop ) { // mem will be the 1st entry.
        RCmappedMemListTop = mem;
    } else {
        RCmappedMemListTail->next = mem;
    }
    RCmappedMemListTail = mem;
}

RCmappedMem*
RCmappedMemQuery(void *pHost) {
    RCmappedMem *mem = RCmappedMemListTop;
    while (mem) {
        if (mem->pHost == pHost) {
            return mem;
        }
        mem = mem->next;
    }
    return NULL; // pHost not found in the list.
}

void
RCmappedMemUnregister(void *pHost) {
    RCmappedMem *mem = RCmappedMemQuery(pHost);
    if (!mem) return;

    if (mem->prev) { // reconnect the linked list.
        mem->prev->next = mem->next;
    } else { // mem was the 1st entry.
        RCmappedMemListTop = mem->next;
        if (mem->next) {
            mem->next->prev = NULL;
        }
    }
    if ( !mem->next ) { // mem was the last entry.
        RCmappedMemListTail = mem->prev;
    }
    free( mem );
}

//#if RC_SUPPORT_STREAM
/*
 * Register a stream array. each component is associated to a stream
 * on each Server[]. User see only the 1st element, streams[0].
 * Others, i.e., streams[1..Nredunddancy-1], are used by this library
 * to handle redundant calculation mechanism.
 */
static void
RCstreamArrayRegister(hipStream_t *streams) {
    RCstreamArray *st = (RCstreamArray *)malloc(sizeof(RCstreamArray));
    if (!st) {
        perror("RCstreamArrayRegister");
    }
    for (int i=0; i<RC_NREDUNDANCYMAX; i++) {
        st->s[i] = streams[i];
    }
    st->prev = RCstreamArrayListTail;
    st->next = NULL;
    if (!RCstreamArrayListTop) { // st will be the 1st entry.
        RCstreamArrayListTop = st;
    } else {
        RCstreamArrayListTail->next = st;
    }
    RCstreamArrayListTail = st;
}

#if 0
static void
showsta(void) {
    RCstreamArray *st = RCstreamArrayListTop;
    while (st) {
        fprintf(stderr, ">>> 0x%08llx    prev:%p  next:%p\n", st, st->prev, st->next);
        st = st->next;
    }
}
#endif

RCstreamArray*
RCstreamArrayQuery(hipStream_t stream0) {
    static RCstreamArray default_stream = { 0,};

    if (stream0 == 0) {
        return &default_stream;
    }

    RCstreamArray *st = RCstreamArrayListTop;
    while (st) {
        if (st->s[0] == stream0) {
            return st;
        }
        st = st->next;
    }
    return NULL;
}

static void
RCstreamArrayUnregister(hipStream_t stream0) {
    RCstreamArray *st = RCstreamArrayQuery(stream0);
    if (!st) return;

    if (st->prev) { // reconnect the linked list.
        st->prev->next = st->next;
    } else { // st was the 1st entry.
        RCstreamArrayListTop = st->next;
        if (st->next) {
            st->next->prev = NULL;
        }
    }
    if (!st->next) { // st was the last entry.
        RCstreamArrayListTail = st->prev;
    }
    free(st);
    //    showsta();
}
//#endif


/*
 * Register a hipArray array. each component is associated to a hipArray
 * on each Server[]. User see only the 1st element, cuarrays[0].
 * Others, i.e., cuarrays[1..Nredunddancy-1], are used by this library
 * to handle redundant calculation mechanism.
 */
void
RCcuarrayArrayRegister(hipArray **cuarrays) {
    RCcuarrayArray *ca = (RCcuarrayArray *)malloc(sizeof(RCcuarrayArray));
    if (!ca) {
        perror("RCcuarrayArrayRegister");
    }
    for (int i = 0; i < RC_NREDUNDANCYMAX; i++) {
        ca->ap[i] = cuarrays[i];
    }
    ca->prev = RCcuarrayArrayListTail;
    ca->next = NULL;
    if (!RCcuarrayArrayListTop) { // ca will be the 1st entry.
        RCcuarrayArrayListTop = ca;
    } else {
        RCcuarrayArrayListTail->next = ca;
    }
    RCcuarrayArrayListTail = ca;
}

RCcuarrayArray*
RCcuarrayArrayQuery(hipArray *cuarray0) {
    RCcuarrayArray *ca = RCcuarrayArrayListTop;
    while (ca) {
        if (ca->ap[0] == cuarray0) {
            return ca;
        }
        ca = ca->next;
    }
    return NULL;
}

void
RCcuarrayArrayUnregister(hipArray *cuarray0) {
    RCcuarrayArray *ca = RCcuarrayArrayQuery(cuarray0);
    if (!ca) return;

    if (ca->prev) { // reconnect the linked list.
        ca->prev->next = ca->next;
    } else { // ca was the 1st entry.
        RCcuarrayArrayListTop = ca->next;
        if (ca->next) {
            ca->next->prev = NULL;
        }
    }
    if (!ca->next) { // ca was the last entry.
        RCcuarrayArrayListTail = ca->prev;
    }
    free(ca);
}


/*
 * Register an event array. each component is associated to an event
 * on each Server[]. User see only the 1st element, events[0].
 * Others, i.e., events[1..Nredunddancy-1], are used by this library
 * to handle redundant calculation mechanism.
 */
void
RCeventArrayRegister(hipEvent_t *events) {
    RCeventArray *ev = (RCeventArray *)malloc(sizeof(RCeventArray));
    if (!ev) {
        perror("RCeventArrayRegister");
    }
    for (int i = 0; i < RC_NREDUNDANCYMAX; i++) {
        ev->e[i] = events[i];
    }
    ev->prev = RCeventArrayListTail;
    ev->next = NULL;
    if (!RCeventArrayListTop) { // ev will be the 1st entry.
        RCeventArrayListTop = ev;
    } else {
        RCeventArrayListTail->next = ev;
    }
    RCeventArrayListTail = ev;
}

RCeventArray*
RCeventArrayQuery(hipEvent_t event0) {
    RCeventArray *ev = RCeventArrayListTop;
    while (ev) {
        if (ev->e[0] == event0) {
            return ev;
        }
        ev = ev->next;
    }
    return NULL;
}

void
RCeventArrayUnregister(hipEvent_t event0) {
    RCeventArray *ev = RCeventArrayQuery(event0);
    if (!ev) return;

    if (ev->prev) { // reconnect the linked list.
        ev->prev->next = ev->next;
    } else { // ev was the 1st entry.
        RCeventArrayListTop = ev->next;
        if (ev->next) {
            ev->next->prev = NULL;
        }
    }
    if (!ev->next) { // ev was the last entry.
        RCeventArrayListTail = ev->prev;
    }
    free(ev);
}
/*
 * Compose UVA from GPU local address and its deviceID.
 */
void*
dscudaUvaOfAdr( void *adr, int devid ) {
    DscudaUva_t adri = (DscudaUva_t)adr;
#if __LP64__
    adri |= ((DscudaUva_t)devid << 48);
#endif
    return (void *)adri;
}
/*====================================================================
 * Get GPU deviceID from UVA.
 */
int
dscudaDevidOfUva( void *adr ) {
#if __LP64__
    DscudaUva_t adri = (DscudaUva_t)adr;
    int devid = adri >> 48;
    return devid;
#else
    return 0;
#endif
}
/*
 * Get GPU local address from UVA.
 */
void*
dscudaAdrOfUva( void *adr ) {
    DscudaUva_t adri = (DscudaUva_t)adr;
#if __LP64__
    adri &= 0x0000ffffffffffffLL;
#endif
    return (void *)adri;
}

static char*
readServerConf(char *fname) {
    FILE *fp = fopen(fname, "r");
    char linebuf[1024];
    int len;
    static char buf[1024*RC_NVDEVMAX];

    buf[0] = 0;
    if (!fp) {
        WARN(0, "cannot open file '%s'\n", fname);
        exit(1);
    }

    while (!feof(fp)) {
        char *s = fgets(linebuf, sizeof(linebuf), fp);
        if (!s) break;
        len = strlen(linebuf);
        if (linebuf[len-1] == '\n') {
            linebuf[len-1] = 0;
        }
        if (sizeof(buf) < strlen(buf) + len) {
            WARN(0, "readServerConf:file %s too long.\n", fname);
            exit(1);
        }
        strncat(buf, linebuf, sizeof(linebuf));
        strcat(buf, " ");
    }
    fclose(fp);
    return buf;
}

/*
 *
 */
void
printVirtualDeviceList( void ) {
    VirDev     *pVdev;
    PhyDev   *pSvr;
    int         i,j;
    
    INFO0("###***   <---Start Virtual Device Construction. (Total %d device%s)\n", St.Nvdev, (St.Nvdev>1)? "s":"" );
    for (i=0, pVdev=St.Vdev; i<St.Nvdev; i++, pVdev++) {
	if (i >= RC_NVDEVMAX) {
	    INFO0("(;_;) Too many virtual devices. %s().\nexit.", __func__);
	    exit(EXIT_FAILURE);
	}
	if (pVdev->nredundancy == 1) {
	    INFO0("    Virt[%d] (MONO)\n", i );
	} else if ( pVdev->nredundancy > 1 ) {
	    INFO0("    Virt[%d] (POLY:%d)\n", i, pVdev->nredundancy );
	} else {
	    INFO0("    Virt[%d] (????:%d)\n", i, pVdev->nredundancy );
	}
	
	for (j=0, pSvr=pVdev->server; j<pVdev->nredundancy; j++, pSvr++) {
	    if (j >= RC_NREDUNDANCYMAX) {
		WARN(0, "(;_;) Too many redundant devices %d. %s().\nexit.\n", __func__);
		exit( EXIT_FAILURE );
	    }
	    INFO0("    +  Phy[%d]: id=%d, cid=%d, IP=%s(%s), uniq=%d.\n", j,
		   pSvr->id, pSvr->cid, pSvr->ip, pSvr->hostname, pSvr->uniq);
	}
    }
    INFO0("###***   --->Stop Virtual Device Construction.\n\n");

    if (St.ft_mode==FT_BYCPY || St.ft_mode==FT_BYTIMER) {
	/*
	 * Device Candidates
	 */
	INFO0("###*** Physical Device Candidates. (Ncand=%d)\n", SvrCand.num );
	for( i=0, pSvr=SvrCand.svr; i < SvrCand.num; i++, pSvr++ ){
	    if (i >= RC_NVDEVMAX) {
		WARN(0, "(;_;) Too many candidate devices. %s().\nexit.", __func__);
		exit( EXIT_FAILURE );
	    }
	    INFO0("###***    - Cand[%2d]: id=%d, cid=%d, IP=%s, uniq=%d.\n", i,
		  pSvr->id, pSvr->cid, pSvr->ip, pSvr->uniq);
	}
	/*
	 * Alternate Devices
	 */
	INFO0("###*** Spare Server Info.(Nspare=%d)\n", SvrSpare.num);
	for( i=0, pSvr=SvrSpare.svr; i < SvrSpare.num; i++, pSvr++ ){
	    if (i >= RC_NVDEVMAX) {
		WARN(0, "(;_;) Too many spare devices. %s().\nexit.", __func__);
		exit( EXIT_FAILURE );
	    }
	    INFO0("###***    - Spare[%d]: id=%d, cid=%d, IP=%s, uniq=%d.\n", i,
		  pSvr->id, pSvr->cid, pSvr->ip, pSvr->uniq);
	}
    }
}

void
VirDev::setConfInfo(int redun) {
    nredundancy = redun; //Update Vdev.nredundancy.
    if (redun == 1) {
	conf = VDEV_MONO;
	sprintf(info, "MONO");
    } else if (redun > 1) {
	conf = VDEV_POLY;
	sprintf(info, "POLY%d", redun);
    } else {
	WARN(0, "Detect invalid nredundancy = %d.\n", redun);
	exit(EXIT_FAILURE);
    }
}

void
VirDev::printModuleList(void) {
    const int len = 256;
    char printbuf[len];
    int valid_cnt = 0;
    
    WARN(5, "====================================================\n");
    WARN(5, "===  VirDev::%s(void)\n", __func__ );
    WARN(5, "====================================================\n");
    WARN(5, "RC_NKMODULEMAX= %d\n", RC_NKMODULEMAX);
    
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	if( modulelist[i].valid==1 || modulelist[i].valid==0 ) {
	    WARN( 5, "Virtual[%d]:modulelist[%d]:\n", id, i);
	    WARN( 5, "    + name= %s\n", modulelist[i].ptx_data->name);
	    //printf("    + send_time= \n", sent_time., sent_time.);
	    //strncpy(printbuf, modulelist[i].ptx_data->ptx_image, len - 1 );
	    //printbuf[255]='\0';
	    //printf("# %s():    + ptx_image=\n%s\n", __func__, printbuf);
	    valid_cnt++;
	}
    }
    WARN(5, "%d valid modules registered.\n",  valid_cnt);
    WARN(5, "====================================================\n");
    
}

void
printModuleList(void) {
    for (int i=0; i<St.Nvdev; i++) {
	St.Vdev[i].printModuleList();
    }
}
uint32_t
dscuda::calcChecksum(void *sta, size_t size_byte) {
    uint32_t *p = (uint32_t *)sta;
    uint32_t  s           = 0;
    uint32_t  s_remain    = 0; // zero padding, ignore sign bit.
    size_t    sum_count   = size_byte / sizeof(uint32_t);
    size_t    size_remain = size_byte - (sum_count * sizeof(uint32_t));
    for (int i=0; i<sum_count; i++) {
	s += *p;
	p++;
    }
    if (size_remain >= sizeof(s_remain)) {
	fprintf(stderr, "Unexpected ERROR: %s()\n", __func__);
	exit(1);
    }
    memcpy( &s_remain, p, size_remain );
    s += s_remain;
    return s;
}
int
dscuda::searchDaemon(void) {
    int sendsock;
    int recvsock;

    char sendbuf[SEARCH_BUFLEN_TX];
    char recvbuf[SEARCH_BUFLEN_RX];
    
    int recvlen;
    int num_daemon = 0;
    int num_device = 0;
    int num_ignore = 0;

    unsigned int adr, mask;
    socklen_t    sin_size;
    int          setsockopt_ret;

    struct sockaddr_in addr, svr;
    struct ifreq ifr[2];
    struct ifconf ifc;
    struct passwd *pwd;

    INFO0("[ Constant             ] RC_DAEMON_IP_PORT = %d\n", RC_DAEMON_IP_PORT);
    sendsock = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
    recvsock = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP);
    if ( sendsock == -1 || recvsock == -1 ) {
	perror("searchDaemon: socket()");
	exit(1);
    }
    {
	int val = 1;
	setsockopt_ret = setsockopt(sendsock, SOL_SOCKET, SO_BROADCAST, &val, sizeof(val));
	if (setsockopt_ret != 0) {
	    perror("searchDaemon: setsockopt()");
	    exit(1);
	}
    }
    ifc.ifc_len = sizeof(ifr) * 2;
    ifc.ifc_ifcu.ifcu_buf = (char *)ifr;
    ioctl(sendsock, SIOCGIFCONF, &ifc);

    ifr[1].ifr_addr.sa_family = AF_INET;
    ioctl(sendsock, SIOCGIFADDR, &ifr[1]);
    adr = ((struct sockaddr_in *)(&ifr[1].ifr_addr))->sin_addr.s_addr;
    ioctl(sendsock, SIOCGIFNETMASK, &ifr[1]);
    mask = ((struct sockaddr_in *)(&ifr[1].ifr_netmask))->sin_addr.s_addr;

    addr.sin_family      = AF_INET;
    addr.sin_port        = htons(RC_DAEMON_IP_PORT - 1);
    addr.sin_addr.s_addr = adr | ~mask;

    strncpy( sendbuf, SEARCH_PING, SEARCH_BUFLEN_TX - 1 );
    sendto( sendsock, sendbuf, SEARCH_BUFLEN_TX, 0, (struct sockaddr *)&addr, sizeof(addr));
    INFO0("Broadcast \"%s\" message\n", SEARCH_PING);
    sin_size = sizeof(struct sockaddr_in);

    svr.sin_family      = AF_INET;
    svr.sin_port        = htons(RC_DAEMON_IP_PORT - 2);
    svr.sin_addr.s_addr = htonl(INADDR_ANY);
    
    // Set timeout for recvsock.
    {
	struct timeval tout;
	tout.tv_sec  = RC_SEARCH_DAEMON_TIMEOUT ;
	tout.tv_usec = 0;
	setsockopt_ret = setsockopt(recvsock, SOL_SOCKET, SO_RCVTIMEO, (char *)&tout, sizeof(tout));
	if (setsockopt_ret != 0) {
	    perror("searchDaemon: setsockopt(recvsock)");
	    exit(1);
	}
    }

    {
	int bind_ret = bind( recvsock, (struct sockaddr *)&svr, sizeof(svr) );
	if( bind_ret != 0 ) {
	    fprintf(stderr, "Error: bind() returned %d. recvsock=%d, port=%d\n",
		    bind_ret, recvsock, svr.sin_port); //port:38655
	    perror("searchDaemon: bind()");
	    return -1;
	}
    }
    
    pwd = getpwuid( getuid() );

    /* Recieve ack message from dscudad running at other host. */
    char *magic_word;
    char *user_name;
    char *host_name;
    char *dev_count;
    char  ipaddr[32];
    int   num_eachdev;

    SvrCand.num = 0;

    memset( recvbuf, 0, SEARCH_BUFLEN_RX );
    while(( recvlen = recvfrom( recvsock, recvbuf, SEARCH_BUFLEN_RX - 1, 0, (struct sockaddr *)&svr, &sin_size)) > 0) {
	INFO0(" + Detect ACK \"%s\" ", recvbuf);
	/*
	 * Analyze message.
	 */
	magic_word = strtok( recvbuf, SEARCH_DELIM );
	user_name  = strtok( NULL,    SEARCH_DELIM );
	host_name  = strtok( NULL,    SEARCH_DELIM );
	dev_count  = strtok( NULL,    SEARCH_DELIM ); // Ndev=4
	sscanf( dev_count, "Ndev=%d", &num_eachdev );
	sprintf( ipaddr, "%s", inet_ntoa( svr.sin_addr )); //192.168.1.1
	if ( magic_word == NULL ) {
	    WARN(0, "\n\n###(ERROR) Unexpected token in %s().\n\n", __func__);
	    exit(1);
	} else {
	    INFO0("from \"%s\" ", ipaddr );
	    if ( strcmp( magic_word, SEARCH_ACK   )==0 &&
		 strcmp( user_name,  pwd->pw_name )==0 ) { /* Found */
		INFO0("valid.\n");
		/*
		 * Updata SvrCand;
		 */
		for (int d=0; d<num_eachdev; d++) {
		    SvrCand.append(ipaddr, d, host_name);
		}
		num_daemon += 1;
		num_device += num_eachdev;
	    } else {
		INFO0("ignored.\n");
		num_ignore++;
	    }
	}
	memset( recvbuf, 0, SEARCH_BUFLEN_RX );
    }

    {
	int close_ret = close( sendsock );
	if ( close_ret != 0 ) {
	    WARN(0, "close(sendsock) failed.\n");
	    exit(EXIT_FAILURE);
	}
	
	close_ret = close( recvsock );
	if ( close_ret != 0 ) {
	    WARN(0, "close(recvsock) failed.\n");
	    exit(EXIT_FAILURE);
	}
    }

    if (num_daemon > 0) {
	INFO0("Found %d valid DSCUDA daemon%s. (%d ignored).\n",
	      num_daemon, (num_daemon>1)? "s":"", num_ignore );
    }
    else if (num_daemon==0) {
	//
	// Even if no daemons found, the servers defined in DSCUDA_SERVER are available.
	//
	WARN( 0, "%s(): Not found DS-CUDA daemon in this network.\n", __func__ );
	WARN( 0, "%s(): And Trying to continue execution.\n", __func__ );
    }
    else { 	/* Terminate program and exit. */
	WARN( 0, "%s(): Detected unexpected trouble; num_daemon=%d?\n", __func__, num_daemon );
	exit(EXIT_FAILURE);
    }
    return num_daemon;
} //---> int dscuda::searchDaemon(void)
//
//
//
#if 0
void
ServerArray::removeArray(ServerArray *sub) {

    for (int i=0; i<num; i++) {
	for (int k=0; k<sub->num; k++) {
	}
    }
}
#endif
static void
updateSpareServerList(void) {
    int         spare_count = 0;;
    VirDev     *pVdev;
    PhyDev *pSvr;

    for (int i=0; i<SvrCand.num; i++) {    // Sweep all Vdev.server[] and compare.
	int found = 0;
	pVdev = St.Vdev;
	for (int j=0; j<St.Nvdev; j++) {
	    pSvr = pVdev->server;
	    for (int k=0; k < pVdev->nredundancy; k++) {
		if ( strcmp( SvrCand.svr[i].ip,  pSvr->ip  )==0 &&
		     SvrCand.svr[i].cid==pSvr->cid ) { /* check same IP */
		    found=1;
		}
		pSvr++;
	    }
	    pVdev++;
	}
	if (found==0) { /* not found */
	    SvrSpare.svr[spare_count].id   = SvrCand.svr[i].id;
	    SvrSpare.svr[spare_count].cid  = SvrCand.svr[i].cid;
	    SvrSpare.svr[spare_count].uniq = SvrCand.svr[i].uniq;
	    SvrSpare.svr[spare_count].ft_health = hl_GOOD;
	    strcpy(SvrSpare.svr[spare_count].ip, SvrCand.svr[i].ip);
	    spare_count++;
	}
    }
    SvrSpare.num = spare_count;
}

static void
getenvDSCUDA_WARNLEVEL(void) {
    char *env = getenv("DSCUDA_WARNLEVEL");
    int val;
    if ( env ) {
        val = atoi(strtok(env, " "));
        if ( val >= 0 ) {
	    dscuda::setWarnLevel( val );
	} else {
	    WARN(0, "(;_;) Invalid DSCUDA_WARNLEVEL(%d), set 0 or positive integer.\n", val);
	    exit(EXIT_FAILURE);
	}
    } else {
	dscuda::setWarnLevel(RC_WARNLEVEL_DEFAULT);
    }

}
static void
extractENV(bool &bool_var, const char *envname) {
    char *env = getenv(envname);
    if (env==NULL) {
	bool_var = false;
    } else {
	int val = atoi(env);
	if (val==0) {
	    bool_var = false;
	} else {
	    bool_var = true;
	}
    }
}
static void
extractENV(int &int_var, const char *envname, int undef) {
    char *env = getenv(envname);
    if (env==NULL) {
	int_var = undef;
    } else {
	int_var = atoi(env);
    }
}
static void
extractENV(char *str_var, const char *envname, int len) {
    char *env = getenv(envname);
    if (env==NULL) {
	strncpy(str_var, "(undef)", len);  
    } else {
	strncpy(str_var, env, len);  
    }
}
/****** CHECK-POINTING THREAD ****************************************
 * Take the data backups of each virtualized GPU to client's host
 * memory after verifying between redundant physical GPUs every
 * specified wall clock
 * time period. The period is defined in second.
 */
void*
periodicCheckpoint(void *arg) {
    int         cp_period = *(int *)arg;
    int         age = 0; //
    int         devid;
    int         errcheck = 1;
    hipError_t cuerr;
    int         pmem_devid;
    BkupMem    *pmem;
    int         pmem_count;
    void       *lsrc;
    void       *ldst;
    int         redun;
    int         size;
    
    int  cmp_result[RC_NREDUNDANCYMAX][RC_NREDUNDANCYMAX]; //verify
    int  regional_match;
    int  snapshot_match = 1;
    int  snapshot_count = 0;
    void *dst_cand[RC_NREDUNDANCYMAX];
    int  dst_color[RC_NREDUNDANCYMAX], next_color;

    int correct_count = 0;
    while (true) {
	//<-- Wait for specified period (sec) passed.
	sleep(cp_period);
	//--> Wait for specified period (sec) passed.

	//<-- Output beginning message.
	WARN_CP(1,
	"=====================================================================\n");
	WARN_CP( 1,"periodicCheckpoint( period = %d sec, age=%d )\n", cp_period, age);
	WARN(9,"CP: periodicCheckpoint( period = %d sec, age=%d ) {\n", cp_period, age);
	//--> Output beginning message.
		
	//<-- mutex locks for avoiding R/W collisions 
	pthread_mutex_lock( &cudaMemcpyD2H_mutex );
	pthread_mutex_lock( &cudaMemcpyH2D_mutex );
	pthread_mutex_lock( &cudaKernelRun_mutex );
	//--> mutex locks for avoiding R/W collisions 

	//<-- copy from all hipMalloc() regions of all devices.
	St.collectEntireRegions();
	//**  calling hierachical as following,
	//**  +---> Vdev[*].collectEntireRegions();
	//**           +---> server[*].collectEntireRegions();
	//--> copy from all hipMalloc() regions of all devices.

	int correct = St.verifyEntireRegions();
	if (correct==1) {
	    correct_count++;
	}
#if 1 // force pseudo error
	if (correct_count % 5 == 4) {
	    correct = 0;
	}
#endif
	if ( correct==1 ) {
	    //***
	    //*** All memory regions on all virtual devices are correct.
	    //*** Then, collect clean device memory regions to host memory.
	    //*** and clear CUDA API called history.
	    //***
	    WARN(1, "CP: (^_^)Ready to update clean backup region.\n");
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].updateMemlist(0); // 0 means server[0].
		St.Vdev[i].clearReclist();
	    }
	}
	else {
	    //***
	    //*** Some memory regions on any virtual devices are currupted.
	    //*** Then, restore clean memory regions to all devices, and
	    //*** redo the historical cuda API calls.
	    //***
	    WARN(1, "CP: (+_+)Detected corrupted device region.\n");
	    WARN(1, "CP: (+_+)Restore the device memory using backup.\n");
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].restoreMemlist();
	    }
	    WARN(1, "CP: (+_+)Redo the CUDA APIs Rollbacked.\n");	    
	    for (int i=0; i<St.Nvdev; i++) {
		St.Vdev[i].reclist.print();
		St.Vdev[i].reclist.recall();
	    }
	}
	//<-- mutex unlocks for following R/W.
	pthread_mutex_unlock( &cudaMemcpyD2H_mutex );
	pthread_mutex_unlock( &cudaMemcpyH2D_mutex );
	pthread_mutex_unlock( &cudaKernelRun_mutex );
	//--> mutex unlocks for following R/W.
	
	//<-- Output ending message.	
	WARN(9,"CP: } periodicCheckpoint().\n");
	//--> Output ending message.	
	pthread_testcancel();/* cancelation available */
	age++;
    }//for (;;)
} // periodicCheckpoint()
void
VirDev::invalidateAllModuleCache(void) {
    for (int i=0; i<RC_NKMODULEMAX; i++) {
        if( modulelist[i].isValid() ){
	    modulelist[i].invalidate();
	} else { 
	    continue;
	}
    }
}

/*
 * public functions
 */
int
dscudaNredundancy(void) {
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    return vdev->nredundancy;
}

void
dscudaSetErrorHandler(void (*handler)(void *), void *handler_arg) {
    errorHandler = handler;
    errorHandlerArg = handler_arg;
}

/*
 * Obtain a mangled symbol name of a function, whose
 * interface is given by 'funcif' and is defined somewhere in 'ptxdata'.
 * The obtained symbol name is returned to 'name'.
 *
 * eg) funcif  : void dscudavecAdd(dim3, dim3, size_t, ihipStream_t*, float*, float*, float*)
 *     ptxdata : .version 1.4
 *               .target sm_10, map_f64_to_f32
 *               ...
 *               .entry _Z6vecAddPfS_S_ (
 *               ...
 *               } // _Z6vecMulPfS_fS_iPi
 */
void
dscudaGetMangledFunctionName(char *name, const char *funcif, const char *ptxdata)
{
    WARN(10, "<---Entering %s()\n", __func__);
    static char mangler[256] = {0, };
    char cmd[4096];
    FILE *outpipe;
    FILE *tmpfp;
    char ptxfile[1024];

    WARN(4, "getMangledFunctionName(%p, %p, %p)  funcif:\"%s\"\n",
         name, funcif, ptxdata, funcif);

    // create a tmporary file that contains 'ptxdata'.
    system("/bin/mkdir /tmp/dscuda 1> /dev/null  2> /dev/null");
    // do not use >& since /bin/sh on some distro does not recognize it.

    sprintf(ptxfile, "/tmp/dscuda/mgl%d", getpid());
    tmpfp = fopen(ptxfile, "w");
    fprintf(tmpfp, "%s", ptxdata);
    fclose(tmpfp);

    // exec 'ptx2symbol' to obtain the mangled name.
    // command output is stored to name.
    if (!mangler[0]) {
        sprintf(mangler, "%s/bin/ptx2symbol", St.dscuda_path);
    }
    sprintf(cmd, "%s %s << EOF\n%s\nEOF", mangler, ptxfile, funcif);
    outpipe = popen(cmd, "r");
    if (!outpipe) {
        perror("getMangledFunctionName()");
        exit(1);
    }
    fgets(name, 256, outpipe);
    pclose(outpipe);
    if (!strlen(name)) {
        WARN(0, "getMangledFunctionName() : %s returned an error. "
             "it could not found any entry, or found multiple candidates. "
             "set DSCUDA_WARNLEVEL 4 or higher and try again to see "
             "error messages from %s.\n", mangler, mangler);
        exit(1);
    }
    WARN(10, "--->Exiting %s()\n", __func__);
}


static pthread_mutex_t LoadModuleMutex = PTHREAD_MUTEX_INITIALIZER;
/*
 * Load a cuda module from a .ptx file, and then, send it to the server.
 * returns id for the module.
 * the module is cached and sent only once for a certain period.
 */

int
dscudaLoadModule(char *name, char *strdata) { // 'strdata' must be NULL terminated.
    WARN(5, "dscudaLoadModule( name=%p(%s), strdata=%p ) {\n", name, name, strdata);
    int idx = vdevidIndex();
    VirDev *vdev = St.Vdev + Vdevid[idx];
    int module_index;

    module_index = vdev->loadModule(name, strdata);
    
    //printModuleList();
    WARN(5, "} //dscudaLoadModule() returned %d.\n", module_index);
    WARN(5, "\n");
    return module_index;
}

#if 0 // backup
int*
dscudaLoadModule(char *name, char *strdata) {// 'strdata' must be NULL terminated.
    int i, j, mid;
    ClientModule *mp;
    int idx;

    if (name != NULL) {
	WARN(5, "dscudaLoadModule(%p) modulename:%s  ...\n", name, name);
#if RC_CACHE_MODULE
	// look for modulename in the module list.
	for (i=0, mp=CltModulelist; i < RC_NKMODULEMAX; i++, mp++) {
	    if ( mp->isInvalid() ) {
		continue;
	    }
	    
	    idx = vdevidIndex();
	    if (mp->vdevid != Vdevid[idx]) {
		continue;
	    }
	    if ( !strcmp(name, mp->name) ) {
		if ( mp->isAlive() ) {
		    WARN(5, "done. found a cached one. id:%d  age:%d  name:%s\n",
			 mp->id[i], time(NULL) - mp->sent_time, mp->name);
		    return mp->id; // module found. i.e, it's already loaded.
		} else {
		    WARN(5, "found a cached one with id:%d, but it is too old (age:%d). resend it.\n",
			 mp->id[i], time(NULL) - mp->sent_time);
		    mp->invalidate(); // invalidate the cache.
		}
	    }
	} //for
#endif // RC_CACHE_MODULE
    } else {
	WARN(5, "dscudaLoadModule(%p) modulename:-\n", name);
    }

    //<---
    char *strdata_found = NULL;
    char *name_found=NULL;
    if (name==NULL && strdata==NULL) {
        for (i=0, mp=CltModulelist; i<RC_NKMODULEMAX; i++, mp++) {
	    WARN(10, "i=%d\n", i);
	    if (mp->isInvalid()) continue;
	    idx = vdevidIndex();
	    if (mp->vdevid != Vdevid[idx]) continue;
	    if (!strcmp(name, mp->name)) {     /* matched */
		strdata_found = mp->ptx_image;
		name_found = mp->name;
		break;
	    }
	}
    } else {
	strdata_found = strdata;
	name_found = name;
    }
    //--->

    // module not found in the module list.
    // really need to send it to the server.
    idx = vdevidIndex();
    VirDev *vdev = St.Vdev + Vdevid[idx];
    PhyDev *sp = vdev->server;

    for (i=0; i < vdev->nredundancy; i++) {
	// mid = dscudaLoadModuleLocal(St.getIpAddress(), getpid(), name, strdata, Vdevid[vi], i);
	// mid = dscudaLoadModuleLocal(St.getIpAddress(), getpid(), name_found, strdata_found, Vdevid[idx], i);
	mid = sp[i].loadModule(St.getIpAddress(), getpid(), name_found, strdata_found);
        WARN(3, "(info) dscudaLoadModuleLocal() returns mid=%d as Vdevid[%d], Redun[%d].\n", mid, idx, i);

        // register a new module into the list,
        // and then, return a module id assigned by the server.
        if (i==0) {
            for (j=0; j<RC_NKMODULEMAX; j++) { /* Search vacant sheet. */
                if( CltModulelist[j].isInvalid() ) break;
            }
	    if( j >= RC_NKMODULEMAX ) {
		WARN(0, "\n\n### (+_+) ERROR in DS-CUDA!\n");
		WARN(0,     "### (+_+) module send buffer is full. and exit.\n");
		WARN(0,     "### (+_+) Check if the array length of CltModulelist[%d].\n\n\n", j);
		exit(1);
	    }
            CltModulelist[j].validate();
            CltModulelist[j].sent_time = time(NULL);
//            CltModulelist[j].setPtxPath(name_found);
//	    CltModulelist[j].setPtxImage(strdata_found);
	    CltModulelist[j].linkPtxData(name_found, strdata_found, &PtxStore);
	    
            WARN(5, "New client module item was registered. id:%d\n", mid);
        }
        CltModulelist[j].id[i] = mid;
    }
    CltModulelist[j].vdevid = Vdevid[idx];
    printModuleList();

    return CltModulelist[j].id; //mp->id;
}
#endif

hipError_t
dscudaFuncGetAttributesWrapper(int *moduleid, struct hipFuncAttributes *attr, const char *func)
{
    hipError_t err = hipSuccess;
    dscudaFuncGetAttributesResult *rp;

    WARN(3, "dscudaFuncGetAttributesWrapper(%d, 0x%08llx, %s)...",
         moduleid, (unsigned long long)attr, func);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {
#warning fill this part in dscudaFuncGetAttributesWrapper().
        } else {
            rp = dscudafuncgetattributesid_1(moduleid[i], (char*)func, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            if (i == 0) {
                attr->binaryVersion      = rp->attr.binaryVersion;
                attr->constSizeBytes     = rp->attr.constSizeBytes;
                attr->localSizeBytes     = rp->attr.localSizeBytes;
                attr->maxThreadsPerBlock = rp->attr.maxThreadsPerBlock;
                attr->numRegs            = rp->attr.numRegs;
                attr->ptxVersion         = rp->attr.ptxVersion;
                attr->sharedSizeBytes    = rp->attr.sharedSizeBytes;
            }
            xdr_free((xdrproc_t)xdr_dscudaFuncGetAttributesResult, (char *)rp);
        }
    }

    WARN(3, "done.\n");
    WARN(3, "  attr->binaryVersion: %d\n", attr->binaryVersion);
    WARN(3, "  attr->constSizeBytes: %zu\n", attr->constSizeBytes);
    WARN(3, "  attr->localSizeBytes: %zu\n", attr->localSizeBytes);
    WARN(3, "  attr->maxThreadsPerBlock: %d\n", attr->maxThreadsPerBlock);
    WARN(3, "  attr->numRegs: %d\n", attr->numRegs);
    WARN(3, "  attr->ptxVersion: %d\n", attr->ptxVersion);
    WARN(3, "  attr->sharedSizeBytes: %zu\n", attr->sharedSizeBytes);

    return err;
}

hipError_t
dscudaMemcpyToSymbolWrapper(int *moduleid, const char *symbol, const void *src,
                           size_t count, size_t offset, enum hipMemcpyKind kind) {
    hipError_t err = hipSuccess;
    int nredundancy;

    WARN(3, "dscudaMemcpyToSymbolWrapper(%p, 0x%08lx, 0x%08lx, %zu, %zu, %s)"
         "symbol:%s  ...",
         moduleid, (unsigned long)symbol, (unsigned long)src,
         count, offset, dscudaMemcpyKindName(kind), symbol);

    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyHostToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolH2D(moduleid[i], (char *)symbol, src, count, offset, Vdevid[vdevidIndex()], i);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolD2D(moduleid[i], (char *)symbol, src, count, offset, Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    if ((St.Vdev + Vdevid[vdevidIndex()])->isRecording() &&
	(kind==hipMemcpyHostToDevice || kind==hipMemcpyDeviceToDevice)) {
        CudaMemcpyToSymbolArgs args;
        args.moduleid = moduleid;
        args.symbol = (char *)symbol;
        args.src = (void *)src;
        args.count = count;
        args.offset = offset;
        args.kind = kind;
        //HISTREC.add(dscudaMemcpyToSymbolH2DId, (void *)&args);
    }

    return err;
}//dscudaMemcpyToSymbolWrapper(int *moduleid, const char *symbol, const void *src,

hipError_t
dscudaMemcpyFromSymbolWrapper(int *moduleid, void *dst, const char *symbol,
                             size_t count, size_t offset,
                             enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    int nredundancy;
    void *dstbuf;

    WARN(3, "dscudaMemcpyFromSymbolWrapper(%p, %p, %p, %zu, %zu, %s)"
         "symbol:%s  ...",
         moduleid, dst, symbol, count, offset, dscudaMemcpyKindName(kind), symbol);

    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyDeviceToHost:
        if (St.isIbv()) {
            dstbuf = calloc(1, count);
            if (!dstbuf) {
                WARN(0, "dscudaMemcpyFromSymbolWrapper:calloc() failed.\n");
                exit(1);
            }
        }

        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolD2H(moduleid[i], &dstbuf, (char *)symbol, count, offset, Vdevid[vdevidIndex()], i);
            if (i == 0) {
                memcpy(dst, dstbuf, count);
            } else if (bcmp(dst, dstbuf, count) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpyFromSymbol() data copied from device%d matched with that from device0.\n", i);
            }
        }
        if (St.isIbv()) {
            free(dstbuf);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolD2D(moduleid[i], dst, (char *)symbol, count, offset, Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
dscudaMemcpyToSymbolAsyncWrapper(int *moduleid, const char *symbol, const void *src,
                                 size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream)
{
    hipError_t err = hipSuccess;
    RCstreamArray *st;
    int nredundancy;

    WARN(3, "sym:%s\n", symbol);
    WARN(3, "dscudaMemcpyToSymbolAsyncWrapper(%p, 0x%08lx, 0x%08lx, %zu, %zu, %s, 0x%08lx) "
         "symbol:%s  ...",
         moduleid, (unsigned long)symbol, (unsigned long)src,
         count, offset, dscudaMemcpyKindName(kind), (unsigned long)stream, symbol);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyHostToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolAsyncH2D(moduleid[i], (char *)symbol, src, count, offset,
                                               (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyToSymbolAsyncD2D(moduleid[i], (char *)symbol, src, count, offset,
                                               (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}


hipError_t
dscudaMemcpyFromSymbolAsyncWrapper(int *moduleid, void *dst, const char *symbol,
                                   size_t count, size_t offset, enum hipMemcpyKind kind, hipStream_t stream)
{
    hipError_t err = hipSuccess;
    RCstreamArray *st;
    int nredundancy;
    void *dstbuf;

    WARN(3, "dscudaMemcpyFromSymbolAsyncWrapper(%d, 0x%08lx, 0x%08lx, %zu, %zu, %s, 0x%08lx)"
         " symbol:%s  ...",
         moduleid, (unsigned long)dst, (unsigned long)symbol,
         count, offset, dscudaMemcpyKindName(kind), (unsigned long)stream, symbol);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    nredundancy = (St.Vdev + Vdevid[vdevidIndex()])->nredundancy;
    switch (kind) {
      case hipMemcpyDeviceToHost:
        if (St.isIbv()) {
            dstbuf = calloc(1, count);
            if (!dstbuf) {
                WARN(0, "dscudaMemcpyFromSymbolAsyncWrapper:calloc() failed.\n");
                exit(1);
            }
        }
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolAsyncD2H(moduleid[i], &dstbuf, (char *)symbol, count, offset,
                                                 (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
            if (i == 0) {
                memcpy(dst, dstbuf, count);
            } else if (bcmp(dst, dstbuf, count) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpyFromSymbol() data copied from device%d matched with that from device0.\n", i);
            }
        }
        if (St.isIbv()) {
            free(dstbuf);
        }
        break;
      case hipMemcpyDeviceToDevice:
        for (int i = 0; i < nredundancy; i++) {
            err = dscudaMemcpyFromSymbolAsyncD2D(moduleid[i], dst, (char *)symbol, count, offset,
                                                 (RCstream)st->s[i], Vdevid[vdevidIndex()], i);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

static void
setTextureParams(RCtexture *texbufp, const struct textureReference *tex, const struct hipChannelFormatDesc *desc)
{
    texbufp->normalized = tex->normalized;
    texbufp->filterMode = tex->filterMode;
    texbufp->addressMode[0] = tex->addressMode[0];
    texbufp->addressMode[1] = tex->addressMode[1];
    texbufp->addressMode[2] = tex->addressMode[2];
    if (desc) {
        texbufp->x = desc->x;
        texbufp->y = desc->y;
        texbufp->z = desc->z;
        texbufp->w = desc->w;
        texbufp->f = desc->f;
    } else {
        texbufp->x = tex->channelDesc.x;
        texbufp->y = tex->channelDesc.y;
        texbufp->z = tex->channelDesc.z;
        texbufp->w = tex->channelDesc.w;
        texbufp->f = tex->channelDesc.f;
    }
}

hipError_t
dscudaBindTextureWrapper(int *moduleid, char *texname,
                        size_t *offset,
                        const struct textureReference *tex,
                        const void *devPtr,
                        const struct hipChannelFormatDesc *desc,
                        size_t size)
{
    hipError_t err = hipSuccess;
    dscudaBindTextureResult *rp;
    RCtexture texbuf;

    WARN(3, "dscudaBindTextureWrapper(%p, %s, %p, %p, %p, %p, %zu)...",
         moduleid, texname,
         offset, tex, devPtr, desc, size);

    setTextureParams(&texbuf, tex, desc);

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {

#warning fill this part in dscudaBindTextureWrapper().
        }
        else {
            rp = dscudabindtextureid_1(moduleid[i], texname,
                                       (RCadr)devPtr, size, (RCtexture)texbuf, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            if (i == 0) {
                if (offset) {
                    *offset = rp->offset;
                }
            }
            xdr_free((xdrproc_t)xdr_dscudaBindTextureResult, (char *)rp);
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTexture2DWrapper(int *moduleid, char *texname,
                          size_t *offset,
                          const struct textureReference *tex,
                          const void *devPtr,
                          const struct hipChannelFormatDesc *desc,
                          size_t width, size_t height, size_t pitch)
{
    hipError_t err = hipSuccess;
    dscudaBindTexture2DResult *rp;
    RCtexture texbuf;

    WARN(3, "dscudaBindTexture2DWrapper(%p, %s, %p, %p, %p, %p, %zu, %zu, %zu)...",
         moduleid, texname,
         offset, tex, devPtr, desc, width, height, pitch);

    setTextureParams(&texbuf, tex, desc);

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {

#warning fill this part in dscudaBindTexture2DWrapper().
        } else {

            rp = dscudabindtexture2did_1(moduleid[i], texname,
                                         (RCadr)devPtr, width, height, pitch, (RCtexture)texbuf, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            if (i == 0) {
                if (offset) {
                    *offset = rp->offset;
                }
            }
            xdr_free((xdrproc_t)xdr_dscudaBindTexture2DResult, (char *)rp);
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTextureToArrayWrapper(int *moduleid, char *texname,
                               const struct textureReference *tex,
                               const struct hipArray *array,
                               const struct hipChannelFormatDesc *desc) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCtexture texbuf;
    RCcuarrayArray *ca;

    WARN(3, "dscudaBindTextureToArrayWrapper(%p, %s, %p, %p)...", moduleid, texname, array, desc);

    setTextureParams(&texbuf, tex, desc);

    ca = RCcuarrayArrayQuery((hipArray *)array);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", array);
        exit(1);
    }

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        if (St.isIbv()) {

#warning fill this part in dscudaBindTextureToArrayWrapper().
        } else {

            rp = dscudabindtexturetoarrayid_1(moduleid[i], texname, (RCadr)ca->ap[i], (RCtexture)texbuf, sp[i].Clnt);
            checkResult(rp, sp[i]);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
        }
    }
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipGetDevice(int *device) {
    hipError_t err = hipSuccess;

    WARN(3, "hipGetDevice(%p)...", device);
    *device = Vdevid[vdevidIndex()];
    WARN(3, "done.\n");

    return err;
}
hipError_t
cudaSetDevice_clnt(int device, int errcheck) {
    hipError_t cuerr = hipSuccess;
    int         vi    = vdevidIndex();
    
    if (0 <= device && device < St.Nvdev ) {
        Vdevid[vi] = device;
    }
    else {
        cuerr = hipErrorInvalidDevice;
	if (errcheck != 0) {
	    fprintf( stderr, "%s(): failed.\n", __func__);
	    exit(1);
	}
    }
    return cuerr;
}

hipError_t
hipSetDevice(int device) {
    hipError_t cuerr    = hipSuccess;
    int         errcheck = 0; 
    WARN(3, "%s(%d) {\n", __func__, device);
    
#if 0
    // hipSetDevice() is not needed on CUDA called record.
    // active target device is recorded on each cuda*() funcitons.
    if (HISTREC.rec_en > 0) {
        CudaSetDeviceArgs args;
        args.device = device;
        //HISTREC.add(dscudaSetDeviceId, (void *)&args);
    }
#endif
    
    cuerr = cudaSetDevice_clnt( device, errcheck );
    WARN(3, "}\n");
    return cuerr;
}

hipError_t
hipChooseDevice(int *device, const struct hipDeviceProp_t *prop) {
    hipError_t err = hipSuccess;

    WARN(3, "hipChooseDevice(%p, %p)...", device, prop);
    *device = 0;
    WARN(3, "done.\n");
    WARN(3, "Note : The current implementation always returns device 0.\n");

    return err;
}

hipError_t
hipGetDeviceCount(int *count) {
    hipError_t err = hipSuccess;

    *count = St.Nvdev;
    WARN(3, "hipGetDeviceCount(%p)  count:%d ...", count, *count);
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceCanAccessPeer(int *canAccessPeer, int device, int peerDevice) {
    hipError_t err = hipSuccess;

    WARN(3, "hipDeviceCanAccessPeer(%p, %d, %d)...", canAccessPeer, device, peerDevice);
    if (device < 0 || St.Nvdev <= device) {
        err = hipErrorInvalidDevice;
    }
    if (peerDevice < 0 || St.Nvdev <= peerDevice) {
        err = hipErrorInvalidDevice;
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceEnablePeerAccess(int peerDevice, unsigned int flags) {
    hipError_t err = hipSuccess;

    WARN(3, "cudaDeviceEnablePeer(%d, %d)...", peerDevice, flags);
    if (peerDevice < 0 || St.Nvdev <= peerDevice) {
        err = hipErrorInvalidDevice;
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceDisablePeerAccess(int peerDevice) {
    hipError_t err = hipSuccess;

    WARN(3, "cudaDeviceDisablePeer(%d)...", peerDevice);
    if (peerDevice < 0 || St.Nvdev <= peerDevice) {
        err = hipErrorInvalidDevice;
    }
    WARN(3, "done.\n");

    return err;
}

/*
 * MEMO: BkupMemList_t::reallocDeviceRegion(PhyDev *svr)
 */
void
VirDev::remallocRegionsGPU(int num_svr) {
    BkupMem *mem = memlist.headPtr();
    //int     verb = St.isAutoVerb();
    int     copy_count = 0;
    int     i = 0;
    
    WARN(1, "%s(PhyDev *sp).\n", __func__);
    //WARN(1, "Num. of realloc region = %d\n", BKUPMEM.length );
    //St.unsetAutoVerb();
    while ( mem != NULL ) {
	/* TODO: select migrateded virtual device, not all region. */
	WARN(5, "mem[%d]->dst = %p, size= %d\n", i, mem->d_region, mem->size);
	//dscudaVerbMalloc(&mem->d_region, mem->size, svr);
	mem = mem->next;
	i++;
    }
    //St.setAutoVerb(verb);
    WARN(1, "+--- Done.\n");
}
