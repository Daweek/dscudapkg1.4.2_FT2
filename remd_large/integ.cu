#include "hip/hip_runtime.h"
//                             -*- Mode: C++ -*-
// Filename         : integ.cu
// Description      : Time integrator.
// Author           : Minoru Oikawa (m_oikawa@amber.plala.or.jp)
// Created On       : 2013-08-25 11:49:27
// Last Modified By : Minoru Oikawa
// Last Modified On : 2014-02-12 20:57:57
// Update Count     : 0.0
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include "switch_float_double.H"
#include "remd_typedef.H"
#include "mytools.H"
#include "init_cond.H"
#include "comm_save.H"
#include "integ.H"  
#include "calc_force.cu" 
#include "dscuda.h"

#define ENERGY_WARNING 0

FaultConf_t FAULT_CONF(5); // fault 1 times.

__device__
void calcVelScale(int t, Real_t *scale, Real_t targ_temp, Real3_t *vel_ar, Real_t mass, int Nmol, Real3_t *shared);
void checkEnergyVal( int t0, const Real_t *h_energy, int len);
void calcHistogram( int *histo_ar, Remd_t &remd, Simu_t &simu);
void saveHistogram( const int *histo_ar, Remd_t &remd, Simu_t &simu);
void exchTemp( int t0, Remd_t &remd, Simu_t &simu);
void saveAccRatio( Remd_t &remd, int step_numkernel);
__device__
int ctrlTemp( int, int, Real3_t *velo_ar, Real_t zeta, int Nmol, Real_t dt);
static void calcZetaSum( Real_t&, Real_t, Real_t);
static Real_t hamiltonian( Real_t, Real_t, Real_t, Real_t, Real_t, Real_t, int);

// Debug
__device__ int chksum( int *start, int count ) {
    int sum, *p, i;
    if ( threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0 ) { /* Do by only 1 thread */
	p=start;
	sum=0;
	for (i=0; i<count; i++) {
	    sum += *(p+i);
	}
    }
    return sum;
}
__device__ void
probePosVelFoc( int t, int num_rep, Real3_t *posi, Real3_t *velo, Real3_t *forc, int Nmol ) {
   if ( blockIdx.x == num_rep ) { 
      if ( threadIdx.x==0 ) {
	 for (int i=0; i<Nmol; i++) {
	    printf("t=%d, Rep=%d [%d]: p={%+6.3f, %+6.3f, %+6.3f}, v={%+6.3f, %+6.3f %+6.3f}, f={%+6.3f, %+6.3f %+6.3f}\n", t, num_rep, i,
		   posi[i].x, posi[i].y, posi[i].z,
		   velo[i].x, velo[i].y, velo[i].z,
		   forc[i].x, forc[i].y, forc[i].z);
	 }
      }
   }
}

extern "C" __global__ void
g_test01( Real3_t *d_work_ar, Real_t *d_poten_ar, int Nmol ) {
   int i;
   Real3_t *shared    = d_work_ar + (Nmol * blockIdx.x);  // originally __shared__
   Real_t  *poten_ar  = d_poten_ar + (Nmol * blockIdx.x); // originally __shared__

   __shared__ Real_t sum;

   //=================================================
   if (threadIdx.x==0) {
      for (i=0;i<1024;i++) {
	 poten_ar[i]=(Real_t)i+1.0;
      }
   }
   __syncthreads();
   
   reductSum1D( &sum, poten_ar, 1 ); // Sum. to sum
   __syncthreads();
   if (threadIdx.x==0) {
      printf("%s():sum=%f\n", __func__, sum);
   }
   //-------------------------------------------------
   //=================================================
   if (threadIdx.x==0) {
      for (i=0;i<1024;i++) {
	 poten_ar[i]=(Real_t)i+1.0;
      }
   }
   __syncthreads();
   
   reductSum1D( &sum, poten_ar, 2 ); // Sum. to sum
   __syncthreads();
   if (threadIdx.x==0) {
      printf("%s():sum=%f\n", __func__, sum);
   }
   //-------------------------------------------------
   //=================================================
   if (threadIdx.x==0) {
      for (i=0;i<1024;i++) {
	 poten_ar[i]=(Real_t)i+1.0;
      }
   }
   __syncthreads();
   
   reductSum1D( &sum, poten_ar, 3 ); // Sum. to sum
   __syncthreads();
   if (threadIdx.x==0) {
      printf("%s():sum=%f\n", __func__, sum);
   }
   //-------------------------------------------------

   if (threadIdx.x==0) {
      for (i=0;i<1024;i++) {
	 poten_ar[i]=(Real_t)i+1.0;
      }
   }
   __syncthreads();
   
   reductSum1D( &sum, poten_ar, 100 ); // Sum. to sum
   __syncthreads();
   if (threadIdx.x==0) {
      printf("%s():sum=%f\n", __func__, sum);
   }

   if (threadIdx.x==0) {
      for (i=0;i<1024;i++) {
	 poten_ar[i]=(Real_t)i+1.0;
      }
   }
   __syncthreads();

   reductSum1D( &sum, poten_ar, 101 ); // Sum. to sum
   __syncthreads();
   if (threadIdx.x==0) {
      printf("%s():sum=%f\n", __func__, sum);
   }

}

//===============================================================================
extern "C" __global__ void
fitVel( int Nmol, int step_exch, Real_t dt, Real_t cellsize, Real_t rcut,
	Real_t lj_sigma, Real_t lj_epsilon, Real_t mass, 
	Real3_t *d_pos_ar, Real3_t *d_vel_ar, Real3_t *d_foc_ar,
	Real3_t *d_work_ar, Real_t *d_poten_ar,
	Real_t  *d_ene_ar, Real_t  *d_temp_ar,Real_t  *d_temp_meas, int *d_exch_ar) {
   /* Point out each data region from bulk data block */
   Real3_t *pos_ar    = d_pos_ar + (Nmol * blockIdx.x);
   Real3_t *vel_ar    = d_vel_ar + (Nmol * blockIdx.x);
   Real3_t *foc_ar    = d_foc_ar + (Nmol * blockIdx.x);
   Real3_t *shared    = d_work_ar + (Nmol * blockIdx.x);  // originally __shared__
   Real_t  *poten_ar  = d_poten_ar + (Nmol * blockIdx.x); // originally __shared__
   Real_t  *ene_ar    = d_ene_ar + (step_exch * blockIdx.x);
   Real_t   temp_targ = d_temp_ar[blockIdx.x];
   //Real_t  *temp_meas = d_temp_meas + (step_exch * blockIdx.x); /* use [0] only. */
   __shared__ Real_t temp_meas;
   __shared__ Real_t vel_scale;
   
   int      t_max = 1000;
   int      ret_code[8];

   if ( blockIdx.x==0 && threadIdx.x==0 ) {
       printf("gridDim.x=%d, blockDim.x=%d\n", gridDim.x, blockDim.x);
   }

   for ( int t=0; t<t_max; t++ ) {
#if 0 //debug
      for (int i=0; i<1; i++) {
	 probePosVelFoc(t, i, pos_ar, vel_ar, foc_ar, Nmol);
      }
#endif
      ret_code[0] = integVel(t, 10, vel_ar, foc_ar, Nmol, mass, dt * 0.5);
      __syncthreads();
      if ( ret_code[0] != 0 ) {
	 printf("(;_;) serious error in function %s(): timestep t = %d, blockIdx.x = %d, threadIdx.x = %d\n", __func__, t, blockIdx.x, threadIdx.x);
	 return;
      }
      
      ret_code[1] = integPos(t, pos_ar, vel_ar, Nmol, dt, cellsize);
      __syncthreads();
      if (ret_code[1] !=0) {
	 printf("(;_;) serious error in function %s(): timestep t = %d, blockIdx.x = %d, threadIdx.x = %d\n", __func__, t, blockIdx.x, threadIdx.x);
	 return;
      }

      __syncthreads();
      killMomentum(t, vel_ar, mass, Nmol, shared );
      __syncthreads();
      calcVelScale(t, &vel_scale, temp_targ, vel_ar, mass, Nmol, shared);
      __syncthreads();
      if (threadIdx.x==0) {
	 printf("%s():t=%d: rep_num=%d, vel_scale=%f, temp_meas=%f, temp_targ=%f\n",
		__func__, t, blockIdx.x, vel_scale, temp_meas, temp_targ);
      }

      scaleVelo(t, vel_ar, vel_scale, Nmol);
      __syncthreads();
      measTemper( &temp_meas, vel_ar, mass, Nmol, shared );
      __syncthreads();
      calcForce( foc_ar, poten_ar, pos_ar, Nmol, rcut, cellsize, lj_sigma, lj_epsilon);
      __syncthreads();
      integVel( t, 11, vel_ar, foc_ar, Nmol, mass, dt * 0.5);
      __syncthreads();
   }
   return;
}// fitVel(...)

//===============================================================
//
//---------------------------------------------------------------
extern "C" __global__ void
integTime(int t0,
	  int Nmol, int step_exch, Real_t dt, Real_t cellsize, Real_t rcut,
	  Real_t lj_sigma, Real_t lj_epsilon, Real_t mass, 
	  Real3_t *d_pos_ar, Real3_t *d_vel_ar, Real3_t *d_foc_ar,
	  Real3_t *d_work_ar, Real_t *d_poten_ar,
	  Real_t  *d_ene_ar, Real_t  *d_temp_ar,Real_t  *d_temp_meas, int *d_exch_ar,
	  FaultConf_t FAULT_CONF) {
   __shared__ Real_t   zeta;
   __shared__ int fault_cnt;
   Real_t calc_err;

   if (threadIdx.x==0 && blockIdx.x==0) {
       printf("Entering %s()\n", __func__);
   }
   if (threadIdx.x==0) {
      fault_cnt = *FAULT_CONF.d_Nfault;
      printf("FAULT_CONF= %d/%d %s.\n", fault_cnt, FAULT_CONF.fault_en, FAULT_CONF.tag);
   }
   
   Real3_t *pos_ar    = d_pos_ar + (Nmol * blockIdx.x);
   Real3_t *vel_ar    = d_vel_ar + (Nmol * blockIdx.x);
   Real3_t *foc_ar    = d_foc_ar + (Nmol * blockIdx.x);
   Real3_t *shared    = d_work_ar + (Nmol * blockIdx.x);
   Real_t  *poten_ar  = d_poten_ar + (Nmol * blockIdx.x);
   Real_t  *ene_ar    = d_ene_ar + (step_exch * blockIdx.x);
   Real_t  *temp_meas = d_temp_meas + (step_exch * blockIdx.x);
   Real_t   temp_targ = d_temp_ar[blockIdx.x];
   int      exch_flag = d_exch_ar[blockIdx.x];
   int    ret_code;
   __shared__ Real_t vel_scale;
   __shared__ Real_t poten_mean;

   __syncthreads();
#if 0
   //<--- checksum
   if (blockIdx.x==0 && threadIdx.x==0) printf("checksum -------------------------------\n");
   int checksum;
   int checksize;
   
   checksize = sizeof(Real3_t) * Nmol;
   checksum  = chksum((int *)pos_ar, checksize);

   if (threadIdx.x==0) printf("checksum(pos_ar[%d])= %+d\n", blockIdx.x, checksum);
   __syncthreads();
   
   checksum  = chksum((int *)vel_ar, checksize);
   if (threadIdx.x==0) printf("checksum(vel_ar[%d])= %+d\n", blockIdx.x, checksum);
   __syncthreads();
   
   checksum  = chksum((int *)foc_ar, checksize);
   if (threadIdx.x==0) printf("checksum(foc_ar[%d])= %+d\n", blockIdx.x, checksum);
   __syncthreads();
   
    //---> checksum
#endif
   Real_t   zeta_sum = 0.0; // unused?
   Real_t   Q        = 70.0;

   // <--- calc LRC
   Real_t   cellsize_pow3 = cellsize * cellsize * cellsize;
   Real_t   sigma_pow3 = lj_sigma * lj_sigma * lj_sigma;
   Real_t   Nmol_pow2 = (Real_t)(Nmol * Nmol);
   Real_t   sigma_rcut = lj_sigma / rcut;
#if defined(REAL_AS_SINGLE)
   Real_t   poten_LRC = (8.0 * M_PI) / (9.0 * cellsize_pow3) *
     Nmol_pow2 * lj_epsilon * sigma_pow3 *
     ( powf(sigma_rcut, 9.0) - 3.0 * powf(sigma_rcut, 3.0) );
   Real_t poten0_LRC = 8.0 *3.1416 * Nmol * Nmol * (2.0 * powf(1.0 / rcut, 9.0)
						    - 3.0*powf(1.0 / rcut, 3.0)) / (9.0 * cellsize_pow3);
#elif defined(REAL_AS_DOUBLE)
   Real_t   poten_LRC = (8.0 * M_PI) / (9.0 * cellsize_pow3) *
     Nmol_pow2 * lj_epsilon * sigma_pow3 *
     ( pow(sigma_rcut, 9.0) - 3.0 * pow(sigma_rcut, 3.0) );
#endif
  // ---> calc_LRC

   __syncthreads();
   if (blockIdx.x==0 && threadIdx.x==0) {
       if (FAULT_CONF.fault_en==0 || fault_cnt==0) {  /* Normal calc */
	   printf("[Normal calculation] (t0=%d)\n", t0);
	   /* nop */
       }
       else { /* Fault calc */
	   printf("[Fault  calculation] (t0=%d)\n", t0);

       }
   }

   /***********************************************************************
    *  <--- FAULT INJECTION
    */
   if (threadIdx.x==0) {
       if (FAULT_CONF.fault_en==0 || fault_cnt==0) {
	   calc_err = 0.0;
       }
       else {
	   if (blockIdx.x==0) { calc_err = +500.0; }
	   if (blockIdx.x==1) { calc_err = -500.0; }
	   if (blockIdx.x==2) { calc_err = +500.0; }
	   if (blockIdx.x==3) { calc_err = -500.0; }
       }
       printf("(%d,%d)calc_err[%d]:blockIdx.x=%d= %f\n", FAULT_CONF.fault_en, fault_cnt, t0, blockIdx.x, calc_err);
   }
   /*
    *  ---> FAULT INJECTION 
    ***********************************************************************/
   
   // if exchanged, scale velocity //
   if (exch_flag == 1) {
      calcVelScale( -1, &vel_scale, temp_targ, vel_ar, mass, Nmol, shared); // vel_scale = shared[0].x
      __syncthreads();
      scaleVelo(-1, vel_ar, vel_scale, Nmol);
      __syncthreads();
      killMomentum(-1, vel_ar, mass, Nmol, shared);
      __syncthreads();
   }

   if (threadIdx.x == 0) zeta = 0.0;

   int t;
   for (t=0; t<step_exch; t++) {            // run "step_exch" steps.
      integVel( 1000*(t0+1) + t, 21, vel_ar, foc_ar, Nmol, mass, dt * 0.5);
      __syncthreads();
      
      ctrlTemp( 1000*(t0+1) + t, 21, vel_ar, zeta,   Nmol, dt * 0.5);
      __syncthreads();
      
      ret_code = integPos( 1000*(t0+1) + t, pos_ar, vel_ar, Nmol, dt, cellsize);
      __syncthreads();
      if (ret_code != 0) {
	 printf("(;_;) serious error in function %s()\n timestep t = %d\n blockIdx.x = %d, threadIdx.x = %d\n", __func__, t, blockIdx.x, threadIdx.x);
	 return;
      }
    
      measTemper( &temp_meas[t], vel_ar, mass, Nmol, shared); // curr_temp => shared[0].x
      __syncthreads();
      //    calcZeta(zeta, temp_meas[t], Q, temp_targ, dt, Nmol);
      if (threadIdx.x == 0) {
	 zeta = (sqrt( temp_meas[t]) - sqrt(temp_targ)) * dt / Q;
	 if ( !isfinite(zeta) ) {
	    printf("t=%d: zeta=%f, blockIdx.x=%d, temp_meas[]=%f, temp_targ=%f\n",
		   t, zeta, blockIdx.x, temp_meas[t], temp_targ);
	 }
      }
      __syncthreads();
      killMomentum(t, vel_ar, mass, Nmol, shared);
      __syncthreads();

      // calculate forces //
      calcForce( foc_ar, poten_ar, pos_ar, Nmol, rcut, cellsize, lj_sigma, lj_epsilon);
      __syncthreads();

      meanPotential( &poten_mean, poten_ar, Nmol, shared); // + (poten_LRC / Nmol);
      __syncthreads();

      if (threadIdx.x == 0) {
	 if (t > step_exch-10) {
	    ene_ar[t] =  poten_mean / 2.0 + calc_err; // to global memory by specified one thread. 2.0;muguruma's paper.
	 } else {
	    ene_ar[t] = poten_mean / 2.0; // to global memory by specified one thread. 2.0;muguruma's paper.
	 }
      }
      __syncthreads();
      integVel( 1000*(t0+1) + t, 22, vel_ar, foc_ar, Nmol, mass, dt * 0.5);
      __syncthreads();
      ctrlTemp( 1000*(t0+1) + t, 22, vel_ar, zeta,   Nmol, dt * 0.5);
      __syncthreads();
   } // for (int t=0; ...
   
   __syncthreads();
   if (blockIdx.x==0 && threadIdx.x==0) {
       if (FAULT_CONF.fault_en>0 && fault_cnt>0) {
	   *FAULT_CONF.d_Nfault = fault_cnt - 1;
       }
   }
	      
} //integTime()
//==============================================================================
static
int checkSum(void *targ, int size) {
    int sum=0;
    int *ptr = (int *)targ;
    for (int s=0; s<size; s+=sizeof(int)) {
	//printf("ptr[%d]= %d\n", s, *ptr);
	sum += *ptr;
	ptr++;
    }
    return sum;
}
// simRemd()
//------------------------------------------------------------------------------
void simRemd( Remd_t &remd, Simu_t &simu ) {
   debug_print(2, "Entering %s().\n", __func__);
   const int MAX_THREADS_PER_BLOCK = 256; // 
   const int MAX_NMOL = 32768;
   
#if !defined(HOST_RUN) && !defined(DEVICE_RUN)
   die("undefined HOST_RUN or DEVICE_RUN.\n");
#endif

   const int    Nrep      = remd.Nrep;
   const int    Nmol      = remd.Nmol;
   const int    Ngpu      = simu.Ngpu;
   const int    Nrep_1dev = simu.Nrep_1dev;
   const int    step_exch = simu.step_exch;
   const Real_t dt        = simu.dt;
   const Real_t cellsize  = remd.cellsize;
   const Real_t rcut      = remd.rcut;
   const Real_t lj_sigma  = remd.lj_sigma;
   const Real_t lj_epsilon = remd.lj_epsilon;
   const Real_t mass      = remd.mass;
   double curr_progress;
   double next_progress;
   double step_progress;
   double elapsed_time_sec;
   int    total_bins = simu.histo_bins;
   int   *histo_ar  = (int *)malloc(sizeof(int) * total_bins * Nrep);
   
   hipError_t cu_err[8];
   
   int  i;
   dim3 blocks(Nrep_1dev, 1, 1);    // GPU grid size
   dim3 threads(1, 1, 1);        // GPU block size

   if ( Nmol < 2 ) {
      die("Nmol is too small.\n");
   } else if ( Nmol <= MAX_THREADS_PER_BLOCK) {
      threads.x = Nmol;
   } else if ( Nmol <= MAX_NMOL) {
      threads.x = MAX_THREADS_PER_BLOCK; // is maximum number.
   } else {
      die("Nmol is too large.\n");
   }

   if( histo_ar == NULL) { die("not enough memory on host.\n"); }
   for ( i=0; i<total_bins * Nrep; i++) histo_ar[i] = 0;
   
   // initialize exch_ar[] //
   for (int rep_i=0; rep_i<Nrep; rep_i++) {
      remd.h_exch_ar[rep_i] = 1;
   }
#if 0 // function test
   g_test01<<<4, 1>>> ( remd.d_work_ar[0], remd.d_poten_ar[0], Nmol);
   return ;
#endif

   copyTempTarg(H2D);
   copyExch(H2D, remd, simu);
   
   // ************************************
   // *  Initialize Temperature on GPU   *
   // ************************************
   printf("[REMD] fitVel() begins.\n");
   for ( i=0; i<Ngpu; i++ ) {
      cu_err[0] = hipSetDevice(i);
      if (cu_err[0] != hipSuccess) { die("hipSetDevice(%d) failed.\n", i ); }
      fitVel <<<blocks, threads>>>
	 (Nmol, step_exch, dt, cellsize, rcut, lj_sigma, lj_epsilon, mass, 
	  remd.d_pos_ar[i], remd.d_vel_ar[i], remd.d_foc_ar[i],
	  remd.d_work_ar[i], remd.d_poten_ar[i],
	  remd.d_energy[i], remd.d_temp_ar[i],remd.d_temp_meas[i],
	  remd.d_exch_ar[i]);
   }
   cu_err[1] = hipGetLastError();
   if (cu_err[1] != hipSuccess) {
       printf("err: %s\n", hipGetErrorString(cu_err[1]));
       exit(1);
   }
   hipDeviceSynchronize();
   printf("[REMD] fitVel() ends.\n");

   // *****************************
   // *  Main integration on GPU  *
   // *****************************
   next_progress = 0.0;
   step_progress = 0.05;


   for ( int t0 = 0; t0 < simu.step_max; t0++ ) {
      printf("###=============================================================\n");
      printf("### t0 = %d / %d\n", t0, simu.step_max-1);
      printf("###=============================================================\n");
      fflush(stdout);
      curr_progress = (double)t0 / (double)simu.step_max;
      if (curr_progress >= next_progress) {
	 printf("---> ******** %s(): simulation progress is now %5.2f %%.\n",
		__func__, curr_progress * 100); fflush(stdout);
	 next_progress += step_progress;
      }
      
#if defined(__DSCUDA__)
      dscudaClearHist();     /*** <--- Clear Recall List.        ***/
      dscudaRecordHistOff();  /*** <--- Enable recording history. ***/ 
#endif
      if (simu.report_posi >= 1)  { savePosAll(t0 * step_exch);      } // cudaMemcpyD2H * Nrep
      if (simu.report_velo >= 1)  { saveVelAll(t0 * step_exch);      } // cudaMemcpyD2H * Nrep
      if (simu.report_force >= 1) { saveFocAll(t0 * step_exch);      } // cudaMemcpyD2H * Nrep
      if (simu.report_temp >= 2)  { saveTempMeasAll(t0 * step_exch); } // cudaMemcpyD2H * Nrep
      //	printf("checksum: Vel[t0=%d]= %d\n",
      //     t0, checkSum((void*)remd.h_vel_ar, sizeof(Real3_t)*Nmol*Nrep)); fflush(stdout);
	
      // Update target temperature of each replica. //
      copyTempTarg( H2D );                                       // cudaMemcpyH2D * ?
      if (simu.report_temp >= 1)   { saveTempTarg(remd, t0); } // cudaMemcpyD2H * ?
      
      // Update exchanging information. //
      copyExch( H2D, remd, simu );                               // cudaMemcpyH2D * ?
      if ( simu.report_ene  >= 1 )   { saveSorted(remd, t0); }   // cudaMemcpyD2H * ?
      
      if ( t0 < 2 ) {
	 FAULT_CONF.fault_en     = 0;
	 FAULT_CONF.overwrite_en = 0;
      } else {
#if defined( FAULT_ON )
	 FAULT_CONF.fault_en     = 1;
#endif
	 FAULT_CONF.overwrite_en = 1;
      }
      
#if defined( __DSCUDA__ )
      dscudaRecordHistOn();  /*** <--- Enable recording history. ***/ 
#endif
      
      //	printf("checksum: Pos[t0=%d before]= %d\n",
      //     t0, checkSum((void*)remd.h_pos_ar, sizeof(Real3_t)*Nmol*Nrep)); fflush(stdout);

      for ( i=0; i<Ngpu; i++ ) {                          // Sweep GPU.
	 cu_err[0] = hipSetDevice(i);
	 if( cu_err[0] != hipSuccess ) { die("hipSetDevice() failed.\n"); }
	 integTime <<< blocks, threads >>>                       // rpcLaunchKernel
	    ( t0, Nmol, step_exch, dt, cellsize, rcut, lj_sigma, lj_epsilon, mass, 
	      remd.d_pos_ar[i], remd.d_vel_ar[i], remd.d_foc_ar[i],
	      remd.d_work_ar[i], remd.d_poten_ar[i],
	      remd.d_energy[i], remd.d_temp_ar[i],remd.d_temp_meas[i],
	      remd.d_exch_ar[i], FAULT_CONF );
      }
      cu_err[1] = hipGetLastError();
      if (cu_err[1] != hipSuccess) {
	 printf("err: %s\n", hipGetErrorString(cu_err[1]));
	 exit(1);
      }

      copyEnergy( D2H, remd, simu );       /* Correct data of potential energy. */
      
#if defined( __DSCUDA__ )
      dscudaRecordHistOff();
#endif
      //savePosAll(t0 * step_exch + 100000);
#if defined( __DSCUDA__ )
      dscudaAutoVerbOn();
      dscudaClearHist();          /*** <--- Clear Recall List.        ***/
#endif
      if( simu.report_ene >= 2)   { saveEne(remd, t0); }
#if 0
      checkEnergyVal( t0, remd.h_energy, Nrep*step_exch);
#endif
      calcHistogram( histo_ar, remd, simu); // struct histogram 
      exchTemp( t0, remd, simu);            // 
   } //for (t = 0; ...
   saveHistogram( histo_ar, remd, simu );
   saveAccRatio( remd, simu.step_max );
   // free
   free(histo_ar);
   debug_print(2, "Exiting  %s().\n", __func__);
}

//===============================================================================
// Parallel Reduction Sum on DEVICE.
//-------------------------------------------------------------------------------
__device__ void reductClear3D( Real3_t *ar, int size ) {
   for (int i = threadIdx.x; i < size; i += blockDim.x) {
      ar[i].x = ar[i].y = ar[i].z = 0.0;
   }
   __syncthreads();
}

__device__ void
reductSet3D( Real3_t *dst, const Real3_t *src, int size ) {
   for (int i = threadIdx.x; i < size; i += blockDim.x) {
      dst[i].x = src[i].x; 
      dst[i].y = src[i].y; 
      dst[i].z = src[i].z;
   }
   __syncthreads();
}
__device__ void
reductSum1D( Real_t *sum, Real_t *ar, int len ) { // must be 2^N, and less than 2049.
   int i, j;
#if 0 // original multi thread
   for (int reduce_num=1024; reduce_num>1; reduce_num /= 2) {
      if (len > reduce_num) {
	 for (int i=threadIdx.x; i<len; i+=blockDim.x) {
	    if (i < reduce_num) {
	       ar[i] += ar[i + reduce_num];
	    }
	 }
	 __syncthreads();
      }
   }
   if (threadIdx.x == 0) { // 2 -> 1
      sum = ar[0] + ar[1];
   }
#elif 1  /*TODO*/
   int jmax;
   
   for ( jmax = len; jmax > 1; jmax = (jmax/2) + (jmax%2)) {
      /* reduct 1st to thread size, len->len/2 or len/2+1 */
      for ( i=threadIdx.x; i<jmax; i+=blockDim.x ) {
	 j = i + (jmax/2) + (jmax%2); //even:(len/2), odd:(len/2+1)
	 if ( j < jmax ) {
	    ar[i] += ar[j];
	 }
      }
      __syncthreads();
   }
   __syncthreads();
   if ( threadIdx.x == 0 ) {
      *sum = ar[0];
   }
#else // single thread; expected slow.
   Real_t buf;
   
   if ( threadIdx.x == 0 ) {
      buf = 0.0;
      for ( i=0; i<len; i++ ) {
	 buf += ar[i];
      }
      *sum = buf;
   }
#endif
}

__device__ void
reductSum3D( Real3_t *sum, Real3_t *ar, int len ) {
   int i,j;

#if 0 // debug
   for (int reduce_num=1024; reduce_num>1; reduce_num /= 2) {  // 2048 -> 2
      if (len > reduce_num) {
	 for ( i=threadIdx.x; i<len; i+=blockDim.x) {
	    if ( i < reduce_num ) {
	       ar[i].x += ar[i + reduce_num].x;
	       ar[i].y += ar[i + reduce_num].y;
	       ar[i].z += ar[i + reduce_num].z;
	    }
	 }
	 __Syncthreads();
      }
   }
   if (threadIdx.x == 0) {                                             // 2 -> 1
      sum.x = ar[0].x + ar[1].x;
      sum.y = ar[0].y + ar[1].y;
      sum.z = ar[0].z + ar[1].z;
   }
#elif 1
   int jmax;
   
   for ( jmax = len; jmax > 1; jmax = (jmax/2) + (jmax%2)) {
      for ( i=threadIdx.x; i<jmax; i+=blockDim.x ) {
	 j = i + (jmax/2) + (jmax%2); //even:(len/2), odd:(len/2+1)
	 if ( j < jmax ) {
	    ar[i].x += ar[j].x;
	    ar[i].y += ar[j].y;
	    ar[i].z += ar[j].z;
	 }
      }
      __syncthreads();
   }
   __syncthreads();
   if ( threadIdx.x == 0 ) {
      sum->x = ar[0].x;
      sum->y = ar[0].y;
      sum->z = ar[0].z;
   }
#else
   Real3_t buf;

   if ( threadIdx.x == 0 ) {
      buf.x = buf.y = buf.z = 0.0; // Reset.
      for ( i=0; i<len; i++ ){
	 buf.x += ar[i].x;
	 buf.y += ar[i].y;
	 buf.z += ar[i].z;
      }
      sum->x = buf.x; 
      sum->y = buf.y; 
      sum->z = buf.z;
   }
#endif
}
//===============================================================================
// integVel(), for HOST and DEVICE.
//-------------------------------------------------------------------------------
__device__ int
integVel( int t, int tag, Real3_t *vel_ar, Real3_t *foc_ar, int Nmol, Real_t mass, Real_t dt) {
   int i;
   
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
#if 1 //debug
      if (!isfinite(vel_ar[i].x) || !isfinite(vel_ar[i].y) || !isfinite(vel_ar[i].z) ||
	  !isfinite(foc_ar[i].x) || !isfinite(foc_ar[i].y) || !isfinite(foc_ar[i].z)) {
	 printf("ERROR: %s(%d), t = %d, vel_ar[%d] = %f %f %f, foc_ar[%d] = %f %f %f. threadIdx.x=%d\n",
		   __func__, tag, t,
		i, vel_ar[i].x, vel_ar[i].y, vel_ar[i].z,
		i, foc_ar[i].x, foc_ar[i].y, foc_ar[i].z, threadIdx.x);
	 return -2;
      }
#endif
      vel_ar[i].x += foc_ar[i].x * dt / mass;
      vel_ar[i].y += foc_ar[i].y * dt / mass;
      vel_ar[i].z += foc_ar[i].z * dt / mass;
   }
   return 0;
}
//==============================================================================
// integPos(), for HOST and DEVECE.
//------------------------------------------------------------------------------
__device__ int
integPos( int t, Real3_t *pos_ar, Real3_t *vel_ar, int Nmol, Real_t dt, Real_t cellsize) {
   Real3_t round;
   int i;
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x) {
#if 1 //debug
      if (!isfinite(vel_ar[i].x) || !isfinite(vel_ar[i].y) || !isfinite(vel_ar[i].z)) {
	 printf("ERROR: %s(%d), vel_ar[%d] = %f %f %f\n",
		__func__, t, i, vel_ar[i].x, vel_ar[i].y, vel_ar[i].z);
	 return -1;
      }
#endif
      pos_ar[i].x += vel_ar[i].x * dt;
      pos_ar[i].y += vel_ar[i].y * dt;
      pos_ar[i].z += vel_ar[i].z * dt;
#if defined(REAL_AS_SINGLE)
      round.x = rintf(pos_ar[i].x / cellsize);
      round.y = rintf(pos_ar[i].y / cellsize);
      round.z = rintf(pos_ar[i].z / cellsize);
#elif defined(REAL_AS_DOUBLE)
      round.x = rint( pos_ar[i].x / cellsize);
      round.y = rint( pos_ar[i].y / cellsize);
      round.z = rint( pos_ar[i].z / cellsize);
#endif
      pos_ar[i].x -= round.x * cellsize;
      pos_ar[i].y -= round.y * cellsize;
      pos_ar[i].z -= round.z * cellsize;

#if 1 // debug
      if (!isfinite(pos_ar[i].x) || !isfinite(pos_ar[i].y) || !isfinite(pos_ar[i].z)) {
	 printf("ERROR: %s(%d), pos_ar[%d] = %f %f %f, threadidx.x=%d\n",
		__func__, t, i, pos_ar[i].x, pos_ar[i].y, pos_ar[i].z, threadIdx.x);
	 return -2;
      }
      if (pos_ar[i].x < (-0.6)*cellsize || pos_ar[i].x > 0.6*cellsize ||
	  pos_ar[i].y < (-0.6)*cellsize || pos_ar[i].y > 0.6*cellsize ||
	  pos_ar[i].z < (-0.6)*cellsize || pos_ar[i].z > 0.6*cellsize) {
	 printf("ERROR:<<<%d,%d>>> %s(%d), pos_ar[%d]= {%f %f %f}, vel_ar[%d]= {%f, %f, %f}, round= {%f %f %f}: threadIdx.x=%d\n",
		blockIdx.x, threadIdx.x, __func__, t,
		i, pos_ar[i].x, pos_ar[i].y, pos_ar[i].z,
		i, vel_ar[i].x, vel_ar[i].y, vel_ar[i].z,
		round.x, round.y, round.z, threadIdx.x);
      return -3;
      }
#endif
   }
   return 0;
}
//===============================================================================
// measTemper()  ! needs reduct !
// molKineticEne(const Real3_t &vel, Real_t mass)
//
__device__ Real_t
molKineticEne( const Real3_t *vel, Real_t mass ) {
   Real_t abs_sq = (vel->x * vel->x) + (vel->y * vel->y) + (vel->z * vel->z);
   Real_t kinetic_ene = 0.5 * mass * abs_sq;
   return kinetic_ene;
}

// *******************/
// **  DEVICE CODE  **/
// *******************/
__device__ void
measTemper( Real_t *temper, const Real3_t *vel_ar, Real_t mass, int Nmol,
	    Real3_t *shared ) {
   Real_t *smem = (Real_t *)shared;
   Real_t meas;
   Real_t scale_factor = UNIT_MASS * (UNIT_LENGTH * UNIT_LENGTH) / (UNIT_TIME * UNIT_TIME);
   Real_t sum;
   int i;

   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      smem[i] = molKineticEne(&vel_ar[i], mass);
   }
   __syncthreads();

   reductSum1D( &sum, smem, Nmol );
   __syncthreads();

   if ( threadIdx.x == 0 ) {
      meas = scale_factor * sum * (2.0 / 3.0) / (Nmol * Boltzmann_constant);
#if 0
      if (meas < 30.0 || meas > 400.0) {
	 printf("%s(): temp_meas[Rep:%d] = %f\n", __func__, blockIdx.x, meas);
	 for (i=0;i<Nmol;i++){
	    printf("%s(): kinetic[%d]=%+6.2f\n", __func__, i, smem[i]);
	 }
      }
#endif
      *temper = meas;
   }
}

//==============================================================================
// calcVelScale()
//------------------------------------------------------------------------------
__device__ void
calcVelScale( int t, Real_t *scale, Real_t targ_temp, Real3_t *vel_ar, Real_t mass,
	      int Nmol, Real3_t *shared ) {
   Real_t *smem = (Real_t *)shared;
   Real_t bunshi = 3.0 / 2.0 * (Real_t)Nmol * targ_temp;
   Real_t unit_scale = (UNIT_TIME / UNIT_LENGTH) * sqrt(Kb / UNIT_MASS);
   Real_t sum, vel_scale;

   for ( int i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      smem[i] = molKineticEne(&vel_ar[i], mass);
   }
   __syncthreads();
#if 0 //Debug
   if (threadIdx.x==0) {
      for (int i=0; i<Nmol; i++) {
	 printf("molkinet[%d]=%f\n", i, smem[i]);
      }
   }
   __syncthreads();
#endif
   reductSum1D( &sum, smem, Nmol ); // shared[0].x <= sum_kinetic_ene
   __syncthreads();

   if ( threadIdx.x == 0 ) {
      vel_scale = sqrt(bunshi / sum) * unit_scale; // = vel_scale
      *scale = vel_scale;
#if 0 //debug
      printf("t=%d: targ_temp= %f, unit_scale= %f, sum=%f, vel_scale= %f\n.", t, targ_temp, unit_scale, sum, vel_scale);
#endif
   }
}
//==============================================================================
// calcMomentum().
//------------------------------------------------------------------------------
__device__ void
calcMomentum(Real3_t *mome, const Real3_t *velo_ar, Real_t mass, int Nmol, Real3_t *shared) {
   Real3_t sum, vel_max, vel_min;
   int i;

   for (i=threadIdx.x; i<Nmol; i+=blockDim.x) {
      shared[i].x = mass * velo_ar[i].x;
      shared[i].y = mass * velo_ar[i].y;
      shared[i].z = mass * velo_ar[i].z;
   }
   __syncthreads();
   
   reductSum3D(&sum, shared, Nmol);                      // shared[0] <= sum.

   if (threadIdx.x == 0) {
      mome->x = sum.x / (Real_t)Nmol;
      mome->y = sum.y / (Real_t)Nmol;
      mome->z = sum.z / (Real_t)Nmol;
#if 0 // debug monitor
      vel_max.x = vel_max.y = vel_max.z = -999.0;
      vel_min.x = vel_min.y = vel_min.z = +999.0;
      for ( i=0; i<Nmol; i++ ) {
	 if ( velo_ar[i].x > vel_max.x ) vel_max.x = velo_ar[i].x;  
	 if ( velo_ar[i].y > vel_max.y ) vel_max.y = velo_ar[i].y;  
	 if ( velo_ar[i].z > vel_max.z ) vel_max.z = velo_ar[i].z;
	 
	 if ( velo_ar[i].x < vel_min.x ) vel_min.x = velo_ar[i].x;  
	 if ( velo_ar[i].y < vel_min.y ) vel_min.y = velo_ar[i].y;  
	 if ( velo_ar[i].z < vel_min.z ) vel_min.z = velo_ar[i].z;  
      }
      printf("momentum= {%e, %e, %e}, Nmol=%d, mass=%f\n",
	     mome->x, mome->y, mome->z, Nmol, mass );
      printf("vel-max= {%+12.6f, %+12.6f, %+12.6f}\n",
	     vel_max.x, vel_max.y, vel_max.z );
      printf("vel-min= {%+12.6f, %+12.6f, %+12.6f}\n",
	     vel_min.x, vel_min.y, vel_min.z );
#endif
   }
   __syncthreads();
}
//==============================================================================
// killMomentum(). 
//------------------------------------------------------------------------------
__device__ void
killMomentum(int t, Real3_t *velo_ar, Real_t mass, int Nmol, Real3_t *shared) {
   int i;
   Real3_t momentum;
   
   calcMomentum( &momentum, velo_ar, mass, Nmol, shared ); // momentum => shared[0]
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      velo_ar[i].x -= momentum.x;
      velo_ar[i].y -= momentum.y;
      velo_ar[i].z -= momentum.z;
#if 0 // debug monitor
      printf("%s(), t=%d, velo_ar[%d] = { %f , %f , %f } --- momentum { %f %f %f }\n",
	     __func__, t, i, velo_ar[i].x, velo_ar[i].y, velo_ar[i].z, momentum->x, momentum->y, momentum->z);
#endif 
   }
   __syncthreads();
}
//==============================================================================
// scaleVelo()
//------------------------------------------------------------------------------
__device__ void
scaleVelo(int t, Real3_t *velo_ar, Real_t scale, int Nmol) {
   int i;
#if 1 // debug: range check of float.
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      if (!isfinite(velo_ar[i].x) || !isfinite(velo_ar[i].y) || !isfinite(velo_ar[i].z)) {
	 printf("[REMD-ERROR] %s(), t = %d, velo_ar[%d] = %f %f %f\n",
		__func__, t, i, velo_ar[i].x, velo_ar[i].y, velo_ar[i].z);
      }
   }
#endif
   if (threadIdx.x==0) {
      printf("t=%d: scale=%f\n", t, scale);
   }
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      velo_ar[i].x *= scale;
      velo_ar[i].y *= scale;
      velo_ar[i].z *= scale;
   }
}

//===============================================================================
__device__ void
debugVel(const char *mes, const Real3_t *vel_ar, int Nmol) {
   if (threadIdx.x == 0) {
      printf("%s(): %s\n", __func__, mes);
      for (int i=0; i<Nmol; i++) {
	 printf("vel_ar[%d]= %f %f %f\n", i, vel_ar[i].x, vel_ar[i].y, vel_ar[i].z);
      }
   }
}

//===============================================================================
// meanPotential
//     Calculate the mean value of each potentials of all atoms.
//     output: 
//         Real_t mean;
//     inputs:
//         Real_t poten_ar[Nmol]: potential values.
//         Real3_t shared[Nmol] : reduct work area.
//-------------------------------------------------------------------------------
__device__ void
meanPotential( Real_t *mean, Real_t *poten_ar, int Nmol, Real3_t *shared ) {
   //Memo: The "mean" must be __shared__ memory pointer.
   Real_t *smem = (Real_t *)shared;
   Real_t sum;
   int i;
   
   /* Clear space */
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      smem[i] = 0.0;
   }
   __syncthreads();
   
   /* Set data */
   for ( i=threadIdx.x; i<Nmol; i+=blockDim.x ) {
      smem[i] = poten_ar[i];
   }
   __syncthreads();
   
   reductSum1D( &sum, smem, Nmol ); // Sum. to sum
   __syncthreads();

   if (threadIdx.x == 0) {
      *mean = sum / (Real_t) Nmol;      // is mean_potential, [J/mol]
   }
}

__device__ int
ctrlTemp(int t, int tag, Real3_t *vel_ar, Real_t zeta, int Nmol, Real_t dt) {
#if defined(REAL_AS_SINGLE)
   //Real_t temp_ctrl = 1.0 - dt * zeta;
   Real_t temp_ctrl = expf(-1.0 * dt * zeta);
#elif defined(REAL_AS_DOUBLE)
   //Real_t temp_ctrl = 1.0 - dt * zeta;
   Real_t temp_ctrl = exp(-1.0 * dt * zeta);
#endif
   for (int i = threadIdx.x; i < Nmol; i += blockDim.x) {
#if 1 //debug
      if (!isfinite(vel_ar[i].x) || !isfinite(vel_ar[i].y) || !isfinite(vel_ar[i].z)) {
	 printf("ERROR: %s(%d) before, t= %d, vel_ar[%d] = %f %f %f, temp_ctrl= %f, zeta = %f\n",
		__func__, tag, t,
		i, vel_ar[i].x, vel_ar[i].y, vel_ar[i].z, temp_ctrl, zeta);
	 return -2;
      }
#endif
      vel_ar[i].x *= temp_ctrl;
      vel_ar[i].y *= temp_ctrl;
      vel_ar[i].z *= temp_ctrl;
#if 1 //debug
      if (!isfinite(vel_ar[i].x) || !isfinite(vel_ar[i].y) || !isfinite(vel_ar[i].z)) {
	 printf("ERROR: %s(%d) after, t= %d, vel_ar[%d] = %f %f %f, temp_ctrl= %f, zeta = %f\n",
	     __func__, tag, t,
		i, vel_ar[i].x, vel_ar[i].y, vel_ar[i].z, temp_ctrl, zeta);
	 return -2;
      }
#endif
   }
   return 0;
}

//==============================================================================
static void
calcZetaSum(Real_t &zeta_sum, Real_t zeta, Real_t dt) {
   zeta_sum += zeta * dt;
}

__device__ void
calcZeta(Real_t &zeta, Real_t curr_temp, Real_t Q, Real_t targ_temp, Real_t dt, int Nmol) {
   // Real_t g = 3.0 * (Real_t)Nmol;
   // zeta += (curr_temp - g * targ_temp) * dt / Q;
   // zeta = (curr_temp - targ_temp) * dt / Q;
   zeta = (sqrt(curr_temp) - sqrt(targ_temp)) * dt / Q;
}
//===============================================================================
//
// histo[Nrep][totol_bins];
// idx_rep[Nrep]
__inline__ static int
histIdxBegin(int rep_i, Simu_t &simu) {
   int total_bins = simu.histo_bins;
   int idx_begin = total_bins * rep_i;
   return idx_begin; 
}
__inline__ static int
histIdxEnd(int rep_i, Simu_t &simu) {
   int total_bins = simu.histo_bins;
   int idx_end = total_bins * (rep_i + 1) - 1;
   return idx_end; 
}

void
checkEnergyVal(int t0, const Real_t *h_energy, int len) {
   //  printf("%s(): check if value of energy is obviously wrong...\n", __func__);
   const Real_t upper_bound = -1.0;
   const int    max_err_cnt = 10;
   int   err_cnt = 0;
   for (int i = 0; i < len; i++) {
      if (h_energy[i] > upper_bound) {
	 if (err_cnt == 0 && i > 0) {
	    printf("%s(): step= %d , h_energy[%d]= %f (just before err).\n", __func__, t0, i-1, h_energy[i-1]);
	 }
	 printf("%s(): step= %d , h_energy[%d]= %f .\n", __func__, t0, i, h_energy[i]);
	 err_cnt++;
	 if (err_cnt > max_err_cnt) {
#if 0
	    die("%s(): detect error %d times\n", __func__, max_err_cnt);
#endif
	 }
      }
   }
  //  printf("%s(): passed.\n", __func__);
}

//===============================================================================
void                                                    // very important //
calcHistogram(int *histo_ar, Remd_t &remd, Simu_t &simu) {
   debug_print(5, "%s(): Entering\n", __func__);

   int    ene_idx;                                    // index of energy array. //
   double offset_ene;
   int    idx_hist;
   int   *sort_temper = remd.sort_temper;

   for (int temp_i = 0; temp_i < remd.Nrep; temp_i++) {
      for (int t = 0; t < simu.step_exch; t++) {
	 ene_idx = (sort_temper[temp_i] * simu.step_exch) + t; //pick a value of energy.
	 if (ene_idx < 0) { 
	    fprintf(stderr, "ERROR in %s: ene_idx must have non-nega value.\n", __func__);
	    continue;
	 }
	 offset_ene = (double)remd.h_energy[ene_idx] - (double)simu.ene_min;
	 debug_print(5, "%s(): remp_i = %d, t = %d, offset_ene = %f\n", __func__, temp_i, t, offset_ene);
	 //idx_hist = (int)(offset_ene / (double)simu.delta_ene) + i * simu.histo_bins;
	 idx_hist = histIdxBegin(temp_i, simu) + (int)(offset_ene / (double)simu.delta_ene);
	 
	 if (idx_hist >= histIdxBegin(temp_i, simu) && idx_hist <= histIdxEnd(temp_i, simu)) {
	    histo_ar[idx_hist] += 1;
	 }
	 else {
#if defined(ENERGY_WARNING) && (ENERGY_WARNING > 0)
	    fprintf(stderr, "(-_-) warning: %s() energy value %f [J/mol] didn't be counted to histogram.\n",
		    __func__, remd.h_energy[ene_idx]);
#endif
	 }
      }
   }
   debug_print(5, "%s(): Exiting\n", __func__);
}
//===============================================================================
void
saveHistogram(const int *hist, Remd_t &remd, Simu_t &simu) {
   debug_print(2, "%s(): Entering\n", __func__);

   FILE *fp;
   char filename[1024];
   int total_bins = simu.histo_bins;
   int ene_i, rep_i;
   int j, j_start, j_stop;

   for (rep_i = 0; rep_i < remd.Nrep; rep_i++) {
      debug_print(5, "%s(): saving rep_i = %d / %d\n", __func__, rep_i, remd.Nrep - 1);
      sprintf(filename, "%s/h%05d.dat", simu.odir_histogram, rep_i);
      if ((fp = fopen(filename, "w")) == NULL) {
	 die("fopen error.\n");
      }
    
      ene_i = 0;
      j_start = histIdxBegin(rep_i, simu);
      j_stop  = histIdxEnd(rep_i, simu);
      for (j=j_start; j<=j_stop; j++) {
	 debug_print(5, "%s(): rep_i = %d, j = %d\n", __func__, rep_i, j);
	 if (j > remd.Nrep * total_bins) {
	    die("too large array_index.--> %d\n", j);
	 }
	 // ******************** //
	 // *  Save HISTOGRAM  * //
	 // ******************** //
	 fprintf(fp, "%7.5f  %15d\n", simu.ene_min + (simu.delta_ene * (Real_t)ene_i), hist[j]);
	 ene_i++;
      }
      fclose(fp);
   }

   debug_print(2, "%s(): Exiting\n", __func__);
}
//===============================================================================
void 
saveAccRatio(Remd_t &remd, int step_numkernel) {
   debug_print(2, "%s(): Entering\n", __func__);

   FILE *fp;
   char savepath[256];
   char filename[256];
   printf("info: saving acceptance ratio to file \"accept_ratio.rep\".\n");
   getPathToSaveRoot(savepath);
   sprintf(filename, "%s/accept_ratio.rep", savepath);
   if ((fp = fopen(filename, "w")) == NULL){
      die("file open error.\n");
   }
   for (int i = 0; i < remd.Nrep - 1; i++) {
      fprintf(fp, "T%d<-->T%d %4.1f\n", i, i + 1,
	      (double)remd.acc_ratio[i] / (double)step_numkernel * 100.0 * 2.0);
   }
   fclose(fp);

   debug_print(2, "%s(): Exiting\n", __func__);
}
//===============================================================================
static void
swapTemp(Real_t *temp_ar, int idx1, int idx2) {
   Real_t buf;
   //printf("%f <--> %f\n", temp_host[idx1], temp_host[idx2]);
   buf = temp_ar[idx1];
   temp_ar[idx1] = temp_ar[idx2];
   temp_ar[idx2] = buf;
}
//===============================================================================
void exchTemp(int t0, Remd_t &remd, Simu_t &simu) {
   int    rep_i, rep_j;
   Real_t ene1, ene2;
   Real_t delta;
   double P, rand;
   int   *sort_temper = remd.sort_temper;

   Real_t  *temp_ar = remd.h_temp_ar;
   Real_t  *energy  = remd.h_energy;
   int      step_exch = simu.step_exch;

   for (int i=0; i<remd.Nrep; i++) {
      remd.h_exch_ar[i] = 0;
   }

   for (int i = t0 % 2; i < remd.Nrep - 1; i+=2) {
      rep_i = sort_temper[i];     // select one neighboring pair
      rep_j = sort_temper[i+1];

      debug_print(5, "%s(): t0= %5d, i= %4d, rep_i= %4d, rep_j= %4d : ", __func__, t0, i, rep_i, rep_j);
	
      delta = (1.0 / temp_ar[rep_j] - 1.0 / temp_ar[rep_i]) *
	      (energy[step_exch * (rep_i + 1) - 1] - energy[step_exch * (rep_j + 1) - 1]);
      P = exp(-delta);
      rand = drand48();
      
      debug_print(5, "%s():P[%3d/%3d]= %f , rand= %f\n", __func__, i, remd.Nrep, P, rand);

      if (P > 1.0 || P > rand) {
	 sort_temper[i]   = rep_j; // swap sort_temper[]
	 sort_temper[i+1] = rep_i; //
	    
	 remd.h_exch_ar[rep_i] = 1;
	 remd.h_exch_ar[rep_j] = 1;	   
 
	 swapTemp(temp_ar, rep_i, rep_j);
	 remd.acc_ratio[rep_i] += 1;
      }
   }
   
   for (int i=0; i<remd.Nrep; i++) {
       printf("%s(): t0=%d, h_exch_ar[%d]= %d\n", __func__, t0, i, remd.h_exch_ar[i]);
   }
   
} // exchTemp

//==============================================================================
// memo: imported from "sub.cu".
//------------------------------------------------------------------------------
static Real_t
hamiltonian(Real_t kinetic, Real_t potential, Real_t mass_stat, Real_t zeta,
	    Real_t zeta_sum, Real_t set_temp, int Nmol) {
   return (kinetic + potential +  
	   0.5 * mass_stat * (zeta * zeta) +
	   3.0 * Nmol * set_temp * zeta_sum);
}

//--- EOF ---
