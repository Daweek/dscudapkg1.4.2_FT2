#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11. 
 *
 */

// Utilities and system includes
#include "sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
#include "shrQATest.h"

#include <hip/hip_runtime.h>

#include "matrixMul.h"

// includes, kernels
#include "matrixMul_kernel.cu"

static char *sSDKsample = "matrixMul";

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line ) {
    if( hipSuccess != err) {
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d.\n", file, line, (int)err);
	exit(-1);
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )  {
    hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d).\n",
                    file, line, errorMessage, (int)err);
            exit(-1);
        }
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId() {
    int current_device   = 0, sm_per_multiproc = 0;
    int max_compute_perf = 0, max_perf_device  = 0;
    int device_count     = 0, best_SM_arch     = 0;
    hipDeviceProp_t deviceProp;

    hipGetDeviceCount( &device_count );
    // Find the best major SM Architecture GPU device
    while ( current_device < device_count ) {
	hipGetDeviceProperties( &deviceProp, current_device );
	if (deviceProp.major > 0 && deviceProp.major < 9999) {
	    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
	}
	current_device++;
    }
    
    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count ) {
	hipGetDeviceProperties( &deviceProp, current_device );
	if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
	    sm_per_multiproc = 1;
	} else {
	    sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	}
	
	int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
	if( compute_perf  > max_compute_perf ) {
	    // If we find GPU with SM major > 2, search only these
	    if ( best_SM_arch > 2 ) {
		// If our device==dest_SM_arch, choose this, or else pass
		if (deviceProp.major == best_SM_arch) {	
		    max_compute_perf  = compute_perf;
		    max_perf_device   = current_device;
		}
	    } else {
		max_compute_perf  = compute_perf;
		max_perf_device   = current_device;
	    }
	}
	++current_device;
    }
    return max_perf_device;
}

// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv) {
    hipDeviceProp_t deviceProp;
    int devID = 0;
    // If the command-line has a device number specified, use it
    if (checkCmdLineFlag(argc, argv, "device")) {
	devID = getCmdLineArgumentInt(argc, argv, "device=");
	if (devID < 0) {
	    printf("Invalid command line parameters\n");
	    exit(-1);
	} else {
	    devID = 0;
	    if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
	    }
	}
    } else {
	// Otherwise pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	checkCudaErrors( hipSetDevice( devID ) );
	checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
	printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
    }
    return devID;
}
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    shrQAStart(argc, argv);
    printf("[ %s ]\n", sSDKsample);

    printf("%s\n\tStarting (CUDA and CUBLAS tests)...\n\n", argv[0]);

    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv) {
    if(checkCmdLineFlag(argc, (const char**)argv, "device"))  {
        int devID = getCmdLineArgumentInt(argc, (const char **)argv, "device=");
        if (devID < 0) {
            printf("Invalid command line parameters\n");
            exit(-1);
        } else {
            devID = 0;
            if (devID < 0) {
               printf("exiting...\n");
               shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
               exit(-1);
            }
        }
    } else {
        checkCudaErrors( hipSetDevice(gpuGetMaxGflopsDeviceId()) );
    }

    int devID = 0;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    // use a larger block size for Fermi and above
    int block_size = (props.major < 2) ? 16 : 32;
    printf("block_size = %d\n", block_size);
    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

    // set seed for rand()
    srand(2006);

    // Optional Command-line multiplier for matrix sizes
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
    int iSizeMultiple = 5;
    if (checkCmdLineFlag( argc, (const char **)argv, "sizemult" )) {
        iSizeMultiple = getCmdLineArgumentInt(argc, (const char**)argv, "sizemult"); 
    }
    
    //iSizeMultiple = CLAMP(iSizeMultiple, 1, 10);
    if ( iSizeMultiple < 1 ) {
	iSizeMultiple = 1;
    } else if ( iSizeMultiple > 128 ) {
	iSizeMultiple = 128;
    }

    // For GPUs with fewer # of SM's, we limit the maximum size of the matrix
    printf("multiProcessorCount = %d\n", props.multiProcessorCount);

    if (props.multiProcessorCount <= 4) {
	uiWA = 2 * block_size * iSizeMultiple;
	uiHA = 4 * block_size * iSizeMultiple;
	uiWB = 2 * block_size * iSizeMultiple;
	uiHB = 4 * block_size * iSizeMultiple;
	uiWC = 2 * block_size * iSizeMultiple;
	uiHC = 4 * block_size * iSizeMultiple;
    } else {
	uiWA = WA * iSizeMultiple;
	uiHA = HA * iSizeMultiple;
	uiWB = WB * iSizeMultiple;
	uiHB = HB * iSizeMultiple;
	uiWC = WC * iSizeMultiple;
	uiHC = HC * iSizeMultiple;
    }
    printf("\nUsing Matrix Sizes: A(%u x %u), B(%u x %u), C(%u x %u)\n\n", 
            uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);

    // allocate host memory for matrices A and B
    unsigned int size_A = uiWA * uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)malloc(mem_size_A);
    unsigned int size_B = uiWB * uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A, *d_B, *d_C;
    unsigned int size_C = uiWC * uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float* h_C      = (float*) malloc(mem_size_C);
    float* h_CUBLAS = (float*) malloc(mem_size_C);

    unsigned int mem_size_DEV = 0;
    checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));
    mem_size_DEV += mem_size_A;
    mem_size_DEV += mem_size_B;

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
    
    checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));
    mem_size_DEV += mem_size_C;
   
    // setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(uiWC / threads.x, uiHC / threads.y);

    // create and start timer
    printf("Runing Kernels...\n\n");

    StopWatchInterface * timer_matrixMul;

    // execute the kernel
    int nIter = 2;
    {
	//Performs warmup operation using matrixMul CUDA kernel
	if (block_size == 16) {
            matrixMul<16><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
        } else {
            matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
        }
        hipDeviceSynchronize();

	// Start Timing	
	sdkCreateTimer(&timer_matrixMul);
	sdkStartTimer(&timer_matrixMul);
	for (int j = 0; j < nIter; j++) {
	    if (block_size == 16) {
		matrixMul<16><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	    } else {
		matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
	    }
	}
	// check if kernel execution generated and error
	getLastCudaError("CUDA matrixMul Kernel execution failed");
	
        hipDeviceSynchronize();
	// stop and destroy timer
	sdkStopTimer(&timer_matrixMul);
	
	double dSeconds0 = sdkGetTimerValue(&timer_matrixMul)/(1000.0);
	double dSeconds = sdkGetTimerValue(&timer_matrixMul)/((double)nIter * 1000.0);
	double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
	double gflops = 1.0e-9 * dNumOps/dSeconds;

	//Log througput, etc
	printf("> CUDA matrixMul %.4f GFlop/s, Time = %.5f s(Elapsed = %.5f s), Size = %.0f Ops.\n", 
	       gflops, dSeconds, dSeconds0, dNumOps);
	
	printf("NumDevsUsed = %d, Workgroup = %u, Elapsed_memsize_DEV = %u [kB]\n", 1, threads.x * threads.y, mem_size_DEV/1000);
	
	sdkDeleteTimer(&timer_matrixMul);
	
	// copy result from device to host
	checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
    }
#if 0 // Bypass comparing with golden.
    // compute reference solution
    printf("\nComparing GPU results with Host computation...\n\n");    
    float* reference = (float*)malloc(mem_size_C);
    computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

    // check result (matrixMul)
    printf("Comparing CUDA matrixMul & Host results\n");
    bool resCUDA = sdkCompareL2fe(reference, h_C, size_C, 1.0e-6f);
    if (resCUDA != true) {
        printDiff(reference, h_C, uiWC, uiHC, 100, 1.0e-5f);
    }
    printf("CUDA matrixMul compares %s\n\n", (true == resCUDA) ? "OK" : "FAIL");
    free(reference);
#endif
    
    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
#if 0 // Bypass comparing with golden.
    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (resCUDA == true) ? QA_PASSED : QA_FAILED);
#endif
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol) {
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;
    for (j = 0; j < height; j++) {
        if (error_count < iListLength) {
            printf("\n  Row %d:\n", j);
        }
        for (i = 0; i < width; i++) {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);
            if (fDiff > fListTol) {                
                if (error_count < iListLength) {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d\n\n", error_count);
}
