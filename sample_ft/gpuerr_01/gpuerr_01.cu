#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>

#define MAX_NGPU (16)

#define MAX_TILES (1024)
#define NUM_TILES (256)
#define TILE_SIZE (1024 * 1024) 

int Ngpu;

__global__
void gpuerr( float *d_tiles, int num_tile, int tile_size ) {
    
}
#if 0
//
//
//
void initTiles( int num_gpu, float **h_tiles, int tile_size ) {
    for (int i=0; i<tile_size; i++) {
	*h_tiles[NUM_TILES*d + i] = (float)i + ((float)num_gpu * NUM_TILES);
    }
}
#endif
//
//
//
void sendTiles( float *h_tiles, float *d_tiles, int num_tiles ) {
    hipError_t cuerr;

    for (int i=0; i<num_tiles; i++) {
	cuerr = hipMemcpy( d_tiles, h_tiles, sizeof(float)*TILE_SIZE, hipMemcpyHostToDevice );
	if (cuerr != hipSuccess) {
	    fprintf( stderr, "hipMemcpy() failed.\n");
	    exit(1);
	}
    }
}
#if 0
//
//
//
void recvTiles( int Ngpu, float *h_tiles, float *d_tiles, int num_tiles ) {
    hipError_t cuerr;

    cuerr = hipMemcpy(, hipMemcpyDeviceToHost );
}
#endif

//
//
//
int main( int argc, char *argv[] )
{
    float *d_tiles[MAX_TILES * MAX_NGPU];
    float *h_tiles[MAX_TILES * MAX_NGPU];

    float *h_val;
    int i, j, k, d;
    hipError_t cuerr;

    Ngpu = 1;

    // malloc@DEV
    for (d=0; d<Ngpu; d++) {
	cuerr = hipSetDevice(d);
	if (cuerr!=hipSuccess) {
	    fprintf( stderr, "hipSetDevice() failed.\n");
	    exit(1);
	}
	for (i=0; i<NUM_TILES; i++) {
	    cuerr = hipMalloc( &d_tiles[NUM_TILES*d + i], sizeof(float)*TILE_SIZE);
	    if (cuerr!=hipSuccess) {
		fprintf( stderr, "hipMalloc() failed.\n");
		exit(1);
	    } else {
		printf(" d_tiles[%3d] = %p\n", NUM_TILES*d+i, d_tiles[i]);
	    }
	}
    }
    // malloc@HOST
    for (d=0; d<Ngpu; d++) {
	for (i=0; i<NUM_TILES; i++) {
	    h_tiles[NUM_TILES*d + i] = (float *)malloc(sizeof(float)*TILE_SIZE);
	    if (h_tiles[NUM_TILES*d + i] == NULL) {
		fprintf(stderr, "malloc() failed.\n");
		exit(1);
	    }
	}
    }

    for (d=0; d<Ngpu; d++) {
	h_val = h_tiles[NUM_TILES * d];
	for (i=0; i<TILE_SIZE; i++) {
	    h_val[i] = (float)i;
	}
    }

#if 0
    for (d=0; d<Ngpu; d++) {
	h_val = h_tiles[NUM_TILES * d];
	for (i=0; i<TILE_SIZE; i++) {
	    printf("h_val[%d] = %f\n", i, h_val[i]);
	}
    }
#endif


    for (d=0; d<Ngpu; d++) {
	cuerr = hipSetDevice(d);
	if (cuerr!=hipSuccess) {
	    fprintf( stderr, "hipSetDevice() failed.\n");
	    exit(1);
	}
	sendTiles( h_tiles, d_tiles, NUM_TILES);
    }
    
#if 0    
    //GPU kernel call
    for (d=0; d<Ngpu; d++) {
	cuerr = hipSetDevice(d);
	if (cuerr!=hipSuccess) {
	    fprintf( stderr, "hipSetDevice() failed.\n");
	    exit(1);
	}
	gpuerr <<<NUM_TILES, 1024>>> ( d_tiles[NUM_TILES*d], NUM_TILES, TILE_SIZE );
    }

    for (d=0; d<Ngpu; d++) {
	cuerr = hipSetDevice(d);
	if (cuerr!=hipSuccess) {
	    fprintf( stderr, "hipSetDevice() failed.\n");
	    exit(1);
	}
	recvTiles( h_tiles, d_tiles, NUM_TILES);
    }
#endif
    // free
    for (d=0; d<Ngpu; d++) {
	cuerr = hipSetDevice(d);
	if (cuerr!=hipSuccess) {
	    fprintf( stderr, "hipSetDevice() failed.\n");
	    exit(1);
	}
	for (i=0; i<16; i++) {
	    cuerr = hipFree( d_tiles[NUM_TILES*d + i] );
	    if (cuerr!=hipSuccess) {
		fprintf( stderr, "%d:hipFree(%d) failed.\n", d, i);
		exit(1);
	    }
	    free( h_tiles[NUM_TILES*d + i] );
	}

    }

    printf("Program completed.\n");
    return 0;
}
