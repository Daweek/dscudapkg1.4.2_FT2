//                             -*- Mode: C++ -*-
// Filename         : libdscuda_rpc.cu
// Description      : DS-CUDA client node library for Remote Procedure Call.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-09-21 17:17:46
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include <stdio.h>
#include <string.h>
#include <netdb.h>
#include <sys/socket.h>
#include <rpc/rpc.h>
#include <time.h>
#include <sys/types.h>
#include <unistd.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <pthread.h>
#include "dscuda.h"
#include "libdscuda.h"

static hipError_t cudaMemcpyD2D(void*, const void*, size_t, VirDev*);
static hipError_t cudaMemcpyP2P(void*, int, const void*, int, size_t);

#define DEBUG 1

int
dscudaRemoteCallType(void) {
    return RC_REMOTECALL_TYPE_RPC;
}

//********************************************************************
// PhyDev::
//********************************************************************
PhyDev::PhyDev(void) {
    id           = -1;
    cid          = -1;
    uniq         = RC_UNIQ_INVALID;
    ft_mode      = FT_UNDEF;
    ft_health    = hl_INIT;
    this->recordON();
    strcpy(ip,       "empty");
    strcpy(hostname, "empty");
    stat_error   = 0;
    stat_correct = 0;
    d_faultconf = NULL;
    Clnt = NULL;
}

void
PhyDev::setIP(const char *ip0) {
    strncpy(this->ip, ip0, sizeof(ip));
}
void
PhyDev::setID(int id0) {
    this->id = id0;
}
void
PhyDev::setCID(int cid0) {
    this->cid = cid0;
}
void
PhyDev::setCID(char *cid_sz) {
    int cid0;
    if ( cid_sz == NULL ) {
	cid0 = 0;
    } else { 
	cid0 = atoi( cid_sz );
    }
    this->cid = cid0;
}
void
PhyDev::setUNIQ(int uniq0) {
    this->uniq = uniq0;
}
void
PhyDev::setFTMODE(FTmode ft_mode0) {
    this->ft_mode = ft_mode0;
}
void
PhyDev::setHealth(FThealth cond) {
    this->ft_health = cond;
}
bool
VirDev::isRecording(void) {
    return this->history_recording;
}
bool
PhyDev::isRecording(void) {
    return this->history_recording;
}
void
VirDev::recordON(void) {
    WARN(10, "<-- VirDev::recordON()\n");
    for (int i=0; i<nredundancy; i++) {
	server[i].recordON();
    }
    history_recording = true;
    WARN(10, "--> VirDev::recordON()\n");
}
void
VirDev::recordOFF(void) {
    for (int i=0; i<nredundancy; i++) {
	server[i].recordOFF();
    }
    history_recording = false;
}
void
PhyDev::recordON(void) {
    history_recording = true;
}
void
PhyDev::recordOFF(void) {
    history_recording = false;
}
void
VirDev::appendRecord(int funcID, void *argp) {
    if (isRecording()) {
	this->reclist.append(funcID, argp);
    }
}
void
PhyDev::appendRecord(int funcID, void *argp) {
    if (isRecording()) {
	this->reclist.append(funcID, argp);
    }
}
int
PhyDev::setupConnection(void)
{
    int  pgid = DSCUDA_PROG;
    char msg[256];

    struct sockaddr_in sockaddr;
    int ssock = RPC_ANYSOCK; // socket to the server for RPC communication.
                             // automatically created by clnttcp_create().
    int sport; // port number of the server. given by the daemon, or calculated from cid.

    St.useRpc();
    if ( St.daemon > 0 ) { // access to the server via daemon.
	WARN(1, "Access port number is informed by daemon.\n");
        sport = requestDaemonForDevice( ip, cid, St.isIbv() );
    } else { // directly access to the server.
	WARN(1, "Access port number is self-defined by client.\n");
        sport = RC_SERVER_IP_PORT + cid;
    }
    if ( sport < 0 ) { // means that maybe daemon program is down.
	return -1;
    }
    
    sockaddr = setupSockaddr( ip, sport );

    this->Clnt = clnttcp_create(&sockaddr,
				pgid,
				DSCUDA_VER,
				&ssock,
				RC_BUFSIZE, RC_BUFSIZE);

    sprintf( msg, "Clnt=%p, %s:%d (port %d) ", Clnt, ip, cid, sport );

    //<--- extend timeout by oikawa
    struct timeval tout;
    tout.tv_sec = 300; // 1000;
    tout.tv_usec = 0;
    clnt_control( this->Clnt, CLSET_TIMEOUT, (char *)&tout);
    //---> extend timeout by oikawa

    if ( Clnt == NULL ) {
        clnt_pcreateerror( msg );
        if ( strcmp(ip, DEFAULT_SVRIP) == 0 ) {
            WARN( 0, "You may need to set an environment variable 'DSCUDA_SERVER'.\n" );
        } else {
            WARN( 0, "DSCUDA server (dscudasrv on %s:%d) may be down.\n", ip, id );
        }
        exit( EXIT_FAILURE );
    }
    WARN(2, "Established a socket connection between %s...\n", msg);
    return 0;
}

void
PhyDev::dupServer(PhyDev *dup) {
    dup->id   = this->id;
    dup->cid  = this->cid;
    dup->uniq = this->uniq;
    dup->ft_mode = this->ft_mode;
    dup->stat_error   = this->stat_error;
    dup->stat_correct = this->stat_correct;
    strcpy( dup->ip, this->ip );
    strcpy( dup->hostname, this->hostname );
}

void
PhyDev::migrateServer(PhyDev *spare) {
    PhyDev tmp;

    dupServer(&tmp);

    this->cid          = spare->cid;
    this->stat_error   = spare->stat_error;
    this->stat_correct = spare->stat_correct;
    strcpy(this->ip, spare->ip);
    strcpy(this->hostname, spare->hostname);

    spare->ft_health    = hl_BAD;
    spare->cid          = tmp.cid;
    spare->stat_error   = tmp.stat_error;
    spare->stat_correct = tmp.stat_correct;
    strcpy(spare->ip, tmp.ip);
    strcpy(spare->hostname, tmp.hostname);

    WARN(1, "***  Reconnect to new physical device\n");
    WARN(1, "***  Old physical device: ip=%s, port=%d\n", spare->ip, spare->cid);
    WARN(1, "***  New physical device: ip=%s, port=%d\n", this->ip,  this->cid); 

    return;
}

void
PhyDev::migrateReallocAllRegions(void) {
    BkupMem *memp = memlist.headPtr();
    int     verb = St.isAutoVerb();
    int     i=0;
    
    WARN(1, "PhyDev::%s(void) {\n", __func__);
    WARN(1, "   + # of realloc region = %d.\n", memlist.getLen());

    struct rpc_err rpc_result;
    while (memp != NULL) {
	this->hipMalloc(&memp->d_region, memp->size, &rpc_result);
	WARN(5, "   + region[%d]: v_ptr=%p, d_ptr=%p(updated), size= %d\n",
	     i, memp->v_region, memp->d_region, memp->size);

	memp = memp->next;
	i++;
    }
    WARN(1, "}\n");
    WARN(1, "\n");
}

void
PhyDev::migrateDeliverAllRegions(void)
{
    BkupMem *memp = memlist.headPtr();
    int     verb = St.isAutoVerb();
    int     copy_count = 0;
    int     i = 0;
    
    WARN(1, "PhyDev::%s(void) {\n", __func__);
    WARN(1, "   + # of deliverd region = %d.\n", memlist.getLen());

    struct rpc_err rpc_result;
    while (memp != NULL) {
	WARN(5, "   + region[%d]: v_ptr=%p, d_ptr=%p, h_ptr=%p, size= %d\n",
	     i, memp->v_region, memp->d_region, memp->h_region, memp->size);
	this->cudaMemcpyH2D(memp->v_region, memp->h_region, memp->size, &rpc_result);
	memp = memp->next;
	i++;
    }
    WARN(1, "}\n");
    WARN(1, "\n");
}

void
PhyDev::migrateDeliverAllModules(void)
{
    WARN(1, "PhyDev::%s(void) {\n", __func__);
    WARN(1, "   + # of deliverd modules = %d.\n", -1000);
    
    WARN(1, "}\n");
    WARN(1, "\n");
}

void
PhyDev::migrateRebuildModulelist(void)
{
    WARN(5, "PhyDev::%s(void) {\n", __func__);
    dscudaLoadModuleResult *rp;
    int module_id;
    struct rpc_err rpc_error;
    
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	if (modulelist[i].valid != 1) {
	    continue;
	}
	WARN(0,"i=%d,checkpoint-0\n", i);
	rp = dscudaloadmoduleid_1(St.getIpAddress(),
				  getpid(),
				  modulelist[i].ptx_data->name,
				  modulelist[i].ptx_data->ptx_image,
				  Clnt);
	WARN(0,"i=%d,checkpoint-1\n",i);
	//<--- RPC Error Hook
	clnt_geterr(Clnt, &rpc_error);
	if (rpc_error.re_status == RPC_SUCCESS) {
	    if (rp == NULL) {
		WARN( 0, "NULL pointer returned, %s(). exit.\n", __func__ );
		clnt_perror(Clnt, ip);
		exit(EXIT_FAILURE);
	    }
	} else {
	    rpcErrorHook(&rpc_error);
	}
	//---> RPC Error Hook.
	
	module_id = rp->id;
	xdr_free((xdrproc_t)xdr_dscudaLoadModuleResult, (char *)rp);
	
	modulelist[i].id        = module_id;
	modulelist[i].sent_time = time(NULL);
    }
    WARN(5, "} //PhyDev::%s(void).\n", __func__);
}

VirDev::VirDev(void)
{
    id          = -1;
    nredundancy = 1;
    ft_mode     = FT_UNDEF;
    conf        = VDEV_INVALID;
    strcpy(info, "INVALID");
    recordOFF();
}

void
VirDev::setFaultMode(enum FTmode fault_mode)
{
    this->ft_mode = fault_mode;
    for (int i=0; i<RC_NREDUNDANCYMAX; i++) {
	server[i].ft_mode = fault_mode;
    }
    return;
}

void
checkResult(void *rp, PhyDev &sp)
{
    if ( rp != NULL ) {
	return;
    } else {
	WARN( 0, "NULL pointer returned, %s(). exit.\n", __func__ );
	clnt_perror( sp.Clnt, sp.ip );
	exit(EXIT_FAILURE);
    }
}
//*
//* Error Handler on RPC communication.
//*
//*
void
PhyDev::rpcErrorHook(struct rpc_err *err)
{
    PhyDev *sp;
    int retval;

    if (err->re_status == RPC_SUCCESS) {
	//Nothing to do.
	//WARN(3, "\"PhyDev::%s():RPC_SUCCESS\".\n", __func__);
	return;
    }
    
    WARN(1, "********************************************************\n");
    WARN(1, "***  detected rpc communication error; ");
    switch (err->re_status) {// *refer to /usr/include/rpc/clnt.h.
    case RPC_CANTSEND: //=3
	WARN0(1, "\"RPC_CANTSEND\".\n");
	break;
    case RPC_CANTRECV: //=4
	WARN0(1, "\"RPC_CANTRECV\".\n");
	break;
    case RPC_TIMEDOUT: //=5
	WARN0(1, "\"RPC_TIMEDOUT\".\n");
	break;
    case RPC_UNKNOWNHOST: //=13
	WARN0(1, "\"RPC_UNKNOWNHOST\".\n");
	break;
    case RPC_UNKNOWNPROTO: //=17
	WARN0(1, "\"RPC_UNKNOWNPROTO\".\n");
	break;
    case RPC_UNKNOWNADDR: //=19
	WARN0(1, "\"RPC_UNKNOWNADDR\".\n");
	break;
    default:
	WARN0(1, "\"RPC_(UNKNOWN-KIND).\n");
	break;
    }
    WARN(1, "***  hostname=\"%s\", ip=%s, server[%d]\n", hostname, ip, id);
    WARN(1, "***  FAULT_TOLERANT_MODE= ");
    
    switch(ft_mode) {
    case FT_NONE: //thru
    case FT_ERRSTAT:
	WARN0(1, "\"FT_NONE\"\n");
	WARN(1, "***  So, I give up to continue calculation, sorry.\n");
	WARN(1, "********************************************************\n");
	exit(EXIT_FAILURE);
    case FT_BYCPY:
	WARN0(1, "\"FT_BYCPY\"\n");
	WARN(1, "***  So, I give up to continue calculation, sorry.\n");
	WARN(1, "********************************************************\n");
	exit(EXIT_FAILURE);
    case FT_BYTIMER:
	WARN0(1, "\"FT_BYTIMER\"\n");
	WARN(1, "***  I am going to migrate to another device.\n");

	do {
	    sp = SvrSpare.findSpareOne();
	    if (sp == NULL) {
		WARN(0, "*** Not found any spare servers.\n");
		exit(EXIT_FAILURE);
	    }
	    WARN(1, "*** Found spare server.\n");
	    WARN(1, "***    + ip = %s:%d\n", sp->ip, sp->cid);
	    migrateServer(sp);
	    retval = setupConnection();
	    if (retval != 0) { //failed to connection.
		sp->ft_health = hl_BAD; // write mark of broken.
		WARN(1, "***    + but looks like broken.\n");
	    }
	} while ( retval != 0 );
	
	this->migrateReallocAllRegions();
	this->migrateRebuildModulelist();
	//
	//this->migrateDeliverAllRegions(); // should be done in VirDevlevel.
	//
	//this->reclist.print();   // should be done in VirDevlevel.
	//this->reclist.recall();  // should be done in VirDevlevel.
	break;
    default:
	WARN0(1, "FT_(UNKNOWN)\n");
	exit(EXIT_FAILURE);
    }
    WARN(1, "********************************************************\n");
    return;
}

/*
 * Dscuda client-side counterpart for CUDA runtime API:
 */

/*
 * Thread Management
 */
hipError_t
hipDeviceReset(void)
{
    dscudaResult *rp;
    int           vid = vdevidIndex();
    hipError_t   err = hipSuccess;

    WARN( 3, "hipDeviceReset() {\n");
    VirDev    *vdev = St.Vdev + Vdevid[vid];  //Focused Vdev
    PhyDev   *sp   = vdev->server;           //Focused Server
//  for (int i = 0; i < vdev->nredundancy; i++, sp++) {
//      rp = dscudathreadexitid_1(Clnt[Vdevid[vid]][sp->id]);
    for ( int i = 0; i < vdev->nredundancy; i++ ) {
        rp = dscudathreadexitid_1( sp[i].Clnt );
        checkResult( rp, sp[i] );
        if ( rp->err != hipSuccess ) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, " } hipDeviceReset()\n\n");

    return err;
}

hipError_t
PhyDev::hipDeviceSynchronize( struct rpc_err *rpc_result)
{
    dscudaResult *rp;
    hipError_t   cuerr = hipSuccess;
    rp = dscudathreadsynchronizeid_1( Clnt );
    clnt_geterr( Clnt, rpc_result );
    if ( rpc_result->re_status == RPC_SUCCESS ) { /*Got response from remote client*/
	if (rp == NULL) {
	    WARN( 0, "NULL pointer returned, %s:%s():L%d.\nexit.\n\n\n",
		  __FILE__, __func__, __LINE__ );
	    clnt_perror(Clnt, ip);
	    exit(EXIT_FAILURE);
	} else {
	    cuerr = (hipError_t)rp->err;
	}
    }
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    return cuerr;
}

hipError_t
VirDev::hipDeviceSynchronize(void)
{
    hipError_t cuerr_phy;
    hipError_t cuerr_vir = hipSuccess;
    struct rpc_err rpc_result;
    
    for (int i=0; i<nredundancy; i++) {
	cuerr_phy = server[i].hipDeviceSynchronize( &rpc_result);
	    server[i].rpcErrorHook( &rpc_result );
	    if ( rpc_result.re_status != RPC_SUCCESS ) {
	    this->restoreMemlist();
	    this->reclist.recall();
	}
	
	if (cuerr_phy != hipSuccess) {
	    WARN(0, "      server[%d].hipDeviceSynchronize() Faild\n", i);
	    cuerr_vir = cuerr_phy;
	    break;
	}
    }
}
    
hipError_t
hipDeviceSynchronize(void)
{
    hipError_t cuerr = hipSuccess;
    int         vid = vdevidIndex();
    VirDev    *vdev = St.Vdev + Vdevid[vid];
    
    WARN( 3, "hipDeviceSynchronize() {\n");
    cuerr = vdev->hipDeviceSynchronize();
    WARN( 3, "} cudaThrreadSynchronize()\n");
    WARN( 3, "\n");
    return cuerr;
}

hipError_t
cudaThreadSetLimit( enum hipLimit_t limit, size_t value)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "cudaThreadSetLimit(%d, %zu)...", limit, value);
    VirDev    *vdev = St.Vdev + Vdevid[vid];
    PhyDev   *sp   = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudathreadsetlimitid_1(limit, value, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
cudaThreadGetLimit(size_t *pValue, enum hipLimit_t limit)
{
    hipError_t err = hipSuccess;
    dscudaThreadGetLimitResult *rp;
    int vid = vdevidIndex();

    WARN(3, "cudaThreadGetLimit(%p, %d)...", pValue, limit);
    VirDev    *vdev = St.Vdev + Vdevid[vid];
    PhyDev   *sp   = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudathreadgetlimitid_1(limit, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pValue = rp->value;
        }
        xdr_free((xdrproc_t)xdr_dscudaThreadGetLimitResult, (char *)rp);
    }
    WARN(3, "done.  *pValue: %zu\n", *pValue);

    return err;
}

hipError_t
hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceSetCacheConfig(%d)...", cacheConfig);
    VirDev    *vdev = St.Vdev + Vdevid[vid];
    PhyDev   *sp   = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudathreadsetcacheconfigid_1(cacheConfig, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig)
{
    hipError_t err = hipSuccess;
    dscudaThreadGetCacheConfigResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceGetCacheConfig(%p)...", pCacheConfig);
    VirDev    *vdev = St.Vdev + Vdevid[vid];
    PhyDev   *sp   = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudathreadgetcacheconfigid_1( sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pCacheConfig = (enum hipFuncCache_t)rp->cacheConfig;
        }
        xdr_free((xdrproc_t)xdr_dscudaThreadGetCacheConfigResult, (char *)rp);
    }
    WARN(3, "done.  *pCacheConfig: %d\n", *pCacheConfig);

    return err;
}


/*
 * Error Handling
 */
hipError_t
hipGetLastError(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(5, "hipGetLastError()...");
    VirDev    *vdev = St.Vdev + Vdevid[vid];
    PhyDev   *sp   = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudagetlasterrorid_1( sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(5, "done.\n");

    return err;
}

hipError_t
hipPeekAtLastError(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(5, "hipPeekAtLastError()...");
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudapeekatlasterrorid_1( sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(5, "done.\n");

    return err;
}

const char
*hipGetErrorString(hipError_t error)
{
    dscudaGetErrorStringResult *rp;
    static char str[4096];
    int vid = vdevidIndex();

    WARN(5, "hipGetErrorString()...");
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudageterrorstringid_1(error, sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (i == 0) {
            strcpy(str, rp->errmsg);
        }
        xdr_free((xdrproc_t)xdr_dscudaGetErrorStringResult, (char *)rp);
    }
    WARN(5, "done.\n");

    return str;
}

/*
 * Device Management
 */

hipError_t
hipSetDeviceFlags(unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipSetDeviceFlags()...");
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev  *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudasetdeviceflagsid_1(flags, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    //invalidateModuleCache();
    for (int i=0; i<St.Nvdev; i++) {
	St.Vdev[i].invalidateAllModuleCache();
    }

    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDriverGetVersion (int *driverVersion)
{
    hipError_t err = hipSuccess;
    dscudaDriverGetVersionResult *rp;
    int vid = vdevidIndex();

    WARN(3, "cudaDriverGetVersionCount(%p)...", driverVersion);
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudadrivergetversionid_1( sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *driverVersion = rp->ver;
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipRuntimeGetVersion(int *runtimeVersion)
{
    hipError_t err = hipSuccess;
    dscudaRuntimeGetVersionResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipRuntimeGetVersion(%p)...", runtimeVersion);
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaruntimegetversionid_1( sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *runtimeVersion = rp->ver;
        }
        xdr_free((xdrproc_t)xdr_dscudaRuntimeGetVersionResult, (char *)rp);
    }

    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceSynchronize(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceSynchronize()...");
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudadevicesynchronize_1( sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceReset(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceReset()...");
    VirDev *vdev = St.Vdev + Vdevid[vid];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudadevicereset_1(sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

/*
 * Execution Control
 */

hipError_t
hipFuncSetCacheConfig(reinterpret_cast<const void*>(const char * func), enum hipFuncCache_t cacheConfig)
{
    hipError_t err = hipSuccess;
    WARN(1, "Current implementation of hipFuncSetCacheConfig() does nothing "
         "but returning hipSuccess.\n");
    err = hipSuccess;
    return err;
}

/**-----------------------------------------------------------------------------
 * Memory Management
 * - "hipMalloc()" was composed hierarchical with same name of methods;
 *   hipMalloc() +--> VirDev::hipMalloc() +--> PhyDev::hipMalloc().
 */
hipError_t
hipMalloc(void **d_ptr, size_t size)
{
    hipError_t cuerr;
    int         vid  = vdevidIndex();
    VirDev     *vdev = St.Vdev + Vdevid[vid];
    void       *adrs;

    WARN(3, "hipMalloc(%p, %zu) on Vdev[%d] {\n", d_ptr, size, Vdevid[vid]);
    cuerr = vdev->hipMalloc(d_ptr, size);
    WARN(3, "}\n", d_ptr, size);
    WARN(3, "\n", d_ptr, size);
    return cuerr;
}
hipError_t
VirDev::hipMalloc(void **d_ptr, size_t size)
{
    hipError_t cuerr_phy;
    hipError_t cuerr_vir = hipSuccess;
    void       *adrs[RC_NREDUNDANCYMAX];
    void       *uva_ptr = NULL;

    WARN(3, "   Vdev[%d].hipMalloc(%p, %zu) nredundancy=%d {\n",
	 id, d_ptr, size, nredundancy);
    /*
     * Record called history of CUDA APIs.
     */
    if (isRecording()) {
	WARN(3, "      ===> Recorded to the rollback history.\n");
	CudaMallocArgs args;
	args.devPtr = uva_ptr;
	args.size   = size;
	appendRecord(dscudaMallocId, &args);
    }
    struct rpc_err rpc_result;
    for (int i=0; i<nredundancy; i++) {
	/*
	 * Virtual device looks like one device but has
	 * nredundancy devices.
	 */
	cuerr_phy = server[i].hipMalloc( &adrs[i], size, &rpc_result);
	server[i].rpcErrorHook( &rpc_result);
	if (rpc_result.re_status != RPC_SUCCESS) {
	    this->restoreMemlist();
	    this->reclist.recall();
	}
	
	WARN(3, "      Phy[%d]: d_ptr=%p\n", i, adrs[i]);
	if (cuerr_phy != hipSuccess) {
	    WARN(0, "      svr[%d].hipMalloc() Faild\n", i);
	    cuerr_vir = cuerr_phy;
	    break;
	}
	
	if (i==0) { // The 1st of redundants servers.
	    uva_ptr = dscudaUvaOfAdr(adrs[0], id);
	}
	server[i].memlist.append( uva_ptr, adrs[i], size); // record two addresses: virtual and physial, and its size. 
	WARN(3, "         + memlist.add(v_ptr=%p, d_ptr=%p, size=%zu)\n", uva_ptr, adrs[i], size);
    }

    this->memlist.append(uva_ptr, NULL, size); // Virtual device does not manage the physical address.

    *d_ptr = uva_ptr; // Return UVA address of physical[0].
    WARN(3, "   }\n");
    return cuerr_vir;
}

hipError_t
PhyDev::hipMalloc(void **d_ptr, size_t size, struct rpc_err *rpc_result)
{
    dscudaMallocResult *rp;
    hipError_t cuerr = hipSuccess;
#if 0
    if (isRecording()) {
	CudaMallocArgs args;
	args.devPtr = uva_ptr;
	args.size   = size;
	this->appendRecord(dscudaMallocId, &args);
    }
#endif
    
    rp = dscudamallocid_1( size, Clnt); //Kick RPC
    clnt_geterr( Clnt, rpc_result);
    if ( rpc_result->re_status == RPC_SUCCESS ) { /*Got response from remote client*/
	if (rp == NULL) {
	    WARN( 0, "NULL pointer returned, %s:%s():L%d.\nexit.\n\n\n", __FILE__, __func__, __LINE__ );
	    clnt_perror(Clnt, ip);
	    exit(EXIT_FAILURE);
	} else {
	    cuerr = (hipError_t)rp->err;
	}
    }
    *d_ptr = (void*)rp->devAdr;
    xdr_free((xdrproc_t)xdr_dscudaMallocResult, (char *)rp);
    return cuerr;
}

/*
 * hipFree() series.
 */
hipError_t
hipFree(void *d_ptr) {
    int          vid = vdevidIndex();
    hipError_t  err = hipSuccess;

    WARN(3, "hipFree(%p) {\n", d_ptr);
    VirDev     *vdev = St.Vdev + Vdevid[vid];

    err = vdev->hipFree(d_ptr);

    WARN(3, "}\n");
    WARN(3, "\n");
    return err;
}
hipError_t
VirDev::hipFree(void *v_ptr) {
    hipError_t  err = hipSuccess;

    WARN(3, "   + Vir[%d].hipFree(%p) {\n", id, v_ptr);
    struct rpc_err rpc_result;
    for (int i=0; i<nredundancy; i++) {
	err = server[i].hipFree(v_ptr, &rpc_result);
	server[i].rpcErrorHook( &rpc_result );
	if (rpc_result.re_status != RPC_SUCCESS ) {
	    this->restoreMemlist();
	    this->reclist.recall();
	}
	
	server[i].memlist.remove(v_ptr);
    }
    this->memlist.remove(v_ptr);

    /*
     * Record called history of CUDA APIs.
     */
    if (isRecording()) {
	CudaFreeArgs args;
	args.devPtr = v_ptr;
	appendRecord(dscudaFreeId, &args);
    }
    WARN(3, "   + }\n");
    return err;
}
hipError_t
PhyDev::hipFree(void *v_ptr, struct rpc_err *rpc_result) {
    hipError_t  err = hipSuccess;
    dscudaResult *rp;
    void *d_ptr = memlist.queryDevicePtr(v_ptr);
    
    WARN(3, "      + Phy[%d].hipFree(%p) { }\n", id, d_ptr);

    rp = dscudafreeid_1((RCadr)d_ptr, Clnt);
    clnt_geterr( Clnt, rpc_result );
    if ( rpc_result->re_status == RPC_SUCCESS ) {
	if (rp == NULL) {
	    WARN( 0, "NULL pointer returned, %s(). exit.\n", __func__ );
	    clnt_perror(Clnt, ip);
	    exit(EXIT_FAILURE);
	} else {
	    err = (hipError_t)rp->err;
	}
    }
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return err;
}


/*
 * hipMemcpy( HostToDevice )
 */
hipError_t
hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind) {
    RCuva      *suva, *duva;
    int         dev0;
    hipError_t err  = hipSuccess;
    void       *lsrc = dscudaAdrOfUva((void *)src);
    void       *ldst = dscudaAdrOfUva(dst);

    int         vdevid = Vdevid[ vdevidIndex() ];
    VirDev     *vdev   = St.Vdev + vdevid;
    switch (kind) {
    case hipMemcpyDeviceToHost:
	WARN(3, "hipMemcpy(%p, %p, %zu, D->H) called vdevid=%d...\n",
	     dst, src, count, vdevid);
	pthread_mutex_lock( &cudaMemcpyD2H_mutex );
	err = vdev->cudaMemcpyD2H(dst, src, count);
	pthread_mutex_unlock( &cudaMemcpyD2H_mutex ); 
	break;
    case hipMemcpyHostToDevice:
	WARN(3, "hipMemcpy(%p, %p, %zu, H->D) called\n", ldst, lsrc, count);
	pthread_mutex_lock( &cudaMemcpyH2D_mutex );
	err = vdev->cudaMemcpyH2D(dst, src, count);
	pthread_mutex_unlock( &cudaMemcpyH2D_mutex );
	break;
    case hipMemcpyDeviceToDevice:
	WARN(3, "hipMemcpy(%p, %p, %zu, DeviceToDevice) called\n", ldst, lsrc, count);
	err = cudaMemcpyD2D(ldst, lsrc, count, vdev );
	break;
    case hipMemcpyDefault: //thru
#if !__LP64__
	WARN(0, "hipMemcpy:In 32-bit environment, hipMemcpyDefault cannot be given as arg4."
             "UVA is supported for 64-bit environment only.\n");
        exit(1);
#endif
    default:
	WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
	exit(1);
    }
    WARN(3, "} %s().\n", __func__);
    WARN(3, "\n");
    return err;
}//-->hipMemcpy()
hipError_t
VirDev::cudaMemcpyH2D(void *v_ptr, const void *h_ptr, size_t count) {
    WARN(4, "   Vdev[%d].H2D() {\n", id);
    hipError_t    cuda_error = hipSuccess;
    struct rpc_err rpc_result;

    //--- Record called history of CUDA APIs.
    if (isRecording()) {
	CudaMemcpyArgs args;
	WARN(4, "      ===> Recorded to the rollback history.\n", id);
	args.dst   = v_ptr;
	args.src   = (void *)h_ptr;
	args.count = count;
	args.kind  = hipMemcpyHostToDevice;
	appendRecord(dscudaMemcpyH2DId, &args);
    }
    //--- Exec each physical devices.
    for (int i=0; i<nredundancy; i++) {
	server[i].cudaMemcpyH2D(v_ptr, h_ptr, count, &rpc_result);
        server[i].rpcErrorHook( &rpc_result );
	if (rpc_result.re_status != RPC_SUCCESS) {
	    this->restoreMemlist();
	    this->reclist.recall();
	}
    }
    WARN(4, "   }\n");
    if (isRecording()) {
#if 1
	reclist.print();
#endif
    }
    return cuda_error;
}//--> VirDev::cudaMemcpyH2D()
hipError_t
PhyDev::cudaMemcpyH2D(void *v_ptr, const void *h_ptr, size_t count, struct rpc_err *rpc_result)
{
    dscudaResult *rp;
    RCbuf srcbuf;
    void *d_ptr;
    hipError_t cuda_error;

    srcbuf.RCbuf_len = count;
    srcbuf.RCbuf_val = (char *)h_ptr;

    //<-- Translate virtual v_ptr to real d_ptr.
    d_ptr = memlist.queryDevicePtr(v_ptr);
    //--> Translate virtual v_ptr to real d_ptr.
    WARN(5, "      + Phy[%d]:H2D(v_ptr=%p ==> d_ptr=%p, size=%zu)\n",
	 id, v_ptr, d_ptr, count);

    rp = dscudamemcpyh2did_1((RCadr)d_ptr, srcbuf, count, Clnt); //Kick RPC.
    
    //<--    RPC fault check.
    clnt_geterr( Clnt, rpc_result );
    if ( rpc_result->re_status == RPC_SUCCESS ) { /*Got response from remote client*/
	if ( rp == NULL ) {
	    WARN( 0, "NULL pointer returned, %s:%s():L%d.\nexit.\n\n\n", __FILE__, __func__, __LINE__ );
	    clnt_perror(Clnt, ip);
	    exit(EXIT_FAILURE);
	} else {
	    cuda_error = (hipError_t)rp->err;
	}
    }
    //-->    RPC fault check.
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    
    //--> RPC communication.

    //****>>> Record called history of CUDA APIs. <<<***********
    CudaMemcpyArgs args;
    if (isRecording()) {
	args.dst   = v_ptr;
	args.src   = (void *)h_ptr;
	args.count = count;
	args.kind  = hipMemcpyHostToDevice;
	reclist.append(dscudaMemcpyH2DId, &args);
    }

    return cuda_error;
}//-->PhyDev::cudaMemcpyH2D()
hipError_t
VirDev::cudaMemcpyD2H(void *dst, const void *src, size_t count) {
    WARN(4, "   Vir[%d]:D2H() {\n", id);
    int i, k;

    hipError_t err = hipSuccess;

    //--- Record called history of CUDA APIs.
    CudaMemcpyArgs args;
    if (isRecording()) {
	WARN(4, "      ===> Recorded to the rollback history.\n", id);
	args.dst   = (void *)dst;
	args.src   = (void *)src;
	args.count = count;
	args.kind  = hipMemcpyDeviceToHost;
	this->reclist.append(dscudaMemcpyD2HId, &args);
    } 
    int matched_count   = 0;
    int unmatched_count = 0;

    int all_matched = 1;
    int recall_result;

    // "Table"
    //   ft_mode   | VDEV_MONO  VDEV_POLY 
    //-------------+-----------------------
    //  FT_NONE    |   svr[0]    svr[0] 
    //  FT_ERRSTAT |   svr[0]    svr[N]
    //  FT_BYCPY   |   svr[0]    svr[N]
    //  FT_BYTIMER |   svr[0]    svr[0]
    //

    //switch behavior by
    //   "ft.d2h_reduncpy", "ft.d2h_compare", "ft.d2h_statics", "ft.d2h_rollback"

    //<--- Copy from physical device
    struct rpc_err rpc_result;
    int num_reduncpy = (this->ft.d2h_reduncpy)? this->nredundancy : 1;
    for (i=0; i < num_reduncpy; i++) {
	server[i].cudaMemcpyD2H( dst, src, count, &rpc_result );
	server[i].rpcErrorHook( &rpc_result );
	if (rpc_result.re_status != RPC_SUCCESS) {
	    this->restoreMemlist();
	    this->reclist.recall();
	}
    }
    if (!ft.d2h_reduncpy) {
	//--- Return device data to user application region.
	memcpy( dst, server[0].memlist.queryHostPtr(src), count );
	WARN(4, "   }\n");
    }
    //---> Copy from physical device

    //<--- Compare gathered data
    int memcmp_ret;
    if (ft.d2h_compare && (num_reduncpy>=2)) {
	for (i=0; i < num_reduncpy-1; i++) {
	    for (k=i+1; k < num_reduncpy; k++) {
		//--- Compare ByteToByte
		memcmp_ret = memcmp( server[i].memlist.queryHostPtr(src),
				     server[k].memlist.queryHostPtr(src), count );
		if (memcmp_ret == 0) {
		    server[k].stat_correct++;
		}
		else {
		    server[k].stat_error++;
		    all_matched = 0;
		    WARN( 2, "   Statistics: \n");
		}
		WARN( 2, "   UNMATCHED redundant device %d/%d with device 0. %s()\n", i, nredundancy - 1, __func__);
	    }//for (k=...
	}//for (i=...	
    }
    //---> Compare gathered data
    


    if (this->ft.d2h_reduncpy) {
	if (all_matched==1) {
	    WARN(5, "   #\\(^_^)/ All %d Redundant device(s) matched. statics OK/NG = %d/%d.\n",
		 nredundancy-1, matched_count, unmatched_count);
	    memcpy( dst, server[0].memlist.queryHostPtr(src), count );
	} else {
	    if ( unmatched_count>0 && matched_count<(nredundancy-1)) {
		WARN( 1, " #   #\n");
		WARN( 1, "  # #\n");
		WARN( 1, "   #  Detected Unmatched result. OK/NG= %d/%d.\n", matched_count, unmatched_count);
		WARN( 1, "  # #\n");
		WARN( 1, " #   #\n");
	    } else {
		WARN(1, "   #(;_;)   All %d Redundant device(s) unmathed. statics OK/NG = %d/%d.\n",
		     nredundancy-1, matched_count, unmatched_count);
	    }
	    if (St.isHistoCalling()==0) {
		St.unsetAutoVerb();    // <=== Must be disabled autoVerb during Historical Call.
	    
		//TODO: rewrite BKUPMEM.restructDeviceRegion();
		//TODO: recall_result = HISTREC.recall();
	    
		if (recall_result != 0) {
		    printModuleList();
		    printVirtualDeviceList();
		}
		//HISTREC.on();  // ---> restore recordHist enable.
		St.setAutoVerb();    // ===> restore autoVerb enabled.
	    }
	}
    }//if (this->ft.d2h_reduncpy)
    
    WARN(4, "   }\n");
    return err;
}//-->VirDev::cudaMemcpyD2H(...)
hipError_t
PhyDev::cudaMemcpyD2H( void *h_ptr, const void *v_ptr, size_t count,
		       struct rpc_err *p_rpc_result) {
    CudaMemcpyArgs args;
    if (isRecording()) {
	args.dst   = (void *)h_ptr;
	args.src   = (void *)v_ptr;
	args.count = count;
	args.kind  = hipMemcpyDeviceToHost;
	reclist.append(dscudaMemcpyD2HId, (void *)&args);
    }
    
    //<-- Translate virtual d_ptr to real d_ptr.
    void *h_lptr = this->memlist.queryHostPtr(v_ptr);
    void *d_ptr  = this->memlist.queryDevicePtr(v_ptr);
    WARN(4, "      + Phy[%d]:D2H( dst=%p, src=%p, count=%zu )\n",
	 id, h_lptr, d_ptr, count);
    if (d_ptr==NULL) {//Unexpected error.
	WARN(0, "%s():d_ptr = NULL.\n", __func__);
	exit(1);
    }
    if (h_lptr==NULL) {//Unexpected error.
	WARN(0, "%s():h_lptr = NULL.\n", __func__);
	exit(1);
    }
    //--> Translate virtual d_ptr to real d_ptr.

    //<-- Kick RPC!
    dscudaMemcpyD2HResult *rp = dscudamemcpyd2hid_1((RCadr)d_ptr, count, Clnt);
    //--> Kick RPC!

    //<--- RPC fault check.
    hipError_t cuda_error;
    clnt_geterr(this->Clnt, p_rpc_result);
    if (p_rpc_result->re_status==RPC_SUCCESS) {//RPC was Completed successfully.
	if (rp==NULL) {//NULL returned from hipMemcpy() executed on remote host.
	    WARN(0, "NULL pointer returned, %s:%s():L%d.\nexit.\n\n\n",
		 __FILE__, __func__, __LINE__ );
	    clnt_perror(Clnt, ip);
	    exit(EXIT_FAILURE);
	}
	else {
	    cuda_error = (hipError_t)rp->err;
	}
    }
    //--> RPC fault check.

    memcpy( h_lptr,  rp->buf.RCbuf_val,  rp->buf.RCbuf_len );
    xdr_free( (xdrproc_t)xdr_dscudaMemcpyD2HResult, (char *)rp );
    return cuda_error;
}//--> PhyDev::cudaMemcpyD2H()
void
PhyDev::collectEntireRegions(void) {
    WARN_CP(9, "      + PhyDev[%d]::%s() {\n", id, __func__);
    dscudaMemcpyD2HResult *rp;
    struct rpc_err rpc_error;
    hipError_t    cuda_error;
    void *d_ptr;
    void *h_ptr;
    int   size;
    int   i = 0;

    BkupMem *bkupmem = memlist.headPtr();
    while (bkupmem != NULL) {
	d_ptr = bkupmem->d_region;
	h_ptr = bkupmem->h_region;
	size  = bkupmem->size;
	WARN_CP(9, "         + correct region[%d], %d[Byte]...", i, size);
	rp = dscudamemcpyd2hid_1( (RCadr)d_ptr, size, Clnt );
	WARN0(9, "done.\n");
	
	//<--- RPC fault check.
	clnt_geterr(Clnt, &rpc_error);
	if (rpc_error.re_status == RPC_SUCCESS) {
	    if (rp == NULL) {
		WARN_CP( 0, "NULL pointer returned, %s:%s():L%d.\nexit.\n\n\n", __FILE__, __func__, __LINE__ );
		clnt_perror(Clnt, ip);
		exit(EXIT_FAILURE);
	    } else {
		cuda_error = (hipError_t)rp->err;
	    }
	}
#if 0 // temporary disable.
	else {
	    rpcErrorHook(&rpc_error);	
	}
#endif
	//--> RPC fault check.
	
	memcpy(h_ptr, rp->buf.RCbuf_val, rp->buf.RCbuf_len);
	xdr_free( (xdrproc_t)xdr_dscudaMemcpyD2HResult, (char *)rp );
	//--> RPC communication.
	
	bkupmem = bkupmem->next;
	i++;
    } // while (...);
    WARN_CP(9, "      + } PhyDev[%d]::%s()\n", id, __func__);
} // ---> void PhyDev::collectEntireRegions(void)

void
VirDev::collectEntireRegions(void) {
    WARN_CP(9, "   + VirDev[%d]::%s()\n", id, __func__);
    for (int n=0; n<nredundancy; n++) {
	server[n].collectEntireRegions();
    }
    WARN_CP(9, "   + } VirDev[%d]::%s()\n", id, __func__);
}

void
ClientState::collectEntireRegions(void) {
    WARN_CP(9, "ClientState::%s() {\n", __func__);
    
    for (int n=0; n<Nvdev; n++) {
	Vdev[n].collectEntireRegions();
    }
    
    WARN_CP(9, "} ClientState::%s()\n", __func__);
}

int
VirDev::verifyEntireRegions(void) {
    WARN_CP(9, "   + VirDev::%s() {\n", __func__);
    void *v_region;
    void *h_ptr_i;
    void *h_ptr_j;
    int   size;
    int   all_matched = 1;

    BkupMem *bkupmem = memlist.headPtr();
    while (bkupmem != NULL) {
	v_region = bkupmem->v_region;
	size     = bkupmem->size;
	for (int i=0; i<nredundancy-1; i++) {
	    h_ptr_i = server[i].memlist.queryHostPtr(v_region);
	    if (h_ptr_i == NULL) {
		WARN_CP(0, "%s():not found host pointer.\n");
		exit(-1);
	    }
	    for (int j=i+1; j<nredundancy; j++) {
		h_ptr_j = server[j].memlist.queryHostPtr(v_region);
		if (h_ptr_j == NULL) {
		    WARN_CP(0, "%s():not found host pointer.\n");
		    exit(-1);
		}
		WARN_CP(9, "      + memcmp(phy[%d], phy[%d], %d[Byte])\n",
		     i, j, size);
		if ( memcmp( h_ptr_i, h_ptr_j, size) != 0 ) {
		    all_matched = -1;
		}
	    }
	}
	bkupmem = bkupmem->next;
    }//while
    if (all_matched) {
	WARN_CP(9, "   + ALL MATCHED\n");
    } else {
	WARN_CP(9, "   + NOT MATCHED *************************\n");
    }
    WARN_CP(9, "   + } VirDev::%s()\n", __func__);
    return all_matched;
}

int
ClientState::verifyEntireRegions(void) {
    WARN_CP(9, "ClientState::%s() {\n", __func__);
    int virdev_matched;
    int all_devices_matched = 1;
    
    for (int n=0; n<Nvdev; n++) {
	virdev_matched = Vdev[n].verifyEntireRegions();
	if (virdev_matched != 1) {
	    all_devices_matched = -1;
	}
    }
    
    WARN_CP(9, "} ClientState::%s()\n", __func__);
    return all_devices_matched;
}

//
// 
// Update the host memory region of reliable data.
// *only called in periodic-checkpointing thread.
//
void
VirDev::updateMemlist(int svr_id) {
    WARN(9, "VirDev::updateMemlist(%d) {\n", svr_id);
    PhyDev *svr_ptr;
    BkupMem    *mem_ptr;
    void       *v_ptr, *h_src, *h_dst;
    int         size;

    if (svr_id >= RC_NREDUNDANCYMAX) {
	WARN(0, "VirDev::updateMemlist(): Too large server index(%d). exit.", svr_id);
	exit(1);
    }
    
    mem_ptr = memlist.headPtr();
    svr_ptr = &server[svr_id];
    int i = 0;
    while (mem_ptr != NULL) {
	h_dst = mem_ptr->h_region;
	size  = mem_ptr->size;
	h_src = svr_ptr->memlist.queryHostPtr(mem_ptr->v_region);
	memcpy( h_dst, h_src, size );
	WARN(3, "   + region[%d]: %d[Byte] updated.\n", i, size);

	mem_ptr = mem_ptr->next;
	i++;
    }
    WARN(9, "VirDev::updateMemlist(%d) {\n", svr_id);
}
//
// Clear all the CUDA API called history in vdevs and pdevs.
// ***> only called in periodic-checkpointing thread <***
//
void VirDev::clearReclist(void) {
    for (int i=0; i<nredundancy; i++) {
	server[i].reclist.clear();
    }
    reclist.clear();
}
//
// Restore reliable data into the global memory region of GPU devices.
// This function is inverse of "unpdateMemlist()".
// ***> only called in periodic-checkpointing thread <***
//
void
VirDev::restoreMemlist(void) {
    WARN(9, "VirDev[%d]::%s() {\n", this->id, __func__);
    BkupMem   *mem_ptr;
    void      *v_ptr;
    void      *h_src;
    int        size;
    int        rec_en_stack;
    struct rpc_err rpc_result;

    mem_ptr = memlist.headPtr();    
    while ( mem_ptr != NULL ) {
	v_ptr = mem_ptr->v_region;
	h_src = mem_ptr->h_region;
	size  = mem_ptr->size;
	for (int i=0; i<nredundancy; i++) {
	    rec_en_stack = server[i].isRecording();
	    server[i].recordOFF();
	    server[i].cudaMemcpyH2D(v_ptr, h_src, size, &rpc_result);
	    if (rec_en_stack) server[i].recordON();
	    else              server[i].recordOFF();
	}
	mem_ptr = mem_ptr->next;
    }
    WARN(9, "} VirDev[%d]::%s().\n", this->id, __func__);
    return;
}
//
// Rollback and redo the CUDA API called histories.
//
//
static hipError_t
cudaMemcpyD2D(void *dst, const void *src, size_t count, VirDev *vdev ) {
    dscudaResult *rp;
    hipError_t err = hipSuccess;

    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudamemcpyd2did_1((RCadr)dst, (RCadr)src, count, sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    //<--- oikawa moved to here from hipMemcpy();
    if (St.isAutoVerb() > 0) {
	CudaMemcpyArgs args( dst, (void *)src, count, hipMemcpyDeviceToDevice );
	//HISTREC.append(dscudaMemcpyD2DId, (void *)&args);
    }
    //--->
    return err;
}

static hipError_t
cudaMemcpyP2P(void *dst, int ddev, const void *src, int sdev, size_t count)
{
    hipError_t err = hipSuccess;
    int dev0;
    int pgsz = 4096;
    static int bufsize = 0;
    static char *buf = NULL;

    if (bufsize < count) {
        bufsize = ((count - 1) / pgsz + 1) * pgsz;
        buf = (char *)realloc(buf, bufsize);
        if (!buf) {
            perror("cudaMemcpyP2P");
            exit(1);
        }
    }

    hipGetDevice(&dev0);

    if (sdev != dev0) {
        hipSetDevice(sdev);
    }
    err = hipMemcpy(buf, src, count, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        if (sdev != dev0) {
            hipSetDevice(dev0);
        }
        return err;
    }
    if (ddev != sdev) {
        hipSetDevice(ddev);
    }
    err = hipMemcpy(dst, buf, count, hipMemcpyHostToDevice);
    if (ddev != dev0) {
        hipSetDevice(dev0);
    }
    return err;
}

hipError_t
hipMemcpyPeer(void *dst, int ddev, const void *src, int sdev, size_t count)
{
    WARN(3, "hipMemcpyPeer(0x%08lx, %d, 0x%08lx, %d, %zu)...",
         (unsigned long)dst, ddev, (unsigned long)src, sdev, count);
    hipError_t cuerr;

    cuerr = cudaMemcpyP2P(dst, ddev, src, sdev, count);

    WARN(3, "done.\n");
    return cuerr;
}

hipError_t
hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device) {
    hipError_t err = hipSuccess;
    dscudaGetDevicePropertiesResult *rp;

    WARN(3, "hipGetDeviceProperties(0x%08lx, %d)...", (unsigned long)prop, device);
    VirDev     *vdev = St.Vdev + device;
    PhyDev *sp   = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudagetdevicepropertiesid_1(device, sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            memcpy(prop, rp->prop.RCbuf_val, rp->prop.RCbuf_len);
        }
        xdr_free((xdrproc_t)xdr_dscudaGetDevicePropertiesResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

/*
 * LoadModule Management. 
 */
int
PhyDev::findModuleOpen(void) {
    int i;
    for (i=0; i<RC_NKMODULEMAX; i++) {
	if ( modulelist[i].valid != 1 ) {
	    return i;
	}
    }
    WARN(0, "%s():Module management array is full. and exit.\n", __func__);
    exit(EXIT_FAILURE);
}

int
PhyDev::queryModuleID(int module_index) {
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	if ( modulelist[i].index == module_index ) {
	    return modulelist[i].id;
	}
    }
    WARN(0, "%s():Not found Module Index in array. and exit.\n", __func__);
    exit(EXIT_FAILURE);
}

int
VirDev::findModuleOpen(void) { //TODO: almost same as above func.
    int i;
    for (i=0; i<RC_NKMODULEMAX; i++) {
	if ( modulelist[i].valid != 1 ) {
	    return i;
	}
    }
    WARN(0, "%s():Module management array is full. and exit.\n", __func__);
    exit(EXIT_FAILURE);
}

int
PhyDev::loadModule(unsigned int ipaddr, pid_t pid, char *modulename,
			 char *modulebuf, int module_index)
{
    WARN(5, "      + PhyDev::%s(modulename=%s, module_index=%d) { \n",
	 __func__, modulename, module_index);
    
    /* send to virtual GPU */
    dscudaLoadModuleResult *rp;
    rp = dscudaloadmoduleid_1(ipaddr, getpid(), modulename, modulebuf, Clnt);

    //<--- RPC Error Hook
    struct rpc_err rpc_error;
    clnt_geterr(Clnt, &rpc_error );
    if (rpc_error.re_status == RPC_SUCCESS) {
	if (rp == NULL) {
	    WARN( 0, "NULL pointer returned, %s(). exit.\n", __func__ );
	    clnt_perror(Clnt, ip);
	    exit(EXIT_FAILURE);
	}
    } else {
	rpcErrorHook(&rpc_error);
    }
    //---> RPC Error Hook.

    int module_id = rp->id;
    xdr_free((xdrproc_t)xdr_dscudaLoadModuleResult, (char *)rp);

    // register a new module into the list,
    // and then, return a module id assigned by the server.
    int n = this->findModuleOpen(); 
    modulelist[n].valid     = 1;
    modulelist[n].index     = module_index;
    modulelist[n].id        = module_id;
    modulelist[n].sent_time = time(NULL);
    modulelist[n].ptx_data  = Ptx.query(modulename);
    WARN(5, "PhyDev[%d]: New client module item was registered. id:%d\n", id, module_id);
    
    if (St.isAutoVerb() ) {
	/*Nop*/
    }

    WARN(5, "      + } // PhyDev::%s()\n", __func__);
    return module_id;
}

int
VirDev::loadModule(char *name, char *strdata)
{
    WARN(5, "   + VirDev::loadModule( name=%p(%s), strdata=%p ) {\n", name, name, strdata);

    if (name != NULL) {
#if RC_CACHE_MODULE
	// look for modulename in the module list.
	for (int i=0; i<RC_NKMODULEMAX; i++) {
	    if ( modulelist[i].isInvalid() ) {
		continue;
	    }
	    if ( strcmp(name, modulelist[i].ptx_data->name) == 0 ) { //Found
		if ( modulelist[i].isAlive() ) {
		    WARN(5, "done. found a cached one. id:%d  age:%d  name:%s\n",
			 modulelist[i].index, time(NULL) - modulelist[i].sent_time, modulelist[i].ptx_data->name);
		    return modulelist[i].index; // module found. i.e, it's already loaded.
		} else {
		    WARN(5, "found a cached one with id:%d, but it is too old (age:%d). resend it.\n",
			 modulelist[i].index, time(NULL) - modulelist[i].sent_time);
		    modulelist[i].invalidate(); // invalidate the cache.
		}
	    }
	}
#endif // RC_CACHE_MODULE
    } else {
	WARN(5, "VirDev::loadModule(%p) modulename:-\n", name);
    }

    //<---
    char *strdata_found = NULL;
    char *name_found=NULL;
    if (name==NULL && strdata==NULL) {
        for (int i=0; i<RC_NKMODULEMAX; i++) {
	    WARN(10, "i=%d\n", i);
	    if (modulelist[i].isInvalid()) continue;
	    if (!strcmp(name, modulelist[i].ptx_data->name)) {     /* matched */
		strdata_found = modulelist[i].ptx_data->ptx_image;
		name_found = modulelist[i].ptx_data->name;
		break;
	    }
	}
    } else {
	strdata_found = strdata;
	name_found = name;
    }
    //--->

    // module not found in the module list.
    // really need to send it to the server.

    // <-- If target .ptx is not registered to PtxStore, then register first.
    PtxRecord *ptxrecord_ptr = Ptx.query(name_found);
    if (ptxrecord_ptr == NULL) {
	Ptx.add( name_found, strdata_found );
    }
    // --> If target .ptx is not registered to PtxStore, then register first.
    
    int j = this->findModuleOpen();
    this->modulelist[j].index     = j;
    this->modulelist[j].id        = j; //dummy; not used.
    this->modulelist[j].valid     = 1;
    this->modulelist[j].sent_time = time(NULL);
    this->modulelist[j].ptx_data  = Ptx.query(name_found);
    Ptx.print(4);
    WARN(5, "      + New client-module item was registered. index=%d\n", j);

    int mid;
    for (int i=0; i<nredundancy; i++) {
	mid = server[i].loadModule(St.getIpAddress(), getpid(), name_found, strdata_found, j);
        WARN(3, "(info) server[%d].loadModule() returns mid=%d.\n", i, mid);
    }

    WARN(5, "   + } // VirDev::loadModule().\n");
    return modulelist[j].index;
}//VirDev::loadModule(

/*
 * launch a kernel function of id 'kid', defined in a module of id 'moduleid'.
 * 'kid' must be unique inside a single module.
 */
void
PhyDev::launchKernel(int module_index, int kid, char *kname,
		       RCdim3 gdim, RCdim3 bdim, RCsize smemsize,
		       RCstream stream, RCargs args,
		       struct rpc_err *rpc_result)
{
    WARN(5, "      + PhyDev[%d]::%s() {\n", id, __func__);
    RCargs lo_args;
    lo_args.RCargs_len = args.RCargs_len;
    lo_args.RCargs_val = (RCarg *)dscuda::xmalloc(args.RCargs_len * sizeof(RCarg));

    for (int k=0; k<lo_args.RCargs_len; k++) {
	lo_args.RCargs_val[k] = args.RCargs_val[k];
    }
    
    RCstreamArray *st = RCstreamArrayQuery((hipStream_t)stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }

    /*
     * Replace v_ptr to d_ptr int args.
     */
    RCarg *argp;
    void  *v_ptr;
    void  *d_ptr;
    for (int i=0; i<lo_args.RCargs_len; i++) {
        argp = &(lo_args.RCargs_val[i]);
	if (argp->val.type == dscudaArgTypeP) {
            v_ptr = (void*)(argp->val.RCargVal_u.address);
	    d_ptr = memlist.queryDevicePtr(v_ptr);
	    WARN(6, "      +    Virtual Address Translate: arg[%d]:v_ptr=%p -> d_ptr=%p\n", i, v_ptr, d_ptr);
	    argp->val.RCargVal_u.address = (RCadr)d_ptr;
	}
    }
    /*
     * Replace module_index with real module id.
     */
    int moduleid = queryModuleID(module_index);

    void *rp = dscudalaunchkernelid_1(moduleid, kid, kname, gdim, bdim,
				      smemsize, (RCstream)st->s[id], lo_args, Clnt);

    //<--- Timed Out
    clnt_geterr(Clnt, rpc_result);
    if (rpc_result->re_status == RPC_SUCCESS) {
	if (rp == NULL) {
	    WARN( 0, "NULL pointer returned, %s(). exit.\n", __func__ );
	    clnt_perror(Clnt, ip);
	    exit( EXIT_FAILURE );
	}
    }
    //--->
    free(lo_args.RCargs_val);

    CudaRpcLaunchKernelArgs args2;
    if (isRecording()) {
        args2.moduleid = module_index;
        args2.kid      = kid;
        args2.kname    = kname;
        args2.gdim     = gdim;
        args2.bdim     = bdim;
        args2.smemsize = smemsize;
        args2.stream   = stream;
        args2.args     = args;
        reclist.append( dscudaLaunchKernelId, (void *)&args2 );
    }

    WARN(5, "      + } PhyDev[%d]::%s()\n", id, __func__);
}

void
VirDev::launchKernel(int module_index, int kid, char *kname,
			    RCdim3 gdim, RCdim3 bdim, RCsize smemsize,
			    RCstream stream, RCargs args)
{
    WARN(5, "   + VirDev::%s() {\n", __func__);
    /*     
     * Automatic Recovery, Register to the called history.
     */
    CudaRpcLaunchKernelArgs args2;
    if (isRecording()) {
        args2.moduleid = module_index;
        args2.kid      = kid;
        args2.kname    = kname;
        args2.gdim     = gdim;
        args2.bdim     = bdim;
        args2.smemsize = smemsize;
        args2.stream   = stream;
        args2.args     = args;
        reclist.append( dscudaLaunchKernelId, (void *)&args2 );
    }

    struct rpc_err rpc_result;
    for (int i=0; i<nredundancy; i++) {
        server[i].launchKernel(module_index, kid, kname, gdim,
			       bdim, smemsize, stream, args, &rpc_result);
	server[i].rpcErrorHook( &rpc_result );
	if (rpc_result.re_status != RPC_SUCCESS ) {
	    this->restoreMemlist();
	    this->reclist.recall();
	}
    }
    WARN(5, "   + } // VirDev::%s()\n", __func__);
}

void
rpcDscudaLaunchKernelWrapper(int module_index, int kid, char *kname,  /* moduleid is got by "dscudaLoadModule()" */
                             RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream,
                             RCargs args)
{
    WARN(5, "%s() {\n", __func__);
    pthread_mutex_lock( &cudaKernelRun_mutex );

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    vdev->launchKernel(module_index, kid, kname, gdim, bdim, smemsize, stream, args);
    
    pthread_mutex_unlock( &cudaKernelRun_mutex );
    WARN(5, "} %s().\n", __func__)
    WARN(5, "\n")
}

hipError_t
hipMallocArray(struct hipArray **array, const struct hipChannelFormatDesc *desc,
                size_t width, size_t height, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaMallocArrayResult *rp;
    RCchanneldesc descbuf;
    hipArray *ca[RC_NREDUNDANCYMAX];

    WARN(3, "hipMallocArray(%p, %p, %zu, %zu, 0x%08x)...",
         array, desc, width, height, flags);

    descbuf.x = desc->x;
    descbuf.y = desc->y;
    descbuf.z = desc->z;
    descbuf.w = desc->w;
    descbuf.f = desc->f;

    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++ ) {
        rp = dscudamallocarrayid_1(descbuf, width, height, flags, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ca[i] = (hipArray *)rp->array;
        xdr_free((xdrproc_t)xdr_dscudaMallocArrayResult, (char *)rp);
    }

    *array = ca[0];
    RCcuarrayArrayRegister(ca);
    WARN(3, "done. *array:%p\n", *array);

    return err;
}

hipError_t
hipFreeArray(struct hipArray *array)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCcuarrayArray *ca;

    WARN(3, "hipFreeArray(%p)...", array);
    ca = RCcuarrayArrayQuery(array);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", array);
        exit( EXIT_FAILURE );
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++ ) {
        rp = dscudafreearrayid_1((RCadr)ca->ap[i], sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCcuarrayArrayUnregister(ca->ap[0]);
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipMemcpyToArray(struct hipArray *dst, size_t wOffset, size_t hOffset, const void *src,
                  size_t count, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCcuarrayArray *ca;
    VirDev *vdev;
    PhyDev *sp;

    WARN(3, "hipMemcpyToArray(%p, %zu, %zu, %p, %zu, %s)...",
         dst, wOffset, hOffset, src, count, dscudaMemcpyKindName(kind));
    ca = RCcuarrayArrayQuery(dst);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", dst);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;

        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            h2drp = dscudamemcpytoarrayh2did_1((RCadr)ca->ap[i], wOffset, hOffset, srcbuf, count, sp[i].Clnt);
            checkResult(h2drp, sp[i]);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            d2drp = dscudamemcpytoarrayd2did_1((RCadr)ca->ap[i], wOffset, hOffset, (RCadr)src, count, sp[i].Clnt );
            checkResult(d2drp, sp[i]);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemset(void *devPtr, int value, size_t count)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipMemset()...");
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudamemsetid_1((RCadr)devPtr, value, count, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMallocPitch(void **devPtr, size_t *pitch, size_t width, size_t height)
{
    hipError_t err = hipSuccess;
    dscudaMallocPitchResult *rp;

    WARN(3, "hipMallocPitch(%p, %p, %zu, %zu)...", devPtr, pitch, width, height);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudamallocpitchid_1(width, height, sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *devPtr = (void*)rp->devPtr;
            *pitch = rp->pitch;
        }
        xdr_free((xdrproc_t)xdr_dscudaMallocPitchResult, (char *)rp);
    }

    WARN(3, "done. *devPtr:%p  *pitch:%zu\n", *devPtr, *pitch);

    return err;
}

hipError_t
hipMemcpy2DToArray(struct hipArray *dst, size_t wOffset, size_t hOffset,
                    const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpy2DToArrayD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCcuarrayArray *ca;
    VirDev *vdev;
    PhyDev *sp;

    WARN(3, "hipMemcpy2DToArray(%p, %zu, %zu, %p, %zu, %zu, %zu, %s)...",
         dst, wOffset, hOffset,
         src, spitch, width, height, dscudaMemcpyKindName(kind));
    ca = RCcuarrayArrayQuery(dst);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", dst);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++ ) {
            d2hrp = dscudamemcpy2dtoarrayd2hid_1(wOffset, hOffset,
                                                 (RCadr)src, spitch, width, height, sp[i].Clnt);
            checkResult( d2hrp, sp[i] );
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            } else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpy2DToArray() data copied from device%d matched with that from device0.\n", i);
            }
            xdr_free((xdrproc_t)xdr_dscudaMemcpy2DToArrayD2HResult, (char *)d2hrp);
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = spitch * height;
        srcbuf.RCbuf_val = (char *)src;
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            h2drp = dscudamemcpy2dtoarrayh2did_1((RCadr)ca->ap[i], wOffset, hOffset,
                                                 srcbuf, spitch, width, height, sp[i].Clnt );
            checkResult(h2drp, sp[i]);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            d2drp = dscudamemcpy2dtoarrayd2did_1((RCadr)ca->ap[i], wOffset, hOffset,
                                                 (RCadr)src, spitch, width, height, sp[i].Clnt );
            checkResult(d2drp, sp[i]);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemcpy2D(void *dst, size_t dpitch,
             const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpy2DD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    VirDev *vdev;
    PhyDev *sp;

    WARN(3, "hipMemcpy2D(%p, %zu, %p, %zu, %zu, %zu, %s)...",
         dst, dpitch,
         src, spitch, width, height, dscudaMemcpyKindName(kind));

    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            d2hrp = dscudamemcpy2dd2hid_1(dpitch,
                                          (RCadr)src, spitch, width, height, sp[i].Clnt );
            checkResult(d2hrp, sp[i]);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            } else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpy() data copied from device%d matched with that from device0.\n", i);
            }
            xdr_free((xdrproc_t)xdr_dscudaMemcpy2DD2HResult, (char *)d2hrp);
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = spitch * height;
        srcbuf.RCbuf_val = (char *)src;
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            h2drp = dscudamemcpy2dh2did_1((RCadr)dst, dpitch,
                                          srcbuf, spitch, width, height, sp[i].Clnt );
            checkResult(h2drp, sp[i] );
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            d2drp = dscudamemcpy2dd2did_1((RCadr)dst, dpitch,
                                          (RCadr)src, spitch, width, height, sp[i].Clnt);
            checkResult(d2drp, sp[i]);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemset2D(void *devPtr, size_t pitch, int value, size_t width, size_t height)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipMemset2D(%p, %zu, %d, %zu, %zu)...",
         devPtr, pitch, value, width, height);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudamemset2did_1((RCadr)devPtr, pitch, value, width, height, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipHostMalloc(void **ptr, size_t size)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaMallocHostResult *rp;

    WARN(3, "hipHostMalloc(%p, %d)...", ptr, size);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudamallochostid_1(size, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *ptr = (void*)rp->ptr;
        }
        xdr_free((xdrproc_t)xdr_dscudaMallocHostResult, (char *)rp);
    }

    WARN(3, "done. *ptr:%p\n", *ptr);
    return err;
#else
    // returned memory is not page locked.
    // it cannot be passed to hipMemcpyAsync().
    *ptr = malloc(size);
    if (*ptr) {
        return hipSuccess;
    } else {
        return hipErrorOutOfMemory;
    }
#endif
}

hipError_t
hipHostAlloc(void **pHost, size_t size, unsigned int flags)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaHostAllocResult *rp;

    WARN(3, "hipHostAlloc(0x%08llx, %d, 0x%08x)...", (unsigned long)pHost, size, flags);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudahostallocid_1(size, flags, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pHost = (void*)rp->pHost;
        }
        xdr_free((xdrproc_t)xdr_dscudaHostAllocResult, (char *)rp);
    }

    WARN(3, "done. *pHost:0x%08llx\n", *pHost);

    return err;
#else
    // returned memory is not page locked.
    // it cannot be passed to hipMemcpyAsync().

    hipError_t err = hipSuccess;
    void *devmem;

    WARN(3, "hipHostAlloc(%p, %zu, 0x%08x)...", pHost, size, flags);

    *pHost = malloc(size);
    if (!*pHost) return hipErrorOutOfMemory;
    if (!(flags & hipHostMallocMapped)) {
        WARN(3, "done. *pHost:%p\n", *pHost);
        return hipSuccess;
    }

    // flags says the host memory must be mapped on to the device memory.
    err = hipMalloc(&devmem, size);
    if (err == hipSuccess) {
        RCmappedMemRegister(*pHost, devmem, size);
    }
    WARN(3, "done. host mem:%p  device mem:%p\n", *pHost, devmem);

    return err;
#endif
}

hipError_t hipHostFree(void *ptr) {
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipHostFree(0x%08llx)...", (unsigned long)ptr);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++ ) {
        rp = dscudafreehostid_1((RCadr)ptr, sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
#else
    hipError_t err = hipSuccess;
    RCmappedMem *mem = RCmappedMemQuery(ptr);
    free(ptr);
    if (mem) { // ptr mapped on to a device memory.
        err = hipFree(mem->pDevice);
        RCmappedMemUnregister(ptr);
        return err;
    } else {
        return hipSuccess;
    }
#endif
}

// flags is not used for now in CUDA3.2. It should always be zero.
hipError_t
hipHostGetDevicePointer(void **pDevice, void*pHost, unsigned int flags) {
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaHostGetDevicePointerResult *rp;

    WARN(3, "hipHostGetDevicePointer(0x%08llx, 0x%08llx, 0x%08x)...",
         (unsigned long)pDevice, (unsigned long)pHost, flags);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudahostgetdevicepointerid_1((RCadr)pHost, flags, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pDevice = (void *)rp->pDevice;
        }
        xdr_free((xdrproc_t)xdr_dscudaHostGetDevicePointerResult, (char *)rp);
    }

    WARN(3, "done. *pDevice:0x%08llx\n", *pDevice);
    return err;
#else
    RCmappedMem *mem = RCmappedMemQuery(pHost);
    if (!mem) return hipErrorInvalidValue; // pHost is not registered as RCmappedMem.
    *pDevice = mem->pDevice;
    return hipSuccess;
#endif
}

hipError_t hipHostGetFlags(unsigned int *pFlags, void *pHost) {
    hipError_t err = hipSuccess;
    dscudaHostGetFlagsResult *rp;

    WARN(3, "hipHostGetFlags(%p %p)...", pFlags, pHost);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudahostgetflagsid_1((RCadr)pHost, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pFlags = rp->flags;
        }
        xdr_free((xdrproc_t)xdr_dscudaHostGetFlagsResult, (char *)rp);
    }
    WARN(3, "done. flags:0x%08x\n", *pFlags);
    return err;    
}

hipError_t
hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind, hipStream_t stream) {
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaMemcpyAsyncD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCstreamArray *st;
    VirDev *vdev;
    PhyDev *sp;

    WARN(3, "hipMemcpyAsync(0x%08llx, 0x%08llx, %d, %s, 0x%08llx)...",
         (unsigned long)dst, (unsigned long)src, count, dscudaMemcpyKindName(kind), st->s[0]);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyDeviceToHost:
        VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
        PhyDev *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            d2hrp = dscudamemcpyasyncd2hid_1((RCadr)src, count, (RCstream)st->s[i], sp[i].Clnt);
            checkResult(d2hrp, sp[i]);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            }
            xdr_free((xdrproc_t)xdr_dscudaMemcpyAsyncD2HResult, (char *)d2hrp);
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;
        VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
        PhyDev *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            h2drp = dscudamemcpyasynch2did_1((RCadr)dst, srcbuf, count, (RCstream)st->s[i], sp[i].Clnt);
            checkResult(h2drp, sp[i]);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
        PhyDev *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++) {
            d2drp = dscudamemcpyasyncd2did_1((RCadr)dst, (RCadr)src, count, (RCstream)st->s[i], sp[i].Clnt);
            checkResult(d2drp, sp[i]);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;

#else
    // this DOES block.
    // this is only for use with a poor implementation of dscudaMallocHost().
    return hipMemcpy(dst, src, count, kind);
#endif
}

hipError_t
dscudaMemcpyToSymbolH2D(int moduleid, char *symbol, const void *src,
                        size_t count, size_t offset, int vdevid, int raidid) {
    dscudaResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    RCbuf srcbuf;
    hipError_t err;

    srcbuf.RCbuf_len = count;
    srcbuf.RCbuf_val = (char *)src;
    rp = dscudamemcpytosymbolh2did_1(moduleid, symbol, srcbuf, count, offset, sp[raidid].Clnt);
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyToSymbolD2D(int moduleid, char *symbol, const void *src,
                        size_t count, size_t offset, int vdevid, int raidid)
{
    dscudaResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    hipError_t err;

    rp = dscudamemcpytosymbold2did_1(moduleid, symbol, (RCadr)src, count, offset, sp[raidid].Clnt );
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolD2H(int moduleid, void **dstbuf, char *symbol,
                          size_t count, size_t offset, int vdevid, int raidid)
{
    dscudaMemcpyFromSymbolD2HResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    hipError_t err;

    rp = dscudamemcpyfromsymbold2hid_1(moduleid, (char *)symbol, count, offset, sp[raidid].Clnt);
    *dstbuf = rp->buf.RCbuf_val;
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaMemcpyFromSymbolD2HResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolD2D(int moduleid, void *dstadr, char *symbol,
                          size_t count, size_t offset, int vdevid, int raidid) {
    dscudaResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    hipError_t err;

    rp = dscudamemcpyfromsymbold2did_1(moduleid, (RCadr)dstadr, (char *)symbol, count, offset, sp[raidid].Clnt );
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyToSymbolAsyncH2D(int moduleid, char *symbol, const void *src,
                             size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    RCbuf srcbuf;
    hipError_t err;

    srcbuf.RCbuf_len = count;
    srcbuf.RCbuf_val = (char *)src;
    rp = dscudamemcpytosymbolasynch2did_1(moduleid, symbol, srcbuf, count, offset, stream, sp[raidid].Clnt);
    checkResult(rp, sp[raidid]);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyToSymbolAsyncD2D(int moduleid, char *symbol, const void *src,
                             size_t count, size_t offset, RCstream stream, int vdevid, int raidid) {
    dscudaResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    hipError_t err;

    rp = dscudamemcpytosymbolasyncd2did_1( moduleid, symbol, (RCadr)src, count, offset, stream, sp[raidid].Clnt);
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolAsyncD2H(int moduleid, void **dstbuf, char *symbol,
                               size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaMemcpyFromSymbolAsyncD2HResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    hipError_t err;

    rp = dscudamemcpyfromsymbolasyncd2hid_1(moduleid, (char *)symbol, count, offset,
                                            stream, sp[raidid].Clnt );
    *dstbuf = rp->buf.RCbuf_val;
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaMemcpyFromSymbolAsyncD2HResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolAsyncD2D(int moduleid, void *dstadr, char *symbol,
                               size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaResult *rp;
    PhyDev *sp = (St.Vdev + vdevid)->server;
    hipError_t err;

    rp = dscudamemcpyfromsymbolasyncd2did_1(moduleid, (RCadr)dstadr, (char *)symbol, count, offset, stream, sp[raidid].Clnt);
    checkResult(rp, sp[raidid] );
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

/*
 * Stream Management
 */

hipError_t hipStreamCreate(hipStream_t *pStream) {
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaStreamCreateResult *rp;
    hipStream_t st[RC_NREDUNDANCYMAX];

    WARN(3, "hipStreamCreate(0x%08llx)...", (unsigned long)pStream);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudastreamcreateid_1(sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        st[i] = (hipStream_t)rp->stream;
        xdr_free((xdrproc_t)xdr_dscudaStreamCreateResult, (char *)rp);
    }

    *pStream = st[0];
    RCstreamArrayRegister(st);
    WARN(3, "done. *pStream:0x%08llx\n", *pStream);

    return err;
#else
    *pStream = 0;
    return hipSuccess;
#endif
}

hipError_t hipStreamDestroy(hipStream_t stream) {
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    WARN(3, "hipStreamDestroy(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudastreamdestroyid_1((RCadr)st->s[i], sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCstreamArrayUnregister(st->s[0]);
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

hipError_t hipStreamSynchronize(hipStream_t stream) {
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    WARN(3, "hipStreamSynchronize(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudastreamsynchronizeid_1((RCadr)st->s[i], sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

hipError_t hipStreamQuery(hipStream_t stream) {
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    WARN(3, "hipStreamQuery(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudastreamqueryid_1((RCadr)st->s[i], sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

/*
 * Event Management
 */

hipError_t hipEventCreate(hipEvent_t *event) {
    hipError_t err = hipSuccess;
    dscudaEventCreateResult *rp;
    hipEvent_t ev[RC_NREDUNDANCYMAX];

    WARN(3, "hipEventCreate(%p)...", event);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaeventcreateid_1(sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ev[i] = (hipEvent_t)rp->event;
        xdr_free((xdrproc_t)xdr_dscudaEventCreateResult, (char *)rp);
    }
    *event = ev[0];
    RCeventArrayRegister(ev);
    WARN(3, "done. *event:%p\n", *event);

    return err;
}

hipError_t
hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags) {
    hipError_t err = hipSuccess;
    dscudaEventCreateResult *rp;
    hipEvent_t ev[RC_NREDUNDANCYMAX];

    WARN(3, "hipEventCreateWithFlags(%p, 0x%08x)...", event, flags);
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaeventcreatewithflagsid_1(flags, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ev[i] = (hipEvent_t)rp->event;
        xdr_free((xdrproc_t)xdr_dscudaEventCreateResult, (char *)rp);
    }
    *event = ev[0];
    RCeventArrayRegister(ev);
    WARN(3, "done. *event:%p\n", *event);

    return err;
}

hipError_t hipEventDestroy( hipEvent_t event ) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    WARN(3, "hipEventDestroy(%p)...", event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++ ) {
        rp = dscudaeventdestroyid_1((RCadr)ev->e[i], sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCeventArrayUnregister(ev->e[0]);
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end) {
    hipError_t err = hipSuccess;
    dscudaEventElapsedTimeResult *rp;
    RCeventArray *es, *ee;

    WARN(3, "hipEventElapsedTime(%p, %p, %p)...", ms, start, end);
    es = RCeventArrayQuery(start);
    if (!es) {
        WARN(0, "invalid start event : %p\n", start);
        exit(1);
    }
    ee = RCeventArrayQuery(end);
    if (!ee) {
        WARN(0, "invalid end event : %p\n", end);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaeventelapsedtimeid_1((RCadr)es->e[i], (RCadr)ee->e[i], sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaEventElapsedTimeResult, (char *)rp);
    }

    *ms = rp->ms;
    WARN(3, "done.\n");
    return err;
}

hipError_t hipEventRecord(hipEvent_t event, hipStream_t stream) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;
    RCeventArray *ev;

    WARN(3, "hipEventRecord(%p, %p)...", event, stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaeventrecordid_1((RCadr)ev->e[i], (RCadr)st->s[i], sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t hipEventSynchronize(hipEvent_t event) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    WARN(3, "hipEventSynchronize(%p)...", event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaeventsynchronizeid_1((RCadr)ev->e[i], sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
}

hipError_t hipEventQuery(hipEvent_t event) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    WARN(3, "hipEventQuery(%p)...", event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudaeventqueryid_1((RCadr)ev->e[i], sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags) {
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;
    RCeventArray *ev;

    WARN(3, "hipStreamWaitEvent(%p, %p, 0x%08x)...", stream, event, flags);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudastreamwaiteventid_1((RCadr)st->s[i], (RCadr)ev->e[i], flags, sp[i].Clnt);
        checkResult(rp, sp[i]);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }

    WARN(3, "done.\n");
    return err;
}

/*
 * Texture Reference Management
 */

hipChannelFormatDesc
hipCreateChannelDesc(int x, int y, int z, int w, enum hipChannelFormatKind f)
{
    dscudaCreateChannelDescResult *rp;
    hipChannelFormatDesc desc;

    WARN(3, "hipCreateChannelDesc()...");
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudacreatechanneldescid_1(x, y, z, w, f, sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (i == 0) {
            desc.x = rp->x;
            desc.y = rp->y;
            desc.z = rp->z;
            desc.w = rp->w;
            desc.f = (enum hipChannelFormatKind)rp->f;
        }
        xdr_free((xdrproc_t)xdr_dscudaCreateChannelDescResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return desc;
}

hipError_t
hipGetChannelDesc(struct hipChannelFormatDesc *desc, const struct hipArray *array)
{
    hipError_t err = hipSuccess;
    dscudaGetChannelDescResult *rp;
    RCcuarrayArray *ca;

    WARN(3, "hipGetChannelDesc()...");
    ca = RCcuarrayArrayQuery((hipArray *)array);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", array);
        exit(1);
    }
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscudagetchanneldescid_1( (RCadr)ca->ap[i], sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            desc->x = rp->x;
            desc->y = rp->y;
            desc->z = rp->z;
            desc->w = rp->w;
            desc->f = (enum hipChannelFormatKind)rp->f;
        }
        xdr_free((xdrproc_t)xdr_dscudaGetChannelDescResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}


hipError_t
hipUnbindTexture(const struct textureReference * texref)
{
    hipError_t err = hipSuccess;

    WARN(4, "Current implementation of hipUnbindTexture() does nothing "
         "but returning hipSuccess.\n");

    err = hipSuccess;

    return err;
}

/*
 * CUFFT library
 */
hipfftResult CUFFTAPI
hipfftPlan3d(hipfftHandle *plan, int nx, int ny, int nz, hipfftType type)
{
    hipfftResult res = HIPFFT_SUCCESS;
    dscufftPlanResult *rp;

    WARN(3, "hipfftPlan3d()...");
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscufftplan3did_1(nx, ny, nz, (unsigned int)type, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
        if (i == 0) {
            *plan = rp->plan;
        }
        xdr_free((xdrproc_t)xdr_dscufftPlanResult, (char *)rp);
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftDestroy(hipfftHandle plan) {
    hipfftResult res = HIPFFT_SUCCESS;
    dscufftResult *rp;

    WARN(3, "hipfftDestroy()...");
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++ ) {
        rp = dscufftdestroyid_1((unsigned int)plan, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if ( rp->err != HIPFFT_SUCCESS ) {
            res = (hipfftResult)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscufftResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecC2C(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata, int direction)
{
    hipfftResult res = HIPFFT_SUCCESS;
    dscufftResult *rp;

    WARN(3, "hipfftExecC2C()...");
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    PhyDev *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++) {
        rp = dscufftexecc2cid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, direction, sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscufftResult, (char *)rp);
    }

    WARN(3, "done.\n");

    return res;
}

#if 0

/*
 * Interface to CUFFT & CUBLAS written by Yoshikawa for old Remote CUDA.
 * some are already ported to DS-CUDA (see 'dscufftXXXid_1_svc' function defs above),
 * but some are not. Maybe someday, when I have time...
 */

hipfftResult CUFFTAPI
hipfftPlan1d(hipfftHandle *plan, int nx, hipfftType type, int batch)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftPlanResult *rp;

    WARN(3, "hipfftPlan1d()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++) {
        rp = rcufftplan1did_1(nx, (unsigned int)type, batch, sp.Clnt );
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }
    *plan = rp->plan;

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftPlan2d(hipfftHandle *plan, int nx, int ny, hipfftType type, int batch)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftPlanResult *rp;

    WARN(3, "hipfftPlan2d()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcufftplan2did_1(nx, ny, (unsigned int)type, sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }
    *plan = rp->plan;

    WARN(3, "done.\n");

    return res;
}

/*
  hipfftResult CUFFTAPI
  hipfftPlanMany(hipfftHandle *plan, int nx, hipfftType type, int batch)
  {
  hipfftResult res = HIPFFT_SUCCESS;
  rcufftPlanResult *rp;

  WARN(3, "hipfftPlan1d()...");
  Server *sp = Serverlist;
  for (int i = 0; i < Nredundancy; i++, sp++) {
  rp = rcufftplan1did_1(nx, (unsigned int)type, Clnt[0][sp->id]);
  checkResult(rp, sp);
  if (rp->err != HIPFFT_SUCCESS) {
  res = (hipfftResult)rp->err;
  }
  }
  *plan = rp->plan;

  WARN(3, "done.\n");

  return res;
  }
*/

hipfftResult CUFFTAPI
hipfftExecR2C(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecR2C()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcufftexecr2cid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecC2R(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata) {
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecC2R()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcufftexecc2rid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecZ2Z(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata, int direction) {
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecZ2Z()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++) {
        rp = rcufftexecz2zid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, direction, sp.Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecD2Z(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecD2Z()...");
    Server *sp = Serverlist;
    for ( int i = 0; i < Nredundancy; i++ ) {
        rp = rcufftexecd2zid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, sp[i].Clnt);
        checkResult(rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecZ2D(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata) {
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecZ2D()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcufftexecz2did_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, sp[i].Clnt );
        checkResult(rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

/*
  hipfftResult CUFFTAPI
  hipfftSetStream(hipfftHandle plan, hipStream_t stream)
  {
  }
*/

hipfftResult CUFFTAPI
cufftSetCompatibilityMode(hipfftHandle plan, cufftCompatibility mode) {
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "cufftSetCompatibilityMode()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcufftsetcompatibilitymodeid_1((unsigned int)plan, (unsigned int)mode, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}


/*
 * CUBLAS Library functions
 */
hipblasStatus_t CUBLASAPI
hipblasCreate(hipblasHandle_t *handle) {
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasCreateResult *rp;

    WARN(3, "hipblasCreate()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcublascreate_v2id_1( sp[i].Clnt );
        checkResult(rp, sp[i]);
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    *handle = (hipblasHandle_t)rp->handle;

    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasDestroy(hipblasHandle_t handle) {
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasResult *rp;

    WARN(3, "hipblasDestroy()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcublasdestroy_v2id_1((RCadr)handle, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasSetVector(int n, int elemSize, const void *x, int incx, void *devicePtr, int incy) {
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasResult *rp;

    RCbuf buf;
    buf.RCbuf_val = (char *)dscuda::xmalloc(n * elemSize);
    buf.RCbuf_len = n;
    memcpy(buf.RCbuf_val, x, n);

    WARN(3, "hipblasSetVector()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++ ) {
        rp = rcublassetvectorid_1(n, elemSize, buf, incx, (RCadr)devicePtr, incy, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasGetVector(int n, int elemSize, const void *x, int incx, void *y, int incy) {
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasGetVectorResult *rp;

    WARN(3, "hipblasGetVector()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++) {
        rp = rcublasgetvectorid_1(n, elemSize, (RCadr)x, incx, incy, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }

    memcpy(y, rp->y.RCbuf_val, n * elemSize);
    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
               const float *alpha, const float *A, int lda,
               const float *B, int ldb, const float *beta, float *C, int ldc) {
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasResult *rp;

    WARN(3, "hipblasSgemm()...");
    Server *sp = Serverlist;
    for ( int i = 0; i < Nredundancy; i++ ) {
        rp = rcublassgemm_v2id_1((RCadr)handle, (unsigned int)transa, (unsigned int)transb, m, n, k,
                                 *alpha, (RCadr)A, lda, (RCadr)B, ldb, *beta, (RCadr)C, ldc, sp[i].Clnt );
        checkResult( rp, sp[i] );
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    WARN(3, "done.\n");

    return res;
}
#endif // CUFFT

