//                             -*- Mode: C++ -*-
// Filename         : dscudaverb.cu
// Description      : DS-CUDA verb function.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-02-12 20:57:57
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include <string.h>
#include "dscuda.h"
#include "dscudarpc.h"
#include "libdscuda.h"
#include "dscudaverb.h"

#define DEBUG

static dscudaVerbHist   *verbHists = NULL;
static int               verbHistNum = 0; /* Number of recorded function calls to be recalled */
static int               verbHistMax = 0; /* Upper bound of "verbHistNum", extensible */
static verbAllocatedMem *verbAllocatedMemListTop = NULL;
static verbAllocatedMem *verbAllocatedMemListTail = NULL;

typedef enum {
    DSCVMethodNone = 0,
    DSCVMethodSetDevice,
    //DSCVMethodGetDeviceProperties,
    DSCVMethodMalloc,
    DSCVMethodMemcpyH2D,
    DSCVMethodMemcpyD2D,
    DSCVMethodMemcpyD2H,
    DSCVMethodMemcpyToSymbolH2D,
    DSCVMethodMemcpyToSymbolD2D,
    DSCVMethodFree,
    //DSCVMethodLoadModule,
    DSCVMethodRpcLaunchKernel,
    DSCVMethodIbvLaunchKernel,
    DSCVMethodEnd
} DSCVMethod;

int
verbGetLengthOfMemList(void)
{
    verbAllocatedMem *pMem = verbAllocatedMemListTop;
    int length = 0;
    while (pMem != NULL) {
	pMem = pMem->next;
	length++;
    }
    return length;
}

verbAllocatedMem *
verbAllocatedMemQuery(void *dst)
{
    verbAllocatedMem *mem = verbAllocatedMemListTop;

    while (mem != NULL) { /* Search */
	if (mem->dst == dst) { /* tagged by its address on GPU */
	    WARN(10, "---> %s(%p): return %p\n", __func__, dst, mem);
	    return mem;
	}
	mem = mem->next;
    }
    return NULL;
}

void *
verbAllocatedMemUpdateQuery(void *dst)
{
    verbAllocatedMem *mem = verbAllocatedMemListTop;
    //WARN(2, "<--- %s(%p):\n", __func__, dst);
    char *d_targ  = (char *)dst;
    char *d_begin;
    char *h_begin;
    char *h_p     = NULL;
    int   i = 0;

    //WARN(2, "   + d_targ  = %p\n", d_targ);
    while (mem) { /* Search */
	d_begin = (char *)mem->dst;
	h_begin = (char *)mem->src;
	//WARN(2, "   + d_begin[%d] = %p\n", i, d_begin);
	//WARN(2, "   + h_begin[%d] = %p\n", i, h_begin);
	if (d_targ >= d_begin &&
	    d_targ < (d_begin + mem->size)) {
	    h_p = h_begin + (d_targ - d_begin);
	    break;
	}
	mem = mem->next;
	i++;
    }
    WARN(10, "---> %s(%p): return %p\n", __func__, dst, h_p);
    return (void *)h_p;
}
static int
checkSum(void *targ, int size) {
    int sum=0, *ptr = (int *)targ;
    
    for (int s=0; s<size; s+=sizeof(int)) {
	//printf("ptr[%d]= %d\n", s, *ptr);
	sum += *ptr;
	ptr++;
    }
    return sum;
}
void
printRegionalCheckSum(void) {
    verbAllocatedMem *pMem = verbAllocatedMemListTop;
    int length = 0;
    while (pMem != NULL) {
	printf("Region[%d](dp=%p, size=%d): checksum=0x%08x\n",
	       length, pMem->dst, pMem->size, checkSum(pMem->src, pMem->size));
	fflush(stdout);
	pMem = pMem->next;
	length++;
    }
}
/*
 * Register hipMalloc()
 */
void
verbAllocatedMemRegister(void *dst, int size)
{
    static int i=0;
    // WARN(10, "<--- %s(dst=%p, size=%d) [%d]\n", __func__, dst, size, i);
    verbAllocatedMem *mem = (verbAllocatedMem *)malloc(sizeof(verbAllocatedMem));
    if (!mem) {
	perror("verbAllocatedMemRegister");
    }
    mem->dst  = dst;
    mem->size = size;
    mem->src  = malloc(size);
    mem->next = NULL;
    
    if (verbAllocatedMemListTop == NULL) {
	verbAllocatedMemListTop = mem;
	mem->prev = NULL;
    }
    else {
	verbAllocatedMemListTail->next = mem;
	mem->prev = verbAllocatedMemListTail;
    }
    verbAllocatedMemListTail = mem;
    // WARN(10, "---> %s(dst=%p, size=%d: mem=%p, src=%p) [%d]\n", __func__, dst, size, mem, mem->src, i);
    i++;
}

void
verbAllocatedMemUnregister(void *dst)
{
    // WARN(10, "<--- %s(dst=%p)\n", __func__, dst);
    verbAllocatedMem *mem = verbAllocatedMemQuery(dst);
    verbAllocatedMem *p_list = verbAllocatedMemListTop;
    int i;
    if (!mem) {
	WARN(0, "%s(): not found requested memory region.\n", __func__);
	WARN(0, "mem. list length= %d \n", verbGetLengthOfMemList());
	i = 0;
	while (p_list != NULL) {
	    WARN(0, "mem-list[%d] = %p\n", i, p_list->dst);
	    p_list = p_list->next;
	    i++;
	}
	return;
    }

    if (mem->prev != NULL) { /* not Top */
	//WARN(2, "not TOP\n");
	mem->prev->next = mem->next;
    }
    else {
	//WARN(2, "is TOP\n");
	verbAllocatedMemListTop = mem->next;
	if (mem->next) {
	    mem->next->prev = NULL;
	}
    }

    if (!mem->next) {
	verbAllocatedMemListTail = mem->prev;
    }

    free(mem->src);
    free(mem);
    // WARN(10, "---> %s(dst=%p)\n", __func__, dst);
}

void
verbAllocatedMemUpdate(void *dst, void *src, int size)
// dst : GPU device memory region
// src : HOST memory region
{
    verbAllocatedMem *mem;
    void             *src_mirrored;

    WARN(10, "    <--- %s(dst=%p, src=%p, size=%d)\n", __func__, dst, src, size);

    if (src == NULL) {
	WARN(0, "(+_+) not found backup target memory region (%p).\n", dst);
	exit(1);
    }
    else {
	//mem = verbAllocatedMemQuery(dst);
	//src_mirrored = mem->src;
	src_mirrored = verbAllocatedMemUpdateQuery(dst);
	memcpy(src_mirrored, src, size); // update historical memory region.
	WARN(10, "        Also copied to backup region (%p), checksum=%d.\n",
	     dst, checkSum(src, size));
	printRegionalCheckSum();
    }
    WARN(10, "    ---> %s(dst=%p, src=%p, size=%d)\n", __func__, dst, src, size); 
}

void
dscudaVerbRealloc(void)
{
    verbAllocatedMem *mem = verbAllocatedMemListTop;
    int               verb = St.isAutoVerb();
    int               copy_count = 0;
    unsigned char    *mon;
    float            *fmon;
    int              *imon;

    WARN(1, "###============================================================\n");
    WARN(1, "### %s() called.\n", __func__);
    WARN(1, "###============================================================\n");

    while (mem != NULL) {
	//hipMalloc(&mem->dst, mem->size); /* To migrate another GPU, you need to do hipMalloc() before following hipMemcpy(). */
	WARN(2, "Restore device memory region[%d] (dst=%p, src=%p, size=%d) . checksum=0x%08x\n",
	     copy_count++, mem->dst, mem->src, mem->size, checkSum(mem->src, mem->size));
	if (mem->size <= 16) {
	    fmon = (float *)mem->src;
	    imon = (int   *)mem->src;
	    for (int i=0; i < (mem->size / sizeof(float)); i++) {
		WARN(2, "  + float[%d]= %f, int[%d]= %d\n", i, *fmon, i, *imon);
		fmon++;
		imon++;
	    }
	}
	hipMemcpy(mem->dst, mem->src, mem->size, hipMemcpyHostToDevice);
	// mon = (unsigned char *)mem->src;
	// for (int i=0; i<16; i++) {
	//     printf("%04d: ", i);
	//     for (int j=0; j<16; j++) {
	// 	printf("%02x ", *mon);
	// 	mon++;
	//     }
	//     printf("\n"); fflush(stdout);
	// }
	mem = mem->next;
    }
    WARN(1, "###============================================================\n");
    WARN(1, "### %s() done.\n", __func__);
    WARN(1, "###============================================================\n");
}


//stubs for store/release args, and recall functions.
static void *(*storeArgsStub[DSCVMethodEnd])(void *);
static void (*releaseArgsStub[DSCVMethodEnd])(void *);
static void (*recallStub[DSCVMethodEnd])(void *);

#define DSCUDAVERB_SET_STUBS(mthd) \
  storeArgsStub[DSCVMethod ## mthd] = store ## mthd; \
  releaseArgsStub[DSCVMethod ## mthd] = release ## mthd; \
  recallStub[DSCVMethod ## mthd] = recall ## mthd;

#define DSCUDAVERB_SET_ARGS(mthd) \
  cuda ## mthd ## Args *argsrc; \
  argsrc = (cuda ## mthd ## Args *)argp;

#define DSCUDAVERB_STORE_ARGS(mthd) \
  DSCUDAVERB_SET_ARGS(mthd); \
  cuda ## mthd ## Args *argdst; \
  argdst = (cuda ## mthd ## Args *)malloc(sizeof(cuda ## mthd ## Args)); \
  *argdst = *(cuda ## mthd ## Args *)argp;


//mapping RPCfunctionID to DSCUDAVerbMethodID
static DSCVMethod funcID2DSCVMethod(int funcID) {
    switch (funcID) {
      case dscudaSetDeviceId:
	return DSCVMethodSetDevice;
      case dscudaMallocId:
	return DSCVMethodMalloc;
      case dscudaMemcpyH2DId:
	return DSCVMethodMemcpyH2D;
      case dscudaMemcpyD2DId:
	return DSCVMethodMemcpyD2D;
      case dscudaMemcpyD2HId:
	return DSCVMethodMemcpyD2H;
      case dscudaMemcpyToSymbolH2DId:
	return DSCVMethodMemcpyToSymbolH2D;
      case dscudaMemcpyToSymbolD2DId:
	return DSCVMethodMemcpyToSymbolD2D;
      case dscudaFreeId:
	return DSCVMethodFree;
	/*    
	      case dscudaLoadModuleId:
	      return DSCVMethodLoadModule;
	*/
      case dscudaLaunchKernelId:
	if (dscudaRemoteCallType() == RC_REMOTECALL_TYPE_IBV) {
	    return DSCVMethodIbvLaunchKernel;
	} else {
	    return DSCVMethodRpcLaunchKernel;
	}
      default:
	return DSCVMethodNone;
    }
}

//stubs for store args
static void *
storeSetDevice(void *argp) {
    WARN(3, "add hist hipSetDevice\n");
    DSCUDAVERB_STORE_ARGS(SetDevice); 
    return argdst;
}

static void *
storeMalloc(void *argp) {
    //nothing to do
    return NULL;
}

static void *
storeMemcpyH2D(void *argp) {
    WARN(3, "add hist cudaMemcpyH2D\n");
    DSCUDAVERB_STORE_ARGS(Memcpy);
    argdst->src = malloc(argsrc->count + 1);
    memcpy(argdst->src, (const void *)argsrc->src, argsrc->count);
    return argdst;
}

static void *
storeMemcpyD2D(void *argp) {
    WARN(3, "add hist cudaMemcpyD2D\n");
    DSCUDAVERB_STORE_ARGS(Memcpy);
    return argdst;
}

static void *
storeMemcpyD2H(void *argp) {
    WARN(3, "add hist cudaMemcpyD2H\n");
    DSCUDAVERB_STORE_ARGS(Memcpy);
    return argdst;
}

static void *
storeMemcpyToSymbolH2D(void *argp) {
    WARN(3, "add hist cudaMemcpyToSymbolH2D\n");
    DSCUDAVERB_STORE_ARGS(MemcpyToSymbol);
    
    int nredundancy = dscudaNredundancy();
    argdst->moduleid = (int *)malloc(sizeof(int) * nredundancy);
    memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);
  
    argdst->symbol = (char *)malloc(sizeof(char) * (strlen(argsrc->symbol) + 1));
    argdst->src = malloc(argsrc->count);
    
    strcpy(argdst->symbol, argsrc->symbol);
    memcpy(argdst->src, argsrc->src, argsrc->count);

    return argdst;
}

static void *
storeMemcpyToSymbolD2D(void *argp) {
    WARN(3, "add hist cudaMemcpyToSymbolD2D\n");
    DSCUDAVERB_STORE_ARGS(MemcpyToSymbol);

    int nredundancy = dscudaNredundancy();
    argdst->moduleid = (int *)malloc(sizeof(int) * nredundancy);
    memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);

    argdst->symbol = (char *)malloc(sizeof(char) * (strlen(argsrc->symbol) + 1));
    strcpy(argdst->symbol, argsrc->symbol);
    
    return argdst;
}

static void *
storeFree(void *argp) {
    //nothing to do
    return NULL;
}

static void *
storeLoadModule(void *argp) {
    DSCUDAVERB_STORE_ARGS(LoadModule);
    argdst->name = (char *)malloc(sizeof(char) * (strlen(argsrc->name) + 1));
    argdst->strdata = (char *)malloc(sizeof(char) * (strlen(argsrc->strdata) + 1));
    strcpy(argdst->name, argsrc->name);
    strcpy(argdst->strdata, argsrc->strdata);
    return argdst;
}

static void *
storeRpcLaunchKernel(void *argp)
{
    WARN(3, "add hist RpcLaunchKernel\n");
    DSCUDAVERB_STORE_ARGS(RpcLaunchKernel);

    int nredundancy = dscudaNredundancy();
    argdst->moduleid = (int *)malloc(sizeof(int) * nredundancy);
    memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);
    
    argdst->kname = (char *)malloc(sizeof(char) * strlen(argsrc->kname) + 1);
    strcpy(argdst->kname, argsrc->kname);
    
    int narg = argsrc->args.RCargs_len;
    RCarg *rpcargbuf = (RCarg *)malloc(sizeof(RCarg) * narg);
    memcpy(rpcargbuf, argsrc->args.RCargs_val, sizeof(RCarg) * narg);
    argdst->args.RCargs_val = rpcargbuf;

    return argdst;
}

static void *
storeIbvLaunchKernel(void *argp)
{
    WARN(3, "add hist IbvLaunchKernel\n");
    DSCUDAVERB_STORE_ARGS(IbvLaunchKernel);

    int nredundancy = dscudaNredundancy();
    argdst->moduleid = (int *)malloc(sizeof(int) * nredundancy);
    memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);

    argdst->kname = (char *)malloc(sizeof(char) * strlen(argsrc->kname) + 1);
    strcpy(argdst->kname, argsrc->kname);

    argdst->gdim = (int *)malloc(sizeof(dim3));
    argdst->bdim = (int *)malloc(sizeof(dim3));
    memcpy(argdst->gdim, argsrc->gdim, sizeof(dim3));
    memcpy(argdst->bdim, argsrc->bdim, sizeof(dim3));
    
    int narg = argsrc->narg;
    IbvArg *ibvargbuf = (IbvArg *)malloc(sizeof(IbvArg) * narg);
    memcpy(ibvargbuf, argsrc->arg, sizeof(IbvArg) * narg);
    argdst->arg = ibvargbuf;
    
    return argdst;
}

//stubs for release args
static void
releaseSetDevice(void *argp) {
    DSCUDAVERB_SET_ARGS(SetDevice);
    free(argsrc);
}

static void
releaseMalloc(void *argp) {
    //nothing to do
}

static void
releaseMemcpyH2D(void *argp) {
    DSCUDAVERB_SET_ARGS(Memcpy);
    free(argsrc->src);
    free(argsrc);
}

static void
releaseMemcpyD2D(void *argp) {
    DSCUDAVERB_SET_ARGS(Memcpy);
    free(argsrc);
}

static void
releaseMemcpyD2H(void *argp) {
    DSCUDAVERB_SET_ARGS(Memcpy);
    free(argsrc);
}

static void
releaseMemcpyToSymbolH2D(void *argp) {
    DSCUDAVERB_SET_ARGS(MemcpyToSymbol);
    free(argsrc->moduleid);
    free(argsrc->symbol);
    free(argsrc->src);
    free(argsrc);
}

static void
releaseMemcpyToSymbolD2D(void *argp) {
    DSCUDAVERB_SET_ARGS(MemcpyToSymbol);
    free(argsrc->moduleid);
    free(argsrc->symbol);
    free(argsrc);

}

static void
releaseFree(void *argp) {
    //nothing to do
}

static void
releaseLoadModule(void *argp) {
    DSCUDAVERB_SET_ARGS(LoadModule);
    free(argsrc->name);
    free(argsrc->strdata);
    free(argsrc);
}

static void
releaseRpcLaunchKernel(void *argp) {
    DSCUDAVERB_SET_ARGS(RpcLaunchKernel);
    free(argsrc->moduleid);
    free(argsrc->kname);
    free(argsrc->args.RCargs_val);
    free(argsrc);
}

static void
releaseIbvLaunchKernel(void *argp) {
    DSCUDAVERB_SET_ARGS(IbvLaunchKernel);
    free(argsrc->moduleid);
    free(argsrc->kname);
    free(argsrc->gdim);
    free(argsrc->bdim);
    free(argsrc->arg);
    free(argsrc);
}

//stubs for recall
static void
recallSetDevice(void *argp) {
    DSCUDAVERB_SET_ARGS(SetDevice);
    WARN(3, "Recall hipSetDevice()...\n");
    hipSetDevice(argsrc->device);
}

static void
recallMalloc(void *argp) {
    //nothing to do
}

static void
recallMemcpyH2D(void *argp) {
    DSCUDAVERB_SET_ARGS(Memcpy);
    WARN(3, "Recall cudaMemcpyH2D()...\n");
    hipMemcpy(argsrc->dst, argsrc->src, argsrc->count, hipMemcpyHostToDevice);
}

static void
recallMemcpyD2D(void *argp) {
    DSCUDAVERB_SET_ARGS(Memcpy);
    WARN(3, "Recall cudaMemcpyD2D()...\n");
    hipMemcpy(argsrc->dst, argsrc->src, argsrc->count, hipMemcpyDeviceToDevice);
}

static void
recallMemcpyD2H(void *argp) {
    DSCUDAVERB_SET_ARGS(Memcpy);
    WARN(3, "Recall cudaMemcpyD2H()...\n");
    hipMemcpy(argsrc->dst, argsrc->src, argsrc->count, hipMemcpyDeviceToHost);
}

static void
recallMemcpyToSymbolH2D(void *argp) {
    DSCUDAVERB_SET_ARGS(MemcpyToSymbol);
    WARN(3, "recall cudaMemcpyToSymbolH2D\n");
    dscudaMemcpyToSymbolWrapper(argsrc->moduleid, argsrc->symbol, argsrc->src, argsrc->count, argsrc->offset, hipMemcpyHostToDevice);
}

static void
recallMemcpyToSymbolD2D(void *argp) {
    DSCUDAVERB_SET_ARGS(MemcpyToSymbol);
    WARN(3, "recall cudaMemcpyToSymbolD2D\n");
    dscudaMemcpyToSymbolWrapper(argsrc->moduleid, argsrc->symbol, argsrc->src, argsrc->count, argsrc->offset, hipMemcpyDeviceToDevice);
}

static void
recallFree(void *argp) {
    //nothing to do
}

static void
recallLoadModule(void *argp) {
    DSCUDAVERB_SET_ARGS(LoadModule);
}

static void
recallIbvLaunchKernel(void *argp) {
    DSCUDAVERB_SET_ARGS(IbvLaunchKernel);
    WARN(3, "recall IbvLaunchKernel\n");
    ibvDscudaLaunchKernelWrapper(argsrc->moduleid, argsrc->kid, argsrc->kname, argsrc->gdim, argsrc->bdim, argsrc->smemsize, argsrc->stream, argsrc->narg, argsrc->arg);
}

static void
recallRpcLaunchKernel(void *argp) {
    DSCUDAVERB_SET_ARGS(RpcLaunchKernel);
    WARN(3, "Recall RpcLaunchKernel()...\n");
    rpcDscudaLaunchKernelWrapper(argsrc->moduleid, argsrc->kid, argsrc->kname, argsrc->gdim, argsrc->bdim, argsrc->smemsize, argsrc->stream, argsrc->args);
}

//initialize redundant unit
void
dscudaVerbInit(void) {
    memset(storeArgsStub,   0, sizeof(DSCVMethod) * DSCVMethodEnd);
    memset(releaseArgsStub, 0, sizeof(DSCVMethod) * DSCVMethodEnd);
    memset(recallStub,      0, sizeof(DSCVMethod) * DSCVMethodEnd);
  
    DSCUDAVERB_SET_STUBS(SetDevice);
    DSCUDAVERB_SET_STUBS(Malloc);
    DSCUDAVERB_SET_STUBS(MemcpyH2D);
    DSCUDAVERB_SET_STUBS(MemcpyD2D);
    DSCUDAVERB_SET_STUBS(MemcpyD2H);
    DSCUDAVERB_SET_STUBS(MemcpyToSymbolH2D);
    DSCUDAVERB_SET_STUBS(MemcpyToSymbolD2D);
    DSCUDAVERB_SET_STUBS(Free);
    //DSCUDAVERB_SET_STUBS(LoadModule);
    DSCUDAVERB_SET_STUBS(RpcLaunchKernel);
    DSCUDAVERB_SET_STUBS(IbvLaunchKernel);

    for (int i=1; i<DSCVMethodEnd; i++) {
	if (!storeArgsStub[i]) {
	    fprintf(stderr, "dscudaVerbInit: storeArgsStub[%d] is not initialized.\n", i);
	    exit(1);
	}
	if (!releaseArgsStub[i]) {
	    fprintf(stderr, "dscudaVerbInit: releaseArgsStub[%d] is not initialized.\n", i);
	    exit(1);
	}
	if (!recallStub[i]) {
	    fprintf(stderr, "dscudaVerbInit: recallStub[%d] is not initialized.\n", i);
	    exit(1);
	}
    }
    St.unsetRecordHist();
}

void
dscudaVerbAddHist(int funcID, void *argp)
{
    int DSCVMethodId;

    if (verbHistNum == verbHistMax) { /* Extend the existing memory region. */
	verbHistMax += DSCUDAVERB_HISTMAX_GROWSIZE;
	verbHists = (dscudaVerbHist *)realloc(verbHists, sizeof(dscudaVerbHist) * verbHistMax);
    }

    DSCVMethodId = funcID2DSCVMethod(funcID);
    verbHists[verbHistNum].args = (storeArgsStub[funcID2DSCVMethod(funcID)])(argp);
    verbHists[verbHistNum].funcID = funcID;
    verbHistNum++; /* Increment the count of cuda call */

    switch (funcID2DSCVMethod(funcID)) {
      case DSCVMethodMemcpyD2D: { /* hipMemcpy(DevicetoDevice) */
	  cudaMemcpyArgs *args = (cudaMemcpyArgs *)argp;
	  verbAllocatedMem *mem = verbAllocatedMemQuery(args->dst);
	  if (!mem) {
	      break;
	  }
	  int verb = St.isAutoVerb();
	  St.unsetAutoVerb();
	  hipMemcpy(mem->dst, args->src, args->count, hipMemcpyDeviceToHost);
	  St.setAutoVerb(verb);
	  break;
      }
    }
    return;
}
/*
 *
 */
void
dscudaVerbClearHist(void)
{
   if (verbHists) {
      for (int i=0; i<verbHistNum; i++) {
         (releaseArgsStub[funcID2DSCVMethod(verbHists[i].funcID)])(verbHists[i].args);
      }
      //free(verbHists);
      //verbHists = NULL;
   }
   verbHistNum = 0;
   
   WARN(3, "\"%s\":%s()> function history cleared.\n", __FILE__, __func__);
   return;
}

void
dscudaClearHist(void)
{
    dscudaVerbClearHist();
}

void
dscudaPrintHist(void)
{
    WARN(1, "%s(): *************************************************\n", __func__);
    if (verbHistNum==0) {
	WARN(1, "%s(): Recall History[]> (Empty).\n", __func__);
	return;
    }
    for (int i=0; i<verbHistNum; i++) { /* Print recall history. */
	WARN(1, "%s(): Recall History[%d]> ", __func__, i);
	switch (verbHists[i].funcID) { /* see "dscudarpc.h" */
	  case 305: WARN(1, "hipSetDevice()\n");        break;
	  case 504: WARN(1, "hipEventRecord()\n");      break;
	  case 505: WARN(1, "hipEventSynchronize()\n"); break;
	  case 600: WARN(1, "kernel-call<<< >>>()\n");   break;
	  case 700: WARN(1, "hipMalloc()\n");           break;
	  case 701: WARN(1, "hipFree()\n");             break;
	  case 703: WARN(1, "hipMemcpy(H2D)\n");        break;
	  case 704: WARN(1, "hipMemcpy(D2H)\n");        break;
	  default:  WARN(1, "/* %d */()\n", verbHists[i].funcID);
	}
    }
    WARN(1, "%s(): *************************************************\n", __func__);
}

void
dscudaVerbRecallHist(void)
{
   char       func_name[256]; 
   static int called_depth=0;

   WARN(1, "#<--- Entering (depth=%d) %d function(s)..., %s().\n", called_depth, verbHistNum, __func__);
   WARN(1, "called_depth= %d.\n", called_depth);
   if (called_depth >= RC_REDUNDANT_GIVEUP_COUNT) {
       WARN(1, "#*****************************************************\n");
       WARN(1, "# (;_;) I give up redundant calculation.             \n"); 
       WARN(1, "#       I have tried %d times and all failed.        \n", RC_REDUNDANT_GIVEUP_COUNT);
       WARN(1, "#*****************************************************\n");
       exit(1);
       called_depth=0;
   }
   else {
       dscudaPrintHist();
       called_depth++;       
       for (int i=0; i<verbHistNum; i++) { /* Do recall history */
	   (recallStub[funcID2DSCVMethod(verbHists[i].funcID)])(verbHists[i].args); /* partially recursive */
       }
       called_depth=0;
   }
   WARN(1, "#---> Exiting (depth=%d) done, %s()\n", called_depth, __func__);
}
