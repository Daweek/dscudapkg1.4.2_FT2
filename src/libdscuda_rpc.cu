//                             -*- Mode: C++ -*-
// Filename         : libdscuda_rpc.cu
// Description      : DS-CUDA client node library for Remote Procedure Call.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-08-24 18:16:31
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include <stdio.h>
#include <string.h>
#include <netdb.h>
#include <sys/socket.h>
#include <rpc/rpc.h>
#include <time.h>
#include <sys/types.h>
#include <unistd.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <pthread.h>
#include "dscuda.h"
#include "libdscuda.h"
#include "dscudaverb.h"

#define DEBUG 1

int dscudaRemoteCallType(void)
{
    return RC_REMOTECALL_TYPE_RPC;
}

void setupConnection(int idev, RCServer_t *sp) {
    int  id   = sp->id;
    int  cid  = sp->cid;
    int  pgid = DSCUDA_PROG;
    char msg[256];

    struct sockaddr_in sockaddr;
    int ssock = RPC_ANYSOCK; // socket to the server for RPC communication.
                             // automatically created by clnttcp_create().
    int sport; // port number of the server. given by the daemon, or calculated from cid.

    St.useRpc();
    if ( St.daemon > 0 ) { // access to the server via daemon.
        sport = requestDaemonForDevice(sp->ip, cid, St.isIbv());
    } else { // directly access to the server.
        sport = RC_SERVER_IP_PORT + cid;
    }
    sockaddr = setupSockaddr(sp->ip, sport);

    Clnt[idev][id] = clnttcp_create(&sockaddr,
                                    pgid,
                                    DSCUDA_VER,
                                    &ssock,
                                    RC_BUFSIZE, RC_BUFSIZE);

    sprintf(msg, "%s:%d (port %d) ", sp->ip, cid, sport);

    if ( !Clnt[idev][id] ) {
        clnt_pcreateerror(msg);
        if ( 0 == strcmp(sp->ip, DEFAULT_SVRIP) ) {
            WARN(0, "You may need to set an environment variable 'DSCUDA_SERVER'.\n");
        } else {
            WARN(0, "DSCUDA server (dscudasrv on %s:%d) may be down.\n", sp->ip, id);
        }
        exit(1);
    }
    WARN(2, "Established a socket connection to %s...\n", msg);
}

void checkResult(void *rp, RCServer_t *sp) {
    if (rp) {
	return;
    } else {
	WARN(0, "NULL pointer returned, %s(). exit.\n", __func__);
	clnt_perror(Clnt[Vdevid[vdevidIndex()]][sp->id], sp->ip);
	exit(1);
    }
}

static
void recoverClntError(RCServer_t *failed, RCServer_t *spare, struct rpc_err *err)
{
    switch ( err->re_status ) {
	/* re_status is "clnt_stat" type.
	 * refer to /usr/include/rpc/clnt.h.
	 */
    case RPC_SUCCESS: //=0
	break;
    case RPC_CANTSEND: //=3
	break;
    case RPC_CANTRECV: //=4
	break;
    case RPC_TIMEDOUT: //=5
	WARN(1, "Detected RPC:Timed Out in  %s().\n", __func__);
	dscudaVerbMigrateDevice( failed, spare );
	break;
    case RPC_UNKNOWNHOST: //=13
	break;
    case RPC_UNKNOWNPROTO: //=17
	break;
    case RPC_UNKNOWNADDR: //=19
	break;
    default:
	break;
    }
}

/*
 * Dscuda client-side counterpart for CUDA runtime API:
 */

/*
 * Thread Management
 */

hipError_t hipDeviceReset(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceReset()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadexitid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if ( rp->err != hipSuccess ) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t hipDeviceSynchronize(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    St.cudaCalled();
    WARN(3, "hipDeviceSynchronize()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadsynchronizeid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t cudaThreadSetLimit(enum hipLimit_t limit, size_t value)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "cudaThreadSetLimit(%d, %zu)...", limit, value);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadsetlimitid_1(limit, value, Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t cudaThreadGetLimit(size_t *pValue, enum hipLimit_t limit)
{
    hipError_t err = hipSuccess;
    dscudaThreadGetLimitResult *rp;
    int vid = vdevidIndex();

    WARN(3, "cudaThreadGetLimit(%p, %d)...", pValue, limit);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadgetlimitid_1(limit, Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pValue = rp->value;
        }
        xdr_free((xdrproc_t)xdr_dscudaThreadGetLimitResult, (char *)rp);
    }
    WARN(3, "done.  *pValue: %zu\n", *pValue);

    return err;
}

hipError_t
hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceSetCacheConfig(%d)...", cacheConfig);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadsetcacheconfigid_1(cacheConfig, Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig)
{
    hipError_t err = hipSuccess;
    dscudaThreadGetCacheConfigResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceGetCacheConfig(%p)...", pCacheConfig);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadgetcacheconfigid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pCacheConfig = (enum hipFuncCache_t)rp->cacheConfig;
        }
        xdr_free((xdrproc_t)xdr_dscudaThreadGetCacheConfigResult, (char *)rp);
    }
    WARN(3, "done.  *pCacheConfig: %d\n", *pCacheConfig);

    return err;
}


/*
 * Error Handling
 */

hipError_t hipGetLastError(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(5, "hipGetLastError()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudagetlasterrorid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(5, "done.\n");

    return err;
}

hipError_t hipPeekAtLastError(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(5, "hipPeekAtLastError()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudapeekatlasterrorid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(5, "done.\n");

    return err;
}

const char *hipGetErrorString(hipError_t error)
{
    dscudaGetErrorStringResult *rp;
    static char str[4096];
    int vid = vdevidIndex();

    WARN(5, "hipGetErrorString()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudageterrorstringid_1(error, Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (i == 0) {
            strcpy(str, rp->errmsg);
        }
        xdr_free((xdrproc_t)xdr_dscudaGetErrorStringResult, (char *)rp);
    }
    WARN(5, "done.\n");

    return str;
}

/*
 * Device Management
 */

hipError_t hipSetDeviceFlags(unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipSetDeviceFlags()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudasetdeviceflagsid_1(flags, Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    invalidateModuleCache();

    WARN(3, "done.\n");

    return err;
}

hipError_t hipDriverGetVersion (int *driverVersion)
{
    hipError_t err = hipSuccess;
    dscudaDriverGetVersionResult *rp;
    int vid = vdevidIndex();

    WARN(3, "cudaDriverGetVersionCount(%p)...", driverVersion);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudadrivergetversionid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *driverVersion = rp->ver;
    WARN(3, "done.\n");

    return err;
}

hipError_t hipRuntimeGetVersion(int *runtimeVersion)
{
    hipError_t err = hipSuccess;
    dscudaRuntimeGetVersionResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipRuntimeGetVersion(%p)...", runtimeVersion);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaruntimegetversionid_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *runtimeVersion = rp->ver;
        }
        xdr_free((xdrproc_t)xdr_dscudaRuntimeGetVersionResult, (char *)rp);
    }

    WARN(3, "done.\n");

    return err;
}

hipError_t hipDeviceSynchronize(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceSynchronize()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudadevicesynchronize_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t hipDeviceReset(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    int vid = vdevidIndex();

    WARN(3, "hipDeviceReset()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudadevicereset_1(Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

/*
 * Execution Control
 */

hipError_t
hipFuncSetCacheConfig(reinterpret_cast<const void*>(const char * func), enum hipFuncCache_t cacheConfig)
{
    hipError_t err = hipSuccess;
    WARN(1, "Current implementation of hipFuncSetCacheConfig() does nothing "
         "but returning hipSuccess.\n");
    err = hipSuccess;
    return err;
}

/*
 * Memory Management
 */

hipError_t hipMalloc(void **devAdrPtr, size_t size) {
    dscudaMallocResult *rp;
    hipError_t err = hipSuccess;
    int vid = vdevidIndex();
    void *adrs[RC_NREDUNDANCYMAX];
    CLIENT *p_clnt;

    WARN(3, "hipMalloc( %p, %zu )...\n", devAdrPtr, size);
    St.cudaCalled();
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for ( int i = 0; i < vdev->nredundancy; i++, sp++ ) {
	p_clnt = Clnt[Vdevid[vid]][sp->id]; 
        rp = dscudamallocid_1( size, p_clnt );
	//recoverClntError( sp, &(SvrSpare.svr[0]),  p_clnt);
        checkResult(rp, sp);
        if ( rp->err != hipSuccess ) {
            err = (hipError_t)rp->err;
        }
        adrs[i] = (void*)rp->devAdr;
	WARN(3, "+--- redun[%d]: devAdrPtr=%p\n", i, adrs[i]);	
        xdr_free((xdrproc_t)xdr_dscudaMallocResult, (char *)rp);
    }

    RCuvaRegister(Vdevid[vid], adrs, size);
    *devAdrPtr = dscudaUvaOfAdr(adrs[0], Vdevid[vid]);
    /*
     * Automatic Recoverly
     */
    if ( St.isAutoVerb() ) {
	cudaMallocArgs args( *devAdrPtr, size );
	BKUPMEM.addRegion(args.devPtr, args.size);  /* Allocate mirroring memory */
    }
    WARN(3, "+--- done. *devAdrPtr:%p, Length of Registered MemList: %d\n", *devAdrPtr, BKUPMEM.countRegion());

    return err;
}

hipError_t hipFree(void *mem) {
    int          vid = vdevidIndex();
    hipError_t  err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipFree(%p)...", mem);
    Vdev_t *vdev = St.Vdev + Vdevid[vid];
    RCServer_t *sp = vdev->server;
    for (int i=0; i < vdev->nredundancy; i++, sp++) {
	rp = dscudafreeid_1((RCadr)dscudaAdrOfUva(mem), Clnt[Vdevid[vid]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCuvaUnregister(mem);

    /*
     * Automatic Recoverly
     */
    if (St.isAutoVerb()) {
	BKUPMEM.removeRegion(mem);
    }
    WARN(3, "+--- done.\n");
    return err;
}

static hipError_t
cudaMemcpyH2D(void *dst, const void *src, size_t count, Vdev_t *vdev, CLIENT **clnt)
{
    WARN( 4, "   libdscuda:%s() called with \"%s(%s)\" recordHist=%d, histoCalling=%d {\n",
	  __func__, St.getFtModeString(), vdev->info, HISTREC.rec_en, St.isHistoCalling() );
    dscudaResult *rp;
    RCServer_t *sp;
    RCbuf srcbuf;
    hipError_t err = hipSuccess;

    St.cudaCalled();
    srcbuf.RCbuf_len = count;
    srcbuf.RCbuf_val = (char *)src;
    sp = vdev->server;
    for ( int i = 0; i < vdev->nredundancy; i++, sp++ ) {
	WARN( 4, "      + Physical[%d] dst=%p\n", i, dst);
        rp = dscudamemcpyh2did_1((RCadr)dst, srcbuf, count, clnt[sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    if ( St.ft_mode==FT_REDUN || St.ft_mode==FT_MIGRA || St.ft_mode==FT_BOTH ) {
	cudaMemcpyArgs args( dst, (void*)src, count, hipMemcpyHostToDevice );
	HISTREC.add(dscudaMemcpyH2DId, (void *)&args);
    }
    WARN( 4, "   } libdscuda:%s().\n", __func__);
    return err;
}

/*
 * hipMemcpy(DeviceToHost)
 */
hipError_t
cudaMemcpyD2H_redundant( void *dst, void *src_uva, size_t count, int redundant ) {
    WARN(3, "%s( dst=%p, src_uva=%p, count=%zu redundant=%d ).\n",
	 __func__, dst, src_uva, count, redundant );
    int vdevid;
    RCServer_t *sp;
    CLIENT **clnt;
    dscudaMemcpyD2HResult *rp;

    hipError_t err = hipSuccess;
    void *src = dscudaAdrOfUva( (void *)src_uva );
    
    vdevid = Vdevid[ vdevidIndex() ];  // Get active device ID#.
    clnt   = Clnt[vdevid];
    /* Get the data from remote GPU(s), then verify */
    sp = &St.Vdev[vdevid].server[redundant];
    rp = dscudamemcpyd2hid_1((RCadr)src, count, clnt[sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    if (rp->err != hipSuccess) {
	err = (hipError_t)rp->err;
    }

    memcpy(dst, rp->buf.RCbuf_val, rp->buf.RCbuf_len);
    xdr_free((xdrproc_t)xdr_dscudaMemcpyD2HResult, (char *)rp);
    WARN(3, "+--- done.\n");
    return err;
}

static hipError_t
cudaMemcpyD2H( void *dst, void *src, size_t count, Vdev_t *vdev, CLIENT **clnt ) {
    WARN( 4, "   libdscuda:%s() called with \"%s(%s)\" {\n",
	  __func__, St.getFtModeString(), vdev->info );

    int matched_count   = 0;
    int unmatched_count = 0;
    int recall_result;

    cudaMemcpyArgs args;
    dscudaMemcpyD2HResult *rp;

    RCServer_t *failed_1st;
    //    int fail_flag[RC_NVDEVMAX]={0};
    hipError_t err = hipSuccess;

    St.cudaCalled();
    /*
     * Register called history.
     */
    switch ( St.ft_mode ) {
    case FT_PLAIN:
	break;
    case FT_REDUN: //thru
    case FT_MIGRA: //thru
    case FT_BOTH:
	args.dst   = (void *)dst;
	args.src   = (void *)src;
	args.count = count;
	args.kind  = hipMemcpyDeviceToHost;
	HISTREC.add(dscudaMemcpyD2HId, (void *)&args); // not needed?
	break;
    default:
	WARN( 0, "Unexpected failure.\n");
	exit( EXIT_FAILURE );
    }

    /* Get the data from remote GPU(s), then verify */
    RCServer_t *sp = vdev->server;
    for ( int i=0; i < vdev->nredundancy; i++, sp++ ) {
	WARN(4, "      + Physical[%d]:hipMemcpy( dst=%p, src=%p, count=%zu )\n", i, dst, src, count);
	/*
	 * Access to Physical GPU Device.
	 */
        rp = dscudamemcpyd2hid_1( (RCadr)src, count, clnt[sp->id] );
        checkResult(rp, sp);
        err = (hipError_t)rp->err;
        if ( rp->err != hipSuccess ) {
            err = (hipError_t)rp->err;
        }
	
        if ( i==0 ) {
	    memcpy( dst, rp->buf.RCbuf_val, rp->buf.RCbuf_len );
        } else {
	    if ( bcmp( dst, rp->buf.RCbuf_val, rp->buf.RCbuf_len ) != 0 ) { // unmatched case
		sp->errcount++; //count up error.
		WARN( 0, "[ERRORSTATICS] Total Error Count: %d\n", sp->errcount );
		unmatched_count++;
		//fail_flag[i]=1;
		failed_1st = sp; // temporary
		WARN(2, "   UNMATCHED redundant device %d/%d with device 0. %s()\n", i, vdev->nredundancy - 1, __func__);
	    } else { /* Matched case */
		matched_count++;
		//fail_flag[i]=0;
		WARN(3, "   Matched   reduncant device %d/%d with device 0. %s()\n", i, vdev->nredundancy - 1, __func__);
		memcpy(dst, rp->buf.RCbuf_val, rp->buf.RCbuf_len); // overwrite matched data
	    }
	}
	xdr_free( (xdrproc_t)xdr_dscudaMemcpyD2HResult, (char *)rp );
    }

    switch ( vdev->conf ) {
    case VDEV_MONO:
	if (( St.ft_mode==FT_REDUN || St.ft_mode==FT_MIGRA || St.ft_mode==FT_BOTH ) && (St.isHistoCalling()==0 )) {
	    BKUPMEM.updateRegion( src, dst, count );
	}
	break;
    case VDEV_POLY:
	if ( unmatched_count==0 && matched_count==(vdev->nredundancy-1)) {
	    WARN(5, "   #\\(^_^)/ All %d Redundant device(s) matched. statics OK/NG = %d/%d.\n",
		 vdev->nredundancy-1, matched_count, unmatched_count);
	    /*
	     * Update backuped memory region.
	     */
	    if (( St.ft_mode==FT_REDUN || St.ft_mode==FT_MIGRA || St.ft_mode==FT_BOTH ) && (St.isHistoCalling()==0 )) {
		WARN( 5, "checkpoint-0\n");
		BKUPMEM.updateRegion(src, dst, count); /* mirroring copy. !!!src and dst is swapped!!! */
		WARN( 5, "checkpoint-1\n");
	    }
	} else { /* redundant failed */
	    if ( unmatched_count>0 && matched_count<(vdev->nredundancy-1)) {
		WARN( 1, " #   #\n");
		WARN( 1, "  # #\n");
		WARN( 1, "   #  Detected Unmatched result. OK/NG= %d/%d.\n", matched_count, unmatched_count);
		WARN( 1, "  # #\n");
		WARN( 1, " #   #\n");
	    } else {
		WARN(1, "   #(;_;)   All %d Redundant device(s) unmathed. statics OK/NG = %d/%d.\n",
		     vdev->nredundancy-1, matched_count, unmatched_count);
	    }

	    if (( St.ft_mode==FT_REDUN || St.ft_mode==FT_MIGRA || St.ft_mode==FT_BOTH ) && (St.isHistoCalling()==0 )) {
		St.unsetAutoVerb();    // <=== Must be disabled autoVerb during Historical Call.
		HISTREC.rec_en = 0; // <--- Must not record Historical call list.
	    
		BKUPMEM.restructDeviceRegion();
		recall_result = HISTREC.recall();
	    
		if (recall_result != 0) {
		    printModuleList();
		    printVirtualDeviceList();
		    //dscudaVerbMigrateDevice(failed_1st, &svrSpare[0]);
		    dscudaVerbMigrateDevice(failed_1st, &(SvrSpare.svr[0]));
		}
		HISTREC.on();  // ---> restore recordHist enable.
		St.setAutoVerb();    // ===> restore autoVerb enabled.
	    }
	}
	break;
    default: //irregular condition.
	WARN(1, "ERROR: # of redundancy is zero or minus value????. %s\n", __func__);
	exit( EXIT_FAILURE );
    }//switch

    WARN(4, "   } libdscuda:%s().\n", __func__);
    return err;
}

static hipError_t
cudaMemcpyD2D(void *dst, const void *src, size_t count, Vdev_t *vdev, CLIENT **clnt)
{
    dscudaResult *rp;
    RCServer_t *sp;
    hipError_t err = hipSuccess;

    sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamemcpyd2did_1((RCadr)dst, (RCadr)src, count, clnt[sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    //<--- oikawa moved to here from hipMemcpy();
    if (St.isAutoVerb() > 0) {
	cudaMemcpyArgs args( dst, (void *)src, count, hipMemcpyDeviceToDevice );
	HISTREC.add(dscudaMemcpyD2DId, (void *)&args);
    }
    //--->
    return err;
}

static hipError_t
cudaMemcpyP2P(void *dst, int ddev, const void *src, int sdev, size_t count)
{
    hipError_t err = hipSuccess;
    int dev0;
    int pgsz = 4096;
    static int bufsize = 0;
    static char *buf = NULL;

    if (bufsize < count) {
        bufsize = ((count - 1) / pgsz + 1) * pgsz;
        buf = (char *)realloc(buf, bufsize);
        if (!buf) {
            perror("cudaMemcpyP2P");
            exit(1);
        }
    }

    hipGetDevice(&dev0);

    if (sdev != dev0) {
        hipSetDevice(sdev);
    }
    err = hipMemcpy(buf, src, count, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        if (sdev != dev0) {
            hipSetDevice(dev0);
        }
        return err;
    }
    if (ddev != sdev) {
        hipSetDevice(ddev);
    }
    err = hipMemcpy(dst, buf, count, hipMemcpyHostToDevice);
    if (ddev != dev0) {
        hipSetDevice(dev0);
    }
    return err;
}

/*
 * 
 */
hipError_t hipMemcpy( void *dst, const void *src,
			size_t count, enum hipMemcpyKind kind ) {
    int         vdevid = Vdevid[ vdevidIndex() ];
    Vdev_t     *vdev   = St.Vdev + vdevid;
    CLIENT    **clnt   = Clnt[vdevid];
    RCuva *suva, *duva;
    int dev0;
    void *lsrc, *ldst;
    hipError_t err    = hipSuccess;

    lsrc = dscudaAdrOfUva((void *)src);
    ldst = dscudaAdrOfUva(dst);
    
    switch ( kind ) {
    case hipMemcpyDeviceToHost:
	WARN(3, "libdscuda:hipMemcpy(%p, %p, %zu, DeviceToHost) called vdevid=%d...\n",
	     ldst, lsrc, count, vdevid);
	// Avoid conflict between CheckPointing thread.
	pthread_mutex_lock( &cudaMemcpyD2H_mutex );
        err = cudaMemcpyD2H( ldst, lsrc, count, vdev, clnt );
	pthread_mutex_unlock( &cudaMemcpyD2H_mutex ); 
        break;
    case hipMemcpyHostToDevice:
	WARN(3, "libdscuda:hipMemcpy(%p, %p, %zu, HostToDevice) called\n", ldst, lsrc, count);
	// Avoid conflict with CheckPointing thread.	
	pthread_mutex_lock( &cudaMemcpyH2D_mutex );
        err = cudaMemcpyH2D( ldst, lsrc, count, vdev, clnt );
	pthread_mutex_unlock( &cudaMemcpyH2D_mutex );
        break;
    case hipMemcpyDeviceToDevice:
	WARN(3, "libdscuda:hipMemcpy(%p, %p, %zu, DeviceToDevice) called\n", ldst, lsrc, count);
        err = cudaMemcpyD2D(ldst, lsrc, count, vdev, clnt);
        break;
    case hipMemcpyDefault:
#if !__LP64__
        WARN(0, "hipMemcpy:In 32-bit environment, hipMemcpyDefault cannot be given as arg4."
             "UVA is supported for 64-bit environment only.\n");
        exit(1);
#endif

        hipGetDevice(&dev0);
        suva = RCuvaQuery((void *)src);
        duva = RCuvaQuery(dst);
        if ( !suva && !duva ) {
            WARN(0, "hipMemcpy:invalid argument.\n");
            exit(1);
        } else if ( !suva ) { // sbuf resides in the client.
            if ( duva->devid != dev0 ) {
                hipSetDevice( duva->devid );
            }
            err = hipMemcpy( dst, src, count, hipMemcpyHostToDevice );
            if ( duva->devid != dev0 ) {
                hipSetDevice( dev0 );
            }
        } else if ( !duva ) { // dbuf resides in the client.
            if ( suva->devid != dev0 ) {
                hipSetDevice( suva->devid );
            }
            err = hipMemcpy( dst, src, count, hipMemcpyDeviceToHost );
            if ( suva->devid != dev0 ) {
                hipSetDevice( dev0 );
            }
        } else {
            err = cudaMemcpyP2P( dst, duva->devid, src, suva->devid, count );
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "} libdscuda:%s().\n", __func__);
    WARN(3, "\n");
    return err;
}

hipError_t
hipMemcpyPeer(void *dst, int ddev, const void *src, int sdev, size_t count) {
    WARN(3, "hipMemcpyPeer(0x%08lx, %d, 0x%08lx, %d, %zu)...",
         (unsigned long)dst, ddev, (unsigned long)src, sdev, count);

    cudaMemcpyP2P(dst, ddev, src, sdev, count);

    WARN(3, "done.\n");
}

hipError_t
hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device)
{
    hipError_t err = hipSuccess;
    dscudaGetDevicePropertiesResult *rp;

    WARN(3, "hipGetDeviceProperties(0x%08lx, %d)...", (unsigned long)prop, device);
    Vdev_t *vdev = St.Vdev + device;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudagetdevicepropertiesid_1(device, Clnt[device][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            memcpy(prop, rp->prop.RCbuf_val, rp->prop.RCbuf_len);
        }
        xdr_free((xdrproc_t)xdr_dscudaGetDevicePropertiesResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

int
dscudaLoadModuleLocal(unsigned int ipaddr, pid_t pid, char *modulename, char *modulebuf, int vdevid, int raidid) {
    //WARN(10, "<---Entering %s()\n", __func__);
    //WARN(10, "ipaddr= %u, modulename= %s\n", ipaddr, modulename);
    
    int ret;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    /* send to virtual GPU */
    dscudaLoadModuleResult *rp = dscudaloadmoduleid_1(St.getIpAddress(), getpid(), modulename, modulebuf, Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    ret = rp->id;
    xdr_free((xdrproc_t)xdr_dscudaLoadModuleResult, (char *)rp);
    
    if (St.isAutoVerb() ) {
	/*Nop*/
    }

    //WARN(10, "--->Exiting  %s()\n", __func__);
    return ret;
}

/*
 * launch a kernel function of id 'kid', defined in a module of id 'moduleid'.
 * 'kid' must be unique inside a single module.
 */

void
rpcDscudaLaunchKernelWrapper(int *moduleid, int kid, char *kname,  /* moduleid is got by "dscudaLoadModule()" */
                             RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream,
                             RCargs args)
{
    WARN(5, "%s().\n", __func__)
    RCmappedMem *mem;
    RCstreamArray *st;
    CLIENT *p_clnt;
    
    pthread_mutex_lock( &cudaKernelRun_mutex ); // Avoid conflict with CheciPointing.p
    /*     
     * Automatic Recovery, Register to the called history.
     */
    if (St.isAutoVerb() ) {
        cudaRpcLaunchKernelArgs args2;
        args2.moduleid = moduleid;
        args2.kid      = kid;
        args2.kname    = kname;
        args2.gdim     = gdim;
        args2.bdim     = bdim;
        args2.smemsize = smemsize;
        args2.stream   = stream;
        args2.args     = args;
        HISTREC.add( dscudaLaunchKernelId, (void *)&args2 );
    }

    st = RCstreamArrayQuery((hipStream_t)stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }

    mem = RCmappedMemListTop;
    while (mem) {
        hipMemcpy(mem->pDevice, mem->pHost, mem->size, hipMemcpyHostToDevice);
        mem = mem->next;
    }

    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    struct rpc_err rpc_error;

    for ( int i = 0; i < vdev->nredundancy; i++, sp++ ) {
	p_clnt = Clnt[Vdevid[vdevidIndex()]][sp->id] ;
        void *rp = dscudalaunchkernelid_1(moduleid[i], kid, kname,
                                          gdim, bdim, smemsize, (RCstream)st->s[i],
                                          args, p_clnt );
	//<--- Timed Out
	clnt_geterr( p_clnt, &rpc_error );
	if ( rpc_error.re_status != RPC_SUCCESS ) {
	    break;
	}
	//--->
        checkResult(rp, sp);
    }

    recoverClntError(sp, &(SvrSpare.svr[0]), &rpc_error );
    
    mem = RCmappedMemListTop;
    while (mem) {
        hipMemcpy(mem->pHost, mem->pDevice, mem->size, hipMemcpyDeviceToHost);
        mem = mem->next;
    }
    //---> Avoid conflict with CheckPointing.
    pthread_mutex_unlock( &cudaKernelRun_mutex ); // Avoid conflict with CheciPointing.
    WARN(5, "+--- done. %s().\n", __func__)
}

#if !defined(RPC_ONLY)
void
ibvDscudaLaunchKernelWrapper(int *moduleid, int kid, char *kname,
                             int *gdim, int *bdim, RCsize smemsize, RCstream stream,
                             int narg, IbvArg *arg)
{
    // a dummy func.
}
#endif

hipError_t
hipMallocArray(struct hipArray **array, const struct hipChannelFormatDesc *desc,
                size_t width, size_t height, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaMallocArrayResult *rp;
    RCchanneldesc descbuf;
    hipArray *ca[RC_NREDUNDANCYMAX];

    WARN(3, "hipMallocArray(%p, %p, %zu, %zu, 0x%08x)...",
         array, desc, width, height, flags);

    descbuf.x = desc->x;
    descbuf.y = desc->y;
    descbuf.z = desc->z;
    descbuf.w = desc->w;
    descbuf.f = desc->f;

    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallocarrayid_1(descbuf, width, height, flags, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ca[i] = (hipArray *)rp->array;
        xdr_free((xdrproc_t)xdr_dscudaMallocArrayResult, (char *)rp);
    }

    *array = ca[0];
    RCcuarrayArrayRegister(ca);
    WARN(3, "done. *array:%p\n", *array);

    return err;
}

hipError_t
hipFreeArray(struct hipArray *array)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCcuarrayArray *ca;

    WARN(3, "hipFreeArray(%p)...", array);
    ca = RCcuarrayArrayQuery(array);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", array);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudafreearrayid_1((RCadr)ca->ap[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCcuarrayArrayUnregister(ca->ap[0]);
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipMemcpyToArray(struct hipArray *dst, size_t wOffset, size_t hOffset, const void *src,
                  size_t count, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCcuarrayArray *ca;
    Vdev_t *vdev;
    RCServer_t *sp;

    WARN(3, "hipMemcpyToArray(%p, %zu, %zu, %p, %zu, %s)...",
         dst, wOffset, hOffset, src, count, dscudaMemcpyKindName(kind));
    ca = RCcuarrayArrayQuery(dst);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", dst);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;

        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpytoarrayh2did_1((RCadr)ca->ap[i], wOffset, hOffset, srcbuf, count, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpytoarrayd2did_1((RCadr)ca->ap[i], wOffset, hOffset, (RCadr)src, count, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemset(void *devPtr, int value, size_t count)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipMemset()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamemsetid_1((RCadr)devPtr, value, count, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMallocPitch(void **devPtr, size_t *pitch, size_t width, size_t height)
{
    hipError_t err = hipSuccess;
    dscudaMallocPitchResult *rp;

    WARN(3, "hipMallocPitch(%p, %p, %zu, %zu)...", devPtr, pitch, width, height);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallocpitchid_1(width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *devPtr = (void*)rp->devPtr;
            *pitch = rp->pitch;
        }
        xdr_free((xdrproc_t)xdr_dscudaMallocPitchResult, (char *)rp);
    }

    WARN(3, "done. *devPtr:%p  *pitch:%zu\n", *devPtr, *pitch);

    return err;
}

hipError_t
hipMemcpy2DToArray(struct hipArray *dst, size_t wOffset, size_t hOffset,
                    const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpy2DToArrayD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCcuarrayArray *ca;
    Vdev_t *vdev;
    RCServer_t *sp;

    WARN(3, "hipMemcpy2DToArray(%p, %zu, %zu, %p, %zu, %zu, %zu, %s)...",
         dst, wOffset, hOffset,
         src, spitch, width, height, dscudaMemcpyKindName(kind));
    ca = RCcuarrayArrayQuery(dst);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", dst);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpy2dtoarrayd2hid_1(wOffset, hOffset,
                                                 (RCadr)src, spitch, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            } else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpy2DToArray() data copied from device%d matched with that from device0.\n", i);
            }
            xdr_free((xdrproc_t)xdr_dscudaMemcpy2DToArrayD2HResult, (char *)d2hrp);
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = spitch * height;
        srcbuf.RCbuf_val = (char *)src;
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpy2dtoarrayh2did_1((RCadr)ca->ap[i], wOffset, hOffset,
                                                 srcbuf, spitch, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpy2dtoarrayd2did_1((RCadr)ca->ap[i], wOffset, hOffset,
                                                 (RCadr)src, spitch, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemcpy2D(void *dst, size_t dpitch,
             const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpy2DD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    Vdev_t *vdev;
    RCServer_t *sp;

    WARN(3, "hipMemcpy2D(%p, %zu, %p, %zu, %zu, %zu, %s)...",
         dst, dpitch,
         src, spitch, width, height, dscudaMemcpyKindName(kind));

    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpy2dd2hid_1(dpitch,
                                          (RCadr)src, spitch, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            } else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            } else {
                WARN(3, "hipMemcpy() data copied from device%d matched with that from device0.\n", i);
            }
            xdr_free((xdrproc_t)xdr_dscudaMemcpy2DD2HResult, (char *)d2hrp);
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = spitch * height;
        srcbuf.RCbuf_val = (char *)src;
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpy2dh2did_1((RCadr)dst, dpitch,
                                          srcbuf, spitch, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = St.Vdev + Vdevid[vdevidIndex()];
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpy2dd2did_1((RCadr)dst, dpitch,
                                          (RCadr)src, spitch, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemset2D(void *devPtr, size_t pitch, int value, size_t width, size_t height)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipMemset2D(%p, %zu, %d, %zu, %zu)...",
         devPtr, pitch, value, width, height);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamemset2did_1((RCadr)devPtr, pitch, value, width, height, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipHostMalloc(void **ptr, size_t size)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaMallocHostResult *rp;

    WARN(3, "hipHostMalloc(%p, %d)...", ptr, size);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallochostid_1(size, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *ptr = (void*)rp->ptr;
        }
        xdr_free((xdrproc_t)xdr_dscudaMallocHostResult, (char *)rp);
    }

    WARN(3, "done. *ptr:%p\n", *ptr);
    return err;
#else
    // returned memory is not page locked.
    // it cannot be passed to hipMemcpyAsync().
    *ptr = malloc(size);
    if (*ptr) {
        return hipSuccess;
    } else {
        return hipErrorOutOfMemory;
    }
#endif
}

hipError_t
hipHostAlloc(void **pHost, size_t size, unsigned int flags)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaHostAllocResult *rp;

    WARN(3, "hipHostAlloc(0x%08llx, %d, 0x%08x)...", (unsigned long)pHost, size, flags);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudahostallocid_1(size, flags, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pHost = (void*)rp->pHost;
        }
        xdr_free((xdrproc_t)xdr_dscudaHostAllocResult, (char *)rp);
    }

    WARN(3, "done. *pHost:0x%08llx\n", *pHost);

    return err;
#else
    // returned memory is not page locked.
    // it cannot be passed to hipMemcpyAsync().

    hipError_t err = hipSuccess;
    void *devmem;

    WARN(3, "hipHostAlloc(%p, %zu, 0x%08x)...", pHost, size, flags);

    *pHost = malloc(size);
    if (!*pHost) return hipErrorOutOfMemory;
    if (!(flags & hipHostMallocMapped)) {
        WARN(3, "done. *pHost:%p\n", *pHost);
        return hipSuccess;
    }

    // flags says the host memory must be mapped on to the device memory.
    err = hipMalloc(&devmem, size);
    if (err == hipSuccess) {
        RCmappedMemRegister(*pHost, devmem, size);
    }
    WARN(3, "done. host mem:%p  device mem:%p\n", *pHost, devmem);

    return err;
#endif
}

hipError_t
hipHostFree(void *ptr)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    WARN(3, "hipHostFree(0x%08llx)...", (unsigned long)ptr);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudafreehostid_1((RCadr)ptr, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
#else
    hipError_t err = hipSuccess;
    RCmappedMem *mem = RCmappedMemQuery(ptr);
    free(ptr);
    if (mem) { // ptr mapped on to a device memory.
        err = hipFree(mem->pDevice);
        RCmappedMemUnregister(ptr);
        return err;
    } else {
        return hipSuccess;
    }
#endif
}

// flags is not used for now in CUDA3.2. It should always be zero.
hipError_t
hipHostGetDevicePointer(void **pDevice, void*pHost, unsigned int flags) {
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaHostGetDevicePointerResult *rp;

    WARN(3, "hipHostGetDevicePointer(0x%08llx, 0x%08llx, 0x%08x)...",
         (unsigned long)pDevice, (unsigned long)pHost, flags);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudahostgetdevicepointerid_1((RCadr)pHost, flags, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pDevice = (void *)rp->pDevice;
        }
        xdr_free((xdrproc_t)xdr_dscudaHostGetDevicePointerResult, (char *)rp);
    }

    WARN(3, "done. *pDevice:0x%08llx\n", *pDevice);
    return err;
#else
    RCmappedMem *mem = RCmappedMemQuery(pHost);
    if (!mem) return hipErrorInvalidValue; // pHost is not registered as RCmappedMem.
    *pDevice = mem->pDevice;
    return hipSuccess;
#endif
}

hipError_t
hipHostGetFlags(unsigned int *pFlags, void *pHost)
{
    hipError_t err = hipSuccess;
    dscudaHostGetFlagsResult *rp;

    WARN(3, "hipHostGetFlags(%p %p)...", pFlags, pHost);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudahostgetflagsid_1((RCadr)pHost, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            *pFlags = rp->flags;
        }
        xdr_free((xdrproc_t)xdr_dscudaHostGetFlagsResult, (char *)rp);
    }
    WARN(3, "done. flags:0x%08x\n", *pFlags);
    return err;    
}

hipError_t
hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind, hipStream_t stream)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaMemcpyAsyncD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCstreamArray *st;
    Vdev_t *vdev;
    RCServer_t *sp;

    WARN(3, "hipMemcpyAsync(0x%08llx, 0x%08llx, %d, %s, 0x%08llx)...",
         (unsigned long)dst, (unsigned long)src, count, dscudaMemcpyKindName(kind), st->s[0]);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyDeviceToHost:
        Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
        RCServer_t *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpyasyncd2hid_1((RCadr)src, count, (RCstream)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            }
            xdr_free((xdrproc_t)xdr_dscudaMemcpyAsyncD2HResult, (char *)d2hrp);
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;
        Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
        RCServer_t *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpyasynch2did_1((RCadr)dst, srcbuf, count, (RCstream)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)h2drp);
        }
        break;
      case hipMemcpyDeviceToDevice:
        Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
        RCServer_t *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpyasyncd2did_1((RCadr)dst, (RCadr)src, count, (RCstream)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
            xdr_free((xdrproc_t)xdr_dscudaResult, (char *)d2drp);
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;

#else
    // this DOES block.
    // this is only for use with a poor implementation of dscudaMallocHost().
    return hipMemcpy(dst, src, count, kind);
#endif
}

hipError_t
dscudaMemcpyToSymbolH2D(int moduleid, char *symbol, const void *src,
                        size_t count, size_t offset, int vdevid, int raidid)
{
    dscudaResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    RCbuf srcbuf;
    hipError_t err;

    srcbuf.RCbuf_len = count;
    srcbuf.RCbuf_val = (char *)src;
    rp = dscudamemcpytosymbolh2did_1(moduleid, symbol, srcbuf, count, offset, Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyToSymbolD2D(int moduleid, char *symbol, const void *src,
                        size_t count, size_t offset, int vdevid, int raidid)
{
    dscudaResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    hipError_t err;

    rp = dscudamemcpytosymbold2did_1(moduleid, symbol, (RCadr)src, count, offset, Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolD2H(int moduleid, void **dstbuf, char *symbol,
                          size_t count, size_t offset, int vdevid, int raidid)
{
    dscudaMemcpyFromSymbolD2HResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    hipError_t err;

    rp = dscudamemcpyfromsymbold2hid_1(moduleid, (char *)symbol, count, offset, Clnt[vdevid][sp->id]);
    *dstbuf = rp->buf.RCbuf_val;
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaMemcpyFromSymbolD2HResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolD2D(int moduleid, void *dstadr, char *symbol,
                          size_t count, size_t offset, int vdevid, int raidid)
{
    dscudaResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    hipError_t err;

    rp = dscudamemcpyfromsymbold2did_1(moduleid, (RCadr)dstadr, (char *)symbol, count, offset, Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyToSymbolAsyncH2D(int moduleid, char *symbol, const void *src,
                             size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    RCbuf srcbuf;
    hipError_t err;

    srcbuf.RCbuf_len = count;
    srcbuf.RCbuf_val = (char *)src;
    rp = dscudamemcpytosymbolasynch2did_1(moduleid, symbol, srcbuf, count, offset, stream, Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyToSymbolAsyncD2D(int moduleid, char *symbol, const void *src,
                             size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    hipError_t err;

    rp = dscudamemcpytosymbolasyncd2did_1(moduleid, symbol, (RCadr)src, count, offset, stream,
                                          Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolAsyncD2H(int moduleid, void **dstbuf, char *symbol,
                               size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaMemcpyFromSymbolAsyncD2HResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    hipError_t err;

    rp = dscudamemcpyfromsymbolasyncd2hid_1(moduleid, (char *)symbol, count, offset,
                                            stream, Clnt[vdevid][sp->id]);
    *dstbuf = rp->buf.RCbuf_val;
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaMemcpyFromSymbolAsyncD2HResult, (char *)rp);

    return (hipError_t)err;
}

hipError_t
dscudaMemcpyFromSymbolAsyncD2D(int moduleid, void *dstadr, char *symbol,
                               size_t count, size_t offset, RCstream stream, int vdevid, int raidid)
{
    dscudaResult *rp;
    RCServer_t *sp = (St.Vdev + vdevid)->server + raidid;
    hipError_t err;

    rp = dscudamemcpyfromsymbolasyncd2did_1(moduleid, (RCadr)dstadr, (char *)symbol, count, offset,
                                            stream, Clnt[vdevid][sp->id]);
    checkResult(rp, sp);
    err = (hipError_t)rp->err;
    xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);

    return (hipError_t)err;
}


/*
 * Stream Management
 */

hipError_t
hipStreamCreate(hipStream_t *pStream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaStreamCreateResult *rp;
    hipStream_t st[RC_NREDUNDANCYMAX];

    WARN(3, "hipStreamCreate(0x%08llx)...", (unsigned long)pStream);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamcreateid_1(Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        st[i] = (hipStream_t)rp->stream;
        xdr_free((xdrproc_t)xdr_dscudaStreamCreateResult, (char *)rp);
    }

    *pStream = st[0];
    RCstreamArrayRegister(st);
    WARN(3, "done. *pStream:0x%08llx\n", *pStream);

    return err;
#else
    *pStream = 0;
    return hipSuccess;
#endif
}

hipError_t
hipStreamDestroy(hipStream_t stream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    WARN(3, "hipStreamDestroy(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamdestroyid_1((RCadr)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCstreamArrayUnregister(st->s[0]);
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

hipError_t
hipStreamSynchronize(hipStream_t stream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    WARN(3, "hipStreamSynchronize(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamsynchronizeid_1((RCadr)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

hipError_t
hipStreamQuery(hipStream_t stream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    WARN(3, "hipStreamQuery(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamqueryid_1((RCadr)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

/*
 * Event Management
 */

hipError_t
hipEventCreate(hipEvent_t *event)
{
    hipError_t err = hipSuccess;
    dscudaEventCreateResult *rp;
    hipEvent_t ev[RC_NREDUNDANCYMAX];

    WARN(3, "hipEventCreate(%p)...", event);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventcreateid_1(Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ev[i] = (hipEvent_t)rp->event;
        xdr_free((xdrproc_t)xdr_dscudaEventCreateResult, (char *)rp);
    }
    *event = ev[0];
    RCeventArrayRegister(ev);
    WARN(3, "done. *event:%p\n", *event);

    return err;
}

hipError_t
hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaEventCreateResult *rp;
    hipEvent_t ev[RC_NREDUNDANCYMAX];

    WARN(3, "hipEventCreateWithFlags(%p, 0x%08x)...", event, flags);
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventcreatewithflagsid_1(flags, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ev[i] = (hipEvent_t)rp->event;
        xdr_free((xdrproc_t)xdr_dscudaEventCreateResult, (char *)rp);
    }
    *event = ev[0];
    RCeventArrayRegister(ev);
    WARN(3, "done. *event:%p\n", *event);

    return err;
}

hipError_t
hipEventDestroy(hipEvent_t event)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    WARN(3, "hipEventDestroy(%p)...", event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventdestroyid_1((RCadr)ev->e[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    RCeventArrayUnregister(ev->e[0]);
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipSuccess;
    dscudaEventElapsedTimeResult *rp;
    RCeventArray *es, *ee;

    WARN(3, "hipEventElapsedTime(%p, %p, %p)...", ms, start, end);
    es = RCeventArrayQuery(start);
    if (!es) {
        WARN(0, "invalid start event : %p\n", start);
        exit(1);
    }
    ee = RCeventArrayQuery(end);
    if (!ee) {
        WARN(0, "invalid end event : %p\n", end);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventelapsedtimeid_1((RCadr)es->e[i], (RCadr)ee->e[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaEventElapsedTimeResult, (char *)rp);
    }

    *ms = rp->ms;
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventRecord(hipEvent_t event, hipStream_t stream)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;
    RCeventArray *ev;

    WARN(3, "hipEventRecord(%p, %p)...", event, stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventrecordid_1((RCadr)ev->e[i], (RCadr)st->s[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    WARN(3, "hipEventSynchronize(%p)...", event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventsynchronizeid_1((RCadr)ev->e[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventQuery(hipEvent_t event)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    WARN(3, "hipEventQuery(%p)...", event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventqueryid_1((RCadr)ev->e[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;
    RCeventArray *ev;

    WARN(3, "hipStreamWaitEvent(%p, %p, 0x%08x)...", stream, event, flags);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : %p\n", stream);
        exit(1);
    }
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : %p\n", event);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamwaiteventid_1((RCadr)st->s[i], (RCadr)ev->e[i], flags, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscudaResult, (char *)rp);
    }

    WARN(3, "done.\n");
    return err;
}

/*
 * Texture Reference Management
 */

hipChannelFormatDesc
hipCreateChannelDesc(int x, int y, int z, int w, enum hipChannelFormatKind f)
{
    dscudaCreateChannelDescResult *rp;
    hipChannelFormatDesc desc;

    WARN(3, "hipCreateChannelDesc()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudacreatechanneldescid_1(x, y, z, w, f, Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (i == 0) {
            desc.x = rp->x;
            desc.y = rp->y;
            desc.z = rp->z;
            desc.w = rp->w;
            desc.f = (enum hipChannelFormatKind)rp->f;
        }
        xdr_free((xdrproc_t)xdr_dscudaCreateChannelDescResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return desc;
}

hipError_t
hipGetChannelDesc(struct hipChannelFormatDesc *desc, const struct hipArray *array)
{
    hipError_t err = hipSuccess;
    dscudaGetChannelDescResult *rp;
    RCcuarrayArray *ca;

    WARN(3, "hipGetChannelDesc()...");
    ca = RCcuarrayArrayQuery((hipArray *)array);
    if (!ca) {
        WARN(0, "invalid hipArray : %p\n", array);
        exit(1);
    }
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudagetchanneldescid_1((RCadr)ca->ap[i], Clnt[Vdevid[vdevidIndex()]][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        if (i == 0) {
            desc->x = rp->x;
            desc->y = rp->y;
            desc->z = rp->z;
            desc->w = rp->w;
            desc->f = (enum hipChannelFormatKind)rp->f;
        }
        xdr_free((xdrproc_t)xdr_dscudaGetChannelDescResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return err;
}


hipError_t
hipUnbindTexture(const struct textureReference * texref)
{
    hipError_t err = hipSuccess;

    WARN(4, "Current implementation of hipUnbindTexture() does nothing "
         "but returning hipSuccess.\n");

    err = hipSuccess;

    return err;
}

/*
 * CUFFT library
 */
hipfftResult CUFFTAPI
hipfftPlan3d(hipfftHandle *plan, int nx, int ny, int nz, hipfftType type)
{
    hipfftResult res = HIPFFT_SUCCESS;
    dscufftPlanResult *rp;

    WARN(3, "hipfftPlan3d()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscufftplan3did_1(nx, ny, nz, (unsigned int)type, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
        if (i == 0) {
            *plan = rp->plan;
        }
        xdr_free((xdrproc_t)xdr_dscufftPlanResult, (char *)rp);
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftDestroy(hipfftHandle plan)
{
    hipfftResult res = HIPFFT_SUCCESS;
    dscufftResult *rp;

    WARN(3, "hipfftDestroy()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscufftdestroyid_1((unsigned int)plan, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscufftResult, (char *)rp);
    }
    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecC2C(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata, int direction)
{
    hipfftResult res = HIPFFT_SUCCESS;
    dscufftResult *rp;

    WARN(3, "hipfftExecC2C()...");
    Vdev_t *vdev = St.Vdev + Vdevid[vdevidIndex()];
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscufftexecc2cid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, direction, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
        xdr_free((xdrproc_t)xdr_dscufftResult, (char *)rp);
    }

    WARN(3, "done.\n");

    return res;
}

#if 0

/*
 * Interface to CUFFT & CUBLAS written by Yoshikawa for old Remote CUDA.
 * some are already ported to DS-CUDA (see 'dscufftXXXid_1_svc' function defs above),
 * but some are not. Maybe someday, when I have time...
 */

hipfftResult CUFFTAPI
hipfftPlan1d(hipfftHandle *plan, int nx, hipfftType type, int batch)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftPlanResult *rp;

    WARN(3, "hipfftPlan1d()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftplan1did_1(nx, (unsigned int)type, batch, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }
    *plan = rp->plan;

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftPlan2d(hipfftHandle *plan, int nx, int ny, hipfftType type, int batch)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftPlanResult *rp;

    WARN(3, "hipfftPlan2d()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftplan2did_1(nx, ny, (unsigned int)type, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }
    *plan = rp->plan;

    WARN(3, "done.\n");

    return res;
}

/*
  hipfftResult CUFFTAPI
  hipfftPlanMany(hipfftHandle *plan, int nx, hipfftType type, int batch)
  {
  hipfftResult res = HIPFFT_SUCCESS;
  rcufftPlanResult *rp;

  WARN(3, "hipfftPlan1d()...");
  Server *sp = Serverlist;
  for (int i = 0; i < Nredundancy; i++, sp++) {
  rp = rcufftplan1did_1(nx, (unsigned int)type, Clnt[0][sp->id]);
  checkResult(rp, sp);
  if (rp->err != HIPFFT_SUCCESS) {
  res = (hipfftResult)rp->err;
  }
  }
  *plan = rp->plan;

  WARN(3, "done.\n");

  return res;
  }
*/

hipfftResult CUFFTAPI
hipfftExecR2C(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecR2C()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftexecr2cid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecC2R(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecC2R()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftexecc2rid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecZ2Z(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata, int direction)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecZ2Z()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftexecz2zid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, direction, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecD2Z(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecD2Z()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftexecd2zid_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

hipfftResult CUFFTAPI
hipfftExecZ2D(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "hipfftExecZ2D()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftexecz2did_1((unsigned int)plan, (RCadr)idata, (RCadr)odata, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}

/*
  hipfftResult CUFFTAPI
  hipfftSetStream(hipfftHandle plan, hipStream_t stream)
  {
  }
*/

hipfftResult CUFFTAPI
cufftSetCompatibilityMode(hipfftHandle plan, cufftCompatibility mode)
{
    hipfftResult res = HIPFFT_SUCCESS;
    rcufftResult *rp;

    WARN(3, "cufftSetCompatibilityMode()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcufftsetcompatibilitymodeid_1((unsigned int)plan, (unsigned int)mode, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->err != HIPFFT_SUCCESS) {
            res = (hipfftResult)rp->err;
        }
    }

    WARN(3, "done.\n");

    return res;
}


/*
 * CUBLAS Library functions
 */
hipblasStatus_t CUBLASAPI
hipblasCreate(hipblasHandle_t *handle)
{
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasCreateResult *rp;

    WARN(3, "hipblasCreate()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcublascreate_v2id_1(Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    *handle = (hipblasHandle_t)rp->handle;

    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasDestroy(hipblasHandle_t handle)
{
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasResult *rp;

    WARN(3, "hipblasDestroy()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcublasdestroy_v2id_1((RCadr)handle, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasSetVector(int n, int elemSize, const void *x, int incx, void *devicePtr, int incy)
{
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasResult *rp;

    RCbuf buf;
    buf.RCbuf_val = (char *)malloc(n * elemSize);
    buf.RCbuf_len = n;
    memcpy(buf.RCbuf_val, x, n);

    WARN(3, "hipblasSetVector()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcublassetvectorid_1(n, elemSize, buf, incx, (RCadr)devicePtr, incy, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasGetVector(int n, int elemSize, const void *x, int incx, void *y, int incy)
{
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasGetVectorResult *rp;

    WARN(3, "hipblasGetVector()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcublasgetvectorid_1(n, elemSize, (RCadr)x, incx, incy, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }

    memcpy(y, rp->y.RCbuf_val, n * elemSize);
    WARN(3, "done.\n");

    return res;
}

hipblasStatus_t CUBLASAPI
hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
               const float *alpha, const float *A, int lda,
               const float *B, int ldb, const float *beta, float *C, int ldc)
{
    hipblasStatus_t res = HIPBLAS_STATUS_SUCCESS;
    rcublasResult *rp;

    WARN(3, "hipblasSgemm()...");
    Server *sp = Serverlist;
    for (int i = 0; i < Nredundancy; i++, sp++) {
        rp = rcublassgemm_v2id_1((RCadr)handle, (unsigned int)transa, (unsigned int)transb, m, n, k,
                                 *alpha, (RCadr)A, lda, (RCadr)B, ldb, *beta, (RCadr)C, ldc, Clnt[0][sp->id]);
        checkResult(rp, sp);
        if (rp->stat != HIPBLAS_STATUS_SUCCESS) {
            res = (hipblasStatus_t)rp->stat;
        }
    }
    WARN(3, "done.\n");

    return res;
}
#endif // CUFFT

