#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <math.h>
#include <cutil.h>
#include <cutil_inline.h>
#include "dscuda.h"
#include "vecadd.H"

#define N (8)

int main(void) {
    int i, t, t_total=3;
    float a[N], b[N], c[N];
    float *d_a, *d_b, *d_c;

    printf("start vecadd\n"); fflush(stdout);
    //dscudaRecordHistOff();    
    FaultConf_t FAULT_CONF(2);
    FAULT_CONF.overwrite_en = 1;
    printf("The size of FAULT_CONF is %d Byte.\n", sizeof(FAULT_CONF));

    cutilSafeCall(hipMalloc((void**) &d_a, sizeof(float) * N));
    cutilSafeCall(hipMalloc((void**) &d_b, sizeof(float) * N));
    cutilSafeCall(hipMalloc((void**) &d_c, sizeof(float) * N));
    //dscudaRecordHistOn();
    for (t=0; t<t_total; t++) {
	printf("#\n");
	printf("# Try: %d/%d\n", t+1, t_total);
	printf("#\n"); fflush(stdout);
        for (i = 0; i < N; i++) {
            a[i] = rand()%64;
            b[i] = rand()%64;
        }

	//	dscudaClearHist();
	//dscudaRecordHistOn();

        hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
        vecAdd<<<N, 1>>>(d_a, d_b, d_c, FAULT_CONF);
        hipMemcpy(c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost); /* verify */

	//dscudaRecordHistOff();

        for (i=0; i<N; i++) {
            printf("% 6.2f + % 6.2f = % 7.2f",
                   a[i], b[i], c[i]);
            if (a[i] + b[i] != c[i]) printf("   NG");
            printf("\n");
        }
        printf("\n"); fflush(stdout);
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
