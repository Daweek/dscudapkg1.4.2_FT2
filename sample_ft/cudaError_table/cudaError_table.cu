
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDAERROR 6 

int main(int argc, char** argv) {
    hipError_t i;

    printf("hipSuccess = %d\n", hipSuccess);
    printf("hipErrorOutOfMemory = %d\n", hipErrorOutOfMemory);
    printf("hipErrorLaunchTimeOut = %d\n", hipErrorLaunchTimeOut);
	
    return 0;
}
