//                             -*- Mode: C++ -*-
// Filename         : dacudasvr.cu
// Description      : DS-CUDA server node.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-09-08 13:17:22
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//------------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <inttypes.h>
#include <time.h>
#include <unistd.h>
#include <rpc/rpc.h>
#include <rpc/pmap_clnt.h>
#include <cutil.h>
// remove definition of some macros which will be redefined in \"cutil_inline.h\".
#ifdef MIN
#undef MIN
#endif
#ifdef MAX
#undef MAX
#endif
#include <cutil_inline.h>
#include <hipfft/hipfft.h>
#include <pthread.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <poll.h>
#include <errno.h>

#include "dscudarpc.h"
#include "dscudasvr.h"
#include "dscudasvr_rpc.h"
#include "dscuda.h"
#include "sockutil.h"

//<-- oikawa
#define FAULT_INJECTION_LEN (32)
//--> oikawa

//********************************************************************
//***  Class Name: "ServerModule_t"
//***  Description:
//***      - CUDA Kernel function module management for Server.
//********************************************************************
ServerModule SvrModulelist[RC_NKMODULEMAX] = {0};

ServerState_t DscudaSvr;
    
int D2Csock = -1; // socket for sideband communication to the client. inherited from the daemon.
int TcpPort = RC_SERVER_IP_PORT;
static int Connected = 0;
static int UseIbv = 0; // use IB Verbs if set to 1. use RPC by default.
int Ndevice = 1;                 // # of devices in the system.
int Devid[RC_NDEVICEMAX] = {0,}; // real device ids of the ones in the system.
int dscuDevice;                   // virtual device id of the one used in the current context.
hipCtx_t dscuContext = NULL;
int Devid2Vdevid[RC_NDEVICEMAX]; // device id conversion table from real to virtual.

static void notifyIamReady(void);
static void showUsage(char *command);
static void showConf(void);
static void parseArgv(int argc, char **argv);
static hipError_t initDscuda(void);

static void initEnv(void);

static void notifyIamReady(void) {
    char msg[] = "ready";
    if (D2Csock >= 0) {
        SWARN(3, "send \"ready\" to the client.\n");
        sendMsgBySocket(D2Csock, msg);
    }
}

static int receiveProtocolPreference(void)
{
    char msg[256], rc[64];

    if (D2Csock >= 0) {
        SWARN(3, "wait for remotecall preference (\"rpc\" or \"ibv\") from the client.\n");
        recvMsgBySocket(D2Csock, msg, sizeof(msg));
        sscanf(msg, "remotecall:%s", rc);
        SWARN(2, "method of remote procedure call: %s\n", rc);
        if (!strncmp("ibv", rc, strlen("ibv"))) {
            return 1;
        } else {
            return 0;
        }
    } else {
        return UseIbv; // do not modify the preference.
    }
}

int main(int argc, char **argv)
{
    parseArgv(argc, argv);
    initEnv();
    initDscuda();
    showConf();

    UseIbv = receiveProtocolPreference();

    if (UseIbv) {
#if !defined(RPC_ONLY)
        setupIbv();
        notifyIamReady();
        ibvMainLoop(NULL);
#endif
    } else {
        setupRpc();
        notifyIamReady();
        svc_run(); // RPC main loop.
    }
    fprintf (stderr, "main loop returned.\n"); // never reached.
    exit (1);
}

static void showUsage(char *command) {
    fprintf(stderr,
            "usage: %s [-s server_id] [-d 'deviceid'] [-p port] [-S socket]\n"
            "       (-p & -S are used by the daemon only.)\n",
            command);
}

static void showConf(void) {
    int i;
    char str[1024], str0[1024];

    SWARN(2, "TCP port : %d (base + %d)\n", TcpPort, TcpPort - RC_SERVER_IP_PORT);
    SWARN(2, "ndevice : %d\n", Ndevice);
    sprintf(str, "real device%s      :", Ndevice > 1 ? "s" : " ");
    for (i = 0; i < Ndevice; i++) {
        sprintf(str0, " %d", Devid[i]);
        strcat(str, str0);
    }
    SWARN(2, "%s\n", str);
    sprintf(str, "virtual device%s   :", Ndevice > 1 ? "s" : " ");
    for (i = 0; i < Ndevice; i++) {
        sprintf(str0, " %d", Devid2Vdevid[Devid[i]]);
        strcat(str, str0);
    }
    SWARN(2, "%s\n", str);
}

extern char *optarg;
extern int optind;
static void
parseArgv(int argc, char **argv) {
    int c, ic;
    char *param = "d:hp:s:S:";
    char *num;
    char buf[256];
    int device_used[RC_NDEVICEMAX] = {0,};
    int tcpport_set = 0;
    int serverid = 0;

    while ((c = getopt(argc, argv, param)) != EOF) {
        switch (c) {
          case 'p':
            TcpPort = atoi(optarg);
            tcpport_set = 1;
            break;

          case 's':
            serverid = atoi(optarg);
            break;

          case 'S':
            D2Csock = atoi(optarg);
            break;

          case 'd':
            Ndevice = 0;
            strncpy(buf, optarg, sizeof(buf));
            num = strtok(buf, " ");
            while (num) {
                ic = atoi(num);
                if (ic < 0 || RC_NDEVICEMAX <= ic ) {
                    fprintf(stderr, "device id out of range: %d\n", ic);
                    exit(2);
                }
                if (!device_used[ic]) { // care for malformed optarg value such as "0 1 2 2 3".
                    device_used[ic] = 1;
                    Devid[Ndevice] = ic;
                    Ndevice++;
                }
                num = strtok(NULL, " ");
            }
            break;

          case 'h':
          default:
            showUsage(argv[0]);
            exit(1);
        }
    }
    if (!tcpport_set) {
        TcpPort = RC_SERVER_IP_PORT + serverid;
        SWARN(3, "TCP port number not given by '-p' option. Use default (%d).\n", TcpPort);
    }
}

// should be called only once in a run.
static hipError_t initDscuda(void) {
    unsigned int flags = 0; // should always be 0.
    hipError_t err;

    SWARN(4, "#(info.)<--- %s()...\n", __func__);

    for (int i = 0; i < Ndevice; i++) {
        Devid2Vdevid[Devid[i]] = i;
    }

    for (unsigned int u=0; u<RC_NKMODULEMAX; u++) { /* Set sequential number. */
	SvrModulelist[u].id = u;
    }

    err = hipInit(flags);
    if (err != hipSuccess) {
        SWARN(0, "hipInit(%d) failed.\n", flags);
        exit(1);
    }
    err = (hipError_t)hipSetValidDevices(Devid, Ndevice);
    if (err != hipSuccess) {
        SWARN(0, "hipSetValidDevices(%p, %d) failed.\n", Devid, Ndevice);
        exit(1);
    }
    dscuDevice = Devid[0];
    SWARN(3, "hipSetValidDevices(%p, %d). dscuDevice:%d\n",
         Devid, Ndevice, dscuDevice);
    SWARN(4, "#(info.)---> %s() done.\n", __func__);
    return (hipError_t)err;
}

hipError_t createDscuContext(void)
{
    //    unsigned int flags = 0; // should always be 0.
    hipDevice_t dev = 0;
    hipError_t err;

    err = hipDeviceGet(&dev, dscuDevice);
    if (err != hipSuccess) {
        SWARN(0, "hipDeviceGet() failed.\n");
        return (hipError_t)err;
    }

#if 0
    err = hipCtxCreate(&dscuContext, flags, dev);
    if (err != hipSuccess) {
        SWARN(0, "hipCtxCreate() failed.\n");
        return (hipError_t)err;
    }
#else // not used. set a dummy value not to be called repeatedly.
    dscuContext = (hipCtx_t)-1;
#endif

    return (hipError_t)err;
}

hipError_t destroyDscuContext(void)
{
#if 0

    hipError_t cuerr;
    bool all = true;

    SWARN(3, "destroyDscuContext(");
    releaseModules(all);

    cuerr = hipCtxDestroy(dscuContext);
    SWARN(4, "hipCtxDestroy(0x%08llx", dscuContext);
    if (cuerr != hipSuccess) {
        SWARN(0, "hipCtxDestroy() failed.\n");
        fatal_error(1);
        return (hipError_t)cuerr;
    }
    dscuContext = NULL;
    SWARN(4, ") done.\n");
    SWARN(3, ") done.\n");

#else

    dscuContext = NULL;

#endif
    return hipSuccess;
}

static void initEnv(void) {
    static int firstcall = 1;
    int tmp, tmp2[FAULT_INJECTION_LEN];
    char *env;

    if (!firstcall) return;

    firstcall = 0;

    // DSCUDA_WARNLEVEL
    env = getenv("DSCUDA_WARNLEVEL");
    if (env) {
        tmp = atoi(strtok(env, " "));
        if (0 <= tmp) {
            dscudaSetWarnLevel(tmp);
        }
        SWARN(1, "WarnLevel: %d\n", dscudaWarnLevel());
    }

    // DSCUDA_REMOTECALL
    env = getenv("DSCUDA_REMOTECALL");
#if defined(RPC_ONLY)
    UseIbv = 0;
    SWARN(2, "method of remote procedure call: RPC\n");
#else
    if (D2Csock >= 0) { // launched by daemon.
        SWARN(3, "A server launched by the daemon "
             "does not use the evironment variable 'DSCUDA_REMOTECALL'.\n");
    }
    else { // launched by hand.
        if (!env) {
            fprintf(stderr, "Set an environment variable 'DSCUDA_REMOTECALL' to 'ibv' or 'rpc'.\n");
            exit(1);
        }
        if (!strcmp(env, "ibv")) {
            UseIbv = 1;
            SWARN(2, "method of remote procedure call: InfiniBand Verbs\n");
        }
        else if (!strcmp(env, "rpc")) {
            UseIbv = 0;
            SWARN(2, "method of remote procedure call: RPC\n");
        }
        else {
            UseIbv = 0;
            SWARN(2, "method of remote procedure call '%s' is not available. use RPC.\n", env);
        }
    }
#endif
    // <-- add by Oikawa
    /* Fault Injection Configuration */
    env = getenv("DSCUDA_FAULT_INJECTION"); // integer type.
    if (env) {
	for (int i=0; i<FAULT_INJECTION_LEN; i++) {
	    tmp2[i]=0;
	}
	tmp2[0] = atoi(strtok(env, ""));
	if (tmp2[0] >= 0) {
	    DscudaSvr.setFaultInjection(tmp2[0]);
	}
    }
    SWARN(1, "Fault Injection Config: 0x%x\n", DscudaSvr.getFaultInjection());

    /* Timed out */
    env = getenv("DSCUDA_FORCE_TIMEOUT"); // integer type.
    if (env) {
	for (int i = 0; i < FAULT_INJECTION_LEN; i++) {
	    tmp2[i]=0;
	}
	tmp2[0] = atoi(strtok(env, ""));
	if (tmp2[0] >= 0) {
	    DscudaSvr.force_timeout = tmp2[0];
	}
    }
    SWARN(1, "Force Timeout Config: 0x%x\n", DscudaSvr.force_timeout);

    // --> add by Oikawa
}

/*
 * Unload Modules never been used for a long time.
 */
void releaseModules(bool releaseall = false)
{
    ServerModule *mp;
    int i;
    for (i=0, mp=SvrModulelist; i < RC_NKMODULEMAX; i++, mp++) {
        if (mp->isInvalid()) continue;
        if (releaseall || time(NULL) - mp->loaded_time > RC_SERVER_CACHE_LIFETIME) {
            hipModuleUnload((hipModule_t)mp->handle);
	    mp->invalidate();
            mp->handle = NULL;
            for (i = 0; i < RC_NKFUNCMAX; i++) {
                mp->kfunc[i] = NULL;
            }
            SWARN(3, "%s() unloaded a module. name:%s pid:%d ip:%s age:%d\n",
		 __func__, mp->name, mp->pid, dscudaGetIpaddrString(mp->ipaddr),
                 time(NULL) - mp->loaded_time);
        }
    }
}

static void
printSvrModuleList(ServerModule *module_list)
{
    for (int i=0; i<RC_NKMODULEMAX; i++) {
	if (module_list[i].isValid()) {
	    SWARN(10, "#--- SvrModulelist[%d]\n", i);
	    SWARN(10, "#---    + ID=%u, ip=%u\n", module_list[i].id, module_list[i].ipaddr);
	    SWARN(10, "#---    + name=%s\n",   module_list[i].name);
	}
    }
}

hipError_t
getFunctionByName(hipFunction_t *kfuncp, const char *kname, int moduleid) {
    SWARN(10, "   + %s(kname=%s) {\n", __func__, kname);
    hipError_t cuerr;
    ServerModule *mp = SvrModulelist + moduleid;

    cuerr = hipModuleGetFunction(kfuncp, mp->handle, kname);
    if (cuerr == hipSuccess) {
        SWARN(3, "(^_^) hipModuleGetFunction() : function '%s' found.\n", kname);
	SWARN(3, "(^_^) moduleid=%d, valid=%d, id=%d, name=%s\n",
	     moduleid, mp->valid, mp->id, mp->name);
	printSvrModuleList(SvrModulelist);
    } else {
        SWARN(0, "(;_;) hipModuleGetFunction() : function:'%s'. %s\n",
             kname, hipGetErrorString((hipError_t)cuerr));
	SWARN(0, "(;_;) moduleid=%d, valid=%d, id=%d, name=%s\n",
	     moduleid, mp->valid, mp->id, mp->name);
	switch (cuerr) {
	  case hipErrorDeinitialized:
	    SWARN(0, "hipErrorDeinitialized.\n");   break;
	  case hipErrorNotInitialized:
	    SWARN(0, "hipErrorNotInitialized.\n"); break;
	  case hipErrorInvalidContext:
	    SWARN(0, "hipErrorInvalidContext.\n"); break;
	  case hipErrorInvalidValue:
	    SWARN(0, "hipErrorInvalidValue.\n");   break;
	  case hipErrorNotFound:
	    SWARN(0, "hipErrorNotFound.\n");       break;
	  case hipErrorInvalidHandle:
	    SWARN(0, "hipErrorInvalidHandle.\n");  break;
	  default:
	    SWARN(0, "(unknown error code: %d)\n", cuerr);
	}
        fatal_error(1);
    }
    SWARN(10, "   + }\n");
    return cuerr;
}

void getGlobalSymbol(int moduleid, char *symbolname, hipDeviceptr_t *dptr, size_t *size)
{
    hipError_t cuerr;
    ServerModule *mp;

    if (moduleid < 0 || RC_NKMODULEMAX <= moduleid) {
        SWARN(0, "getGlobalSymbol() : invalid module id:%d.\n", moduleid);
        fatal_error(1);
    }
    mp = SvrModulelist + moduleid;
    cuerr = hipModuleGetGlobal(dptr, size, mp->handle, symbolname);
    if (cuerr == hipSuccess) {
	SWARN(3, "hipModuleGetGlobal(%p, %p, %p, %s) done."
	 " modulename:%s  symbolname:%s  *dptr:0x%08lx\n",
	 dptr, size, mp->handle, symbolname,
	 mp->name, symbolname, *dptr);
    }
    else {
        SWARN(0, "hipModuleGetGlobal(%p, %p, %p, 0x%08llx) failed."
             " modulename:%s  symbolname:%s  %s\n",
             dptr, size, mp->handle, symbolname,
             mp->name, symbolname, hipGetErrorString((hipError_t)cuerr));
        fatal_error(1);
    }
}

int dscudaLoadModule(RCipaddr ipaddr, RCpid pid, char *mname, char *image)
{
    hipError_t cuerr;
    ServerModule   *mp;
    int      i;
    SWARN(10, "<---Entering %s()\n", __func__);

#if RC_CACHE_MODULE
    // look for mname in the module list, which may found if the client
    // resent multiple requests for the same mname:pid:ipaddr.
    int found = 0;
    mp = SvrModulelist;
    for (i=0; i < RC_NKMODULEMAX; i++) {
        if (mp->isInvalid()) continue;
        if ((unsigned int)ipaddr == mp->ipaddr && pid == mp->pid &&
            !strcmp(mname, mp->name)) {
            found = 1;
            break;
        }
	SWARN(4, "ip:%x  %x    pid:%d  %d    name:%s  %s\n",
	     (unsigned int)ipaddr, mp->ipaddr, pid, mp->pid, mname, mp->name);
	mp++;
    }

    if (found) { // module found. i.e, it's already loaded.
        SWARN(3, "\n\n------------------------------------------------------------------\n"
             "dscudaloadmoduleid_1_svc() got multiple requests for\n"
             "  the same module name : %s,\n"
             "  the same process id  : %d, and\n"
             "  the same IP address  : %s,\n"
             "which means a client resent the same module twice or more.\n"
             "If you see this message too often, you may want to increase\n"
             "$dscuda/include/dscudadefs.h:RC_CLIENT_CACHE_LIFETIME\n"
             "for better performance.\n"
             "------------------------------------------------------------------\n\n",
             mname, pid, dscudaGetIpaddrString(ipaddr));
        SWARN(3, "hipModuleLoadData() : a module found in the cache. id:%d  name:%s  age:%d\n",
             mp->id, mname, time(NULL) - mp->loaded_time);
    }
    else  // module not found in the cache. load it from image.
#endif // RC_CACHE_MODULE

    {
        for (i=0, mp=SvrModulelist; i < RC_NKMODULEMAX; i++, mp++) { /* look for .valid==0 */
            if (mp->isInvalid()) break;
            if (i >= RC_NKMODULEMAX) { SWARN(0, "(+_+) module cache is full.\n"); fatal_error(1); }
        }
	/* Register new SvrModulelist[i] */
        /* mp->id = i; */
        cuerr = hipModuleLoadData(&mp->handle, image); /* load .ptx string */
        if (cuerr != hipSuccess) {
            SWARN(0, "hipModuleLoadData() failed. %s\n", hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        mp->validate(); /* mp->valid  = 1;*/
        mp->ipaddr = ipaddr;
        mp->pid    = pid;
        strncpy(mp->name, mname, sizeof(SvrModulelist[0].name));
        for (i=0; i<RC_NKFUNCMAX; i++) { mp->kfunc[i] = NULL; }
        SWARN(3, "hipModuleLoadData() : a module loaded. id:%d  name:%s\n", mp->id, mname);
    }
    mp->loaded_time = time(NULL); // (re)set the lifetime of the cache.
    releaseModules();

    SWARN(10, "--->Exiting  %s()\n", __func__);
    return mp->id;
}

void *dscudaLaunchKernel(int moduleid, int kid, const char *kname /*kernel func name*/,
                   RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream, RCargs args)
{
    static int called_count = 0;
    static int dummyres     = 123;
    SWARN(10, "%s(int moduleid=%d, int kid=%d, char *kname=%s), %d called.\n",
	 __func__, moduleid, kid, kname, called_count);

    if ( DscudaSvr.force_timeout > 0 ) {
	if ( called_count >= 3 ) {
	    SWARN(2, "sleeping 60 sec...\n");
	    sleep(60);
	}
    }
    
    int paramsize;
    hipError_t cuerr;

#if !RC_SUPPORT_CONCURRENT_EXEC
    stream = 0;
#endif

    if (!dscuContext) createDscuContext();

    // load a kernel function into Module[moduleid].kfunc[kid]
    // form Module[moduleid].handle.
    if (moduleid < 0 || RC_NKMODULEMAX <= moduleid) {
        SWARN(0, "dscudalaunchkernelid_1_svc() : invalid module id:%d.\n", moduleid);
        fatal_error(1);
    }

#if 1 // look inside a module for a function by name.
    hipFunction_t kfunc;
    getFunctionByName(&kfunc, kname, moduleid);
#else // look for a function by its ID.
    // this is faster, but not used since it would cause a problem
    // when called from a kernel function that uses C++ template.
    // in that case kid might not be unique for each instance of the template.
    ServerModule *mp = SvrModulelist + moduleid;
    hipFunction_t kfunc = mp->kfunc[kid];
    if (!kfunc) {
        getFunctionByName(&kfunc, kname, moduleid);
        mp->kfunc[kid] = kfunc;
    }
#endif

    // a kernel function found.
    // now make it run.
    if (UseIbv) {
#if !defined(RPC_ONLY)
	SWARN(10, "ibvUnpackKernelParam()\n");
        paramsize = ibvUnpackKernelParam(&kfunc, args.RCargs_len, (IbvArg *)args.RCargs_val);
#endif
    } else {
	SWARN(10, "rpcUnpackKernelParam()\n");
        paramsize = rpcUnpackKernelParam(&kfunc, &args);
    }
    cuerr = cuParamSetSize(kfunc, paramsize);
    if (cuerr != hipSuccess) {
        SWARN(0, "cuParamSetSize() failed. size:%d %s\n",
             paramsize, hipGetErrorString((hipError_t)cuerr));
        fatal_error(1);
    }
    SWARN(5, "cuParamSetSize() done.\n");

    cuerr = cuFuncSetBlockShape(kfunc, bdim.x, bdim.y, bdim.z);
    if (cuerr != hipSuccess) {
        SWARN(0, "cuFuncSetBlockShape() failed. %s\n", hipGetErrorString((hipError_t)cuerr));
        fatal_error(1);
    }
    SWARN(5, "cuFuncSetBlockShape() done.\n");

    if (smemsize != 0) {
        cuerr = cuFuncSetSharedSize(kfunc, smemsize);
        if (cuerr != hipSuccess) {
            SWARN(0, "cuFuncSetSharedSize() failed. %s\n", hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        SWARN(5, "cuFuncSetSharedSize() done.\n");
    }

    if ((hipStream_t)stream == NULL) {
        cuerr = cuLaunchGrid(kfunc, gdim.x, gdim.y);  /* Launch Kernel Function */
        if (cuerr != hipSuccess) {
            SWARN(0, "cuLaunchGrid() failed. kname:%s %s\n",
                 kname, hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        SWARN(3, "cuLaunchGrid() done. kname:%s\n", kname);
    } else {
        cuerr = cuLaunchGridAsync(kfunc, gdim.x, gdim.y, (hipStream_t)stream);
        if (cuerr != hipSuccess) {
            SWARN(0, "cuLaunchGridAsync() failed. kname:%s  %s\n",
                 kname, hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        SWARN(3, "cuLaunchGridAsync() done.  kname:%s  stream:0x%08llx\n", kname, stream);
    }
    SWARN(10, "+--- Done. %s() %d called.\n", __func__, called_count );
    called_count++;
    return &dummyres; // seems necessary to return something even if it's not used by the client.
}

hipError_t
setTextureParams(hipTexRef texref, RCtexture texbuf, char *texname, HIP_ARRAY_DESCRIPTOR *descp)
{
    hipError_t err;
    int ncomponent, i;
    unsigned int texref_flags = 0;
    unsigned int fmt_high, fmt_low, fmt_index;

    hipArray_Format fmt[] = {
        HIP_AD_FORMAT_UNSIGNED_INT8,    // 00_00
        HIP_AD_FORMAT_UNSIGNED_INT16,   // 00_01
        HIP_AD_FORMAT_UNSIGNED_INT32,   // 00_10
        (hipArray_Format)0,
        HIP_AD_FORMAT_SIGNED_INT8,      // 01_00
        HIP_AD_FORMAT_SIGNED_INT16,     // 01_01
        HIP_AD_FORMAT_SIGNED_INT32,     // 01_10
        (hipArray_Format)0,
        (hipArray_Format)0,
        HIP_AD_FORMAT_HALF,             // 10_01
        (hipArray_Format)0,
        (hipArray_Format)0,
        (hipArray_Format)0,
        (hipArray_Format)0,
        HIP_AD_FORMAT_FLOAT,            // 11_10
        (hipArray_Format)0,
    };

    // set addressmode (wrap/clamp/mirror/border)
    //
    for (i = 0; i < 3; i++) {
        err = (hipError_t)hipTexRefSetAddressMode(texref, i, (HIPaddress_mode_enum)texbuf.addressMode[i]);
        if (err != hipSuccess) {
            check_cuda_error(err);
            return err;
        }
    }

    // set filtermode (point/linear)
    //
    err = (hipError_t)hipTexRefSetFilterMode(texref, (HIPfilter_mode_enum)texbuf.filterMode);
    if (err != hipSuccess) {
        check_cuda_error(err);
        return err;
    }

    // set flags (integer/normalized)
    //
    if (texbuf.normalized) {
        texref_flags |= HIP_TRSF_NORMALIZED_COORDINATES;
    }
    else {
        texref_flags |= HIP_TRSF_READ_AS_INTEGER;
    }
    err = (hipError_t)hipTexRefSetFlags(texref, texref_flags);
    if (err != hipSuccess) {
        check_cuda_error(err);
        return err;
    }

    // set format (unsigned/signed/float, 32/16/8-bit)
    //
    switch (texbuf.x) {
      case 8:
        fmt_low = 0;
        break;
      case 16:
        fmt_low = 1;
        break;
      case 32:
        fmt_low = 2;
        break;
      default:
        SWARN(0, "hipModuleGetTexRef() invalid channel format. texture name:%s descriptor.x:%d\n",
             texname, texbuf.x);
        err = hipErrorInvalidValue;
        return err;
    }
    switch (texbuf.f) {
      case hipChannelFormatKindUnsigned:
        fmt_high = 0;
        break;

      case hipChannelFormatKindSigned:
        fmt_high = 1;
        break;

      case hipChannelFormatKindFloat:
        fmt_high = 3;
        break;

      case hipChannelFormatKindNone:
        SWARN(0, "hipModuleGetTexRef() invalid channel format. texture name:%s descriptor.f:%s\n",
             texname, "hipChannelFormatKindNone");
        err = hipErrorInvalidValue;
        return err;

      default:
        SWARN(0, "hipModuleGetTexRef() invalid channel format. texture name:%s descriptor.f:%s\n",
             texname, texbuf.f);
        err = hipErrorInvalidValue;
        return err;
    }
    fmt_index = fmt_high << 2 | fmt_low;
    ncomponent = 1;
    if (texbuf.y) ncomponent = 2;
    if (texbuf.z) ncomponent = 3;
    if (texbuf.w) ncomponent = 4;
    if (descp) {
        descp->Format = fmt[fmt_index];
        descp->NumChannels = ncomponent;
    }
    SWARN(4, "hipTexRefSetFormat(%p, %d, %d)\n", texref, fmt[fmt_index], ncomponent);
    err = (hipError_t)hipTexRefSetFormat(texref, fmt[fmt_index], ncomponent);
    if (err != hipSuccess) {
        check_cuda_error(err);
        return err;
    }

    return hipSuccess;
}
