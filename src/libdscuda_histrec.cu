//                             -*- Mode: C++ -*-
// Filename         : libdscuda_histrec.cu
// Description      : DS-CUDA verb function.
// Author           : A.Kawai, K.Yoshikawa, T.Narumi
// Created On       : 2011-01-01 00:00:00
// Last Modified By : M.Oikawa
// Last Modified On : 2014-09-17 11:50:29
// Update Count     : 0.1
// Status           : Unknown, Use with caution!
//----------------------------------------------------------------------
#include <string.h>
#include <sys/types.h>
#include <unistd.h>
#include "dscuda.h"
#include "dscudarpc.h"
#include "libdscuda.h"

#define DEBUG

static int
checkSum(void *targ, int size)
{
    int sum=0, *ptr = (int *)targ;
    
    for (int s=0; s<size; s+=sizeof(int)) {
	//printf("ptr[%d]= %d\n", s, *ptr);
	sum += *ptr;
	ptr++;
    }
    return sum;
}

#define DSCUDAVERB_SET_STUBS(mthd) \
  storeArgsStub[DSCVMethod ## mthd] = store ## mthd; \
  releaseArgsStub[DSCVMethod ## mthd] = release ## mthd; \
  recallStub[DSCVMethod ## mthd] = recall ## mthd;

#define DSCUDAVERB_STORE_ARGS(mthd) \
  Cuda ## mthd ## Args *argsrc;		\
  argsrc = (Cuda ## mthd ## Args *)argp; \
  Cuda ## mthd ## Args *argdst; \
  argdst = (Cuda ## mthd ## Args *)malloc(sizeof(Cuda ## mthd ## Args)); \
  *argdst = *(Cuda ## mthd ## Args *)argp;

//mapping RPCfunctionID to DSCUDAVerbMethodID
static DSCVMethod funcID2DSCVMethod(int funcID) {
    switch (funcID) {
      case dscudaSetDeviceId:
	return DSCVMethodSetDevice;
      case dscudaMallocId:
	return DSCVMethodMalloc;
      case dscudaMemcpyH2DId:
	return DSCVMethodMemcpyH2D;
      case dscudaMemcpyD2DId:
	return DSCVMethodMemcpyD2D;
      case dscudaMemcpyD2HId:
	return DSCVMethodMemcpyD2H;
      case dscudaMemcpyToSymbolH2DId:
	return DSCVMethodMemcpyToSymbolH2D;
      case dscudaMemcpyToSymbolD2DId:
	return DSCVMethodMemcpyToSymbolD2D;
      case dscudaFreeId:
	return DSCVMethodFree;
	/*    
	      case dscudaLoadModuleId:
	      return DSCVMethodLoadModule;
	*/
      case dscudaLaunchKernelId:
	return DSCVMethodRpcLaunchKernel;
      default:
	return DSCVMethodNone;
    }
}

//stubs for store args
static void*
storeSetDevice(void *argp)
{
    DSCUDAVERB_STORE_ARGS(SetDevice); 
    return argdst;
}

static void *
storeMalloc(void *argp)
{
    //nothing to do
    return NULL;
}

static void *
storeMemcpyH2D(void *argp)
{
    DSCUDAVERB_STORE_ARGS(Memcpy);
    argdst->src = malloc(argsrc->count + 1);
    memcpy(argdst->src, (const void *)argsrc->src, argsrc->count);
    return argdst;
}

static void*
storeMemcpyD2D(void *argp)
{
    DSCUDAVERB_STORE_ARGS(Memcpy);
    return argdst;
}

static void*
storeMemcpyD2H(void *argp)
{
    DSCUDAVERB_STORE_ARGS(Memcpy);
    return argdst;
}

static void*
storeMemcpyToSymbolH2D(void *argp)
{
    WARN(3, "add hist cudaMemcpyToSymbolH2D\n");
    DSCUDAVERB_STORE_ARGS(MemcpyToSymbol);
    
    int nredundancy = dscudaNredundancy();
    argdst->moduleid = (int *)malloc(sizeof(int) * nredundancy);
    memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);
  
    argdst->symbol = (char *)malloc(sizeof(char) * (strlen(argsrc->symbol) + 1));
    argdst->src = malloc(argsrc->count);
    
    strcpy(argdst->symbol, argsrc->symbol);
    memcpy(argdst->src, argsrc->src, argsrc->count);

    return argdst;
}

static void*
storeMemcpyToSymbolD2D(void *argp)
{
    WARN(3, "add hist cudaMemcpyToSymbolD2D\n");
    DSCUDAVERB_STORE_ARGS(MemcpyToSymbol);

    int nredundancy = dscudaNredundancy();
    argdst->moduleid = (int *)dscuda::xmalloc(sizeof(int) * nredundancy);
    
    memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);

    argdst->symbol = (char *)dscuda::xmalloc(sizeof(char) * (strlen(argsrc->symbol) + 1));

    strcpy(argdst->symbol, argsrc->symbol);
    
    return argdst;
}

static void*
storeFree(void *argp)
{
    //nothing to do
    return NULL;
}

static void*
storeLoadModule(void *argp)
{
    DSCUDAVERB_STORE_ARGS(LoadModule);
    argdst->name = (char *)malloc(sizeof(char) * (strlen(argsrc->name) + 1));
    argdst->strdata = (char *)malloc(sizeof(char) * (strlen(argsrc->strdata) + 1));
    strcpy(argdst->name, argsrc->name);
    strcpy(argdst->strdata, argsrc->strdata);
    return argdst;
}

static void*
storeRpcLaunchKernel(void *argp)
{
    WARN(3, "add hist RpcLaunchKernel\n");
    DSCUDAVERB_STORE_ARGS(RpcLaunchKernel);

    int nredundancy = dscudaNredundancy();
    //argdst->moduleid = (int *)malloc(sizeof(int) * nredundancy);
    //memcpy(argdst->moduleid, argsrc->moduleid, sizeof(int) * nredundancy);
    argdst->moduleid = argsrc->moduleid;
    
    argdst->kname = (char *)malloc(sizeof(char) * strlen(argsrc->kname) + 1);
    strcpy(argdst->kname, argsrc->kname);
    
    int narg = argsrc->args.RCargs_len;
    RCarg *rpcargbuf = (RCarg *)malloc(sizeof(RCarg) * narg);
    memcpy(rpcargbuf, argsrc->args.RCargs_val, sizeof(RCarg) * narg);
    argdst->args.RCargs_val = rpcargbuf;

    return argdst;
}

//stubs for release args
static void
releaseSetDevice(void *argp)
{
    CudaSetDeviceArgs *argsrc;
    argsrc = (CudaSetDeviceArgs *)argp;
    free(argsrc);
}

static void
releaseMalloc(void *argp)
{
    //nothing to do
}

static void
releaseMemcpyH2D(void *argp)
{
    CudaMemcpyArgs *argsrc;
    argsrc = (CudaMemcpyArgs *)argp;
    free(argsrc->src);
    free(argsrc);
}

static void
releaseMemcpyD2D(void *argp)
{
    CudaMemcpyArgs *argsrc;
    argsrc = (CudaMemcpyArgs *)argp;
    free(argsrc);
}

static void
releaseMemcpyD2H(void *argp)
{
    CudaMemcpyArgs *argsrc;
    argsrc = (CudaMemcpyArgs *)argp;
    free(argsrc);
}

static void
releaseMemcpyToSymbolH2D(void *argp)
{
    CudaMemcpyToSymbolArgs *argsrc;
    argsrc = (CudaMemcpyToSymbolArgs *)argp;
    
    free(argsrc->moduleid);
    free(argsrc->symbol);
    free(argsrc->src);
    free(argsrc);
}

static void
releaseMemcpyToSymbolD2D(void *argp)
{
    CudaMemcpyToSymbolArgs *argsrc;
    argsrc = (CudaMemcpyToSymbolArgs *)argp;

    free(argsrc->moduleid);
    free(argsrc->symbol);
    free(argsrc);
}

static void
releaseFree(void *argp)
{
    //nothing to do
}

static void
releaseLoadModule(void *argp)
{
    CudaLoadModuleArgs *argsrc;
    argsrc = (CudaLoadModuleArgs *)argp;
    
    free(argsrc->name);
    free(argsrc->strdata);
    free(argsrc);
}

static void
releaseRpcLaunchKernel(void *argp)
{
    CudaRpcLaunchKernelArgs *argsrc;
    argsrc = (CudaRpcLaunchKernelArgs *)argp;
    
    //free(argsrc->moduleid);
    free(argsrc->kname);
    free(argsrc->args.RCargs_val);
    free(argsrc);
}

//stubs for recall
static void
recallSetDevice(void *argp)
{
    CudaSetDeviceArgs *argsrc;
    argsrc = (CudaSetDeviceArgs *)argp;

    WARN(3, "Recall hipSetDevice()...\n");
    hipSetDevice(argsrc->device);
}

static void
recallMalloc(void *argp)
{
    //nothing to do
}

static void
recallMemcpyH2D(void *argp)
{
    // note: dont insert pthread_mutex_lock or unlock.
    CudaMemcpyArgs *argsrc;
    int         vdevid = Vdevid[ vdevidIndex() ];
    VirDev     *vdev   = St.Vdev + vdevid;
    int         rec_en_stack;
    
    argsrc = (CudaMemcpyArgs *)argp;
    WARN(3, "Recall cudaMemcpyH2D() \n");

    rec_en_stack = vdev->setRecord(0);
    vdev->cudaMemcpyH2D(argsrc->dst, argsrc->src, argsrc->count);
    vdev->setRecord(rec_en_stack);
    WARN(3, "\n");
}

static void
recallMemcpyD2D(void *argp)
{
    // note: dont insert pthread_mutex_lock or unlock.
    CudaMemcpyArgs *argsrc;
    argsrc = (CudaMemcpyArgs *)argp;
    
    WARN(3, "Recall cudaMemcpyD2D()...\n");
    hipMemcpy(argsrc->dst, argsrc->src, argsrc->count, hipMemcpyDeviceToDevice);
}

static void
recallMemcpyD2H(void *argp)
{
    // note: dont insert pthread_mutex_lock or unlock.
    CudaMemcpyArgs *argsrc;
    int         vdevid = Vdevid[ vdevidIndex() ];
    VirDev     *vdev   = St.Vdev + vdevid;
    int         rec_en_stack;

    argsrc = (CudaMemcpyArgs *)argp;
    WARN(3, "Recall cudaMemcpyD2H()...\n");

    rec_en_stack = vdev->setRecord(0);
    vdev->cudaMemcpyD2H(argsrc->dst, argsrc->src, argsrc->count);
    vdev->setRecord(rec_en_stack);
    WARN(3, "\n");
}

static void
recallMemcpyToSymbolH2D(void *argp)
{
    CudaMemcpyToSymbolArgs *argsrc;
    argsrc = (CudaMemcpyToSymbolArgs *)argp;
    WARN(3, "recall cudaMemcpyToSymbolH2D\n");
    dscudaMemcpyToSymbolWrapper(argsrc->moduleid, argsrc->symbol, argsrc->src, argsrc->count, argsrc->offset, hipMemcpyHostToDevice);
}

static void
recallMemcpyToSymbolD2D(void *argp)
{
    CudaMemcpyToSymbolArgs *argsrc;
    argsrc = (CudaMemcpyToSymbolArgs *)argp;
    WARN(3, "recall cudaMemcpyToSymbolD2D\n");
    dscudaMemcpyToSymbolWrapper(argsrc->moduleid, argsrc->symbol, argsrc->src, argsrc->count, argsrc->offset, hipMemcpyDeviceToDevice);
}

static void
recallFree(void *argp)
{
    //nothing to do
}

static void
recallLoadModule(void *argp)
{
    CudaLoadModuleArgs *argsrc;
    argsrc = (CudaLoadModuleArgs *)argp;
}

static void
recallRpcLaunchKernel(void *argp)
{
    // note: dont insert pthread_mutex_lock or unlock.
    CudaRpcLaunchKernelArgs *argsrc;
    argsrc = (CudaRpcLaunchKernelArgs *)argp;
    WARN(3, "Recall RpcLaunchKernel((int)moduleid=%d, (int)kid=%d, (char*)kname=%s, ...)...\n",
	 argsrc->moduleid, argsrc->kid, argsrc->kname);
#if 0
    rpcDscudaLaunchKernelWrapper(argsrc->moduleid, argsrc->kid, argsrc->kname, argsrc->gdim, argsrc->bdim, argsrc->smemsize, argsrc->stream, argsrc->args);
#else
    int         rec_en_stack;
    VirDev *vdev = St.Vdev + Vdevid[vdevidIndex()];
    rec_en_stack = vdev->setRecord(0);
    vdev->launchKernel(argsrc->moduleid, argsrc->kid, argsrc->kname, argsrc->gdim, argsrc->bdim, argsrc->smemsize, argsrc->stream, argsrc->args);
    vdev->setRecord(rec_en_stack);
#endif
}

/*
 *  CONSTRUCTOR
 */
HistList::HistList(void)
{
    add_count = 0;
    length    = 0;
    byte_size = 0;
    max_len   = 32;
    
    histrec = (HistCell*)dscuda::xmalloc( sizeof(HistCell) * max_len );

    //<-- import from dscudaVerbInit()
    memset(storeArgsStub,   0, sizeof(DSCVMethod) * DSCVMethodEnd);
    memset(releaseArgsStub, 0, sizeof(DSCVMethod) * DSCVMethodEnd);
    memset(recallStub,      0, sizeof(DSCVMethod) * DSCVMethodEnd);

    DSCUDAVERB_SET_STUBS(SetDevice);
    DSCUDAVERB_SET_STUBS(Malloc);
    DSCUDAVERB_SET_STUBS(MemcpyH2D);
    DSCUDAVERB_SET_STUBS(MemcpyD2D);
    DSCUDAVERB_SET_STUBS(MemcpyD2H);
    DSCUDAVERB_SET_STUBS(MemcpyToSymbolH2D);
    DSCUDAVERB_SET_STUBS(MemcpyToSymbolD2D);
    DSCUDAVERB_SET_STUBS(Free);
    //DSCUDAVERB_SET_STUBS(LoadModule);
    DSCUDAVERB_SET_STUBS(RpcLaunchKernel);
    //DSCUDAVERB_SET_STUBS(IbvLaunchKernel); // in kaust debug, 17Aug2014

    for (int i=1; i<DSCVMethodEnd; i++) {
	if (!storeArgsStub[i]) {
	    fprintf(stderr, "HistList(constructor): storeArgsStub[%d] is not initialized.\nexit.\n\n", i);
	    exit(1);
	}
	if (!releaseArgsStub[i]) {
	    fprintf(stderr, "HistList(constructor): releaseArgsStub[%d] is not initialized.\nexit.\n\n", i);
	    exit(1);
	}
	if (!recallStub[i]) {
	    fprintf(stderr, "HistList(constructor): recallStub[%d] is not initialized.\nexit.\n\n", i);
	    exit(1);
	}
    }
    //HISTREC.on();
    //--> import from dscudaVerbInit()
    //WARN( 5, "The constructor %s() called.\n", __func__ );
} // HistList::HistList()

void
HistList::extendLen(void)
{
    max_len += EXTEND_LEN;
    histrec = (HistCell*)realloc( histrec, sizeof(HistCell) * max_len );
    if (histrec == NULL) {
	WARN( 0, "%s():realloc() failed.\n", __func__ );
	exit(EXIT_FAILURE);
    }
    return;
}

/*
 * Add one item to called histry of CUDA API. 
 */
void
HistList::add(int funcID, void *argp)
{
    int DSCVMethodId;

    if (length == max_len) { /* Extend the existing memory region. */
	extendLen();
    }

    DSCVMethodId = funcID2DSCVMethod(funcID);
    histrec[length].seq_num = add_count;
    histrec[length].args    = (storeArgsStub[funcID2DSCVMethod(funcID)])(argp);
    histrec[length].funcID  = funcID;
    
    length++; /* Increment the count of cuda call */
    byte_size += sizeof(funcID);
    byte_size += sizeof(int);// dev_id
    add_count++; // count up.
    
    switch (funcID) {
    case dscudaSetDeviceId:
	byte_size += sizeof( CudaSetDeviceArgs );
	break;
    case dscudaMallocId:
	byte_size += sizeof( CudaMallocArgs );
	break;
    case dscudaMemcpyH2DId: //thru
	byte_size += sizeof( CudaMemcpyArgs );
	break;
    case dscudaMemcpyD2HId: //thru
	byte_size += sizeof( CudaMemcpyArgs );
	break;
    case dscudaMemcpyD2DId:
	byte_size += sizeof( CudaMemcpyArgs );
	WARN(3, "add hist cudaMemcpyD2D\n");
	break;
    case dscudaMemcpyToSymbolH2DId: //thru
    case dscudaMemcpyToSymbolD2DId:
	byte_size += sizeof( CudaMemcpyToSymbolArgs );
	break;
    case dscudaFreeId:
	byte_size += sizeof( CudaFreeArgs );
	break;
	/*    
	      case dscudaLoadModuleId:
	      return DSCVMethodLoadModule;
	*/
    case dscudaLaunchKernelId:
	byte_size += sizeof( CudaRpcLaunchKernelArgs ) + 32;
	// 32 is pseudo length of *kname.
	break;
    default:
	WARN(0, "%s():unknown kind of cuda api.\n", __func__);
	exit(1);
    }
}
/*
 * Clear all hisotry of calling cuda functions.
 */
void
HistList::clear(void)
{
   if (histrec != NULL) {
      for (int i=0; i < length; i++) {
         (releaseArgsStub[funcID2DSCVMethod( histrec[i].funcID)])(histrec[i].args);
      }
   }
   length = 0;
   byte_size = 0;
}

void
HistList::setRecallFlag(void)
{
    recall_flag = 1;
}

void
HistList::clrRecallFlag(void)
{
    recall_flag = 0;
}

void
HistList::print(void)
{
    WARN0(1, "<--- Record of CUDA API history Stack  *******\n");
    if (this->length == 0) {
	WARN0(1, "%s(): RecList[]> (Empty).\n", __func__);
	return;
    }
    for (int i=0; i<length; i++) { /* Print recall history. */
	WARN0(1, "        [%d] = #%lld:", i, histrec[i].seq_num);
	switch (histrec[i].funcID) { /* see "dscudarpc.h" */
	case 305: WARN0(1, "hipSetDevice()\n");        break;
	case 504: WARN0(1, "hipEventRecord()\n");      break;
	case 505: WARN0(1, "hipEventSynchronize()\n"); break;
	case 600: WARN0(1, "kernel-call<<< >>>()\n");   break;
	case 700: WARN0(1, "hipMalloc()\n");           break;
	case 701: WARN0(1, "hipFree()\n");             break;
	case 703: WARN0(1, "hipMemcpy(H2D)\n");        break;
	case 704: WARN0(1, "hipMemcpy(D2H)\n");        break;
	default:  WARN0(1, "/* %d */()\n", histrec[i].funcID);
	}
    }
    WARN0(1, "Occupied memory size is %d Byte.\n",  byte_size);
    WARN0(1, "---> Record of CUDA API history  *************\n");
}
/*
 * Rerun the recorded history of cuda function series.
 */
int
HistList::recall(void)
{
    WARN(9, "HistList::%s() {\n", __func__);
    static int called_depth = 0;
    int result;
    int verb_curr = St.autoverb;
   
    setRecallFlag();

    WARN(1, "called_depth= %d.\n", called_depth);
    if (called_depth < 0) {       /* irregal error */
	WARN(1, "#**********************************************************************\n");
	WARN(1, "# (;_;) DS-CUDA gave up the redundant calculation.                    *\n"); 
	WARN(1, "#       Unexpected error occured. called_depth=%d in %s()             *\n", called_depth, __func__);
	WARN(1, "#**********************************************************************\n\n");
	exit(1);
    } else if (called_depth < RC_REDUNDANT_GIVEUP_COUNT) { /* redundant calculation.*/
	called_depth++;       
	for (int i=0; i< length; i++) { /* Do recall history */
	    WARN(3, "(._.)Rollback API[%4d/%d]................................\n", i, length-1);
	    (recallStub[funcID2DSCVMethod( histrec[i].funcID )])(histrec[i].args); /* partially recursive */
	}
	called_depth=0;
	result = 0;
    } else { /* try migraion or not. */
	WARN(1, "#**********************************************************************\n");
	WARN(1, "# (;_;) DS-CUDA gave up the redundant calculation.                    *\n"); 
	WARN(1, "#       I have tried %2d times but never matched.                    *\n", RC_REDUNDANT_GIVEUP_COUNT);
	WARN(1, "#**********************************************************************\n\n");
	called_depth=0;
	result = 1;
   }

    WARN(9, "} HistList::%s()\n", __func__);
    St.autoverb = verb_curr;
    clrRecallFlag();
    
   return result;
} // HistList::recall(void)

